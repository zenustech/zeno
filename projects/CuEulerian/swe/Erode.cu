#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include <cassert>
#include <cstdlib>
#include <cstring>
#include <random>
#include <zeno/types/DummyObject.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/UserData.h>
#include <zeno/utils/log.h>
#include <zeno/utils/parallel_reduce.h>
#include <zeno/utils/vec.h>
#include <zeno/zeno.h>

namespace zs {

    template <typename T, zs::enable_if_t<std::is_floating_point_v<T>> = 0>
    __forceinline__ __device__ T clamp(T v, const T vmin, const T vmax) {
        return zs::max(vmin, zs::min(v, vmax));
    }

    template <typename TOut, typename T, zs::enable_if_t<std::is_integral_v<TOut> && std::is_integral_v<T>> = 0>
    __forceinline__ __device__ TOut clamp(TOut v, const T vmin, const T vmax) {
        v = v < vmin ? vmin : v;
        v = v > vmax ? vmax : v;
        return v;
    }

    __forceinline__ __device__ zs::vec<float, 3> normalizeSafe(const zs::vec<float, 3> &a,
                                                               float b = zs::detail::deduce_numeric_epsilon<float>()) {
        return a * (1 / zs::max(b, a.length()));
    }

    template <typename T, execspace_e space = deduce_execution_space(), enable_if_t<std::is_floating_point_v<T>> = 0>
    constexpr T tan(T v, wrapv<space> = {}) noexcept {
        if constexpr (space == execspace_e::cuda) {
#if ZS_ENABLE_CUDA && defined(__HIPCC__)
            if constexpr (is_same_v<T, float>)
                return ::tanf(v);
            else
                return ::tan((double)v);
#else
            static_assert(space != execspace_e::cuda, "cuda implementation of [tan] is missing!");
        return 0;
#endif
        } else
            return std::tan(v);
    }

} // namespace zs

namespace zeno {

    template <typename T>
    auto to_device_vector(const std::vector<T> &hv, bool copy = true) {
        using namespace zs;
        if constexpr (zs::is_vec<T>::value) {
            zs::Vector<zs::vec<typename T::value_type, std::tuple_size_v<T>>> dv{hv.size(), memsrc_e::device};
            if (copy) {
                Resource::copy(MemoryEntity{dv.memoryLocation(), (void *)dv.data()},
                               MemoryEntity{MemoryLocation{memsrc_e::host, -1}, (void *)hv.data()}, hv.size() * sizeof(T));
            }
            return dv;
        } else {
            zs::Vector<T> dv{hv.size(), memsrc_e::device};
            if (copy) {
                Resource::copy(MemoryEntity{dv.memoryLocation(), (void *)dv.data()},
                               MemoryEntity{MemoryLocation{memsrc_e::host, -1}, (void *)hv.data()}, hv.size() * sizeof(T));
            }
            return dv;
        }
    }

    template <typename T0, typename T1, zs::enable_if_t<sizeof(T0) == sizeof(T1)> = 0>
    void retrieve_device_vector(std::vector<T0> &hv, const zs::Vector<T1> &dv) {
        using namespace zs;
        Resource::copy(MemoryEntity{MemoryLocation{memsrc_e::host, -1}, (void *)hv.data()},
                       MemoryEntity{dv.memoryLocation(), (void *)dv.data()}, dv.size() * sizeof(T1));
    }

    __forceinline__ __device__ int Pos2Idx(const int x, const int z, const int nx) {
        return z * nx + x;
    }

    __forceinline__ __device__ unsigned int erode_random(float seed, int idx) {
        unsigned int s = *(unsigned int *)(&seed);
        s ^= idx << 3;
        s *= 179424691; // a magic prime number
        s ^= s << 13 | s >> (32 - 13);
        s ^= s >> 17 | s << (32 - 17);
        s ^= s << 23;
        s *= 179424691;
        return s;
    }

// 降水/蒸发
    struct zs_erode_value2cond : INode {
        void apply() override {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化网格
            auto terrain = get_input<PrimitiveObject>("prim_2DGrid");
            int nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->verts;
//            vec3f p0 = pos[0];
//            vec3f p1 = pos[1];
//            float cellSize = length(p1 - p0);
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto value = get_input2<float>("value");
            auto seed = get_input2<float>("seed");

            // 初始化网格属性
            if (!terrain->verts.has_attr("cond")) {
                auto &_cond = terrain->verts.add_attr<float>("cond");
                std::fill(_cond.begin(), _cond.end(), 0.0);
            }
            auto &attr_cond = terrain->verts.attr<float>("cond");
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////
            /// @brief  accelerate cond computation using cuda
            auto pol = cuda_exec();

            auto zs_attr_cond = to_device_vector(attr_cond, false);

            pol(range((std::size_t)nz * (std::size_t)nx),
                [=, attr_cond = view<space>(zs_attr_cond)] __device__(std::size_t idx) mutable {
                    if (value >= 1.0f) {
                        attr_cond[idx] = 1;
                    } else {
                        value = value < 0 ? 0 : (value > 1 ? 1 : value);
                        unsigned int cutoff = (unsigned int)(value * 4294967295.0);
                        unsigned int randval = erode_random(seed, idx + nx * nz);
                        attr_cond[idx] = randval < cutoff;
                    }
                });

            /// @brief  write back to host-side attribute
            retrieve_device_vector(attr_cond, zs_attr_cond);

            set_output("prim_2DGrid", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_erode_value2cond, {
        /* inputs: */
        {
            "prim_2DGrid",
            {"float", "value", "1.0"}, // 0.0 ~ 1.0
            {"float", "seed", "0.0"},
        },
        /* outputs: */
        {
            "prim_2DGrid",
        },
        /* params: */ {}, /* category: */
        {
            "erode",
        }});

    struct zs_erode_smooth_flow : INode {
        void apply() override {
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化网格
            auto terrain = get_input<PrimitiveObject>("prim_2DGrid");
            int nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->verts;
//            vec3f p0 = pos[0];
//            vec3f p1 = pos[1];
//            float cellSize = length(p1 - p0);
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto smooth_rate = get_input2<float>("smoothRate");
            auto flowName = get_input2<std::string>("flowName");
            // 初始化网格属性
            auto &flow = terrain->verts.attr<float>(flowName);
            auto &_lap = terrain->verts.add_attr<float>("_lap");
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////
            /// @brief  accelerate cond computation using cuda
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();
            /// @brief  copy host-side attribute
            auto zs_flow = to_device_vector(flow);
            auto zs_lap = to_device_vector(_lap);

            ///
            pol(range((std::size_t)nz * (std::size_t)nx),
                [flow = view<space>(zs_flow), _lap = view<space>(zs_lap), nx, nz] __device__(std::size_t idx) mutable {
                    auto id_z = idx / nx; // outer index
                    auto id_x = idx % nx; // inner index
                    float net_diff = 0.0f;
                    net_diff += flow[idx - 1 * (id_x > 0)];
                    net_diff += flow[idx + 1 * (id_x < nx - 1)];
                    net_diff += flow[idx - nx * (id_z > 0)];
                    net_diff += flow[idx + nx * (id_z < nz - 1)];
                    net_diff *= 0.25f;
                    net_diff -= flow[idx];
                    _lap[idx] = net_diff;
                });

            pol(range((std::size_t)nz * (std::size_t)nx), [flow = view<space>(zs_flow), _lap = view<space>(zs_lap),
                    smooth_rate, nx, nz] __device__(std::size_t idx) mutable {
                auto id_z = idx / nx; // outer index
                auto id_x = idx % nx; // inner index
                float net_diff = 0.0f;
                net_diff += _lap[idx - 1 * (id_x > 0)];
                net_diff += _lap[idx + 1 * (id_x < nx - 1)];
                net_diff += _lap[idx - nx * (id_z > 0)];
                net_diff += _lap[idx + nx * (id_z < nz - 1)];
                net_diff *= 0.25f;
                net_diff -= _lap[idx];
                flow[idx] -= smooth_rate * 0.5f * net_diff;
            });

            /// @brief  write back to host-side attribute
            retrieve_device_vector(flow, zs_flow);
            retrieve_device_vector(_lap, zs_lap);

            terrain->verts.erase_attr("_lap");
            set_output("prim_2DGrid", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_erode_smooth_flow, {
        /* inputs: */
        {
            "prim_2DGrid",
            {"float", "smoothRate", "1.0"},
            {"string", "flowName", "flow"},
        },
        /* outputs: */
        {
            "prim_2DGrid",
        },
        /* params: */ {}, /* category: */
        {
            "erode",
        }});


// 热侵蚀
    struct zs_erode_tumble_material_v0 : INode {
        void apply() override {

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<PrimitiveObject>("prim_2DGrid");
            int nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->verts;
//            vec3f p0 = pos[0];
//            vec3f p1 = pos[1];
//            float cellSize = length(p1 - p0);
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();
            auto cut_angle = get_input<NumericObject>("cut_angle")->get<float>();
            auto global_erosionrate = get_input<NumericObject>("global_erosionrate")->get<float>();
            auto erosionrate = get_input<NumericObject>("erosionrate")->get<float>();
            auto erodability = get_input<NumericObject>("erodability")->get<float>();
            auto removalrate = get_input<NumericObject>("removalrate")->get<float>();
            auto maxdepth = get_input<NumericObject>("maxdepth")->get<float>();

            std::uniform_real_distribution<float> distr(0.0, 1.0); // 设置随机分布
            auto seed = get_input<NumericObject>("seed")->get<float>();

            auto iterations = get_input<NumericObject>("iterations")->get<int>(); // 外部迭代总次数      10
            auto iter = get_input<NumericObject>("iter")->get<int>();             // 外部迭代当前次数    1~10
            auto i = get_input<NumericObject>("i")->get<int>();                   // 内部迭代当前次数    0~7
            auto openborder = get_input<NumericObject>("openborder")->get<int>(); // 获取边界标记

            auto perm = get_input<ListObject>("perm")->get2<int>(); //std::vector<int>
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            // 初始化网格属性
            if (!terrain->verts.has_attr("_height") || !terrain->verts.has_attr("_debris") ||
                !terrain->verts.has_attr("_temp_height") || !terrain->verts.has_attr("_temp_debris")) {
                zeno::log_error("Node [erode_tumble_material_v0], no such data layer named '{}' or '{}' or '{}' or '{}'.",
                                "_height", "_debris", "_temp_height", "_temp_debris");
            }
            auto &_height = terrain->verts.attr<float>("_height"); // 计算用的临时属性
            auto &_debris = terrain->verts.attr<float>("_debris");
            auto &_temp_height = terrain->verts.attr<float>("_temp_height"); // 备份用的临时属性
            auto &_temp_debris = terrain->verts.attr<float>("_temp_debris");

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////
            /// @brief  accelerate cond computation using cuda
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();
            /// @brief  copy host-side attribute
            auto zs_height = to_device_vector(_height);
            auto zs_debris = to_device_vector(_debris);
            auto zs_temp_height = to_device_vector(_temp_height);
            auto zs_temp_debris = to_device_vector(_temp_debris);
            auto zs_perm = to_device_vector(perm);
            auto zs_p_dirs = to_device_vector(p_dirs);
            auto zs_x_dirs = to_device_vector(x_dirs);

            pol(range((std::size_t)nz * (std::size_t)nx),
                [=, _height = view<space>(zs_height), _debris = view<space>(zs_debris),
                        _temp_height = view<space>(zs_temp_height), _temp_debris = view<space>(zs_temp_debris),
                        perm = view<space>(zs_perm), p_dirs = view<space>(zs_p_dirs),
                        x_dirs = view<space>(zs_x_dirs)] __device__(std::size_t idx) mutable {
                    auto id_z = idx / nx; // outer index
                    auto id_x = idx % nx; // inner index

                    int iterseed = iter * 134775813;
                    int color = perm[i];

                    int is_red = ((id_z & 1) == 1) && (color == 1);
                    int is_green = ((id_x & 1) == 1) && (color == 2);
                    int is_blue = ((id_z & 1) == 0) && (color == 3);
                    int is_yellow = ((id_x & 1) == 0) && (color == 4);
                    int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                    int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                    int dxs[] = {0, p_dirs[0], 0, p_dirs[0], x_dirs[0], x_dirs[1], x_dirs[0], x_dirs[1]};
                    int dzs[] = {p_dirs[1], 0, p_dirs[1], 0, x_dirs[0], -x_dirs[1], x_dirs[0], -x_dirs[1]};

                    if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                        int idx = Pos2Idx(id_x, id_z, nx);
                        int dx = dxs[color - 1];
                        int dz = dzs[color - 1];
                        int bound_x = nx;
                        int bound_z = nz;
                        int clamp_x = bound_x - 1;
                        int clamp_z = bound_z - 1;

                        float i_debris = _temp_debris[idx];
                        float i_height = _temp_height[idx];

                        int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                        int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                        int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);
                        if (validsource) {
                            validsource = validsource || !openborder;
                            int j_idx = Pos2Idx(samplex, samplez, nx);
                            float j_debris = validsource ? _temp_debris[j_idx] : 0.0f;
                            float j_height = _temp_height[j_idx];

                            int cidx = 0;
                            int cidz = 0;

                            float c_height = 0.0f;
                            float c_debris = 0.0f;
                            float n_debris = 0.0f;

                            int c_idx = 0;
                            int n_idx = 0;

                            int dx_check = 0;
                            int dz_check = 0;

                            float h_diff = 0.0f;

                            if ((j_height - i_height) > 0.0f) {
                                cidx = samplex;
                                cidz = samplez;

                                c_height = j_height;
                                c_debris = j_debris;
                                n_debris = i_debris;

                                c_idx = j_idx;
                                n_idx = idx;

                                dx_check = -dx;
                                dz_check = -dz;

                                h_diff = j_height - i_height;
                            } else {
                                cidx = id_x;
                                cidz = id_z;

                                c_height = i_height;
                                c_debris = i_debris;
                                n_debris = j_debris;

                                c_idx = idx;
                                n_idx = j_idx;

                                dx_check = dx;
                                dz_check = dz;

                                h_diff = i_height - j_height;
                            }

                            float max_diff = 0.0f;
                            float dir_prob = 0.0f;

                            for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++) {
                                for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++) {
                                    if (!tmp_dx && !tmp_dz)
                                        continue;

                                    int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                    int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);
                                    int tmp_validsource =
                                            (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));
                                    tmp_validsource = tmp_validsource || !openborder;
                                    int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                    float n_height = _temp_height[tmp_j_idx];

                                    float tmp_diff = n_height - (c_height);

                                    float _gridbias = zs::clamp(gridbias, -1.0f, 1.0f);

                                    if (tmp_dx && tmp_dz)
                                        tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                    else
                                        tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                    if (tmp_diff <= 0.0f) {
                                        if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                            dir_prob = tmp_diff;
                                        if (tmp_diff < max_diff)
                                            max_diff = tmp_diff;
                                    }
                                }
                            }
                            if (max_diff > 0.001f || max_diff < -0.001f)
                                dir_prob = dir_prob / max_diff;

                            int cond = 0;
                            if (dir_prob >= 1.0f)
                                cond = 1;
                            else {
                                dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                                unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                                unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                                cond = randval < cutoff;
                            }

                            if (cond) {
                                float abs_h_diff = h_diff < 0.0f ? -h_diff : h_diff;
                                float _cut_angle = zs::clamp(cut_angle, 0.0f, 90.0f);
                                float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);
                                float height_removed =
                                        _cut_angle < 90.0f ? zs::tan(_cut_angle * M_PI / 180) * delta_x : 1e10f;
                                float height_diff = abs_h_diff - height_removed;
                                if (height_diff < 0.0f)
                                    height_diff = 0.0f;
                                float prob = ((n_debris + c_debris) != 0.0f)
                                             ? zs::clamp((height_diff / (n_debris + c_debris)), 0.0f, 1.0f)
                                             : 1.0f;
                                unsigned int cutoff = (unsigned int)(prob * 4294967295.0);
                                unsigned int randval = erode_random(seed * 3.14, (idx + nx * nz) * 8 + color + iterseed);
                                int do_erode = randval < cutoff;

                                float height_removal_amt =
                                        do_erode * zs::clamp(global_erosionrate * erosionrate * erodability, 0.0f, height_diff);

                                _height[c_idx] -= height_removal_amt;

                                float bedrock_density = 1.0f - (removalrate);
                                if (bedrock_density > 0.0f) {
                                    float newdebris = bedrock_density * height_removal_amt;
                                    if (n_debris + newdebris > maxdepth) {
                                        float rollback = n_debris + newdebris - maxdepth;
                                        rollback = zs::min(rollback, newdebris);
                                        _height[c_idx] += rollback / bedrock_density;
                                        newdebris -= rollback;
                                    }
                                    _debris[c_idx] += newdebris;
                                }
                            }
                        }
                    }
                });

            /// @brief  write back to host-side attribute
            retrieve_device_vector(_height, zs_height);
            retrieve_device_vector(_debris, zs_debris);

            set_output("prim_2DGrid", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_erode_tumble_material_v0, {
        /* inputs: */
        {
            "prim_2DGrid",

            {"ListObject", "perm"},
            {"ListObject", "p_dirs"},
            {"ListObject", "x_dirs"},

            {"float", "seed", "9676.79"},
            {"int", "iterations", "0"},
            {"int", "iter", "0"},
            {"int", "i", "0"},

            {"int", "openborder", "0"},
            {"float", "gridbias", "0.0"},

            {"float", "cut_angle", "35"},
            {"float", "global_erosionrate", "1.0"},
            {"float", "erosionrate", "0.03"},
            {"float", "erodability", "0.4"},
            {"float", "removalrate", "0.7"},
            {"float", "maxdepth", "5.0"},
        },
        /* outputs: */
        {
            "prim_2DGrid",
        },
        /* params: */
        {

        },
        /* category: */
        {
            "erode",
        }});

// 崩塌
    struct zs_erode_tumble_material_v2 : INode {
        void apply() override {

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<PrimitiveObject>("prim_2DGrid");
            int nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->verts;
//            vec3f p0 = pos[0];
//            vec3f p1 = pos[1];
//            float cellSize = length(p1 - p0);
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();
            auto repose_angle = get_input<NumericObject>("repose_angle")->get<float>();
            auto quant_amt = get_input<NumericObject>("quant_amt")->get<float>();
            auto flow_rate = get_input<NumericObject>("flow_rate")->get<float>();

            std::uniform_real_distribution<float> distr(0.0, 1.0);
            auto seed = get_input<NumericObject>("seed")->get<float>();

            auto iterations = get_input<NumericObject>("iterations")->get<int>();
            auto iter = get_input<NumericObject>("iter")->get<int>();
            auto i = get_input<NumericObject>("i")->get<int>();
            auto openborder = get_input<NumericObject>("openborder")->get<int>();

            auto perm = get_input<ListObject>("perm")->get2<int>();
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            // 初始化网格属性
            auto stablilityMaskName = get_input2<std::string>("stabilitymask");
            if (!terrain->verts.has_attr(stablilityMaskName)) {
                auto &_sta = terrain->verts.add_attr<float>(stablilityMaskName);
                std::fill(_sta.begin(), _sta.end(), 0.0);
            }
            auto &stabilitymask = terrain->verts.attr<float>(stablilityMaskName);

            if (!terrain->verts.has_attr("height") || !terrain->verts.has_attr("_material") ||
                !terrain->verts.has_attr("_temp_material")) {
                zeno::log_error("Node [erode_tumble_material_v2], no such data layer named '{}' or '{}' or '{}'.", "height",
                                "_material", "_temp_material");
            }
            auto &height = terrain->verts.attr<float>("height");
            auto &_material = terrain->verts.attr<float>("_material");
            auto &_temp_material = terrain->verts.attr<float>("_temp_material");

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////
            /// @brief  accelerate cond computation using cuda
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();
            /// @brief  copy host-side attribute
            auto zs_material = to_device_vector(_material);
            auto zs_height = to_device_vector(height);
            auto zs_temp_material = to_device_vector(_temp_material);
            auto zs_stabilitymask = to_device_vector(stabilitymask);
            auto zs_perm = to_device_vector(perm);
            auto zs_p_dirs = to_device_vector(p_dirs);
            auto zs_x_dirs = to_device_vector(x_dirs);

            pol(range((std::size_t)nz * (std::size_t)nx),
                [=, _material = view<space>(zs_material), height = view<space>(zs_height),
                        _temp_material = view<space>(zs_temp_material), stabilitymask = view<space>(zs_stabilitymask),
                        perm = view<space>(zs_perm), p_dirs = view<space>(zs_p_dirs),
                        x_dirs = view<space>(zs_x_dirs)] __device__(std::size_t idx) mutable {
                    auto id_z = idx / nx; // outer index
                    auto id_x = idx % nx; // inner index

                    int iterseed = iter * 134775813;
                    int color = perm[i];

                    int is_red = ((id_z & 1) == 1) && (color == 1);
                    int is_green = ((id_x & 1) == 1) && (color == 2);
                    int is_blue = ((id_z & 1) == 0) && (color == 3);
                    int is_yellow = ((id_x & 1) == 0) && (color == 4);
                    int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                    int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                    int dxs[] = {0, p_dirs[0], 0, p_dirs[0], x_dirs[0], x_dirs[1], x_dirs[0], x_dirs[1]};
                    int dzs[] = {p_dirs[1], 0, p_dirs[1], 0, x_dirs[0], -x_dirs[1], x_dirs[0], -x_dirs[1]};

                    if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                        int idx = Pos2Idx(id_x, id_z, nx);
                        int dx = dxs[color - 1];
                        int dz = dzs[color - 1];
                        int bound_x = nx;
                        int bound_z = nz;
                        int clamp_x = bound_x - 1;
                        int clamp_z = bound_z - 1;

                        flow_rate = zs::clamp(flow_rate, 0.0f, 1.0f);

                        float i_material = _temp_material[idx];
                        float i_height = height[idx];

                        int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                        int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                        int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);

                        if (validsource) {
                            int same_node = !validsource;

                            validsource = validsource || !openborder;

                            int j_idx = Pos2Idx(samplex, samplez, nx);

                            float j_material = validsource ? _temp_material[j_idx] : 0.0f;
                            float j_height = height[j_idx];

                            float _repose_angle = repose_angle;
                            _repose_angle = zs::clamp(_repose_angle, 0.0f, 90.0f);
                            float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);
                            float static_diff =
                                    _repose_angle < 90.0f ? zs::tan(_repose_angle * M_PI / 180.0) * delta_x : 1e10f;
                            float m_diff = (j_height + j_material) - (i_height + i_material);
                            int cidx = 0;
                            int cidz = 0;

                            float c_height = 0.0f;
                            float c_material = 0.0f;
                            float n_material = 0.0f;

                            int c_idx = 0;
                            int n_idx = 0;

                            int dx_check = 0;
                            int dz_check = 0;

                            if (m_diff > 0.0f) {
                                cidx = samplex;
                                cidz = samplez;

                                c_height = j_height;
                                c_material = j_material;
                                n_material = i_material;

                                c_idx = j_idx;
                                n_idx = idx;

                                dx_check = -dx;
                                dz_check = -dz;
                            } else {
                                cidx = id_x;
                                cidz = id_z;

                                c_height = i_height;
                                c_material = i_material;
                                n_material = j_material;

                                c_idx = idx;
                                n_idx = j_idx;

                                dx_check = dx;
                                dz_check = dz;
                            }

                            float sum_diffs[] = {0.0f, 0.0f};
                            float dir_probs[] = {0.0f, 0.0f};
                            float dir_prob = 0.0f;
                            for (int diff_idx = 0; diff_idx < 2; diff_idx++) {
                                for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++) {
                                    for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++) {
                                        if (!tmp_dx && !tmp_dz)
                                            continue;

                                        int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                        int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);
                                        int tmp_validsource =
                                                (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));
                                        tmp_validsource = tmp_validsource || !openborder;
                                        int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                        float n_material = tmp_validsource ? _temp_material[tmp_j_idx] : 0.0f;
                                        float n_height = height[tmp_j_idx];
                                        float tmp_h_diff = n_height - (c_height);
                                        float tmp_m_diff = (n_height + n_material) - (c_height + c_material);
                                        float tmp_diff = diff_idx == 0 ? tmp_h_diff : tmp_m_diff;
                                        float _gridbias = gridbias;
                                        _gridbias = zs::clamp(_gridbias, -1.0f, 1.0f);

                                        if (tmp_dx && tmp_dz)
                                            tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                        else
                                            tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                        if (tmp_diff <= 0.0f) {
                                            if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                                dir_probs[diff_idx] = tmp_diff;

                                            if (diff_idx && dir_prob > tmp_diff)
                                                dir_prob = tmp_diff;

                                            sum_diffs[diff_idx] += tmp_diff;
                                        }
                                    }
                                }

                                if (diff_idx && (dir_prob > 0.001f || dir_prob < -0.001f))
                                    dir_prob = dir_probs[diff_idx] / dir_prob;

                                if (sum_diffs[diff_idx] > 0.001f || sum_diffs[diff_idx] < -0.001f)
                                    dir_probs[diff_idx] = dir_probs[diff_idx] / sum_diffs[diff_idx];
                            }

                            float movable_mat = (m_diff < 0.0f) ? -m_diff : m_diff;
                            float stability_val = 0.0f;
                            stability_val = zs::clamp(stabilitymask[c_idx], 0.0f, 1.0f);

                            if (stability_val > 0.01f)
                                movable_mat = zs::clamp(movable_mat * (1.0f - stability_val) * 0.5f, 0.0f, c_material);
                            else
                                movable_mat = zs::clamp((movable_mat - static_diff) * 0.5f, 0.0f, c_material);

                            float l_rat = dir_probs[1];
                            if (quant_amt > 0.001)
                                movable_mat =
                                        zs::clamp(quant_amt * zs::ceil((movable_mat * l_rat) / quant_amt), 0.0f, c_material);
                            else
                                movable_mat *= l_rat;

                            float diff = (m_diff > 0.0f) ? movable_mat : -movable_mat;

                            int cond = 0;
                            if (dir_prob >= 1.0f)
                                cond = 1;
                            else {
                                dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                                unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                                unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                                cond = randval < cutoff;
                            }

                            if (!cond || same_node)
                                diff = 0.0f;

                            diff *= flow_rate;
                            float abs_diff = (diff < 0.0f) ? -diff : diff;
                            _material[c_idx] = c_material - abs_diff;
                            _material[n_idx] = n_material + abs_diff;
                        }
                    }
                });

            /// @brief  write back to host-side attribute
            retrieve_device_vector(_material, zs_material);

            set_output("prim_2DGrid", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_erode_tumble_material_v2, {
        /* inputs: */
        {
            "prim_2DGrid",

            {"string", "stabilitymask", "_stability"},
            {"ListObject", "perm"},
            {"ListObject", "p_dirs"},
            {"ListObject", "x_dirs"},

            {"float", "seed", "15231.3"},
            {"int", "iterations", "0"},
            {"int", "iter", "0"},
            {"int", "i", "0"},

            {"int", "openborder", "0"},
            {"float", "gridbias", "0.0"},

            // 崩塌流淌相关
            {"float", "repose_angle", "15.0"},
            {"float", "quant_amt", "0.25"},
            {"float", "flow_rate", "1.0"},
        },
        /* outputs: */
        {
            "prim_2DGrid",
        },
        /* params: */
        {
            //{"string", "stabilitymask", "_stability"},
        },
        /* category: */
        {
            "erode",
        }});

// 崩塌 + flow
    struct zs_erode_tumble_material_v3 : INode {
        void apply() override {

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<PrimitiveObject>("prim_2DGrid");
            int nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->verts;
//            vec3f p0 = pos[0];
//            vec3f p1 = pos[1];
//            float cellSize = length(p1 - p0);
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();
            auto repose_angle = get_input<NumericObject>("repose_angle")->get<float>();
            auto quant_amt = get_input<NumericObject>("quant_amt")->get<float>();
            auto flow_rate = get_input<NumericObject>("flow_rate")->get<float>();

            std::uniform_real_distribution<float> distr(0.0, 1.0);
            auto seed = get_input<NumericObject>("seed")->get<float>();

            auto iterations = get_input<NumericObject>("iterations")->get<int>();
            auto iter = get_input<NumericObject>("iter")->get<int>();
            auto i = get_input<NumericObject>("i")->get<int>();
            auto openborder = get_input<NumericObject>("openborder")->get<int>();

            auto perm = get_input<ListObject>("perm")->get2<int>();
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            // 初始化网格属性
            auto stablilityMaskName = get_input2<std::string>("stabilitymask");
            if (!terrain->verts.has_attr(stablilityMaskName)) {
                auto &_sta = terrain->verts.add_attr<float>(stablilityMaskName);
                std::fill(_sta.begin(), _sta.end(), 0.0);
            }
            auto &stabilitymask = terrain->verts.attr<float>(stablilityMaskName);

            if (!terrain->verts.has_attr("height") || !terrain->verts.has_attr("_material") ||
                !terrain->verts.has_attr("_temp_material") || !terrain->verts.has_attr("flowdir")) {
                zeno::log_error("Node [erode_tumble_material_v3], no such data layer named '{}' or '{}' or '{}' or "
                                "'{}'.",
                                "height", "_material", "_temp_material", "flowdir");
            }
            auto &height = terrain->verts.attr<float>("height");
            auto &_material = terrain->verts.attr<float>("_material");
            auto &_temp_material = terrain->verts.attr<float>("_temp_material");
            auto &flowdir = terrain->verts.attr<vec3f>("flowdir");

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////
            /// @brief  accelerate cond computation using cuda
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();
            /// @brief  copy host-side attribute
            auto zs_material = to_device_vector(_material);
            auto zs_height = to_device_vector(height);
            auto zs_temp_material = to_device_vector(_temp_material);
            auto zs_flowdir = to_device_vector(flowdir);
            auto zs_stabilitymask = to_device_vector(stabilitymask);
            auto zs_perm = to_device_vector(perm);
            auto zs_p_dirs = to_device_vector(p_dirs);
            auto zs_x_dirs = to_device_vector(x_dirs);

            pol(range((std::size_t)nz * (std::size_t)nx),
                [=, _material = view<space>(zs_material), height = view<space>(zs_height),
                        _temp_material = view<space>(zs_temp_material), flowdir = view<space>(zs_flowdir),
                        stabilitymask = view<space>(zs_stabilitymask), perm = view<space>(zs_perm),
                        p_dirs = view<space>(zs_p_dirs), x_dirs = view<space>(zs_x_dirs)] __device__(std::size_t idx) mutable {
                    auto id_z = idx / nx; // outer index
                    auto id_x = idx % nx; // inner index

                    int iterseed = iter * 134775813;
                    int color = perm[i];

                    int is_red = ((id_z & 1) == 1) && (color == 1);
                    int is_green = ((id_x & 1) == 1) && (color == 2);
                    int is_blue = ((id_z & 1) == 0) && (color == 3);
                    int is_yellow = ((id_x & 1) == 0) && (color == 4);
                    int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                    int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                    int dxs[] = {0, p_dirs[0], 0, p_dirs[0], x_dirs[0], x_dirs[1], x_dirs[0], x_dirs[1]};
                    int dzs[] = {p_dirs[1], 0, p_dirs[1], 0, x_dirs[0], -x_dirs[1], x_dirs[0], -x_dirs[1]};

                    if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                        int idx = Pos2Idx(id_x, id_z, nx);
                        int dx = dxs[color - 1];
                        int dz = dzs[color - 1];
                        int bound_x = nx;
                        int bound_z = nz;
                        int clamp_x = bound_x - 1;
                        int clamp_z = bound_z - 1;

                        flow_rate = zs::clamp(flow_rate, 0.0f, 1.0f);

                        // CALC_FLOW
                        float diff_x = 0.0f;
                        float diff_z = 0.0f;

                        float i_material = _temp_material[idx];
                        float i_height = height[idx];

                        int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                        int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                        int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);

                        if (validsource) {
                            int same_node = !validsource;

                            validsource = validsource || !openborder;

                            int j_idx = Pos2Idx(samplex, samplez, nx);

                            float j_material = validsource ? _temp_material[j_idx] : 0.0f;
                            float j_height = height[j_idx];

                            float _repose_angle = repose_angle;
                            _repose_angle = zs::clamp(_repose_angle, 0.0f, 90.0f);
                            float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);

                            float static_diff =
                                    _repose_angle < 90.0f ? zs::tan(_repose_angle * M_PI / 180.0) * delta_x : 1e10f;

                            float m_diff = (j_height + j_material) - (i_height + i_material);

                            int cidx = 0;
                            int cidz = 0;

                            float c_height = 0.0f;
                            float c_material = 0.0f;
                            float n_material = 0.0f;

                            int c_idx = 0;
                            int n_idx = 0;

                            int dx_check = 0;
                            int dz_check = 0;

                            if (m_diff > 0.0f) {
                                cidx = samplex;
                                cidz = samplez;

                                c_height = j_height;
                                c_material = j_material;
                                n_material = i_material;

                                c_idx = j_idx;
                                n_idx = idx;

                                dx_check = -dx;
                                dz_check = -dz;
                            } else {
                                cidx = id_x;
                                cidz = id_z;

                                c_height = i_height;
                                c_material = i_material;
                                n_material = j_material;

                                c_idx = idx;
                                n_idx = j_idx;

                                dx_check = dx;
                                dz_check = dz;
                            }

                            float sum_diffs[] = {0.0f, 0.0f};
                            float dir_probs[] = {0.0f, 0.0f};
                            float dir_prob = 0.0f;
                            for (int diff_idx = 0; diff_idx < 2; diff_idx++) {
                                for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++) {
                                    for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++) {
                                        if (!tmp_dx && !tmp_dz)
                                            continue;

                                        int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                        int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);
                                        int tmp_validsource =
                                                (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));

                                        tmp_validsource = tmp_validsource || !openborder;
                                        int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                        float n_material = tmp_validsource ? _temp_material[tmp_j_idx] : 0.0f;
                                        float n_height = height[tmp_j_idx];
                                        float tmp_h_diff = n_height - (c_height);
                                        float tmp_m_diff = (n_height + n_material) - (c_height + c_material);
                                        float tmp_diff = diff_idx == 0 ? tmp_h_diff : tmp_m_diff;
                                        float _gridbias = gridbias;

                                        _gridbias = zs::clamp(_gridbias, -1.0f, 1.0f);

                                        if (tmp_dx && tmp_dz)
                                            tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                        else
                                            tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                        if (tmp_diff <= 0.0f) {
                                            if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                                dir_probs[diff_idx] = tmp_diff;

                                            if (diff_idx && dir_prob > tmp_diff)
                                                dir_prob = tmp_diff;

                                            sum_diffs[diff_idx] += tmp_diff;
                                        }
                                    }
                                }

                                if (diff_idx && (dir_prob > 0.001f || dir_prob < -0.001f))
                                    dir_prob = dir_probs[diff_idx] / dir_prob;

                                if (sum_diffs[diff_idx] > 0.001f || sum_diffs[diff_idx] < -0.001f)
                                    dir_probs[diff_idx] = dir_probs[diff_idx] / sum_diffs[diff_idx];
                            }

                            float movable_mat = (m_diff < 0.0f) ? -m_diff : m_diff;
                            float stability_val = 0.0f;
                            stability_val = zs::clamp(stabilitymask[c_idx], 0.0f, 1.0f);

                            if (stability_val > 0.01f)
                                movable_mat = zs::clamp(movable_mat * (1.0f - stability_val) * 0.5f, 0.0f, c_material);
                            else
                                movable_mat = zs::clamp((movable_mat - static_diff) * 0.5f, 0.0f, c_material);

                            float l_rat = dir_probs[1];
                            if (quant_amt > 0.001)
                                movable_mat =
                                        zs::clamp(quant_amt * zs::ceil((movable_mat * l_rat) / quant_amt), 0.0f, c_material);
                            else
                                movable_mat *= l_rat;

                            float diff = (m_diff > 0.0f) ? movable_mat : -movable_mat;

                            int cond = 0;
                            if (dir_prob >= 1.0f)
                                cond = 1;
                            else {
                                dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                                unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                                unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                                cond = randval < cutoff;
                            }

                            if (!cond || same_node)
                                diff = 0.0f;

                            diff *= flow_rate;

                            // CALC_FLOW
                            diff_x += (float)dx * diff;
                            diff_z += (float)dz * diff;
                            diff_x *= -1.0f;
                            diff_z *= -1.0f;

                            float abs_diff = (diff < 0.0f) ? -diff : diff;
                            _material[c_idx] = c_material - abs_diff;
                            _material[n_idx] = n_material + abs_diff;

                            // CALC_FLOW
                            float abs_c_x = flowdir[c_idx][0];
                            abs_c_x = (abs_c_x < 0.0f) ? -abs_c_x : abs_c_x;
                            float abs_c_z = flowdir[c_idx][2];
                            abs_c_z = (abs_c_z < 0.0f) ? -abs_c_z : abs_c_z;
                            flowdir[c_idx][0] += diff_x * 1.0f / (1.0f + abs_c_x);
                            flowdir[c_idx][2] += diff_z * 1.0f / (1.0f + abs_c_z);
                        }
                    }
                });

            /// @brief  write back to host-side attribute
            retrieve_device_vector(_material, zs_material);
            retrieve_device_vector(flowdir, zs_flowdir);

            set_output("prim_2DGrid", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_erode_tumble_material_v3, {
        /* inputs: */
        {
            "prim_2DGrid",

            {"string", "stabilitymask", "_stability"},
            {"ListObject", "perm"},
            {"ListObject", "p_dirs"},
            {"ListObject", "x_dirs"},

            {"float", "seed", "15231.3"},
            {"int", "iterations", "0"},
            {"int", "iter", "0"},
            {"int", "i", "0"},

            {"int", "openborder", "0"},
            {"float", "gridbias", "0.0"},

            // 崩塌流淌相关
            {"float", "repose_angle", "0.0"},
            {"float", "quant_amt", "0.0"},
            {"float", "flow_rate", "1.0"},
        },
        /* outputs: */
        {
            "prim_2DGrid",
        },
        /* params: */
        {
            //{"string", "stabilitymask", "_stability"},
        },
        /* category: */
        {
            "erode",
        }});

// 崩塌 + 侵蚀
    struct zs_erode_tumble_material_v4 : INode {
        void apply() override {

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<PrimitiveObject>("prim_2DGrid");
            int nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->verts;
//            vec3f p0 = pos[0];
//            vec3f p1 = pos[1];
//            float cellSize = length(p1 - p0);
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            // 侵蚀主参数
            auto global_erosionrate = get_input<NumericObject>("global_erosionrate")->get<float>(); // 1 全局侵蚀率
            auto erodability = get_input<NumericObject>("erodability")->get<float>();               // 1.0 侵蚀能力
            auto erosionrate = get_input<NumericObject>("erosionrate")->get<float>();               // 0.4 侵蚀率
            auto bank_angle = get_input<NumericObject>("bank_angle")->get<float>(); // 70.0 河堤侵蚀角度
            auto seed = get_input<NumericObject>("seed")->get<float>();             // 12.34

            // 高级参数
            auto removalrate = get_input<NumericObject>("removalrate")->get<float>(); // 0.0 风化率/水吸收率
            auto max_debris_depth = get_input<NumericObject>("max_debris_depth")->get<float>(); // 5	碎屑最大深度
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();                 // 0.0

            // 侵蚀能力调整
            auto max_erodability_iteration = get_input<NumericObject>("max_erodability_iteration")->get<int>();     // 5
            auto initial_erodability_factor = get_input<NumericObject>("initial_erodability_factor")->get<float>(); // 0.5
            auto slope_contribution_factor = get_input<NumericObject>("slope_contribution_factor")->get<float>();   // 0.8

            // 河床参数
            auto bed_erosionrate_factor =
                    get_input<NumericObject>("bed_erosionrate_factor")->get<float>();           // 1 河床侵蚀率因子
            auto depositionrate = get_input<NumericObject>("depositionrate")->get<float>(); // 0.01 沉积率
            auto sedimentcap = get_input<NumericObject>("sedimentcap")
                    ->get<float>(); // 10.0 高度差转变为沉积物的比率 / 泥沙容量，每单位流动水可携带的泥沙量

            // 河堤参数
            auto bank_erosionrate_factor =
                    get_input<NumericObject>("bank_erosionrate_factor")->get<float>(); // 1.0 河堤侵蚀率因子
            auto max_bank_bed_ratio = get_input<NumericObject>("max_bank_bed_ratio")
                    ->get<float>(); // 0.5 The maximum of bank to bed water column height ratio
            // 高于这个比值的河岸将不会在侵蚀中被视为河岸，会停止侵蚀
            // 河流控制
            auto quant_amt = get_input<NumericObject>("quant_amt")->get<float>(); // 0.05 流量维持率，越高流量越稳定
            auto iterations = get_input<NumericObject>("iterations")->get<int>(); // 流淌的总迭代次数

            //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
            std::uniform_real_distribution<float> distr(0.0, 1.0);
            auto iter = get_input<NumericObject>("iter")->get<int>();
            auto i = get_input<NumericObject>("i")->get<int>();
            auto openborder = get_input<NumericObject>("openborder")->get<int>();

            auto perm = get_input<ListObject>("perm")->get2<int>();
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            // 初始化网格属性
            if (!terrain->verts.has_attr("_height") || !terrain->verts.has_attr("_temp_height") ||
                !terrain->verts.has_attr("_material") || !terrain->verts.has_attr("_temp_material") ||
                !terrain->verts.has_attr("_debris") || !terrain->verts.has_attr("_temp_debris") ||
                !terrain->verts.has_attr("_sediment")) {
                zeno::log_error("Node [erode_tumble_material_v4], no such data layer named '{}' or '{}' or '{}' or '{}' or "
                                "'{}' or '{}' or '{}'.",
                                "_height", "_temp_height", "_material", "_temp_material", "_debris", "_temp_debris",
                                "_sediment");
            }
            auto &_height = terrain->verts.attr<float>("_height");
            auto &_temp_height = terrain->verts.attr<float>("_temp_height");
            auto &_material = terrain->verts.attr<float>("_material");
            auto &_temp_material = terrain->verts.attr<float>("_temp_material");
            auto &_debris = terrain->verts.attr<float>("_debris");
            auto &_temp_debris = terrain->verts.attr<float>("_temp_debris");
            auto &_sediment = terrain->verts.attr<float>("_sediment");

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////
            /// @brief  accelerate cond computation using cuda
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();
            /// @brief  copy host-side attribute
            auto zs_height = to_device_vector(_height);
            auto zs_temp_height = to_device_vector(_temp_height);
            auto zs_material = to_device_vector(_material);
            auto zs_temp_material = to_device_vector(_temp_material);
            auto zs_debris = to_device_vector(_debris);
            auto zs_temp_debris = to_device_vector(_temp_debris);
            auto zs_sediment = to_device_vector(_sediment);
            auto zs_perm = to_device_vector(perm);
            auto zs_p_dirs = to_device_vector(p_dirs);
            auto zs_x_dirs = to_device_vector(x_dirs);

            pol(range((std::size_t)nz * (std::size_t)nx),
                [=, _height = view<space>(zs_height), _temp_height = view<space>(zs_temp_height),
                        _material = view<space>(zs_material), _temp_material = view<space>(zs_temp_material),
                        _debris = view<space>(zs_debris), _temp_debris = view<space>(zs_temp_debris),
                        _sediment = view<space>(zs_sediment), perm = view<space>(zs_perm), p_dirs = view<space>(zs_p_dirs),
                        x_dirs = view<space>(zs_x_dirs)] __device__(std::size_t idx) mutable {
                    auto id_z = idx / nx; // outer index
                    auto id_x = idx % nx; // inner index

                    int iterseed = iter * 134775813;
                    int color = perm[i];
                    int is_red = ((id_z & 1) == 1) && (color == 1);
                    int is_green = ((id_x & 1) == 1) && (color == 2);
                    int is_blue = ((id_z & 1) == 0) && (color == 3);
                    int is_yellow = ((id_x & 1) == 0) && (color == 4);
                    int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                    int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                    int dxs[] = {0, p_dirs[0], 0, p_dirs[0], x_dirs[0], x_dirs[1], x_dirs[0], x_dirs[1]};
                    int dzs[] = {p_dirs[1], 0, p_dirs[1], 0, x_dirs[0], -x_dirs[1], x_dirs[0], -x_dirs[1]};

                    if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                        int idx = Pos2Idx(id_x, id_z, nx);
                        int dx = dxs[color - 1];
                        int dz = dzs[color - 1];
                        int bound_x = nx;
                        int bound_z = nz;
                        int clamp_x = bound_x - 1;
                        int clamp_z = bound_z - 1;

                        float i_height = _temp_height[idx];
                        float i_material = _temp_material[idx];
                        float i_debris = _temp_debris[idx];
                        float i_sediment = _sediment[idx];

                        int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                        int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                        int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);

                        if (validsource) {
                            validsource = validsource || !openborder;

                            int j_idx = Pos2Idx(samplex, samplez, nx);

                            float j_height = _temp_height[j_idx];
                            float j_material = validsource ? _temp_material[j_idx] : 0.0f;
                            float j_debris = validsource ? _temp_debris[j_idx] : 0.0f;

                            float j_sediment = validsource ? _sediment[j_idx] : 0.0f;
                            float m_diff = (j_height + j_debris + j_material) - (i_height + i_debris + i_material);
                            float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);

                            int cidx = 0;
                            int cidz = 0;

                            float c_height = 0.0f;

                            float c_material = 0.0f;
                            float n_material = 0.0f;

                            float c_sediment = 0.0f;
                            float n_sediment = 0.0f;

                            float c_debris = 0.0f;
                            float n_debris = 0.0f;

                            float h_diff = 0.0f;

                            int c_idx = 0;
                            int n_idx = 0;
                            int dx_check = 0;
                            int dz_check = 0;
                            int is_mh_diff_same_sign = 0;

                            if (m_diff > 0.0f) {
                                cidx = samplex;
                                cidz = samplez;

                                c_height = j_height;
                                c_material = j_material;
                                n_material = i_material;
                                c_sediment = j_sediment;
                                n_sediment = i_sediment;
                                c_debris = j_debris;
                                n_debris = i_debris;

                                c_idx = j_idx;
                                n_idx = idx;

                                dx_check = -dx;
                                dz_check = -dz;

                                h_diff = j_height + j_debris - (i_height + i_debris);
                                is_mh_diff_same_sign = (h_diff * m_diff) > 0.0f;
                            } else {
                                cidx = id_x;
                                cidz = id_z;

                                c_height = i_height;
                                c_material = i_material;
                                n_material = j_material;
                                c_sediment = i_sediment;
                                n_sediment = j_sediment;
                                c_debris = i_debris;
                                n_debris = j_debris;

                                c_idx = idx;
                                n_idx = j_idx;

                                dx_check = dx;
                                dz_check = dz;

                                h_diff = i_height + i_debris - (j_height + j_debris);
                                is_mh_diff_same_sign = (h_diff * m_diff) > 0.0f;
                            }
                            h_diff = (h_diff < 0.0f) ? -h_diff : h_diff;

                            float sum_diffs[] = {0.0f, 0.0f};
                            float dir_probs[] = {0.0f, 0.0f};
                            float dir_prob = 0.0f;
                            for (int diff_idx = 0; diff_idx < 2; diff_idx++) {
                                for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++) {
                                    for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++) {
                                        if (!tmp_dx && !tmp_dz)
                                            continue;

                                        int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                        int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);

                                        int tmp_validsource =
                                                (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));
                                        tmp_validsource = tmp_validsource || !openborder;
                                        int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                        float tmp_n_material = tmp_validsource ? _temp_material[tmp_j_idx] : 0.0f;
                                        float tmp_n_debris = tmp_validsource ? _temp_debris[tmp_j_idx] : 0.0f;

                                        float n_height = _temp_height[tmp_j_idx];
                                        float tmp_h_diff = n_height + tmp_n_debris - (c_height + c_debris);
                                        float tmp_m_diff =
                                                (n_height + tmp_n_debris + tmp_n_material) - (c_height + c_debris + c_material);
                                        float tmp_diff = diff_idx == 0 ? tmp_h_diff : tmp_m_diff;
                                        float _gridbias = gridbias;
                                        _gridbias = zs::clamp(_gridbias, -1.0f, 1.0f);

                                        if (tmp_dx && tmp_dz)
                                            tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                        else
                                            tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                        if (tmp_diff <= 0.0f) {
                                            if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                                dir_probs[diff_idx] = tmp_diff;

                                            if (diff_idx && (tmp_diff < dir_prob))
                                                dir_prob = tmp_diff;

                                            sum_diffs[diff_idx] += tmp_diff;
                                        }
                                    }
                                }

                                if (diff_idx && (dir_prob > 0.001f || dir_prob < -0.001f))
                                    dir_prob = dir_probs[diff_idx] / dir_prob;
                                else
                                    dir_prob = 0.0f;

                                if (sum_diffs[diff_idx] > 0.001f || sum_diffs[diff_idx] < -0.001f)
                                    dir_probs[diff_idx] = dir_probs[diff_idx] / sum_diffs[diff_idx];
                                else
                                    dir_probs[diff_idx] = 0.0f;
                            }

                            float movable_mat = (m_diff < 0.0f) ? -m_diff : m_diff;
                            movable_mat = zs::clamp(movable_mat * 0.5f, 0.0f, c_material);
                            float l_rat = dir_probs[1];

                            if (quant_amt > 0.001)
                                movable_mat =
                                        zs::clamp(quant_amt * zs::ceil((movable_mat * l_rat) / quant_amt), 0.0f, c_material);
                            else
                                movable_mat *= l_rat;

                            float diff = (m_diff > 0.0f) ? movable_mat : -movable_mat;

                            int cond = 0;
                            if (dir_prob >= 1.0f)
                                cond = 1;
                            else {
                                dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                                unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                                unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                                cond = randval < cutoff;
                            }

                            if (!cond)
                                diff = 0.0f;

                            float slope_cont = (delta_x > 0.0f) ? (h_diff / delta_x) : 0.0f;
                            float kd_factor = zs::clamp((1 / (1 + (slope_contribution_factor * slope_cont))), 0.0f, 1.0f);
                            float norm_iter = zs::clamp(((float)iter / (float)max_erodability_iteration), 0.0f, 1.0f);
                            float ks_factor = zs::clamp((1 - (slope_contribution_factor * zs::exp(-slope_cont))) *
                                                        zs::sqrt(dir_probs[0]) *
                                                        (initial_erodability_factor +
                                                         ((1.0f - initial_erodability_factor) * zs::sqrt(norm_iter))),
                                                        0.0f, 1.0f);

                            float c_ks = global_erosionrate * erosionrate * erodability * ks_factor;

                            float n_kd = depositionrate * kd_factor;
                            n_kd = zs::clamp(n_kd, 0.0f, 1.0f);

                            float _removalrate = removalrate;

                            float bedrock_density = 1.0f - _removalrate;
                            float abs_diff = (diff < 0.0f) ? -diff : diff;
                            float sediment_limit = sedimentcap * abs_diff;
                            float ent_check_diff = sediment_limit - c_sediment;

                            if (ent_check_diff > 0.0f) {
                                float dissolve_amt = c_ks * bed_erosionrate_factor * abs_diff;
                                float dissolved_debris = zs::min(c_debris, dissolve_amt);
                                _debris[c_idx] -= dissolved_debris;
                                _height[c_idx] -= (dissolve_amt - dissolved_debris);
                                _sediment[c_idx] -= c_sediment / 2;
                                if (bedrock_density > 0.0f) {
                                    float newsediment = c_sediment / 2 + (dissolve_amt * bedrock_density);
                                    if (n_sediment + newsediment > max_debris_depth) {
                                        float rollback = n_sediment + newsediment - max_debris_depth;
                                        rollback = zs::min(rollback, newsediment);
                                        _height[c_idx] += rollback / bedrock_density;
                                        newsediment -= rollback;
                                    }
                                    _sediment[n_idx] += newsediment;
                                }
                            } else {
                                float c_kd = depositionrate * kd_factor;
                                c_kd = zs::clamp(c_kd, 0.0f, 1.0f);
                                {
                                    _debris[c_idx] += (c_kd * -ent_check_diff);
                                    _sediment[c_idx] = (1 - c_kd) * -ent_check_diff;

                                    n_sediment += sediment_limit;
                                    _debris[n_idx] += (n_kd * n_sediment);
                                    _sediment[n_idx] = (1 - n_kd) * n_sediment;
                                }

                                int b_idx = 0;
                                int r_idx = 0;
                                float b_material = 0.0f;
                                float r_material = 0.0f;
                                float b_debris = 0.0f;
                                float r_debris = 0.0f;
                                float r_sediment = 0.0f;

                                if (is_mh_diff_same_sign) {
                                    b_idx = c_idx;
                                    r_idx = n_idx;

                                    b_material = c_material;
                                    r_material = n_material;

                                    b_debris = c_debris;
                                    r_debris = n_debris;

                                    r_sediment = n_sediment;
                                } else {
                                    b_idx = n_idx;
                                    r_idx = c_idx;

                                    b_material = n_material;
                                    r_material = c_material;

                                    b_debris = n_debris;
                                    r_debris = c_debris;

                                    r_sediment = c_sediment;
                                }

                                float erosion_per_unit_water =
                                        global_erosionrate * erosionrate * bed_erosionrate_factor * erodability * ks_factor;
                                if (r_material != 0.0f && (b_material / r_material) < max_bank_bed_ratio &&
                                    r_sediment > (erosion_per_unit_water * max_bank_bed_ratio)) {
                                    float height_to_erode = global_erosionrate * erosionrate * bank_erosionrate_factor *
                                                            erodability * ks_factor;

                                    float _bank_angle = bank_angle;

                                    _bank_angle = zs::clamp(_bank_angle, 0.0f, 90.0f);
                                    float safe_diff =
                                            _bank_angle < 90.0f ? zs::tan(_bank_angle * M_PI / 180.0) * delta_x : 1e10f;
                                    float target_height_removal = (h_diff - safe_diff) < 0.0f ? 0.0f : h_diff - safe_diff;

                                    float dissolve_amt = zs::clamp(height_to_erode, 0.0f, target_height_removal);
                                    float dissolved_debris = zs::min(b_debris, dissolve_amt);

                                    _debris[b_idx] -= dissolved_debris;

                                    float division = 1 / (1 + safe_diff);

                                    _height[b_idx] -= (dissolve_amt - dissolved_debris);

                                    if (bedrock_density > 0.0f) {
                                        float newdebris = (1 - division) * (dissolve_amt * bedrock_density);
                                        if (b_debris + newdebris > max_debris_depth) {
                                            float rollback = b_debris + newdebris - max_debris_depth;
                                            rollback = zs::min(rollback, newdebris);
                                            _height[b_idx] += rollback / bedrock_density;
                                            newdebris -= rollback;
                                        }
                                        _debris[b_idx] += newdebris;

                                        newdebris = division * (dissolve_amt * bedrock_density);

                                        if (r_debris + newdebris > max_debris_depth) {
                                            float rollback = r_debris + newdebris - max_debris_depth;
                                            rollback = zs::min(rollback, newdebris);
                                            _height[b_idx] += rollback / bedrock_density;
                                            newdebris -= rollback;
                                        }
                                        _debris[r_idx] += newdebris;
                                    }
                                }
                            }

                            _material[idx] = i_material + diff;
                            _material[j_idx] = j_material - diff;
                        }
                    }
                });

            /// @brief  write back to host-side attribute
            retrieve_device_vector(_height, zs_height);
            retrieve_device_vector(_material, zs_material);
            retrieve_device_vector(_debris, zs_debris);
            retrieve_device_vector(_sediment, zs_sediment);

            set_output("prim_2DGrid", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_erode_tumble_material_v4,
               {/* inputs: */ {
                       "prim_2DGrid",

                       {"ListObject", "perm"},
                       {"ListObject", "p_dirs"},
                       {"ListObject", "x_dirs"},

                       {"float", "seed", "12.34"},
                       {"int", "iterations", "40"}, // 流淌的总迭代次数
                       {"int", "iter", "0"},
                       {"int", "i", "0"},

                       {"int", "openborder", "0"},
                       {"float", "gridbias", "0.0"},

                       // 侵蚀主参数
                       {"float", "global_erosionrate", "1.0"}, // 全局侵蚀率
                       {"float", "erodability", "1.0"},        // 侵蚀能力
                       {"float", "erosionrate", "0.4"},        // 侵蚀率
                       {"float", "bank_angle", "70.0"},        // 河堤侵蚀角度

                       // 高级参数
                       {"float", "removalrate", "0.1"},      // 风化率/水吸收率
                       {"float", "max_debris_depth", "5.0"}, // 碎屑最大深度

                       // 侵蚀能力调整
                       {"int", "max_erodability_iteration", "5"},      // 最大侵蚀能力迭代次数
                       {"float", "initial_erodability_factor", "0.5"}, // 初始侵蚀能力因子
                       {"float", "slope_contribution_factor",
                           "0.8"}, // “地面斜率”对“侵蚀”和“沉积”的影响，“地面斜率大” -> 侵蚀因子大，沉积因子小

                       // 河床参数
                       {"float", "bed_erosionrate_factor", "1.0"}, // 河床侵蚀率因子
                       {"float", "depositionrate", "0.01"},        // 沉积率
                       {"float", "sedimentcap", "10.0"}, // 高度差转变为沉积物的比率 / 泥沙容量，每单位流动水可携带的泥沙量

                       // 河堤参数
                       {"float", "bank_erosionrate_factor", "1.0"}, // 河堤侵蚀率因子
                       {"float", "max_bank_bed_ratio", "0.5"}, // 高于这个比值的河岸将不会在侵蚀中被视为河岸，会停止侵蚀

                       // 河网控制
                       {"float", "quant_amt", "0.05"}, // 流量维持率，越高河流流量越稳定
                   },
                   /* outputs: */
                   {
                       "prim_2DGrid",
                   },
                   /* params: */
                   {

                   },
                   /* category: */
                   {
                       "erode",
                   }});



    __forceinline__ __device__ float fit(const float data, const float ss, const float se, const float ds, const float de) {
        float b = zs::detail::deduce_numeric_epsilon<float>();
        b = zs::max(zs::abs(se - ss), b);
        b = se - ss >= 0 ? b : -b;
        float alpha = (data - ss) / b;
        return ds + (de - ds) * alpha;
    }

    __forceinline__ __device__ float chramp(const float inputData) {
        float data = zs::min(zs::max(inputData, 0.0f), 1.0f);
        float outputData = 0;
        if (data <= 0.1) {
            outputData = fit(data, 0, 0.1, 0, 1);
        } else if (data >= 0.9) {
            outputData = fit(data, 0.9, 1.0, 1, 0);
        } else {
            outputData = 1;
        }
        return outputData;
    }

//    struct zs_HF_maskByFeature : INode {
//        void apply() override {
//
//            ////////////////////////////////////////////////////////////////////////////////////////
//            ////////////////////////////////////////////////////////////////////////////////////////
//            // 初始化
//            ////////////////////////////////////////////////////////////////////////////////////////
//
//            // 初始化网格
//            auto terrain = get_input<PrimitiveObject>("HeightField");
//            int nx, nz;
//            auto &ud = terrain->userData();
//            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
//                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
//            nx = ud.get2<int>("nx");
//            nz = ud.get2<int>("nz");
//            auto &pos = terrain->verts;
////            vec3f p0 = pos[0];
////            vec3f p1 = pos[1];
////            float cellSize = length(p1 - p0);
//            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
//            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
//            float cellSize = zeno::max(pos_delta_x, pos_delta_z);
//
//            // 获取面板参数
//            auto heightLayer = get_input2<std::string>("height_layer");
//            auto maskLayer = get_input2<std::string>("mask_layer");
//            auto smoothRadius = get_input2<int>("smooth_radius");
//
//            auto useSlope = get_input2<bool>("use_slope");
//            auto minSlope = get_input2<float>("min_slopeangle");
//            auto maxSlope = get_input2<float>("max_slopeangle");
//
//            auto useDir = get_input2<bool>("use_direction");
//            auto goalAngle = get_input2<float>("goal_angle");
//            auto angleSpread = get_input2<float>("angle_spread");
//
//            auto useHeight = get_input2<bool>("use_height");
//            auto minHeight = get_input2<float>("min_height");
//            auto maxHeight = get_input2<float>("max_height");
//
//            // 初始化网格属性
//            if (!terrain->verts.has_attr(heightLayer) || !terrain->verts.has_attr(maskLayer)) {
//                zeno::log_error("Node [HF_maskByFeature], no such data layer named '{}' or '{}'.", heightLayer, maskLayer);
//            }
//            auto &height = terrain->verts.attr<float>(heightLayer);
//            auto &mask = terrain->verts.attr<float>(maskLayer);
//
//            auto &_grad = terrain->verts.add_attr<vec3f>("_grad");
//            std::fill(_grad.begin(), _grad.end(), vec3f(0, 0, 0));
//
//            ////////////////////////////////////////////////////////////////////////////////////////
//            ////////////////////////////////////////////////////////////////////////////////////////
//            // 计算
//            ////////////////////////////////////////////////////////////////////////////////////////
//
//            /// @brief  accelerate cond computation using cuda
//            using namespace zs;
//            constexpr auto space = execspace_e::cuda;
//            auto pol = cuda_exec();
//            /// @brief  copy host-side attribute
//            auto zs_height = to_device_vector(height);
//            auto zs_mask = to_device_vector(mask, false);
//            auto zs_grad = to_device_vector(_grad);
//
//            pol(range((std::size_t)nz * (std::size_t)nx),
//                [=, height = view<space>(zs_height), mask = view<space>(zs_mask),
//                        _grad = view<space>(zs_grad)] __device__(std::size_t idx) mutable {
//                    using vec3f = zs::vec<float, 3>;
//
//                    auto id_z = idx / nx; // outer index
//                    auto id_x = idx % nx; // inner index
//
//                    // int idx = Pos2Idx(id_x, id_z, nx);
//                    int idx_xl, idx_xr, idx_zl, idx_zr, scale = 0;
//
//                    if (id_x == 0) {
//                        idx_xl = idx;
//                        idx_xr = Pos2Idx(id_x + 1, id_z, nx);
//                        scale = 1;
//                    } else if (id_x == nx - 1) {
//                        idx_xl = Pos2Idx(id_x - 1, id_z, nx);
//                        idx_xr = idx;
//                        scale = 1;
//                    } else {
//                        idx_xl = Pos2Idx(id_x - 1, id_z, nx);
//                        idx_xr = Pos2Idx(id_x + 1, id_z, nx);
//                        scale = 2;
//                    }
//
//                    if (id_z == 0) {
//                        idx_zl = idx;
//                        idx_zr = Pos2Idx(id_x, id_z + 1, nx);
//                        scale = 1;
//                    } else if (id_x == nz - 1) {
//                        idx_zl = Pos2Idx(id_x, id_z - 1, nx);
//                        idx_zr = idx;
//                        scale = 1;
//                    } else {
//                        idx_zl = Pos2Idx(id_x, id_z - 1, nx);
//                        idx_zr = Pos2Idx(id_x, id_z + 1, nx);
//                        scale = 2;
//                    }
//
//                    _grad[idx][0] = (height[idx_xr] - height[idx_xl]) / (scale * cellSize);
//                    _grad[idx][2] = (height[idx_zr] - height[idx_zl]) / (scale * cellSize);
//
//                    vec3f dx = zs::normalizeSafe(vec3f(1, 0, _grad[idx][0]));
//                    vec3f dy = zs::normalizeSafe(vec3f(0, 1, _grad[idx][2]));
//                    vec3f n = zs::normalizeSafe(dx.cross(dy));
//
//                    mask[idx] = 1;
//                    if (!useSlope && !useDir && !useHeight) // &&
//                        //                    //!useCurvature &&
//                        //                    //!useOcclusion)
//                    {
//                        mask[idx] = 0;
//                    }
//
//                    if (useSlope) {
//                        float slope = 180 * zs::acos(n[2]) / M_PI;
//                        slope = fit(slope, minSlope, maxSlope, 0, 1);
//                        slope = chramp(slope);
//                        mask[idx] *= slope;
//                    }
//
//                    if (useDir) {
//                        float direction = 180 * zs::atan2(n[0], n[1]) / M_PI;
//                        direction -= goalAngle;
//                        direction -= 360 * zs::floor(direction / 360); // Get in range -180 to 180
//                        direction -= 180;
//                        direction = fit(direction, -angleSpread, angleSpread, 0, 1);
//                        direction = chramp(direction);
//                        mask[idx] *= direction;
//                    }
//
//                    if (useHeight) {
//                        float h = fit(height[idx], minHeight, maxHeight, 0, 1);
//                        mask[idx] *= chramp(h);
//                    }
//                });
//
//            /// @brief  write back to host-side attribute
//            retrieve_device_vector(mask, zs_mask);
//            retrieve_device_vector(_grad, zs_grad);
//
//            set_output("HeightField", std::move(terrain));
//        }
//    };
//    ZENDEFNODE(zs_HF_maskByFeature, {/* inputs: */ {
//            "HeightField",
//            {"string", "height_layer", "height"},
//            {"string", "mask_layer", "mask"},
//            {"int", "smooth_radius", "1"},
//            {"bool", "use_slope", "0"},
//            {"float", "min_slopeangle", "0"},
//            {"float", "max_slopeangle", "90"},
//            //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//            {"bool", "use_direction", "0"},
//            {"float", "goal_angle", "0"},
//            {"float", "angle_spread", "30"},
//            //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//            {"bool", "use_height", "0"},
//            {"float", "min_height", "0"},
//            {"float", "max_height", "1"},
//        },
//        /* outputs: */
//        {
//            "HeightField",
//        },
//        /* params: */
//        {},
//        /* category: */
//        {
//            "erode",
//        }});



    ////////////////////////////////////
    ///////// ZJH //////////////////////
    ////////////////////////////////////
    struct zs_tumble_material_erosion : public INode {
        void apply() override {
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<ZenoParticles>("zs_HeightField");
            auto &pars = terrain->getParticles();

            size_t nx, nz;
            auto &ud = static_cast<IObject *>(terrain.get())->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->prim->verts;
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();
            auto cut_angle = get_input<NumericObject>("cutangle")->get<float>();
            auto global_erosionrate = get_input<NumericObject>("global_erosionrate")->get<float>();
            auto erosionrate = get_input<NumericObject>("erosionrate")->get<float>();
            auto erodability = get_input<NumericObject>("erodability")->get<float>();
            auto removalrate = get_input<NumericObject>("removalrate")->get<float>();
            auto maxdepth = get_input<NumericObject>("maxdepth")->get<float>();

            std::uniform_real_distribution<float> distr(0.0, 1.0); // 设置随机分布
            auto seed = get_input<NumericObject>("seed")->get<float>();

            auto iterations = get_input<NumericObject>("iterations")->get<int>(); // 外部迭代总次数      10
            auto iter = get_input<NumericObject>("iter")->get<int>();             // 外部迭代当前次数    1~10
            auto i = get_input<NumericObject>("i")->get<int>();                   // 内部迭代当前次数    0~7
            auto openborder = get_input<NumericObject>("openborder")->get<int>(); // 获取边界标记

            auto perm = get_input<ListObject>("perm")->get2<int>();
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            int iterseed = iter * 134775813;
            int color = perm[i];
            int p_dirs_0 = p_dirs[0];
            int p_dirs_1 = p_dirs[1];
            int x_dirs_0 = x_dirs[0];
            int x_dirs_1 = x_dirs[1];

            // 初始化地形遮罩
            auto erodabilitymask_name = get_input2<std::string>("erodability_mask_layer");
            auto removalratemask_name = get_input2<std::string>("removalrate_mask_layer");
            auto cutanglemask_name = get_input2<std::string>("cutangle_mask_layer");
            auto gridbiasmask_name = get_input2<std::string>("gridbias_mask_layer");
            if (!terrain->prim->verts.has_attr(erodabilitymask_name) ||
                !terrain->prim->verts.has_attr(removalratemask_name) ||
                !terrain->prim->verts.has_attr(cutanglemask_name) ||
                !terrain->prim->verts.has_attr(gridbiasmask_name)) {
                zeno::log_error("Node [zs_tumble_material_erosion], no such data layer named '{}' or '{}' or '{}' or '{}'.",
                                erodabilitymask_name, removalratemask_name, cutanglemask_name, gridbiasmask_name);
            }
            auto _erodabilitymask = pars.begin(erodabilitymask_name);
            auto _removalratemask = pars.begin(removalratemask_name);
            auto _cutanglemask = pars.begin(cutanglemask_name);
            auto _gridbiasmask = pars.begin(gridbiasmask_name);

            // 初始化地形数据
            if (!terrain->prim->verts.has_attr("_height") || !terrain->prim->verts.has_attr("_debris") ||
                !terrain->prim->verts.has_attr("_temp_height") || !terrain->prim->verts.has_attr("_temp_debris")) {
                zeno::log_error("Node [zs_tumble_material_erosion], no such data layer named '{}' or '{}' or '{}' or '{}'.",
                                "_height", "_debris", "_temp_height", "_temp_debris");
            }
            auto _height = pars.begin("_height");
            auto _debris = pars.begin("_debris");
            auto _temp_height = pars.begin("_temp_height");
            auto _temp_debris = pars.begin("_temp_debris");

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////

            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();

            pol( range(nx * nz), [=] __device__ (size_t id) mutable {
                const auto id_z = id / nx; // outer index
                const auto id_x = id % nx; // inner index

                const int idx = Pos2Idx(id_x, id_z, nx);
//                _temp_height[idx] = _height[idx];
//                _temp_debris[idx] = _debris[idx];

                int is_red = ((id_z & 1) == 1) && (color == 1);
                int is_green = ((id_x & 1) == 1) && (color == 2);
                int is_blue = ((id_z & 1) == 0) && (color == 3);
                int is_yellow = ((id_x & 1) == 0) && (color == 4);
                int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                int dxs[] = { 0, p_dirs_0, 0, p_dirs_0, x_dirs_0, x_dirs_1, x_dirs_0, x_dirs_1 };
                int dzs[] = { p_dirs_1, 0, p_dirs_1, 0, x_dirs_0,-x_dirs_1, x_dirs_0,-x_dirs_1 };
                if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                    int dx = dxs[color - 1];
                    int dz = dzs[color - 1];
                    int bound_x = nx;
                    int bound_z = nz;
                    int clamp_x = bound_x - 1;
                    int clamp_z = bound_z - 1;

                    float i_debris = _temp_debris[idx];
                    float i_height = _temp_height[idx];

                    const int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                    const int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                    int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);
                    if (validsource) {
                        validsource = validsource || !openborder;
                        int j_idx = Pos2Idx(samplex, samplez, nx);
                        float j_debris = validsource ? _temp_debris[j_idx] : 0.0f;
                        float j_height = _temp_height[j_idx];

                        int cidx = 0;
                        int cidz = 0;

                        float c_height = 0.0f;
                        float c_debris = 0.0f;
                        float n_debris = 0.0f;

                        int c_idx = 0;
                        int n_idx = 0;

                        int dx_check = 0;
                        int dz_check = 0;

                        float h_diff = 0.0f;

                        if ((j_height - i_height) > 0.0f)
                        {
                            cidx = samplex;
                            cidz = samplez;

                            c_height = j_height;
                            c_debris = j_debris;
                            n_debris = i_debris;

                            c_idx = j_idx;
                            n_idx = idx;

                            dx_check = -dx;
                            dz_check = -dz;

                            h_diff = j_height - i_height;
                        }
                        else
                        {
                            cidx = id_x;
                            cidz = id_z;

                            c_height = i_height;
                            c_debris = i_debris;
                            n_debris = j_debris;

                            c_idx = idx;
                            n_idx = j_idx;

                            dx_check = dx;
                            dz_check = dz;

                            h_diff = i_height - j_height;
                        }

                        float max_diff = 0.0f;
                        float dir_prob = 0.0f;
                        float c_gridbiasmask = _gridbiasmask[c_idx];
                        for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++)
                        {
                            for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++)
                            {
                                if (!tmp_dx && !tmp_dz)
                                    continue;

                                int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);
                                int tmp_validsource = (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));
                                tmp_validsource = tmp_validsource || !openborder;
                                int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                float n_height = _temp_height[tmp_j_idx];

                                float tmp_diff = n_height - (c_height);

                                //float _gridbias = clamp(gridbias, -1.0f, 1.0f);
                                float _gridbias = zs::clamp(gridbias * c_gridbiasmask, -1.0f, 1.0f);

                                if (tmp_dx && tmp_dz)
                                    tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                else
                                    tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                if (tmp_diff <= 0.0f)
                                {
                                    if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                        dir_prob = tmp_diff;
                                    if (tmp_diff < max_diff)
                                        max_diff = tmp_diff;
                                }
                            }
                        }
                        if (max_diff > 0.001f || max_diff < -0.001f)
                            dir_prob = dir_prob / max_diff;

                        int cond = 0;
                        if (dir_prob >= 1.0f)
                            cond = 1;
                        else
                        {
                            dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                            unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                            unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                            cond = randval < cutoff;
                        }

                        if (cond)
                        {
                            float abs_h_diff = h_diff < 0.0f ? -h_diff : h_diff;
                            //float _cut_angle = clamp(cut_angle, 0.0f, 90.0f);
                            float _cut_angle = zs::clamp(cut_angle * _cutanglemask[n_idx], 0.0f, 90.0f);
                            float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);
                            float height_removed = _cut_angle < 90.0f ? zs::tan(_cut_angle * M_PI / 180) * delta_x : 1e10f;
                            float height_diff = abs_h_diff - height_removed;
                            if (height_diff < 0.0f)
                                height_diff = 0.0f;
                            float prob = ((n_debris + c_debris) != 0.0f) ? zs::clamp((height_diff / (n_debris + c_debris)), 0.0f, 1.0f) : 1.0f;
                            unsigned int cutoff = (unsigned int)(prob * 4294967295.0);
                            unsigned int randval = erode_random(seed * 3.14, (idx + nx * nz) * 8 + color + iterseed);
                            int do_erode = randval < cutoff;

                            float height_removal_amt = do_erode * zs::clamp(global_erosionrate * erosionrate * erodability * _erodabilitymask[c_idx], 0.0f, height_diff);

                            _height[c_idx] -= height_removal_amt;

                            //float bedrock_density = 1.0f - (removalrate);
                            float bedrock_density = 1.0f - (removalrate * _removalratemask[c_idx]);
                            if (bedrock_density > 0.0f)
                            {
                                float newdebris = bedrock_density * height_removal_amt;
                                if (n_debris + newdebris > maxdepth)
                                {
                                    float rollback = n_debris + newdebris - maxdepth;
                                    rollback = zs::min(rollback, newdebris);
                                    _height[c_idx] += rollback / bedrock_density;
                                    newdebris -= rollback;
                                }
                                _debris[c_idx] += newdebris;
                            }
                        }
                    }
                }
            });

            set_output("zs_HeightField", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_tumble_material_erosion, {
        /* inputs: */
        {
            "zs_HeightField",
            {"ListObject", "perm"},
            {"ListObject", "p_dirs"},
            {"ListObject", "x_dirs"},

            {"float", "seed", "9676.79"},
            {"int", "iterations", "0"},
            {"int", "iter", "0"},
            {"int", "i", "0"},

            {"int", "openborder", "0"},
            {"float", "maxdepth", "5.0"},
            {"float", "global_erosionrate", "1.0"},
            {"float", "erosionrate", "0.03"},

            {"float", "cutangle", "35"},
            {"string", "cutangle_mask_layer", "_cutangle_mask"},

            {"float", "erodability", "0.4"},
            {"string", "erodability_mask_layer", "_erodability_mask"},

            {"float", "removalrate", "0.7"},
            {"string", "removalrate_mask_layer", "_removalrate_mask"},

            {"float", "gridbias", "0.0"},
            {"string", "gridbias_mask_layer", "_gridbias_mask"},
        },
        /* outputs: */
        {
            "zs_HeightField",
        },
        /* params: */
        {},
        /* category: */
        {
            "erode",
        }});


    struct zs_tumble_material_v1 : public INode {
        void apply() override {
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<ZenoParticles>("zs_HeightField");
            auto &pars = terrain->getParticles();

            size_t nx, nz;
            auto &ud = static_cast<IObject *>(terrain.get())->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->prim->verts;
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto openborder = get_input<NumericObject>("openborder")->get<int>();
            auto repose_angle = get_input<NumericObject>("repose_angle")->get<float>();
            auto flow_rate = get_input<NumericObject>("flow_rate")->get<float>();
            auto height_factor = get_input<NumericObject>("height_factor")->get<float>();
            auto entrainmentrate = get_input<NumericObject>("entrainmentrate")->get<float>();

            // 初始化地形遮罩
            auto write_back_material_layer = get_input2<std::string>("write_back_material_layer");
            if (!terrain->prim->verts.has_attr(write_back_material_layer)) {
                zeno::log_error("Node [zs_tumble_material_v1], no such data layer named '{}'.",
                                write_back_material_layer);
            }
            auto write_back_material = pars.begin(write_back_material_layer);

            // 初始化地形数据
            if (!terrain->prim->verts.has_attr("_height") ||
                !terrain->prim->verts.has_attr("_material") ||
                !terrain->prim->verts.has_attr("flowdir")) {
                zeno::log_error("Node [zs_tumble_material_v1], no such data layer named '{}' or '{}' or '{}'.",
                                "height", "_material", "flowdir");
            }
            auto _height   = pars.begin("_height");
            auto _material = pars.begin("_material");
            auto flowName  = zs::SmallString("flowdir");
//            auto flowdir   = pars3.begin("flowdir");

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////

            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();

            pol( range(nx * nz), [=, pars3 = zs::proxy<space>({ "flowdir" }, pars)] __device__ (size_t id) mutable {
                auto id_z = id / nx; // outer index
                auto id_x = id % nx; // inner index

                int idx = Pos2Idx(id_x, id_z, nx);
//                printf("idx = %d", idx);
                int bound_x = nx;
                int bound_z = nz;
                int clamp_x = bound_x - 1;
                int clamp_z = bound_z - 1;

                // Validate parameters
                flow_rate = zs::clamp(flow_rate, 0.0f, 1.0f);
                repose_angle = zs::clamp(repose_angle, 0.0f, 90.0f);
                height_factor = zs::clamp(height_factor, 0.0f, 1.0f);

                // The maximum slope at which we stop slumping
                float static_diff = repose_angle < 90.0f ? zs::tan(repose_angle * M_PI / 180.0) * cellSize : 1e10f;

                // Initialize accumulation of flow
                float net_diff = 0.0f;
                float net_entrained = 0.0f;

                float net_diff_x = 0.0f;
                float net_diff_z = 0.0f;

                // Get the current height level
                float i_material = _material[idx];
//                printf("%d: %f", (int)id, i_material);
                float i_entrained = 0;
                float i_height = height_factor * _height[idx] + i_material + i_entrained;

                bool moved = false;

                // For each of the 8 neighbours, we get the difference in total
                // height and add to our flow values.
                for (int dz = -1; dz <= 1; dz++) {
                    for (int dx = -1; dx <= 1; dx++) {
                        if (!dx && !dz)
                            continue;

                        int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                        int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                        int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);
                        // If we have closed borders, pretend a valid source to create
                        // a streak condition
                        validsource = validsource || !openborder;
                        int j_idx = samplex + samplez * nx;
                        float j_material = validsource ? _material[j_idx] : 0.0f;
                        float j_entrained = 0;

                        float j_height = height_factor * _height[j_idx] + j_material + j_entrained;

                        float diff = j_height - i_height;

                        // Calculate the distance to this neighbour
                        float distance = (dx && dz) ? 1.4142136f : 1.0f;
                        // Cutoff at the repose angle
                        float static_cutoff = distance * static_diff;
                        diff = diff > 0.0f ? zs::max(diff - static_cutoff, 0.0f) : zs::min(diff + static_cutoff, 0.0f);

                        // Weight the difference by the inverted distance
                        diff = distance > 0.0f ? diff / distance : 0.0f;

                        // Clamp within the material levels of the voxels
                        diff = zs::clamp(diff, -i_material, j_material);

                        // Some percentage of the material flow will drag
                        // the entrained material instead.
                        float entrained_diff = diff * entrainmentrate;

                        // Clamp entrained diff by the entrained levels.
                        entrained_diff = zs::clamp(entrained_diff, -i_entrained, j_entrained);

                        // Flow uses total diff, including entrained material
                        net_diff_x += (float) dx * diff;
                        net_diff_z += (float) dz * diff;

                        // And reduce the material diff by the amount of entrained substance
                        // moved so total height updates as expected.
                        diff -= entrained_diff;

                        // Accumulate the diff
                        net_diff += diff;
                        net_entrained += entrained_diff;
                    }
                }

                // 0.17 is to keep us in the circle of stability
                float weight = flow_rate * 0.17;
                net_diff *= weight;
                net_entrained *= weight;

                // Negate the directional flow so that they are positive in their axis direction
                net_diff_x *= -weight;
                net_diff_z *= -weight;

                // Ensure diff cannot bring the material level negative
                net_diff = zs::max(net_diff, -i_material);
                net_entrained = zs::max(net_entrained, -i_entrained);

                // Update the material level
                write_back_material[idx] = i_material + net_diff;

                auto prev_dir = pars3.template pack<3>(flowName, idx);
                prev_dir[0] += net_diff_x;
                prev_dir[2] += net_diff_x;
                auto flowdir = pars3.template tuple<3>(flowName, idx) = prev_dir;
            });

            set_output("zs_HeightField", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_tumble_material_v1, {
        /* inputs: */
        {
            "zs_HeightField",
            {"string", "write_back_material_layer", "_write_back_material"},
            {"int", "openborder", "0"},
            {"float", "repose_angle", "15.0"},
            {"float", "flow_rate", "1.0"},
            {"float", "height_factor", "1.0"},
            {"float", "entrainmentrate", "0.0"},
        },
        /* outputs: */
        {
            "zs_HeightField",
        },
        /* params: */
        {},
        /* category: */
        {
            "erode",
        }});


    struct zs_tumble_material_v2 : public INode {
        void apply() override {
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<ZenoParticles>("zs_HeightField");
            auto &pars = terrain->getParticles();

            size_t nx, nz;
            auto &ud = static_cast<IObject *>(terrain.get())->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto& pos = terrain->prim->verts;
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();
            auto repose_angle = get_input<NumericObject>("repose_angle")->get<float>();
            auto quant_amt = get_input<NumericObject>("quant_amt")->get<float>();
            auto flow_rate = get_input<NumericObject>("flow_rate")->get<float>();

            std::uniform_real_distribution<float> distr(0.0, 1.0);
            auto seed = get_input<NumericObject>("seed")->get<float>();

            auto iterations = get_input<NumericObject>("iterations")->get<int>();
            auto iter = get_input<NumericObject>("iter")->get<int>();
            auto i = get_input<NumericObject>("i")->get<int>();
            auto openborder = get_input<NumericObject>("openborder")->get<int>();

            auto perm = get_input<ListObject>("perm")->get2<int>();
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            int iterseed = iter * 134775813;
            int color = perm[i];
            int p_dirs_0 = p_dirs[0];
            int p_dirs_1 = p_dirs[1];
            int x_dirs_0 = x_dirs[0];
            int x_dirs_1 = x_dirs[1];
//            printf("==============\n");
//            printf("i = %i, perm[i] = %i\n", i, perm[i]);
//            printf("-------\n");
//            printf("perm list start\n");
//            for(int t = 0; t < 8; t++)
//            {
//                printf("%i ", perm[t]);
//            }
//            printf("\nperm list end\n");
//            printf("==============\n");

            // 初始化地形遮罩
            auto reposeanglemask_name = get_input2<std::string>("reposeangle_mask_layer");
            auto gridbiasmask_name = get_input2<std::string>("gridbias_mask_layer");
            auto stablilitymask_name = get_input2<std::string>("stability_mask_layer");
            if (!terrain->prim->verts.has_attr(reposeanglemask_name) ||
                !terrain->prim->verts.has_attr(gridbiasmask_name) ||
                !terrain->prim->verts.has_attr(stablilitymask_name)) {
                zeno::log_error("Node [erode_tumble_material_v2], no such data layer named '{}' or '{}' or '{}'.",
                                reposeanglemask_name, gridbiasmask_name, stablilitymask_name);
            }
            auto _reposeanglemask = pars.begin(reposeanglemask_name);
            auto _gridbiasmask = pars.begin(gridbiasmask_name);
            auto _stabilitymask = pars.begin(stablilitymask_name);

            //  初始化地形数据
            if (!terrain->prim->verts.has_attr("_height") ||
                !terrain->prim->verts.has_attr("_material") ||
                !terrain->prim->verts.has_attr("_temp_material")) {
                zeno::log_error("Node [erode_tumble_material_v2], no such data layer named '{}' or '{}' or '{}'.",
                                "_height", "_material", "_temp_material");
            }
            auto _height           = pars.begin("_height");
            auto _material         = pars.begin("_material");
            auto _temp_material    = pars.begin("_temp_material");


            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();

            pol( range(nx * nz), [=] __device__ (size_t id) mutable {
                auto id_z = id / nx; // outer index
                auto id_x = id % nx; // inner index

                int idx = Pos2Idx(id_x, id_z, nx);
                //_temp_material[idx] = _material[idx];

                int is_red = ((id_z & 1) == 1) && (color == 1);
                int is_green = ((id_x & 1) == 1) && (color == 2);
                int is_blue = ((id_z & 1) == 0) && (color == 3);
                int is_yellow = ((id_x & 1) == 0) && (color == 4);
                int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                int dxs[] = { 0, p_dirs_0, 0, p_dirs_0, x_dirs_0, x_dirs_1, x_dirs_0, x_dirs_1 };
                int dzs[] = { p_dirs_1, 0, p_dirs_1, 0, x_dirs_0,-x_dirs_1, x_dirs_0,-x_dirs_1 };
                if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                    int dx = dxs[color - 1];
                    int dz = dzs[color - 1];
                    int bound_x = nx;
                    int bound_z = nz;
                    int clamp_x = bound_x - 1;
                    int clamp_z = bound_z - 1;

                    flow_rate = zs::clamp(flow_rate, 0.0f, 1.0f);

                    float i_material = _temp_material[idx];
                    float i_height = _height[idx];

                    int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                    int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                    int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);

                    if (validsource) {
                        int same_node = !validsource;

                        validsource = validsource || !openborder;

                        int j_idx = Pos2Idx(samplex, samplez, nx);

                        float j_material = validsource ? _temp_material[j_idx] : 0.0f;
                        float j_height = _height[j_idx];


                        float _repose_angle = repose_angle;
                        _repose_angle *= _reposeanglemask[idx];
                        _repose_angle = zs::clamp(_repose_angle, 0.0f, 90.0f);
                        float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);
                        float static_diff = _repose_angle < 90.0f ? zs::tan(_repose_angle * M_PI / 180.0) * delta_x : 1e10f;
                        float m_diff = (j_height + j_material) - (i_height + i_material);
                        int cidx = 0;
                        int cidz = 0;

                        float c_height = 0.0f;
                        float c_material = 0.0f;
                        float n_material = 0.0f;

                        int c_idx = 0;
                        int n_idx = 0;

                        int dx_check = 0;
                        int dz_check = 0;

                        if (m_diff > 0.0f) {
                            cidx = samplex;
                            cidz = samplez;

                            c_height = j_height;
                            c_material = j_material;
                            n_material = i_material;

                            c_idx = j_idx;
                            n_idx = idx;

                            dx_check = -dx;
                            dz_check = -dz;
                        } else {
                            cidx = id_x;
                            cidz = id_z;

                            c_height = i_height;
                            c_material = i_material;
                            n_material = j_material;

                            c_idx = idx;
                            n_idx = j_idx;

                            dx_check = dx;
                            dz_check = dz;
                        }

                        float sum_diffs[] = { 0.0f, 0.0f };
                        float dir_probs[] = { 0.0f, 0.0f };
                        float dir_prob = 0.0f;
                        float c_gridbiasmask = _gridbiasmask[c_idx];
                        for (int diff_idx = 0; diff_idx < 2; diff_idx++)
                        {
                            for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++)
                            {
                                for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++)
                                {
                                    if (!tmp_dx && !tmp_dz)
                                        continue;

                                    int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                    int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);
                                    int tmp_validsource = (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));
                                    tmp_validsource = tmp_validsource || !openborder;
                                    int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                    float n_material = tmp_validsource ? _temp_material[tmp_j_idx] : 0.0f;
                                    float n_height = _height[tmp_j_idx];
                                    float tmp_h_diff = n_height - (c_height);
                                    float tmp_m_diff = (n_height + n_material) - (c_height + c_material);
                                    float tmp_diff = diff_idx == 0 ? tmp_h_diff : tmp_m_diff;
                                    float _gridbias = gridbias;
                                    _gridbias *= c_gridbiasmask;
                                    _gridbias = zs::clamp(_gridbias, -1.0f, 1.0f);

                                    if (tmp_dx && tmp_dz)
                                        tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                    else
                                        tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                    if (tmp_diff <= 0.0f)
                                    {
                                        if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                            dir_probs[diff_idx] = tmp_diff;

                                        if (diff_idx && dir_prob > tmp_diff)
                                            dir_prob = tmp_diff;

                                        sum_diffs[diff_idx] += tmp_diff;
                                    }
                                }
                            }

                            if (diff_idx && (dir_prob > 0.001f || dir_prob < -0.001f))
                                dir_prob = dir_probs[diff_idx] / dir_prob;

                            if (sum_diffs[diff_idx] > 0.001f || sum_diffs[diff_idx] < -0.001f)
                                dir_probs[diff_idx] = dir_probs[diff_idx] / sum_diffs[diff_idx];
                        }

                        float movable_mat = (m_diff < 0.0f) ? -m_diff : m_diff;
                        float stability_val = 0.0f;
                        stability_val = zs::clamp(_stabilitymask[c_idx], 0.0f, 1.0f);

                        if (stability_val > 0.01f)
                            movable_mat = zs::clamp(movable_mat * (1.0f - stability_val) * 0.5f, 0.0f, c_material);
                        else
                            movable_mat = zs::clamp((movable_mat - static_diff) * 0.5f, 0.0f, c_material);

                        float l_rat = dir_probs[1];
                        if (quant_amt > 0.001)
                            movable_mat = zs::clamp(quant_amt * zs::ceil<float, space>((movable_mat * l_rat) / quant_amt), 0.0f, c_material);
                        else
                            movable_mat *= l_rat;

                        float diff = (m_diff > 0.0f) ? movable_mat : -movable_mat;

                        int cond = 0;
                        if (dir_prob >= 1.0f)
                            cond = 1;
                        else
                        {
                            dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                            unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                            unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                            cond = randval < cutoff;
                        }

                        if (!cond || same_node)
                            diff = 0.0f;

                        diff *= flow_rate;
                        float abs_diff = (diff < 0.0f) ? -diff : diff;
                        _material[c_idx] = c_material - abs_diff;
                        _material[n_idx] = n_material + abs_diff;
                    }
                }
            });

            set_output("zs_HeightField", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_tumble_material_v2, {
        /* inputs: */
        {
            "zs_HeightField",

            {"string", "stability_mask_layer", "_stability_mask"},      //~~~~~mask
            {"ListObject", "perm"},
            {"ListObject", "p_dirs"},
            {"ListObject", "x_dirs"},

            {"float", "seed", "15231.3"},
            {"int", "iterations", "0"},
            {"int", "iter", "0"},
            {"int", "i", "0"},

            {"int", "openborder", "0"},

            {"float", "gridbias", "0.0"},
            {"string", "gridbias_mask_layer", "_gridbias_mask"},        //~~~~~mask

            // 崩塌流淌相关
            {"float", "repose_angle", "15.0"},
            {"string", "reposeangle_mask_layer", "_reposeangle_mask"},  //~~~~~mask

            {"float", "quant_amt", "0.25"},
            {"float", "flow_rate", "1.0"},
        },
        /* outputs: */
        {
            "zs_HeightField",
        },
        /* params: */
        {},
        /* category: */
        {
            "erode",
        }});


    struct zs_tumble_material_v3 : public INode {
        void apply() override {
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<ZenoParticles>("zs_HeightField");
            auto &pars = terrain->getParticles();

            size_t nx, nz;
            auto &ud = static_cast<IObject *>(terrain.get())->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->prim->verts;
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();
            auto repose_angle = get_input<NumericObject>("repose_angle")->get<float>();
            auto quant_amt = get_input<NumericObject>("quant_amt")->get<float>();
            auto flow_rate = get_input<NumericObject>("flow_rate")->get<float>();

            std::uniform_real_distribution<float> distr(0.0, 1.0);
            auto seed = get_input<NumericObject>("seed")->get<float>();

            auto iterations = get_input<NumericObject>("iterations")->get<int>();
            auto iter = get_input<NumericObject>("iter")->get<int>();
            auto i = get_input<NumericObject>("i")->get<int>();
            auto openborder = get_input<NumericObject>("openborder")->get<int>();

            auto perm = get_input<ListObject>("perm")->get2<int>();
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            int iterseed = iter * 134775813;
            int color = perm[i];
            int p_dirs_0 = p_dirs[0];
            int p_dirs_1 = p_dirs[1];
            int x_dirs_0 = x_dirs[0];
            int x_dirs_1 = x_dirs[1];

            // 初始化地形遮罩
            auto reposeanglemask_name = get_input2<std::string>("reposeangle_mask_layer");
            auto gridbiasmask_name = get_input2<std::string>("gridbias_mask_layer");
            auto stablilitymask_name = get_input2<std::string>("stability_mask_layer");
            if (!terrain->prim->verts.has_attr(reposeanglemask_name) ||
                !terrain->prim->verts.has_attr(gridbiasmask_name) ||
                !terrain->prim->verts.has_attr(stablilitymask_name)) {
                zeno::log_error("Node [erode_tumble_material_v3], no such data layer named '{}' or '{}' or '{}'.",
                                reposeanglemask_name, gridbiasmask_name, stablilitymask_name);
            }
            auto _reposeanglemask = pars.begin(reposeanglemask_name);
            auto _gridbiasmask = pars.begin(gridbiasmask_name);
            auto _stabilitymask = pars.begin(stablilitymask_name);

            // 初始化地形数据
            if (!terrain->prim->verts.has_attr("_height") ||
                !terrain->prim->verts.has_attr("_material") ||
                !terrain->prim->verts.has_attr("_temp_material") ||
                !terrain->prim->verts.has_attr("flowdir")) {
                zeno::log_error("Node [erode_tumble_material_v3], no such data layer named '{}' or '{}' or '{}' or '{}'.",
                                "_height", "_material", "_temp_material", "flowdir");
            }
            auto _height            = pars.begin("_height");
            auto _material          = pars.begin("_material");
            auto _temp_material     = pars.begin("_temp_material");
            auto flowName           = zs::SmallString("flowdir");


            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////

            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();

            pol(range(nx * nz), [=, pars3 = zs::proxy<space>({ "flowdir" }, pars)] __device__ (size_t id) mutable {
                auto id_z = id / nx; // outer index
                auto id_x = id % nx; // inner index

                int idx = Pos2Idx(id_x, id_z, nx);

                int is_red = ((id_z & 1) == 1) && (color == 1);
                int is_green = ((id_x & 1) == 1) && (color == 2);
                int is_blue = ((id_z & 1) == 0) && (color == 3);
                int is_yellow = ((id_x & 1) == 0) && (color == 4);
                int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                int dxs[] = { 0, p_dirs_0, 0, p_dirs_0, x_dirs_0, x_dirs_1, x_dirs_0, x_dirs_1 };
                int dzs[] = { p_dirs_1, 0, p_dirs_1, 0, x_dirs_0,-x_dirs_1, x_dirs_0,-x_dirs_1 };
                if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                    int dx = dxs[color - 1];
                    int dz = dzs[color - 1];
                    int bound_x = nx;
                    int bound_z = nz;
                    int clamp_x = bound_x - 1;
                    int clamp_z = bound_z - 1;

                    flow_rate = zs::clamp(flow_rate, 0.0f, 1.0f);

                    // CALC_FLOW
                    float diff_x = 0.0f;
                    float diff_z = 0.0f;

                    float i_material = _temp_material[idx];
                    float i_height = _height[idx];

                    int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                    int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                    int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);

                    if (validsource)
                    {
                        int same_node = !validsource;

                        validsource = validsource || !openborder;

                        int j_idx = Pos2Idx(samplex, samplez, nx);

                        float j_material = validsource ? _temp_material[j_idx] : 0.0f;
                        float j_height = _height[j_idx];

                        float _repose_angle = repose_angle;
                        _repose_angle *= _reposeanglemask[idx];
                        _repose_angle = zs::clamp(_repose_angle, 0.0f, 90.0f);
                        float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);

                        float static_diff = _repose_angle < 90.0f ? zs::tan(_repose_angle * M_PI / 180.0) * delta_x : 1e10f;

                        float m_diff = (j_height + j_material) - (i_height + i_material);

                        int cidx = 0;
                        int cidz = 0;

                        float c_height = 0.0f;
                        float c_material = 0.0f;
                        float n_material = 0.0f;

                        int c_idx = 0;
                        int n_idx = 0;

                        int dx_check = 0;
                        int dz_check = 0;

                        if (m_diff > 0.0f) {
                            cidx = samplex;
                            cidz = samplez;

                            c_height = j_height;
                            c_material = j_material;
                            n_material = i_material;

                            c_idx = j_idx;
                            n_idx = idx;

                            dx_check = -dx;
                            dz_check = -dz;
                        } else {
                            cidx = id_x;
                            cidz = id_z;

                            c_height = i_height;
                            c_material = i_material;
                            n_material = j_material;

                            c_idx = idx;
                            n_idx = j_idx;

                            dx_check = dx;
                            dz_check = dz;
                        }

                        float sum_diffs[] = {0.0f, 0.0f};
                        float dir_probs[] = {0.0f, 0.0f};
                        float dir_prob = 0.0f;
                        float c_gridbiasmask = _gridbiasmask[c_idx];

                        for (int diff_idx = 0; diff_idx < 2; diff_idx++) {
                            for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++) {
                                for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++) {
                                    if (!tmp_dx && !tmp_dz)
                                        continue;

                                    int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                    int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);
                                    int tmp_validsource = (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));

                                    tmp_validsource = tmp_validsource || !openborder;
                                    int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                    float n_material = tmp_validsource ? _temp_material[tmp_j_idx] : 0.0f;
                                    float n_height = _height[tmp_j_idx];
                                    float tmp_h_diff = n_height - (c_height);
                                    float tmp_m_diff = (n_height + n_material) - (c_height + c_material);
                                    float tmp_diff = diff_idx == 0 ? tmp_h_diff : tmp_m_diff;
                                    float _gridbias = gridbias;
                                    _gridbias *= c_gridbiasmask;
                                    _gridbias = zs::clamp(_gridbias, -1.0f, 1.0f);

                                    if (tmp_dx && tmp_dz)
                                        tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                    else
                                        tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                    if (tmp_diff <= 0.0f)
                                    {
                                        if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                            dir_probs[diff_idx] = tmp_diff;

                                        if (diff_idx && dir_prob > tmp_diff)
                                            dir_prob = tmp_diff;

                                        sum_diffs[diff_idx] += tmp_diff;
                                    }
                                }
                            }

                            if (diff_idx && (dir_prob > 0.001f || dir_prob < -0.001f))
                                dir_prob = dir_probs[diff_idx] / dir_prob;

                            if (sum_diffs[diff_idx] > 0.001f || sum_diffs[diff_idx] < -0.001f)
                                dir_probs[diff_idx] = dir_probs[diff_idx] / sum_diffs[diff_idx];
                        }

                        float movable_mat = (m_diff < 0.0f) ? -m_diff : m_diff;
                        float stability_val = 0.0f;
                        stability_val = zs::clamp(_stabilitymask[c_idx], 0.0f, 1.0f);

                        if (stability_val > 0.01f)
                            movable_mat = zs::clamp(movable_mat * (1.0f - stability_val) * 0.5f, 0.0f, c_material);
                        else
                            movable_mat = zs::clamp((movable_mat - static_diff) * 0.5f, 0.0f, c_material);

                        float l_rat = dir_probs[1];
                        if (quant_amt > 0.001)
                            movable_mat = zs::clamp(quant_amt * zs::ceil((movable_mat * l_rat) / quant_amt), 0.0f, c_material);
                        else
                            movable_mat *= l_rat;

                        float diff = (m_diff > 0.0f) ? movable_mat : -movable_mat;

                        int cond = 0;
                        if (dir_prob >= 1.0f)
                            cond = 1;
                        else {
                            dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                            unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                            unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                            cond = randval < cutoff;
                        }

                        if (!cond || same_node)
                            diff = 0.0f;

                        diff *= flow_rate;

                        // CALC_FLOW
                        diff_x += (float)dx * diff;
                        diff_z += (float)dz * diff;
                        diff_x *= -1.0f;
                        diff_z *= -1.0f;

                        float abs_diff = (diff < 0.0f) ? -diff : diff;
                        _material[c_idx] = c_material - abs_diff;
                        _material[n_idx] = n_material + abs_diff;

                        auto prev_dir = pars3.template pack<3>(flowName, idx);
                        float abs_c_x = prev_dir[0];
                        abs_c_x = (abs_c_x < 0.0f) ? -abs_c_x : abs_c_x;
                        float abs_c_z = prev_dir[2];
                        abs_c_z = (abs_c_z < 0.0f) ? -abs_c_z : abs_c_z;
                        prev_dir[0] += diff_x * 1.0f / (1.0f + abs_c_x);
                        prev_dir[2] += diff_z * 1.0f / (1.0f + abs_c_z);
                        pars3.template tuple<3>(flowName, idx) = prev_dir;
                    }
                }
            });

            set_output("zs_HeightField", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_tumble_material_v3, {
        /* inputs: */
        {
            "zs_HeightField",

            {"string", "stability_mask_layer", "_stability_mask"},  //~~~~~mask
            {"ListObject", "perm"},
            {"ListObject", "p_dirs"},
            {"ListObject", "x_dirs"},

            {"float", "seed", "15231.3"},
            {"int", "iterations", "0"},
            {"int", "iter", "0"},
            {"int", "i", "0"},

            {"int", "openborder", "0"},

            {"float", "gridbias", "0.0"},
            {"string", "gridbias_mask_layer", "_gridbias_mask"},        //~~~~~mask

            // 崩塌流淌相关
            {"float", "repose_angle", "0.0"},
            {"string", "reposeangle_mask_layer", "_reposeangle_mask"},  //~~~~~mask

            {"float", "quant_amt", "0.0"},
            {"float", "flow_rate", "1.0"},
        },
        /* outputs: */
        {
            "zs_HeightField",
        },
        /* params: */
        {
        },
        /* category: */
        {
            "erode",
        }});


    struct zs_tumble_material_v4 : public INode {
        void apply() override {
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 初始化
            ////////////////////////////////////////////////////////////////////////////////////////

            // 初始化网格
            auto terrain = get_input<ZenoParticles>("zs_HeightField");
            auto &pars = terrain->getParticles();

            size_t nx, nz;
            auto &ud = static_cast<IObject *>(terrain.get())->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->prim->verts;
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

            // 获取面板参数
            // 侵蚀主参数
            auto global_erosionrate = get_input<NumericObject>("global_erosionrate")->get<float>(); // 1 全局侵蚀率
            auto erodability = get_input<NumericObject>("erodability")->get<float>();               // 1.0 侵蚀能力
            auto erosionrate = get_input<NumericObject>("erosionrate")->get<float>();               // 0.4 侵蚀率
            auto bank_angle = get_input<NumericObject>("bank_angle")->get<float>(); // 70.0 河堤侵蚀角度
            auto seed = get_input<NumericObject>("seed")->get<float>();             // 12.34

            // 高级参数
            auto removalrate = get_input<NumericObject>("removalrate")->get<float>(); // 0.0 风化率/水吸收率
            auto max_debris_depth = get_input<NumericObject>("max_debris_depth")->get<float>(); // 5	碎屑最大深度
            auto gridbias = get_input<NumericObject>("gridbias")->get<float>();                 // 0.0

            // 侵蚀能力调整
            auto max_erodability_iteration = get_input<NumericObject>("max_erodability_iteration")->get<int>();     // 5
            auto initial_erodability_factor = get_input<NumericObject>("initial_erodability_factor")->get<float>(); // 0.5
            auto slope_contribution_factor = get_input<NumericObject>("slope_contribution_factor")->get<float>();   // 0.8

            // 河床参数
            auto bed_erosionrate_factor =
                    get_input<NumericObject>("bed_erosionrate_factor")->get<float>();           // 1 河床侵蚀率因子
            auto depositionrate = get_input<NumericObject>("depositionrate")->get<float>(); // 0.01 沉积率
            auto sedimentcap = get_input<NumericObject>("sedimentcap")
                    ->get<float>(); // 10.0 高度差转变为沉积物的比率 / 泥沙容量，每单位流动水可携带的泥沙量

            // 河堤参数
            auto bank_erosionrate_factor =
                    get_input<NumericObject>("bank_erosionrate_factor")->get<float>(); // 1.0 河堤侵蚀率因子
            auto max_bank_bed_ratio = get_input<NumericObject>("max_bank_bed_ratio")
                    ->get<float>(); // 0.5 The maximum of bank to bed water column height ratio
            // 高于这个比值的河岸将不会在侵蚀中被视为河岸，会停止侵蚀
            // 河流控制
            auto quant_amt = get_input<NumericObject>("quant_amt")->get<float>(); // 0.05 流量维持率，越高流量越稳定
            auto iterations = get_input<NumericObject>("iterations")->get<int>(); // 流淌的总迭代次数

            //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
            std::uniform_real_distribution<float> distr(0.0, 1.0);
            auto iter = get_input<NumericObject>("iter")->get<int>();
            auto i = get_input<NumericObject>("i")->get<int>();
            auto openborder = get_input<NumericObject>("openborder")->get<int>();

            auto perm = get_input<ListObject>("perm")->get2<int>();
            auto p_dirs = get_input<ListObject>("p_dirs")->get2<int>();
            auto x_dirs = get_input<ListObject>("x_dirs")->get2<int>();

            int iterseed = iter * 134775813;
            int color = perm[i];
            int p_dirs_0 = p_dirs[0];
            int p_dirs_1 = p_dirs[1];
            int x_dirs_0 = x_dirs[0];
            int x_dirs_1 = x_dirs[1];

            // 初始化地形遮罩
            auto gridbiasmask_name = get_input2<std::string>("gridbias_mask_layer");
            auto erodabilitymask_name = get_input2<std::string>("erodability_mask_layer");
            auto bankanglemask_name = get_input2<std::string>("bankangle_mask_layer");
            auto removalratemask_name = get_input2<std::string>("removalrate_mask_layer");
            auto depositionratemask_name = get_input2<std::string>("depositionrate_mask_layer");
            if (!terrain->prim->verts.has_attr(gridbiasmask_name) ||
                !terrain->prim->verts.has_attr(erodabilitymask_name) ||
                !terrain->prim->verts.has_attr(bankanglemask_name) ||
                !terrain->prim->verts.has_attr(removalratemask_name) ||
                !terrain->prim->verts.has_attr(depositionratemask_name)) {
                zeno::log_error("Node [erode_tumble_material_v4], no such data layer named '{}' or '{}' or '{}' or '{}' or '{}'.",
                                gridbiasmask_name, erodabilitymask_name, bankanglemask_name, removalratemask_name, depositionratemask_name);
            }
            auto _gridbiasmask = pars.begin(gridbiasmask_name);
            auto _erodabilitymask = pars.begin(erodabilitymask_name);
            auto _bankanglemask = pars.begin(bankanglemask_name);
            auto _removalratemask = pars.begin(removalratemask_name);
            auto _depositionratemask = pars.begin(depositionratemask_name);

            // 初始化地形数据
            if (!terrain->prim->verts.has_attr("_height") || !terrain->prim->verts.has_attr("_temp_height") ||
                !terrain->prim->verts.has_attr("_material") || !terrain->prim->verts.has_attr("_temp_material") ||
                !terrain->prim->verts.has_attr("_debris") || !terrain->prim->verts.has_attr("_temp_debris") ||
                !terrain->prim->verts.has_attr("_sediment")) {
                zeno::log_error("Node [erode_tumble_material_v4], no such data layer named '{}' or '{}' or '{}' or '{}' or '{}' or '{}' or '{}'.",
                                "_height", "_temp_height", "_material", "_temp_material", "_debris", "_temp_debris", "_sediment");
            }
            auto _height = pars.begin("_height");
            auto _temp_height = pars.begin("_temp_height");
            auto _material = pars.begin("_material");
            auto _temp_material = pars.begin("_temp_material");
            auto _debris = pars.begin("_debris");
            auto _temp_debris = pars.begin("_temp_debris");
            auto _sediment = pars.begin("_sediment");

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            // 计算
            ////////////////////////////////////////////////////////////////////////////////////////

            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();

            pol( range(nx * nz), [=] __device__ (size_t id) mutable {
                auto id_z = id / nx; // outer index
                auto id_x = id % nx; // inner index

                int idx = Pos2Idx(id_x, id_z, nx);
//            _temp_height[idx] = _height[idx];
//            _temp_material[idx] = _material[idx];
//            _temp_debris[idx] = _debris[idx];

                int is_red = ((id_z & 1) == 1) && (color == 1);
                int is_green = ((id_x & 1) == 1) && (color == 2);
                int is_blue = ((id_z & 1) == 0) && (color == 3);
                int is_yellow = ((id_x & 1) == 0) && (color == 4);
                int is_x_turn_x = ((id_x & 1) == 1) && ((color == 5) || (color == 6));
                int is_x_turn_y = ((id_x & 1) == 0) && ((color == 7) || (color == 8));
                int dxs[] = { 0, p_dirs_0, 0, p_dirs_0, x_dirs_0, x_dirs_1, x_dirs_0, x_dirs_1 };
                int dzs[] = { p_dirs_1, 0, p_dirs_1, 0, x_dirs_0,-x_dirs_1, x_dirs_0,-x_dirs_1 };
                if (is_red || is_green || is_blue || is_yellow || is_x_turn_x || is_x_turn_y) {
                    int dx = dxs[color - 1];
                    int dz = dzs[color - 1];
                    int bound_x = nx;
                    int bound_z = nz;
                    int clamp_x = bound_x - 1;
                    int clamp_z = bound_z - 1;

                    float i_height = _temp_height[idx];
                    float i_material = _temp_material[idx];
                    float i_debris = _temp_debris[idx];
                    float i_sediment = _sediment[idx];

                    int samplex = zs::clamp(id_x + dx, 0, clamp_x);
                    int samplez = zs::clamp(id_z + dz, 0, clamp_z);
                    int validsource = (samplex == id_x + dx) && (samplez == id_z + dz);

                    if (validsource) {
                        validsource = validsource || !openborder;

                        int j_idx = Pos2Idx(samplex, samplez, nx);

                        float j_height = _temp_height[j_idx];
                        float j_material = validsource ? _temp_material[j_idx] : 0.0f;
                        float j_debris = validsource ? _temp_debris[j_idx] : 0.0f;

                        float j_sediment = validsource ? _sediment[j_idx] : 0.0f;
                        float m_diff = (j_height + j_debris + j_material) - (i_height + i_debris + i_material);
                        float delta_x = cellSize * (dx && dz ? 1.4142136f : 1.0f);

                        int cidx = 0;
                        int cidz = 0;

                        float c_height = 0.0f;

                        float c_material = 0.0f;
                        float n_material = 0.0f;

                        float c_sediment = 0.0f;
                        float n_sediment = 0.0f;

                        float c_debris = 0.0f;
                        float n_debris = 0.0f;

                        float h_diff = 0.0f;

                        int c_idx = 0;
                        int n_idx = 0;
                        int dx_check = 0;
                        int dz_check = 0;
                        int is_mh_diff_same_sign = 0;

                        if (m_diff > 0.0f) {
                            cidx = samplex;
                            cidz = samplez;

                            c_height = j_height;
                            c_material = j_material;
                            n_material = i_material;
                            c_sediment = j_sediment;
                            n_sediment = i_sediment;
                            c_debris = j_debris;
                            n_debris = i_debris;

                            c_idx = j_idx;
                            n_idx = idx;

                            dx_check = -dx;
                            dz_check = -dz;

                            h_diff = j_height + j_debris - (i_height + i_debris);
                            is_mh_diff_same_sign = (h_diff * m_diff) > 0.0f;
                        } else {
                            cidx = id_x;
                            cidz = id_z;

                            c_height = i_height;
                            c_material = i_material;
                            n_material = j_material;
                            c_sediment = i_sediment;
                            n_sediment = j_sediment;
                            c_debris = i_debris;
                            n_debris = j_debris;

                            c_idx = idx;
                            n_idx = j_idx;

                            dx_check = dx;
                            dz_check = dz;

                            h_diff = i_height + i_debris - (j_height + j_debris);
                            is_mh_diff_same_sign = (h_diff * m_diff) > 0.0f;
                        }
                        h_diff = (h_diff < 0.0f) ? -h_diff : h_diff;

                        float sum_diffs[] = { 0.0f, 0.0f };
                        float dir_probs[] = { 0.0f, 0.0f };
                        float dir_prob = 0.0f;
                        float c_gridbiasmask = _gridbiasmask[c_idx];

                        for (int diff_idx = 0; diff_idx < 2; diff_idx++) {
                            for (int tmp_dz = -1; tmp_dz <= 1; tmp_dz++) {
                                for (int tmp_dx = -1; tmp_dx <= 1; tmp_dx++) {
                                    if (!tmp_dx && !tmp_dz)
                                        continue;

                                    int tmp_samplex = zs::clamp(cidx + tmp_dx, 0, clamp_x);
                                    int tmp_samplez = zs::clamp(cidz + tmp_dz, 0, clamp_z);

                                    int tmp_validsource = (tmp_samplex == (cidx + tmp_dx)) && (tmp_samplez == (cidz + tmp_dz));
                                    tmp_validsource = tmp_validsource || !openborder;
                                    int tmp_j_idx = Pos2Idx(tmp_samplex, tmp_samplez, nx);

                                    float tmp_n_material = tmp_validsource ? _temp_material[tmp_j_idx] : 0.0f;
                                    float tmp_n_debris = tmp_validsource ? _temp_debris[tmp_j_idx] : 0.0f;

                                    float n_height = _temp_height[tmp_j_idx];
                                    float tmp_h_diff = n_height + tmp_n_debris - (c_height + c_debris);
                                    float tmp_m_diff = (n_height + tmp_n_debris + tmp_n_material) - (c_height + c_debris + c_material);
                                    float tmp_diff = diff_idx == 0 ? tmp_h_diff : tmp_m_diff;
                                    float _gridbias = zs::clamp(gridbias * c_gridbiasmask, -1.0f, 1.0f);

                                    if (tmp_dx && tmp_dz)
                                        tmp_diff *= zs::clamp(1.0f - _gridbias, 0.0f, 1.0f) / 1.4142136f;
                                    else
                                        tmp_diff *= zs::clamp(1.0f + _gridbias, 0.0f, 1.0f);

                                    if (tmp_diff <= 0.0f)
                                    {
                                        if ((dx_check == tmp_dx) && (dz_check == tmp_dz))
                                            dir_probs[diff_idx] = tmp_diff;

                                        if (diff_idx && (tmp_diff < dir_prob))
                                            dir_prob = tmp_diff;

                                        sum_diffs[diff_idx] += tmp_diff;
                                    }
                                }
                            }

                            if (diff_idx && (dir_prob > 0.001f || dir_prob < -0.001f))
                                dir_prob = dir_probs[diff_idx] / dir_prob;
                            else
                                dir_prob = 0.0f;

                            if (sum_diffs[diff_idx] > 0.001f || sum_diffs[diff_idx] < -0.001f)
                                dir_probs[diff_idx] = dir_probs[diff_idx] / sum_diffs[diff_idx];
                            else
                                dir_probs[diff_idx] = 0.0f;
                        }

                        float movable_mat = (m_diff < 0.0f) ? -m_diff : m_diff;
                        movable_mat = zs::clamp(movable_mat * 0.5f, 0.0f, c_material);
                        float l_rat = dir_probs[1];

                        if (quant_amt > 0.001)
                            movable_mat = zs::clamp(quant_amt * zs::ceil<float, space>((movable_mat * l_rat) / quant_amt), 0.0f, c_material);
                        else
                            movable_mat *= l_rat;

                        float diff = (m_diff > 0.0f) ? movable_mat : -movable_mat;

                        int cond = 0;
                        if (dir_prob >= 1.0f)
                            cond = 1;
                        else {
                            dir_prob = dir_prob * dir_prob * dir_prob * dir_prob;
                            unsigned int cutoff = (unsigned int)(dir_prob * 4294967295.0);
                            unsigned int randval = erode_random(seed, (idx + nx * nz) * 8 + color + iterseed);
                            cond = randval < cutoff;
                        }

                        if (!cond)
                            diff = 0.0f;

                        float slope_cont = (delta_x > 0.0f) ? (h_diff / delta_x) : 0.0f;
                        float kd_factor = zs::clamp((1 / (1 + (slope_contribution_factor * slope_cont))), 0.0f, 1.0f);
                        float norm_iter = zs::clamp(((float)iter / (float)max_erodability_iteration), 0.0f, 1.0f);
                        float ks_factor = zs::clamp((1 - (slope_contribution_factor * zs::exp<float, space>(-slope_cont))) * zs::sqrt<float, space>(dir_probs[0]) *
                                                    (initial_erodability_factor + ((1.0f - initial_erodability_factor) * zs::sqrt<float, space>(norm_iter))),
                                                    0.0f, 1.0f);

                        float c_ks = global_erosionrate * erosionrate * erodability * ks_factor;
                        float n_kd = depositionrate * kd_factor;
                        c_ks *= _erodabilitymask[c_idx];
                        n_kd *= _depositionratemask[n_idx];
                        n_kd = zs::clamp(n_kd, 0.0f, 1.0f);

                        float _removalrate = removalrate;
                        _removalrate *= _removalratemask[n_idx];

                        float bedrock_density = 1.0f - _removalrate;
                        float abs_diff = (diff < 0.0f) ? -diff : diff;
                        float sediment_limit = sedimentcap * abs_diff;
                        float ent_check_diff = sediment_limit - c_sediment;

                        if (ent_check_diff > 0.0f) {
                            float dissolve_amt = c_ks * bed_erosionrate_factor * abs_diff;
                            float dissolved_debris = zs::min(c_debris, dissolve_amt);
                            _debris[c_idx] -= dissolved_debris;
                            _height[c_idx] -= (dissolve_amt - dissolved_debris);
                            _sediment[c_idx] -= c_sediment / 2;
                            if (bedrock_density > 0.0f) {
                                float newsediment = c_sediment / 2 + (dissolve_amt * bedrock_density);
                                if (n_sediment + newsediment > max_debris_depth) {
                                    float rollback = n_sediment + newsediment - max_debris_depth;
                                    rollback = zs::min(rollback, newsediment);
                                    _height[c_idx] += rollback / bedrock_density;
                                    newsediment -= rollback;
                                }
                                _sediment[n_idx] += newsediment;
                            }
                        } else {
                            float c_kd = depositionrate * kd_factor;
                            c_kd *= _depositionratemask[c_idx];
                            c_kd = zs::clamp(c_kd, 0.0f, 1.0f);
                            {
                                _debris[c_idx] += (c_kd * -ent_check_diff);
                                _sediment[c_idx] = (1 - c_kd) * -ent_check_diff;

                                n_sediment += sediment_limit;
                                _debris[n_idx] += (n_kd * n_sediment);
                                _sediment[n_idx] = (1 - n_kd) * n_sediment;
                            }

                            int b_idx = 0;
                            int r_idx = 0;
                            float b_material = 0.0f;
                            float r_material = 0.0f;
                            float b_debris = 0.0f;
                            float r_debris = 0.0f;
                            float r_sediment = 0.0f;

                            if (is_mh_diff_same_sign) {
                                b_idx = c_idx;
                                r_idx = n_idx;

                                b_material = c_material;
                                r_material = n_material;

                                b_debris = c_debris;
                                r_debris = n_debris;

                                r_sediment = n_sediment;
                            } else {
                                b_idx = n_idx;
                                r_idx = c_idx;

                                b_material = n_material;
                                r_material = c_material;

                                b_debris = n_debris;
                                r_debris = c_debris;

                                r_sediment = c_sediment;
                            }

                            float erosion_per_unit_water = global_erosionrate * erosionrate * bed_erosionrate_factor * erodability * ks_factor;
                            erosion_per_unit_water *= _erodabilitymask[r_idx];
                            if (r_material != 0.0f &&
                                (b_material / r_material) < max_bank_bed_ratio &&
                                r_sediment > (erosion_per_unit_water * max_bank_bed_ratio))
                            {
                                float height_to_erode = global_erosionrate * erosionrate * bank_erosionrate_factor * erodability * ks_factor;
                                height_to_erode *= _erodabilitymask[b_idx];
                                float _bank_angle = bank_angle;
                                _bank_angle *= _bankanglemask[b_idx];
                                _bank_angle = zs::clamp(_bank_angle, 0.0f, 90.0f);
                                float safe_diff = _bank_angle < 90.0f ? zs::tan(_bank_angle * M_PI / 180.0) * delta_x : 1e10f;
                                float target_height_removal = (h_diff - safe_diff) < 0.0f ? 0.0f : h_diff - safe_diff;

                                float dissolve_amt = zs::clamp(height_to_erode, 0.0f, target_height_removal);
                                float dissolved_debris = zs::min(b_debris, dissolve_amt);

                                _debris[b_idx] -= dissolved_debris;

                                float division = 1 / (1 + safe_diff);

                                _height[b_idx] -= (dissolve_amt - dissolved_debris);

                                if (bedrock_density > 0.0f)
                                {
                                    float newdebris = (1 - division) * (dissolve_amt * bedrock_density);
                                    if (b_debris + newdebris > max_debris_depth)
                                    {
                                        float rollback = b_debris + newdebris - max_debris_depth;
                                        rollback = zs::min(rollback, newdebris);
                                        _height[b_idx] += rollback / bedrock_density;
                                        newdebris -= rollback;
                                    }
                                    _debris[b_idx] += newdebris;

                                    newdebris = division * (dissolve_amt * bedrock_density);

                                    if (r_debris + newdebris > max_debris_depth)
                                    {
                                        float rollback = r_debris + newdebris - max_debris_depth;
                                        rollback = zs::min(rollback, newdebris);
                                        _height[b_idx] += rollback / bedrock_density;
                                        newdebris -= rollback;
                                    }
                                    _debris[r_idx] += newdebris;
                                }
                            }
                        }

                        _material[idx] = i_material + diff;
                        _material[j_idx] = j_material - diff;
                    }
                }
            });

            set_output("zs_HeightField", std::move(terrain));
        }
    };
    ZENDEFNODE(zs_tumble_material_v4, {/* inputs: */
        {
            "zs_HeightField",

            {"ListObject", "perm"},
            {"ListObject", "p_dirs"},
            {"ListObject", "x_dirs"},

            {"float", "seed", "12.34"},
            {"int", "iterations", "40"}, // 流淌的总迭代次数
            {"int", "iter", "0"},
            {"int", "i", "0"},

            {"int", "openborder", "0"},

            {"float", "gridbias", "0.0"},
            {"string", "gridbias_mask_layer", "_gridbias_mask"},                //~~~~~mask

            // 侵蚀主参数
            {"float", "global_erosionrate", "1.0"}, // 全局侵蚀率
            {"float", "erosionrate", "0.4"},        // 侵蚀率


            {"float", "erodability", "1.0"},        // 侵蚀能力
            {"string", "erodability_mask_layer", "_erodability_mask"},          //~~~~~mask

            {"float", "bank_angle", "70.0"},        // 河堤侵蚀角度
            {"string", "bankangle_mask_layer", "_bankangle_mask"},              //~~~~~mask

            // 高级参数
            {"float", "removalrate", "0.1"},      // 风化率/水吸收率
            {"string", "removalrate_mask_layer", "_removalrate_mask"},          //~~~~~mask

            {"float", "max_debris_depth", "5.0"}, // 碎屑最大深度

            // 侵蚀能力调整
            {"int", "max_erodability_iteration", "5"},      // 最大侵蚀能力迭代次数
            {"float", "initial_erodability_factor", "0.5"}, // 初始侵蚀能力因子
            {"float", "slope_contribution_factor", "0.8"}, // “地面斜率”对“侵蚀”和“沉积”的影响，“地面斜率大” -> 侵蚀因子大，沉积因子小

            // 河床参数
            {"float", "bed_erosionrate_factor", "1.0"}, // 河床侵蚀率因子

            {"float", "depositionrate", "0.01"},        // 沉积率
            {"string", "depositionrate_mask_layer", "_depositionrate_mask"},    //~~~~~mask

            {"float", "sedimentcap", "10.0"}, // 高度差转变为沉积物的比率 / 泥沙容量，每单位流动水可携带的泥沙量

            // 河堤参数
            {"float", "bank_erosionrate_factor", "1.0"}, // 河堤侵蚀率因子
            {"float", "max_bank_bed_ratio", "0.5"}, // 高于这个比值的河岸将不会在侵蚀中被视为河岸，会停止侵蚀

            // 河网控制
            {"float", "quant_amt", "0.05"}, // 流量维持率，越高河流流量越稳定
        },
        /* outputs: */
        {
            "zs_HeightField",
        },
        /* params: */
        {},
        /* category: */
        {
            "erode",
        }});

    struct zs_HF_maskbyOcclusion : INode {
        void apply() override {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            auto terrain = get_input<PrimitiveObject>("prim");

            auto invert_mask = get_input2<bool>("invert mask");
            auto view_radius = get_input2<int>("view distance");
            auto step_scale = get_input2<float>("step scale");
            auto axis_count = get_input2<int>("num of searches");
            auto dohemisphere = get_input2<bool>("dohemisphere");

            int nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");
            auto &pos = terrain->verts;
            float pos_delta_x = zeno::abs(pos[0][0]-pos[1][0]);
            float pos_delta_z = zeno::abs(pos[0][2]-pos[1][2]);
            float cellSize = zeno::max(pos_delta_x, pos_delta_z);

//        auto heightLayer = get_input2<std::string>("height_layer");
//        if (!terrain->verts.has_attr(heightLayer)) {
//            zeno::log_error("Node [HF_maskByFeature], no such data layer named '{}'.",
//                            heightLayer);
//        }
            auto &height = terrain->verts.attr<float>("height");

            auto &ao = terrain->verts.add_attr<float>("ao");
            std::fill(ao.begin(), ao.end(), 0.0f);
//            auto &attr_ao = terrain->verts.attr<float>("ao");
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            auto pol = cuda_exec();
            auto zs_height = to_device_vector(height);
            auto zs_ao = to_device_vector(ao, false);

            pol(range((std::size_t)nz * (std::size_t)nx), [=, _height = view<space>(zs_height), _ao = view<space>(zs_ao)] __device__(std::size_t idx) mutable {
                    auto id_x = idx % nx; // inner index
                    auto id_z = idx / nx; // outer index
                    float h_start = _height[idx];

                    step_scale = zs::max(step_scale, 0.5f);
                    if (view_radius) step_scale = zs::min(step_scale, 0.499f * view_radius);

                    int step_limit = view_radius > 0.0f ?
                                     zs::ceil(view_radius / (cellSize * step_scale) ) :
                                     zs::ceil(zs::sqrt((float)(nx*nx+nz*nz)) / step_scale);

                    float sweep_angle = 3.14159f / (float) zs::max((float)axis_count, 1.0f);
                    float cur_angle = 0.0f;

                    float total_fov = 0.0f;
                    float successful_rays = 0;


                    float z_step = zs::sin(cur_angle);
                    float x_step = zs::cos(cur_angle);
                    x_step *= step_scale;
                    z_step *= step_scale;

                    for (int i = 0; i < axis_count; i++) {
                        float z_step = zs::sin(cur_angle);
                        float x_step = zs::cos(cur_angle);
                        x_step *= step_scale;
                        z_step *= step_scale;

                        float speed = zs::sqrt(x_step*x_step+z_step*z_step) * cellSize;

                        for (int j = 0; j < 2; j++) {
                            float x = id_x + x_step;
                            float z = id_z + z_step;
                            float distance = speed;
                            int steps = 1;

                            float start_slope;

                            float finalslope = 0.0f;
                            float maxslope = -1e10f;
                            while (steps < step_limit &&
                                   x > 0 && x < (nx-1) &&
                                   z > 0 && z < (nz-1)) {

                                x = zs::clamp(x, 0.0f, (float)(nx-1));
                                z = zs::clamp(z, 0.0f, (float)(nz-1));

                                const int int_x = (int)zs::floor(x);
                                const int int_z = (int)zs::floor(z);

                                const float fract_x = x - int_x;
                                const float fract_z = z - int_z;

                                int srcidx = Pos2Idx(int_x, int_z, nx);
                                const float i00 = _height[srcidx];
                                const float i10 = _height[srcidx + 1];
                                const float i01 = _height[srcidx + nz];
                                const float i11 = _height[srcidx + nz + 1];

                                float h_current = (i00 * (1-fract_x) + i10 * (fract_x)) * (1-fract_z) +
                                                  (i01 * (1-fract_x) + i11 * (fract_x)) * (  fract_z);

                                // Calculate the slope
                                float dh = h_current - h_start;
                                float curslope = dh / distance;
                                if (steps == 1) start_slope = curslope;
                                maxslope = zs::max(maxslope, curslope);
                                finalslope = maxslope;

                                x += x_step;
                                z += z_step;
                                distance += speed;
                                steps++;
                            }

                            if (steps > 1) {
                                successful_rays += 1.0f;

                                if (dohemisphere) start_slope = 0;
                                float slope = zs::max(start_slope, finalslope);
                                total_fov += 1 - slope / zs::sqrt(slope*slope+1.0f);
                            }

                            x_step = -x_step;
                            z_step = -z_step;
                        }

                        cur_angle += sweep_angle;
                    }

                    if (successful_rays != 0) total_fov /= successful_rays;

                    total_fov = zs::clamp(total_fov, 0.0f, 1.0f);
                    _ao[idx] = invert_mask ? 1-total_fov : total_fov;
            });

            retrieve_device_vector(ao, zs_ao);

            set_output("prim", get_input("prim"));
        }
    };
    ZENDEFNODE(zs_HF_maskbyOcclusion,
               { /* inputs: */ {
                       "prim",
                       {"bool", "invert mask", "0"},
                       {"int", "view distance", "200"},
                       {"float", "step scale", "1"},
                       {"int", "num of searches", "16"},
                       {"bool", "dohemisphere", "0"},
                   }, /* outputs: */ {
                       "prim",
                   }, /* params: */ {
                   }, /* category: */ {
                       "erode",
                   } });


    struct zs_HF_Clip : INode {
        void apply() override {

            auto terrain = get_input<PrimitiveObject>("HeightField");
            auto Minclip = get_input2<bool>("MinClip");
            auto Maxclip = get_input2<bool>("MaxClip");
            auto MinclipValue = get_input2<float>("Minclipheight");
            auto MaxclipValue = get_input2<float>("Maxclipheight");
            auto SoftClip = get_input2<bool>("SoftClip");
            auto SoftClipStrength = get_input2<float>("SoftClip Strength");
            auto SoftClipScale = get_input2<float>("SoftClip Scale");

            size_t nx, nz;
            auto &ud = terrain->userData();
            if ((!ud.has<int>("nx")) || (!ud.has<int>("nz")))
                zeno::log_error("no such UserData named '{}' and '{}'.", "nx", "nz");
            nx = ud.get2<int>("nx");
            nz = ud.get2<int>("nz");


            auto &height = terrain->verts.attr<float>("height");
            if (!terrain->verts.has_attr("mask")) {
                auto &mask = terrain->verts.add_attr<float>("mask");
                std::fill(mask.begin(), mask.end(), 0.0);
            }
            auto &mask = terrain->verts.attr<float>("mask");
            ////////////////////////////////////////////////////////////////////////////////////////
            ////////////////////////////////////////////////////////////////////////////////////////
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            auto pol = cuda_exec();
            auto zs_height = to_device_vector(height);
            auto zs_mask = to_device_vector(mask, false);
            pol(range(nz * nx), [=, _height = view<space>(zs_height), _mask = view<space>(zs_mask)] __device__(std::size_t idx) mutable {
                    float i_height = _height[idx];
                    if(Minclip && i_height < MinclipValue){
                        if(SoftClip){
                            float f = (MinclipValue - i_height) / SoftClipScale;
                            float compression = SoftClipScale * SoftClipStrength;
                            float out = f;
                            if(f>0 && compression>0)
                            {
                                float ki = 1.0 / compression;
                                float w = 1.0 / (compression * zs::log(10.0));
                                float k = log10(zs::pow(w,ki));
                                out = log10(zs::pow((f+w), ki)) - k;
                            }
                            _height[idx] = MinclipValue - out;
                        }else{
                            _height[idx] = MinclipValue;
                        }
                        _mask[idx] = 1.0f;
                    }
                    if(Maxclip && i_height > MaxclipValue){
                        if(SoftClip){
                            float f = (i_height - MaxclipValue) / SoftClipScale;
                            float compression = SoftClipScale * SoftClipStrength;
                            float out = f;
                            if(f>0 && compression>0)
                            {
                                float ki = 1.0 / compression;
                                float w = 1.0 / (compression * zs::log(10.0));
                                float k = log10(zs::pow(w,ki));
                                out = log10(zs::pow((f+w), ki)) - k;
                            }
                            _height[idx] = MaxclipValue + out;
                        }else{
                            _height[idx] = MaxclipValue;
                        }
                        _mask[idx] = 1.0f;
                    }
                        
            });
            retrieve_device_vector(height, zs_height);
            retrieve_device_vector(mask, zs_mask);

            set_output("HeightField", get_input("HeightField"));
        }
    };
    ZENDEFNODE(zs_HF_Clip, {
        /* inputs: */
        {
            "HeightField",
            {"bool", "MinClip", "0"},
            {"float", "Minclipheight", "0"},
            {"bool", "MaxClip", "1"},
            {"float", "Maxclipheight", "100"},
            {"bool", "SoftClip", "1"},
            {"float", "SoftClip Strength", "0.1"},
            {"float", "SoftClip Scale", "1"},
        },
        /* outputs: */
        {
            "HeightField",
        },
        /* params: */
        {},
        /* category: */
        {
            "erode",
        }});
}   // namespace zeno