#include "hip/hip_runtime.h"
#include "../Structures.hpp"
#include "../Utils.hpp"

#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/ParticleIO.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"
#include <zeno/types/DictObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

namespace zeno {

struct ConfigConstitutiveModel : INode {
  void apply() override {
    auto out = std::make_shared<ZenoConstitutiveModel>();

    float dx = get_input2<float>("dx");

    // volume
    out->volume = dx * dx * dx / get_input2<float>("ppc");
    out->dx = dx;

    // density
    out->density = get_input2<float>("density");

    // constitutive models
    auto params = has_input("params") ? get_input<DictObject>("params")
                                      : std::make_shared<DictObject>();
    float E = get_input2<float>("E");

    float nu = get_input2<float>("nu");

    auto typeStr = get_input2<std::string>("type");
    // elastic model
    auto &model = out->getElasticModel();

    if (typeStr == "fcr")
      model = zs::FixedCorotated<float>{E, nu};
    else if (typeStr == "nhk")
      model = zs::NeoHookean<float>{E, nu};
    else if (typeStr == "stvk")
      model = zs::StvkWithHencky<float>{E, nu};
    else
      throw std::runtime_error(fmt::format(
          "unrecognized (isotropic) elastic model [{}]\n", typeStr));

    // aniso elastic model
    const auto get_arg = [&params](const char *const tag, auto type) {
      using T = typename RM_CVREF_T(type)::type;
      std::optional<T> ret{};
      if (auto it = params->lut.find(tag); it != params->lut.end())
        ret = safe_any_cast<T>(it->second);
      return ret;
    };
    auto anisoTypeStr = get_input2<std::string>("aniso");
    if (anisoTypeStr == "arap") { // a (fiber direction)
      float strength = get_arg("strength", zs::wrapt<float>{}).value_or(10.f);
      out->getAnisoElasticModel() = zs::AnisotropicArap<float>{E, nu, strength};
    } else
      out->getAnisoElasticModel() = std::monostate{};

    // plastic model
    auto plasticTypeStr = get_input2<std::string>("plasticity");
    if (plasticTypeStr == "nadp") {
      model = zs::StvkWithHencky<float>{E, nu};
      float fa = get_arg("friction_angle", zs::wrapt<float>{}).value_or(35.f);
      out->getPlasticModel() = zs::NonAssociativeDruckerPrager<float>{fa};
    } else if (plasticTypeStr == "navm") {
      model = zs::StvkWithHencky<float>{E, nu};
      float ys = get_arg("yield_stress", zs::wrapt<float>{}).value_or(1e5f);
      out->getPlasticModel() = zs::NonAssociativeVonMises<float>{ys};
    } else if (plasticTypeStr == "nacc") { // logjp
      model = zs::StvkWithHencky<float>{E, nu};
      float fa = get_arg("friction_angle", zs::wrapt<float>{}).value_or(35.f);
      float beta = get_arg("beta", zs::wrapt<float>{}).value_or(2.f);
      float xi = get_arg("xi", zs::wrapt<float>{}).value_or(1.f);
      out->getPlasticModel() =
          zs::NonAssociativeCamClay<float>{fa, beta, xi, 3, true};
    } else
      out->getPlasticModel() = std::monostate{};

    set_output("ZSModel", out);
  }
};

ZENDEFNODE(ConfigConstitutiveModel,
           {
               {{"float", "dx", "0.1"},
                {"float", "ppc", "8"},
                {"float", "density", "1000"},
                {"string", "type", "fcr"},
                {"string", "aniso", "none"},
                {"string", "plasticity", "none"},
                {"float", "E", "10000"},
                {"float", "nu", "0.4"},
                {"DictObject:NumericObject", "params"}},
               {"ZSModel"},
               {},
               {"MPM"},
           });

struct ToTrackerParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToTrackerParticles\n");

    // primitive
    auto inParticles = get_input<PrimitiveObject>("prim");
    auto &obj = inParticles->attr<vec3f>("pos");
    vec3f *velsPtr{nullptr};
    if (inParticles->has_attr("vel"))
      velsPtr = inParticles->attr<vec3f>("vel").data();

    auto outParticles = std::make_shared<ZenoParticles>();

    // primitive binding
    outParticles->prim = inParticles;

    /// category, size
    std::size_t size{obj.size()};
    outParticles->category = ZenoParticles::category_e::tracker;

    // per vertex (node) vol, pos, vel
    using namespace zs;
    auto ompExec = zs::omp_exec();

    // attributes
    std::vector<zs::PropertyTag> tags{{"pos", 3}, {"vel", 3}};
    {
      outParticles->particles =
          std::make_shared<typename ZenoParticles::particles_t>(tags, size,
                                                                memsrc_e::host);
      auto &pars = outParticles->getParticles(); // tilevector
      ompExec(zs::range(size), [pars = proxy<execspace_e::host>({}, pars),
                                velsPtr, &obj](size_t pi) mutable {
        using vec3 = zs::vec<float, 3>;
        using mat3 = zs::vec<float, 3, 3>;

        // pos
        pars.tuple<3>("pos", pi) = obj[pi];

        // vel
        if (velsPtr != nullptr)
          pars.tuple<3>("vel", pi) = velsPtr[pi];
        else
          pars.tuple<3>("vel", pi) = vec3::zeros();
      });

      pars = pars.clone({memsrc_e::um, 0});
    }
    if (inParticles->tris.size()) {
      const auto eleSize = inParticles->tris.size();
      std::vector<zs::PropertyTag> tags{{"pos", 3}, {"vel", 3}, {"inds", 3}};
      outParticles->elements =
          typename ZenoParticles::particles_t{tags, eleSize, memsrc_e::host};
      auto &eles = outParticles->getQuadraturePoints();

      auto &tris = inParticles->tris.values;
      ompExec(zs::range(eleSize), [eles = proxy<execspace_e::host>({}, eles),
                                   &obj, &tris, velsPtr](size_t ei) mutable {
        using vec3 = zs::vec<float, 3>;
        // inds
        int inds[3] = {(int)tris[ei][0], (int)tris[ei][1], (int)tris[ei][2]};
        for (int d = 0; d != 3; ++d)
          eles("inds", d, ei) = reinterpret_bits<float>(inds[d]);
        // pos
        eles.tuple<3>("pos", ei) =
            (obj[inds[0]] + obj[inds[1]] + obj[inds[2]]) / 3.f;

        // vel
        if (velsPtr != nullptr) {
          eles.tuple<3>("vel", ei) =
              (velsPtr[inds[0]] + velsPtr[inds[1]] + velsPtr[inds[2]]) / 3.f;
        } else
          eles.tuple<3>("vel", ei) = vec3::zeros();
      });

      eles = eles.clone({memsrc_e::um, 0});
    }

    fmt::print(fg(fmt::color::cyan), "done executing ToTrackerParticles\n");
    set_output("ZSParticles", outParticles);
  }
};

ZENDEFNODE(ToTrackerParticles, {
                                   {"prim"},
                                   {"ZSParticles"},
                                   {},
                                   {"MPM"},
                               });

struct ConstructBendingSprings : INode {
  // vertex
  std::shared_ptr<ZenoParticles>
  addVertexBendingSprings(zs::CudaExecutionPolicy &cudaPol,
                          const ZenoParticles &surf, float stiffness) {
    if (surf.category != ZenoParticles::surface)
      return {};
    using namespace zs;
    using TableT = HashTable<int, 2, int>;     //
    using VertTableT = HashTable<int, 1, int>; //
    using key_t = typename TableT::key_t;
    using vec1i = zs::vec<int, 1>;
    using vec3 = zs::vec<float, 3>;
    using mat3 = zs::vec<float, 3, 3>;
    auto &surfPars = surf.getParticles();
    auto numV = surfPars.size(); // i.e. sprayedOffset
    auto &surfEles = surf.getQuadraturePoints();
    auto numE = surfEles.size();

    fmt::print("surface mesh: {} verts, {} tris.\n", numV, numE);
    TableT edgeTable{surfPars.get_allocator(), numE * 3}; // edge -> eleid
    edgeTable.reset(cudaPol, true);
    //
    constexpr auto space = execspace_e::cuda;
    cudaPol(range(numE),
            [table = proxy<space>(edgeTable),
             eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
              auto tri =
                  eles.pack<3>("inds", ei).template reinterpret_bits<int>();
              auto vi = tri[2];
              for (int v = 0; v != 3; ++v) {
                auto vj = tri[v];
                if (vi < vj)
                  table.insert(key_t{vi, vj});
                vi = vj;
              }
            });
    std::size_t numRegisteredEdges = edgeTable.size();
    Vector<int> edgeToEles{surfPars.get_allocator(), numRegisteredEdges};
    cudaPol(
        range(numE),
        [table = proxy<space>(edgeTable), edgeToEles = proxy<space>(edgeToEles),
         eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
          auto tri = eles.pack<3>("inds", ei).template reinterpret_bits<int>();
          auto vi = tri[2];
          for (int v = 0; v != 3; ++v) {
            auto vj = tri[v];
            if (vi < vj) {
              auto no = table.query(key_t{vi, vj});
              edgeToEles[no] = ei;
            }
            vi = vj;
          }
        });
    //
    using VertPair = zs::vec<int, 2>;
    Vector<int> cnt{surfPars.get_allocator(), 1};
    cnt.setVal(0);
    Vector<VertPair> vertPairs{surfPars.get_allocator(), numRegisteredEdges};
    Vector<VertPair> elePairs{surfPars.get_allocator(), numRegisteredEdges};
    VertTableT vertTable{surfPars.get_allocator(), numRegisteredEdges * 2};
    vertTable.reset(cudaPol, true);
    cudaPol(
        range(numE),
        [table = proxy<space>(edgeTable), vertTable = proxy<space>(vertTable),
         edgeToEles = proxy<space>(edgeToEles), cnt = proxy<space>(cnt),
         vertPairs = proxy<space>(vertPairs), elePairs = proxy<space>(elePairs),
         eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
          using table_t = RM_CVREF_T(table);
          auto tri = eles.pack<3>("inds", ei).template reinterpret_bits<int>();
          auto vi = tri[2];
          for (int v = 0; v != 3; ++v) {
            auto vj = tri[v];
            if (vi > vj) { // check opposite
              if (auto edgeNo = table.query(key_t{vj, vi});
                  edgeNo != table_t::sentinel_v) {
                auto neighborEleNo = edgeToEles[edgeNo];
                auto neighborTri = eles.pack<3>("inds", neighborEleNo)
                                       .template reinterpret_bits<int>();
                int neighborV = -1, selfV = -1;
                for (int d = 0; d != 3; ++d)
                  if (neighborTri[d] != vi && neighborTri[d] != vj) {
                    neighborV = neighborTri[d];
                    break;
                  }
                for (int d = 0; d != 3; ++d)
                  if (tri[d] != vi && tri[d] != vj) {
                    selfV = tri[d];
                    break;
                  }
                vertTable.insert(vec1i{neighborV});
                vertTable.insert(vec1i{selfV});
                auto no = atomic_add(exec_cuda, &cnt[0], 1);
                vertPairs[no] = VertPair{neighborV, selfV};
                elePairs[no] = VertPair{neighborEleNo, ei};
              }
            }
            vi = vj;
          }
        });
    std::size_t numVertPairs = cnt.getVal();
    vertPairs.resize(numVertPairs);
    elePairs.resize(numVertPairs);
    //
    auto ret = std::make_shared<ZenoParticles>();
    FixedCorotated<float> fcr{};
    zs::match([&fcr](auto &model) {
      fcr.mu = model.mu;
      fcr.lam = model.lam;
    })(surf.getModel().getElasticModel());
    ret->getModel() = surf.getModel();
    ret->getModel().getElasticModel() = fcr;
    ret->category = ZenoParticles::curve;

    std::vector<zs::PropertyTag> tags{{"mass", 1}, {"pos", 3}, {"vel", 3},
                                      {"vol", 1},  {"C", 9},   {"beta", 1}};
    std::vector<zs::PropertyTag> eleTags{
        {"mass", 1},      {"pos", 3}, {"vel", 3}, {"vol", 1},
        {"C", 9},         {"F", 9},   {"d", 9},   {"DmInv", 9},
        {"inds", (int)2}, {"mu", 1},  {"lam", 1}};

    std::size_t numSpringVerts = vertTable.size();
    ret->sprayedOffset = numSpringVerts;
    ret->particles = std::make_shared<typename ZenoParticles::particles_t>(
        surfPars.get_allocator(), tags, numSpringVerts);
    auto &pars = ret->getParticles(); // tilevector
    // springs have no inertial
    cudaPol(range(numSpringVerts),
            [pars = proxy<space>({}, pars),
             surfPars = proxy<space>({}, surfPars),
             vertTable = proxy<space>(vertTable)] __device__(int pi) mutable {
              using mat3 = zs::vec<float, 3, 3>;
              auto opid = vertTable._activeKeys[pi][0];
              pars("mass", pi) = 0.f;
              pars("vol", pi) = surfPars("vol", opid);
              pars("beta", pi) = 0.f;
              pars.tuple<3>("pos", pi) = surfPars.pack<3>("pos", opid);
              pars.tuple<3>("vel", pi) = vec3::zeros();
              pars.tuple<3 * 3>("C", pi) = mat3::zeros();
            });

    ret->elements = typename ZenoParticles::particles_t{
        surfPars.get_allocator(), eleTags, numVertPairs};
    auto &eles = ret->getQuadraturePoints();
    cudaPol(range(numVertPairs), [pars = proxy<space>({}, pars),
                                  eles = proxy<space>({}, eles),
                                  surfEles = proxy<space>({}, surfEles),
                                  vertPairs = proxy<space>(vertPairs),
                                  elePairs = proxy<space>(elePairs),
                                  vertTable = proxy<space>(vertTable),
                                  stiffness] __device__(int ei) mutable {
      using mat3 = zs::vec<float, 3, 3>;
      eles("mass", ei) = 0.f;

      {
        auto eids = elePairs[ei];
        auto mu = zs::min(surfEles("mu", eids[0]), surfEles("mu", eids[1]));
        auto lam = zs::min(surfEles("lam", eids[0]), surfEles("lam", eids[1]));
        eles("mu", ei) = mu * stiffness;
        eles("lam", ei) = lam * stiffness;
      }

      auto inds = vertPairs[ei];
      inds[0] = vertTable.query(vec1i{inds[0]});
      inds[1] = vertTable.query(vec1i{inds[1]});
      vec3 xs[2];
      xs[0] = pars.pack<3>("pos", inds[0]);
      xs[1] = pars.pack<3>("pos", inds[1]);
      eles.tuple<3>("pos", ei) = (xs[0] + xs[1]) / 2;
      eles("vol", ei) = (pars("vol", inds[0]) + pars("vol", inds[1])) / 2;
      eles.tuple<3>("vel", ei) = vec3::zeros();

      eles.tuple<3 * 3>("C", ei) = mat3::zeros();

      auto tangent = xs[1] - xs[0];
      auto tn = tangent.norm();
      auto nrm = tangent.orthogonal().normalized();
      auto binrm = tangent.cross(nrm).normalized();
      auto d = mat3{tangent[0], nrm[0],     binrm[0], tangent[1], nrm[1],
                    binrm[1],   tangent[2], nrm[2],   binrm[2]};
      eles.tuple<3 * 3>("d", ei) = d;
      auto invDstar = mat3::identity();
      invDstar(0, 0) = 1. / tn;
      if (tn <= 10 * limits<float>::epsilon()) {
        eles("mu", ei) = 0.f;
        eles("lam", ei) = 0.f;
      }
      eles.tuple<3 * 3>("DmInv", ei) = invDstar;
      eles.tuple<3 * 3>("F", ei) = d * invDstar;

      eles.tuple<2>("inds", ei) = inds.template reinterpret_bits<float>();
    });

    fmt::print("bending spring mesh: {} verts, {} tris.\n", numSpringVerts,
               numVertPairs);
    return ret;
  }
  // element
  std::shared_ptr<ZenoParticles>
  addElementBendingSprings(zs::CudaExecutionPolicy &cudaPol,
                           const ZenoParticles &surf, float stiffness) {
    if (surf.category != ZenoParticles::surface)
      return {};
    using namespace zs;
    using TableT = HashTable<int, 2, int>;        //
    using ElementTableT = HashTable<int, 1, int>; //
    using key_t = typename TableT::key_t;
    using vec1i = zs::vec<int, 1>;
    using vec3 = zs::vec<float, 3>;
    using mat3 = zs::vec<float, 3, 3>;
    auto &surfPars = surf.getParticles();
    auto numV = surfPars.size(); // i.e. sprayedOffset
    auto &surfEles = surf.getQuadraturePoints();
    auto numE = surfEles.size();

    fmt::print("surface mesh: {} verts, {} tris.\n", numV, numE);
    TableT edgeTable{surfPars.get_allocator(), numE * 3}; // edge -> eleid
    edgeTable.reset(cudaPol, true);
    //
    constexpr auto space = execspace_e::cuda;
    cudaPol(range(numE),
            [table = proxy<space>(edgeTable),
             eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
              auto tri =
                  eles.pack<3>("inds", ei).template reinterpret_bits<int>();
              auto vi = tri[2];
              for (int v = 0; v != 3; ++v) {
                auto vj = tri[v];
                if (vi < vj)
                  table.insert(key_t{vi, vj});
                vi = vj;
              }
            });
    std::size_t numRegisteredEdges = edgeTable.size();
    Vector<int> edgeToEles{surfPars.get_allocator(), numRegisteredEdges};
    cudaPol(
        range(numE),
        [table = proxy<space>(edgeTable), edgeToEles = proxy<space>(edgeToEles),
         eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
          auto tri = eles.pack<3>("inds", ei).template reinterpret_bits<int>();
          auto vi = tri[2];
          for (int v = 0; v != 3; ++v) {
            auto vj = tri[v];
            if (vi < vj) {
              auto no = table.query(key_t{vi, vj});
              edgeToEles[no] = ei;
            }
            vi = vj;
          }
        });
    //
    using ElePair = zs::vec<int, 2>;
    Vector<int> cnt{surfPars.get_allocator(), 1};
    cnt.setVal(0);
    Vector<ElePair> elePairs{surfPars.get_allocator(), numRegisteredEdges};
    ElementTableT eleTable{surfPars.get_allocator(), numRegisteredEdges};
    eleTable.reset(cudaPol, true);
    cudaPol(range(numE),
            [table = proxy<space>(edgeTable), eleTable = proxy<space>(eleTable),
             edgeToEles = proxy<space>(edgeToEles), cnt = proxy<space>(cnt),
             elePairs = proxy<space>(elePairs),
             eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
              using table_t = RM_CVREF_T(table);
              auto tri =
                  eles.pack<3>("inds", ei).template reinterpret_bits<int>();
              auto vi = tri[2];
              for (int v = 0; v != 3; ++v) {
                auto vj = tri[v];
                if (vi > vj) { // check opposite
                  if (auto edgeNo = table.query(key_t{vj, vi});
                      edgeNo != table_t::sentinel_v) {
                    auto neighborEleNo = edgeToEles[edgeNo];
                    eleTable.insert(vec1i{ei});
                    eleTable.insert(vec1i{neighborEleNo});
                    auto no = atomic_add(exec_cuda, &cnt[0], 1);
                    elePairs[no] = ElePair{neighborEleNo, ei};
                  }
                }
                vi = vj;
              }
            });
    std::size_t numElePairs = cnt.getVal();
    elePairs.resize(numElePairs);
    //
    auto ret = std::make_shared<ZenoParticles>();
    FixedCorotated<float> fcr{};
    zs::match([&fcr](auto &model) {
      fcr.mu = model.mu;
      fcr.lam = model.lam;
    })(surf.getModel().getElasticModel());
    ret->getModel() = surf.getModel();
    ret->getModel().getElasticModel() = fcr;
    ret->category = ZenoParticles::curve;

    std::vector<zs::PropertyTag> tags{{"mass", 1}, {"pos", 3}, {"vel", 3},
                                      {"vol", 1},  {"C", 9},   {"beta", 1}};
    std::vector<zs::PropertyTag> eleTags{
        {"mass", 1},      {"pos", 3}, {"vel", 3}, {"vol", 1},
        {"C", 9},         {"F", 9},   {"d", 9},   {"DmInv", 9},
        {"inds", (int)2}, {"mu", 1},  {"lam", 1}};

    std::size_t numSpringVerts = eleTable.size();
    ret->sprayedOffset = numSpringVerts;
    ret->particles = std::make_shared<typename ZenoParticles::particles_t>(
        surfPars.get_allocator(), tags, numSpringVerts);
    auto &pars = ret->getParticles(); // tilevector
    // springs have no inertial
    cudaPol(range(numSpringVerts),
            [pars = proxy<space>({}, pars),
             surfEles = proxy<space>({}, surfEles),
             eleTable = proxy<space>(eleTable)] __device__(int pi) mutable {
              using mat3 = zs::vec<float, 3, 3>;
              auto opid = eleTable._activeKeys[pi][0];
              pars("mass", pi) = 0.f;
              pars("vol", pi) = surfEles("vol", opid);
              pars("beta", pi) = 0.f;
              pars.tuple<3>("pos", pi) = surfEles.pack<3>("pos", opid);
              pars.tuple<3>("vel", pi) = vec3::zeros();
              pars.tuple<3 * 3>("C", pi) = mat3::zeros();
            });

    ret->elements = typename ZenoParticles::particles_t{
        surfPars.get_allocator(), eleTags, numElePairs};
    auto &eles = ret->getQuadraturePoints();
    cudaPol(range(numElePairs), [pars = proxy<space>({}, pars),
                                 eles = proxy<space>({}, eles),
                                 surfEles = proxy<space>({}, surfEles),
                                 elePairs = proxy<space>(elePairs),
                                 eleTable = proxy<space>(eleTable),
                                 stiffness] __device__(int ei) mutable {
      using mat3 = zs::vec<float, 3, 3>;
      eles("mass", ei) = 0.f;

      auto eids = elePairs[ei];
      auto mu = zs::min(surfEles("mu", eids[0]), surfEles("mu", eids[1]));
      auto lam = zs::min(surfEles("lam", eids[0]), surfEles("lam", eids[1]));
      eles("mu", ei) = mu * stiffness;
      eles("lam", ei) = lam * stiffness;

      eids[0] = eleTable.query(vec1i{eids[0]});
      eids[1] = eleTable.query(vec1i{eids[1]});
      vec3 xs[2];
      xs[0] = pars.pack<3>("pos", eids[0]);
      xs[1] = pars.pack<3>("pos", eids[1]);
      eles.tuple<3>("pos", ei) = (xs[0] + xs[1]) / 2;
      eles("vol", ei) = (pars("vol", eids[0]) + pars("vol", eids[1])) / 2;
      eles.tuple<3>("vel", ei) = vec3::zeros();

      eles.tuple<3 * 3>("C", ei) = mat3::zeros();

      auto tangent = xs[1] - xs[0];
      auto tn = tangent.norm();
      auto nrm = tangent.orthogonal().normalized();
      auto binrm = tangent.cross(nrm).normalized();
      auto d = mat3{tangent[0], nrm[0],     binrm[0], tangent[1], nrm[1],
                    binrm[1],   tangent[2], nrm[2],   binrm[2]};
      eles.tuple<3 * 3>("d", ei) = d;
      auto invDstar = mat3::identity();
      invDstar(0, 0) = 1. / tn;
      if (tn <= 10 * limits<float>::epsilon()) {
        eles("mu", ei) = 0.f;
        eles("lam", ei) = 0.f;
      }
      eles.tuple<3 * 3>("DmInv", ei) = invDstar;
      eles.tuple<3 * 3>("F", ei) = d * invDstar;

      eles.tuple<2>("inds", ei) = eids.template reinterpret_bits<float>();
    });

    fmt::print("bending spring mesh: {} verts, {} tris.\n", numSpringVerts,
               numElePairs);
    return ret;
  }
  // angle
  std::shared_ptr<ZenoParticles>
  addAngleBendingSprings(zs::CudaExecutionPolicy &cudaPol, ZenoParticles &surf,
                         float stiffness) {
    if (surf.category != ZenoParticles::surface)
      return {};
    using namespace zs;
    using TableT = HashTable<int, 2, int>;        //
    using ElementTableT = HashTable<int, 1, int>; //
    using key_t = typename TableT::key_t;
    using vec1i = zs::vec<int, 1>;
    using vec3 = zs::vec<float, 3>;
    using mat3 = zs::vec<float, 3, 3>;
    float thickness = surf.getModel().dx;
    auto &surfPars = surf.getParticles();
    auto numV = surfPars.size(); // i.e. sprayedOffset
    auto &surfEles = surf.getQuadraturePoints();
    auto numE = surfEles.size();

    fmt::print("surface mesh: {} verts, {} tris.\n", numV, numE);
    TableT edgeTable{surfPars.get_allocator(), numE * 3}; // edge -> eleid
    edgeTable.reset(cudaPol, true);
    //
    constexpr auto space = execspace_e::cuda;
    cudaPol(range(numE),
            [table = proxy<space>(edgeTable),
             eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
              auto tri =
                  eles.pack<3>("inds", ei).template reinterpret_bits<int>();
              auto vi = tri[2];
              for (int v = 0; v != 3; ++v) {
                auto vj = tri[v];
                if (vi < vj)
                  table.insert(key_t{vi, vj});
                vi = vj;
              }
            });
    std::size_t numRegisteredEdges = edgeTable.size();
    Vector<int> edgeToEles{surfPars.get_allocator(), numRegisteredEdges};
    cudaPol(
        range(numE),
        [table = proxy<space>(edgeTable), edgeToEles = proxy<space>(edgeToEles),
         eles = proxy<space>({}, surfEles)] __device__(int ei) mutable {
          auto tri = eles.pack<3>("inds", ei).template reinterpret_bits<int>();
          auto vi = tri[2];
          for (int v = 0; v != 3; ++v) {
            auto vj = tri[v];
            if (vi < vj) {
              auto no = table.query(key_t{vi, vj});
              edgeToEles[no] = ei;
            }
            vi = vj;
          }
        });
    //
    using ElePair = zs::vec<int, 4>;
    Vector<int> cnt{surfPars.get_allocator(), 1};
    cnt.setVal(0);
    Vector<ElePair> elePairs{surfPars.get_allocator(), numRegisteredEdges};
    Vector<float> kBends{surfPars.get_allocator(), numRegisteredEdges};
    cudaPol(
        range(numE),
        [table = proxy<space>(edgeTable), edgeToEles = proxy<space>(edgeToEles),
         cnt = proxy<space>(cnt), elePairs = proxy<space>(elePairs),
         eles = proxy<space>({}, surfEles), kBends = proxy<space>(kBends),
         thickness] __device__(int ei) mutable {
          using table_t = RM_CVREF_T(table);
          auto [E_self, nu_self] =
              E_nu_from_lame_parameters(eles("mu", ei), eles("lam", ei));
          auto tri = eles.pack<3>("inds", ei).template reinterpret_bits<int>();
          // <vi, vj, vk>
          auto vi = tri[1];
          auto vj = tri[2];
          for (int v = 0; v != 3; ++v) {
            auto vk = tri[v];
            if (vi > vj) { // check opposite
              if (auto edgeNo = table.query(key_t{vj, vi});
                  edgeNo != table_t::sentinel_v) {
                auto neighborEleNo = edgeToEles[edgeNo];
                auto [E_nei, nu_nei] = E_nu_from_lame_parameters(
                    eles("mu", neighborEleNo), eles("lam", neighborEleNo));
                auto neighborTri = eles.pack<3>("inds", neighborEleNo)
                                       .template reinterpret_bits<int>();
                int neighborV = -1;
                for (int d = 0; d != 3; ++d)
                  if (neighborTri[d] != vi && neighborTri[d] != vj) {
                    neighborV = neighborTri[d];
                    break;
                  }
                auto no = atomic_add(exec_cuda, &cnt[0], 1);
                /**
                 *             vi --- vk
                 *            /  \    /
                 *           /    \  /
                 *         nei --- vj
                 */
                elePairs[no] = ElePair{vj, vi, neighborV, vk};
                auto E = zs::min(E_self, E_nei);
                auto nu = zs::min(nu_self, nu_nei);
                kBends[no] = E / (24 * (1.0 - nu * nu)) * thickness *
                             thickness * thickness;
              }
            }
            vi = vj;
            vj = vk;
          }
        });
    std::size_t numElePairs = cnt.getVal();
    elePairs.resize(numElePairs);
    kBends.resize(numElePairs);
    //
    auto ret = std::make_shared<ZenoParticles>();
    ret->getModel() = surf.getModel();
    ret->category = ZenoParticles::bending;

    // k: stiffness
    // ra: rest angle
    std::vector<zs::PropertyTag> eleTags{{"vinds", 4}, {"k", 1}, {"ra", 1}};

    std::size_t numSpringVerts = numElePairs;
    ret->sprayedOffset = numSpringVerts;
    ret->particles = std::shared_ptr<typename ZenoParticles::particles_t>(
        &surfPars, [](...) {}); // no deletion upon dtor
    ret->elements = typename ZenoParticles::particles_t{
        surfPars.get_allocator(), eleTags, numElePairs};
    auto &eles = ret->getQuadraturePoints();
    cudaPol(range(numElePairs), [eles = proxy<space>({}, eles),
                                 surfPars = proxy<space>({}, surfPars),
                                 elePairs = proxy<space>(elePairs),
                                 kBends = proxy<space>(kBends),
                                 stiffness] __device__(int ei) mutable {
      using mat3 = zs::vec<float, 3, 3>;
      // bending_stiffness =
      // E / (24 * (1.0 - nu * nu)) * thickness^3
      eles("k", ei) = stiffness * kBends[ei];

      auto vinds = elePairs[ei];
      eles.tuple<4>("vinds", ei) = vinds.reinterpret_bits<float>();
      /**
       *             v1 --- v3
       *            /  \    /
       *           /    \  /
       *          v2 --- v0
       */
      auto v0 = surfPars.pack<3>("pos", vinds[0]);
      auto v1 = surfPars.pack<3>("pos", vinds[1]);
      auto v2 = surfPars.pack<3>("pos", vinds[2]);
      auto v3 = surfPars.pack<3>("pos", vinds[3]);
      auto n1 = (v0 - v2).cross(v1 - v2);
      auto n2 = (v1 - v3).cross(v0 - v3); // <v2, v1, v3>
      auto DA = zs::acos(
          zs::max(-1.f, zs::min(1.f, n1.dot(n2) / zs::sqrt(n1.l2NormSqr() *
                                                           n2.l2NormSqr()))));
      if (n2.cross(n1).dot(v0 - v1) < 0) // towards "closing"
        DA = -DA;
      eles("ra", ei) = 0;
    });

    fmt::print("bending spring mesh: {} verts, {} tris.\n", numSpringVerts,
               numElePairs);
    return ret;
  }

  void apply() override {
    using namespace zs;
    fmt::print(fg(fmt::color::green),
               "begin executing ConstructBendingSprings\n");

    float stiffness = get_input2<float>("bending_stiffness");
    auto typeStr = get_param<std::string>("type");
    auto cudaPol = cuda_exec();
    if (has_input<ZenoParticles>("ZSSurfPrim")) {
      if (typeStr == "vertex")
        set_output(
            "ZSSpringPrim",
            addVertexBendingSprings(
                cudaPol, *get_input<ZenoParticles>("ZSSurfPrim"), stiffness));
      else if (typeStr == "element")
        set_output(
            "ZSSpringPrim",
            addElementBendingSprings(
                cudaPol, *get_input<ZenoParticles>("ZSSurfPrim"), stiffness));
      else if (typeStr == "angle")
        set_output(
            "ZSSpringPrim",
            addAngleBendingSprings(
                cudaPol, *get_input<ZenoParticles>("ZSSurfPrim"), stiffness));
    } else if (has_input<ListObject>("ZSSurfPrim")) {
      auto list = std::make_shared<ListObject>();
      auto &ret = list->arr;
      auto &objSharedPtrLists = *get_input<zeno::ListObject>("ZSSurfPrim");
      if (typeStr == "vertex")
        for (auto &&objSharedPtr : objSharedPtrLists.get()) {
          if (auto ptr = dynamic_cast<ZenoParticles *>(objSharedPtr.get());
              ptr != nullptr)
            ret.push_back(addVertexBendingSprings(cudaPol, *ptr, stiffness));
        }
      else if (typeStr == "element")
        for (auto &&objSharedPtr : objSharedPtrLists.get()) {
          if (auto ptr = dynamic_cast<ZenoParticles *>(objSharedPtr.get());
              ptr != nullptr)
            ret.push_back(addElementBendingSprings(cudaPol, *ptr, stiffness));
        }
      else if (typeStr == "angle")
        for (auto &&objSharedPtr : objSharedPtrLists.get())
          if (auto ptr = dynamic_cast<ZenoParticles *>(objSharedPtr.get());
              ptr != nullptr)
            ret.push_back(addAngleBendingSprings(cudaPol, *ptr, stiffness));
      set_output("ZSSpringPrim", list);
    }

    fmt::print(fg(fmt::color::cyan),
               "done executing ConstructBendingSprings\n");
  }
};

ZENDEFNODE(ConstructBendingSprings,
           {
               {"ZSSurfPrim", {"float", "bending_stiffness", "0.01"}},
               {"ZSSpringPrim"},
               {{"enum vertex element angle", "type", "element"}},
               {"MPM"},
           });

struct BuildPrimitiveSequence : INode {
  void apply() override {
    using namespace zs;
    fmt::print(fg(fmt::color::green),
               "begin executing BuildPrimitiveSequence\n");

    std::shared_ptr<ZenoParticles> zsprimseq{};

    if (!has_input<ZenoParticles>("ZSParticles"))
      throw std::runtime_error(
          fmt::format("no incoming prim for prim sequence!\n"));
    auto next = get_input<ZenoParticles>("ZSParticles");
    if (!next->asBoundary)
      throw std::runtime_error(
          fmt::format("incoming prim is not used as a boundary!\n"));

    auto cudaPol = cuda_exec().device(0);
    if (has_input<ZenoParticles>("ZSPrimitiveSequence")) {
      zsprimseq = get_input<ZenoParticles>("ZSPrimitiveSequence");
      auto numV = zsprimseq->numParticles();
      auto numE = zsprimseq->numElements();
      auto sprayedOffset = zsprimseq->sprayedOffset;
      auto sprayedSize = numV - sprayedOffset;
      auto size = sprayedOffset;
      if (size != next->numParticles() || numE != next->numElements()) {
        fmt::print(
            "current numVerts ({} + {}) (i.e. {}), numEles ({}).\nIncoming "
            "boundary primitive numVerts ({}), numEles ({})\n",
            size, sprayedSize, numV, numE, next->numParticles(),
            next->numElements());
        throw std::runtime_error(
            fmt::format("prim size mismatch with current sequence prim!\n"));
      }

      fmt::print("{} verts (including {} sprayed), {} elements\n", numV,
                 sprayedSize, numE);

      auto dt = get_input2<float>("framedt"); // framedt
      /// update velocity
      // update mesh verts
      cudaPol(Collapse{size},
              [prev = proxy<execspace_e::cuda>({}, zsprimseq->getParticles()),
               next = proxy<execspace_e::cuda>({}, next->getParticles()),
               dt] __device__(int pi) mutable {
                prev.tuple<3>("vel", pi) =
                    (next.pack<3>("pos", pi) - prev.pack<3>("pos", pi)) / dt;
              });
      // update elements
      cudaPol(Collapse{numE},
              [prev = proxy<execspace_e::cuda>(
                   {}, zsprimseq->getQuadraturePoints()),
               next = proxy<execspace_e::cuda>({}, next->getQuadraturePoints()),
               dt] __device__(int ei) mutable {
                prev.tuple<3>("vel", ei) =
                    (next.pack<3>("pos", ei) - prev.pack<3>("pos", ei)) / dt;
              });
      if (size != numV) { // update sprayed mesh verts
        cudaPol(
            Collapse{sprayedSize},
            [verts = proxy<execspace_e::cuda>({}, zsprimseq->getParticles()),
             eles =
                 proxy<execspace_e::cuda>({}, zsprimseq->getQuadraturePoints()),
             sprayedOffset] __device__(int pi) mutable {
              auto dst = pi + sprayedOffset;

              int eid = reinterpret_bits<int>(verts("eid", dst));
              auto tri = eles.pack<3>("inds", eid).reinterpret_bits<int>();
              auto ws = verts.pack<3>("weights", dst);
              {
                auto v0 = verts.pack<3>("vel", tri[0]);
                auto v1 = verts.pack<3>("vel", tri[1]);
                auto v2 = verts.pack<3>("vel", tri[2]);

                verts.tuple<3>("vel", dst) =
                    ws[0] * v0 + ws[1] * v1 + ws[2] * v2;
              }
              {
                auto p0 = verts.pack<3>("pos", tri[0]);
                auto p1 = verts.pack<3>("pos", tri[1]);
                auto p2 = verts.pack<3>("pos", tri[2]);

                verts.tuple<3>("pos", dst) =
                    ws[0] * p0 + ws[1] * p1 + ws[2] * p2;
              }
            });
      }
    } else {
      zsprimseq = std::make_shared<ZenoParticles>(*next);
    }

    fmt::print(fg(fmt::color::cyan), "done executing BuildPrimitiveSequence\n");
    set_output("ZSPrimitiveSequence", zsprimseq);
  }
};
ZENDEFNODE(BuildPrimitiveSequence, {
                                       {"ZSPrimitiveSequence",
                                        {"float", "framedt", "0.1"},
                                        "ZSParticles"},
                                       {"ZSPrimitiveSequence"},
                                       {},
                                       {"MPM"},
                                   });

/// this requires further polishing
struct UpdatePrimitiveFromZSParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green),
               "begin executing UpdatePrimitiveFromZSParticles\n");

    auto parObjPtrs = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");

    using namespace zs;
    auto ompExec = zs::omp_exec();

    for (auto &&parObjPtr : parObjPtrs) {
      auto &pars = parObjPtr->getParticles();
      if (parObjPtr->prim.get() == nullptr)
        continue;

      auto &prim = *parObjPtr->prim;
      // const auto category = parObjPtr->category;
      auto &pos = prim.attr<vec3f>("pos");
      auto size = pos.size(); // in case zsparticle-mesh is refined
      vec3f *velsPtr{nullptr};
      if (prim.has_attr("vel") && pars.hasProperty("vel"))
        velsPtr = prim.attr<vec3f>("vel").data();

      if (pars.hasProperty("id")) {
        ompExec(range(pars.size()),
                [&, pars = proxy<execspace_e::host>({}, pars)](auto pi) {
                  auto id = (int)pars("id", pi);
                  if (id >= size)
                    return;
                  pos[id] = pars.array<3>("pos", pi);
                  if (velsPtr != nullptr)
                    velsPtr[id] = pars.array<3>("vel", pi);
                });
      } else {
        // currently only write back pos and vel (if exists)
        ompExec(range(size),
                [&, pars = proxy<execspace_e::host>({}, pars)](auto pi) {
                  pos[pi] = pars.array<3>("pos", pi);
                  if (velsPtr != nullptr)
                    velsPtr[pi] = pars.array<3>("vel", pi);
                });
      }
      const auto cnt = pars.size();
    }

    fmt::print(fg(fmt::color::cyan),
               "done executing UpdatePrimitiveFromZSParticles\n");
    set_output("ZSParticles", get_input("ZSParticles"));
  }
};

ZENDEFNODE(UpdatePrimitiveFromZSParticles, {
                                               {"ZSParticles"},
                                               {"ZSParticles"},
                                               {},
                                               {"MPM"},
                                           });

struct MakeZSPartition : INode {
  void apply() override {
    auto partition = std::make_shared<ZenoPartition>();
    partition->get() = typename ZenoPartition::table_t{(std::size_t)1,
                                                       zs::memsrc_e::device, 0};
    partition->requestRebuild = false;
    partition->rebuilt = false;
    set_output("ZSPartition", partition);
  }
};
ZENDEFNODE(MakeZSPartition, {
                                {},
                                {"ZSPartition"},
                                {},
                                {"MPM"},
                            });

struct MakeZSGrid : INode {
  void apply() override {
    auto dx = get_input2<float>("dx");

    std::vector<zs::PropertyTag> tags{{"m", 1}, {"v", 3}};

    auto grid = std::make_shared<ZenoGrid>();
    grid->transferScheme = get_input2<std::string>("transfer");
    // default is "apic"
    if (grid->transferScheme == "flip")
      tags.emplace_back(zs::PropertyTag{"vstar", 3});
    else if (grid->transferScheme == "apic")
      ;
    else if (grid->transferScheme == "aflip")
      tags.emplace_back(zs::PropertyTag{"vstar", 3});
    else if (grid->transferScheme == "boundary")
      tags.emplace_back(zs::PropertyTag{"nrm", 3});
    else
      throw std::runtime_error(fmt::format(
          "unrecognized transfer scheme [{}]\n", grid->transferScheme));

    grid->get() =
        typename ZenoGrid::grid_t{tags, dx, 1, zs::memsrc_e::device, 0};

    using traits = zs::grid_traits<typename ZenoGrid::grid_t>;
    fmt::print("grid of dx [{}], side_length [{}], block_size [{}]\n",
               grid->get().dx, traits::side_length, traits::block_size);
    set_output("ZSGrid", grid);
  }
};
ZENDEFNODE(MakeZSGrid,
           {
               {{"float", "dx", "0.1"}, {"string", "transfer", "apic"}},
               {"ZSGrid"},
               {},
               {"MPM"},
           });

struct MakeZSLevelSet : INode {
  void apply() override {
    auto dx = get_input2<float>("dx");

    std::vector<zs::PropertyTag> tags{{"sdf", 1}};

    auto ls = std::make_shared<ZenoLevelSet>();
    ls->transferScheme = get_param<std::string>("transfer");
    auto cateStr = get_param<std::string>("category");

    // default is "cellcentered"
    if (cateStr == "staggered")
      tags.emplace_back(zs::PropertyTag{"vel", 3});
    // default is "unknown"
    if (ls->transferScheme == "unknown")
      ;
    else if (ls->transferScheme == "flip")
      tags.emplace_back(zs::PropertyTag{"vstar", 3});
    else if (ls->transferScheme == "apic")
      ;
    else if (ls->transferScheme == "aflip")
      tags.emplace_back(zs::PropertyTag{"vstar", 3});
    else if (ls->transferScheme == "boundary")
      tags.emplace_back(zs::PropertyTag{"nrm", 3});
    else
      throw std::runtime_error(fmt::format(
          "unrecognized transfer scheme [{}]\n", ls->transferScheme));

    if (cateStr == "collocated") {
      auto tmp = typename ZenoLevelSet::template spls_t<zs::grid_e::collocated>{
          tags, dx, 1, zs::memsrc_e::device, 0};
      tmp.reset(zs::cuda_exec(), 0);
      ls->getLevelSet() = std::move(tmp);
    } else if (cateStr == "cellcentered") {
      auto tmp =
          typename ZenoLevelSet::template spls_t<zs::grid_e::cellcentered>{
              tags, dx, 1, zs::memsrc_e::device, 0};
      tmp.reset(zs::cuda_exec(), 0);
      ls->getLevelSet() = std::move(tmp);
    } else if (cateStr == "staggered") {
      auto tmp = typename ZenoLevelSet::template spls_t<zs::grid_e::staggered>{
          tags, dx, 1, zs::memsrc_e::device, 0};
      tmp.reset(zs::cuda_exec(), 0);
      ls->getLevelSet() = std::move(tmp);
    } else if (cateStr == "const_velocity") {
      auto v = get_input<zeno::NumericObject>("aux")->get<zeno::vec3f>();
      ls->getLevelSet() = typename ZenoLevelSet::uniform_vel_ls_t{
          zs::vec<float, 3>{v[0], v[1], v[2]}};
    } else
      throw std::runtime_error(
          fmt::format("unknown levelset (grid) category [{}].", cateStr));

    zs::match([](const auto &lsPtr) {
      if constexpr (zs::is_spls_v<typename RM_CVREF_T(lsPtr)::element_type>) {
        using spls_t = typename RM_CVREF_T(lsPtr)::element_type;
        fmt::print(
            "levelset [{}] of dx [{}, {}], side_length [{}], block_size [{}]\n",
            spls_t::category, 1.f / lsPtr->_i2wSinv(0, 0), lsPtr->_grid.dx,
            spls_t::side_length, spls_t::block_size);
      } else if constexpr (zs::is_same_v<
                               typename RM_CVREF_T(lsPtr)::element_type,
                               typename ZenoLevelSet::uniform_vel_ls_t>) {
        fmt::print("uniform velocity field: {}, {}, {}\n", lsPtr->vel[0],
                   lsPtr->vel[1], lsPtr->vel[2]);
      } else {
        throw std::runtime_error(
            fmt::format("invalid levelset [{}] initialized in basicls.",
                        zs::get_var_type_str(lsPtr)));
      }
    })(ls->getBasicLevelSet()._ls);
    set_output("ZSLevelSet", std::move(ls));
  }
};
ZENDEFNODE(MakeZSLevelSet,
           {
               {{"float", "dx", "0.1"}, "aux"},
               {"ZSLevelSet"},
               {{"enum unknown apic flip aflip boundary", "transfer",
                 "unknown"},
                {"enum cellcentered collocated staggered const_velocity",
                 "category", "cellcentered"}},
               {"SOP"},
           });

struct ToZSBoundary : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToZSBoundary\n");
    auto boundary = std::make_shared<ZenoBoundary>();

    auto type = get_param<std::string>("type");
    auto queryType = [&type]() -> zs::collider_e {
      if (type == "sticky" || type == "Sticky")
        return zs::collider_e::Sticky;
      else if (type == "slip" || type == "Slip")
        return zs::collider_e::Slip;
      else if (type == "separate" || type == "Separate")
        return zs::collider_e::Separate;
      return zs::collider_e::Sticky;
    };

    boundary->zsls = get_input<ZenoLevelSet>("ZSLevelSet");

    boundary->type = queryType();

    // translation
    if (has_input("translation")) {
      auto b = get_input<NumericObject>("translation")->get<vec3f>();
      boundary->b = zs::vec<float, 3>{b[0], b[1], b[2]};
    }
    if (has_input("translation_rate")) {
      auto dbdt = get_input<NumericObject>("translation_rate")->get<vec3f>();
      boundary->dbdt = zs::vec<float, 3>{dbdt[0], dbdt[1], dbdt[2]};
      // fmt::print("dbdt assigned as {}, {}, {}\n", boundary->dbdt[0],
      //            boundary->dbdt[1], boundary->dbdt[2]);
    }
    // scale
    if (has_input("scale")) {
      auto s = get_input<NumericObject>("scale")->get<float>();
      boundary->s = s;
    }
    if (has_input("scale_rate")) {
      auto dsdt = get_input<NumericObject>("scale_rate")->get<float>();
      boundary->dsdt = dsdt;
    }
    // rotation
    if (has_input("ypr_angles")) {
      auto yprAngles = get_input<NumericObject>("ypr_angles")->get<vec3f>();
      auto rot = zs::Rotation<float, 3>{yprAngles[0], yprAngles[1],
                                        yprAngles[2], zs::degree_c, zs::ypr_c};
      boundary->R = rot;
    }
    { boundary->omega = zs::AngularVelocity<float, 3>{}; }

    fmt::print(fg(fmt::color::cyan), "done executing ToZSBoundary\n");
    set_output("ZSBoundary", boundary);
  }
};
ZENDEFNODE(ToZSBoundary, {
                             {"ZSLevelSet", "translation", "translation_rate",
                              "scale", "scale_rate", "ypr_angles"},
                             {"ZSBoundary"},
                             {{"string", "type", "sticky"}},
                             {"MPM"},
                         });

struct StepZSBoundary : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing StepZSBoundary\n");

    auto boundary = get_input<ZenoBoundary>("ZSBoundary");
    auto dt = get_input2<float>("dt");

    // auto oldB = boundary->b;

    boundary->s += boundary->dsdt * dt;
    boundary->b += boundary->dbdt * dt;

#if 0
    auto b = boundary->b;
    auto dbdt = boundary->dbdt;
    auto delta = dbdt * dt;
    fmt::print("({}, {}, {}) + ({}, {}, {}) * {} -> ({}, {}, {})\n", oldB[0],
               oldB[1], oldB[2], dbdt[0], dbdt[1], dbdt[2], dt, delta[0],
               delta[1], delta[2]);
#endif

    fmt::print(fg(fmt::color::cyan), "done executing StepZSBoundary\n");
    set_output("ZSBoundary", boundary);
  }
};
ZENDEFNODE(StepZSBoundary, {
                               {"ZSBoundary", {"float", "dt", "0"}},
                               {"ZSBoundary"},
                               {},
                               {"MPM"},
                           });

/// conversion

struct ZSParticlesToPrimitiveObject : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing "
                                      "ZSParticlesToPrimitiveObject\n");
    auto zsprim = get_input<ZenoParticles>("ZSParticles");
    auto &zspars = zsprim->getParticles();
    const auto size = zspars.size();

    auto prim = std::make_shared<PrimitiveObject>();
    prim->resize(size);

    using namespace zs;
    auto cudaExec = cuda_exec().device(0);

    static_assert(sizeof(zs::vec<float, 3>) == sizeof(zeno::vec3f),
                  "zeno::vec3f != zs::vec<float, 3>");
    /// verts
    for (auto &&prop : zspars.getPropertyTags()) {
      if (prop.numChannels == 3) {
        zs::Vector<zs::vec<float, 3>> dst{size, memsrc_e::device, 0};
        cudaExec(zs::range(size),
                 [zspars = zs::proxy<execspace_e::cuda>({}, zspars),
                  dst = zs::proxy<execspace_e::cuda>(dst),
                  name = prop.name] __device__(size_t pi) mutable {
                   // dst[pi] = zspars.pack<3>(name, pi);
                   dst[pi] = zspars.pack<3>(name, pi);
                 });
        copy(zs::mem_device,
             prim->add_attr<zeno::vec3f>(prop.name.asString()).data(),
             dst.data(), sizeof(zeno::vec3f) * size);
      } else if (prop.numChannels == 1) {
        zs::Vector<float> dst{size, memsrc_e::device, 0};
        cudaExec(zs::range(size),
                 [zspars = zs::proxy<execspace_e::cuda>({}, zspars),
                  dst = zs::proxy<execspace_e::cuda>(dst),
                  name = prop.name] __device__(size_t pi) mutable {
                   dst[pi] = zspars(name, pi);
                 });
        copy(zs::mem_device, prim->add_attr<float>(prop.name.asString()).data(),
             dst.data(), sizeof(float) * size);
      }
    }
/// elements
#if 1
    if (zsprim->isMeshPrimitive()) {
      auto &zseles = zsprim->getQuadraturePoints();
      int nVertsPerEle = static_cast<int>(zsprim->category) + 1;
      auto numEle = zseles.size();
      switch (zsprim->category) {
      case ZenoParticles::curve: {
        zs::Vector<zs::vec<int, 2>> dst{numEle, memsrc_e::device, 0};
        cudaExec(zs::range(numEle),
                 [zseles = zs::proxy<execspace_e::cuda>({}, zseles),
                  dst = zs::proxy<execspace_e::cuda>(
                      dst)] __device__(size_t ei) mutable {
                   dst[ei] = zseles.pack<2>("inds", ei).reinterpret_bits<int>();
                 });

        prim->lines.resize(numEle);
        auto &lines = prim->lines.values;
        copy(zs::mem_device, lines.data(), dst.data(),
             sizeof(zeno::vec2i) * numEle);
      } break;
      case ZenoParticles::surface: {
        zs::Vector<zs::vec<int, 3>> dst{numEle, memsrc_e::device, 0};
        cudaExec(zs::range(numEle),
                 [zseles = zs::proxy<execspace_e::cuda>({}, zseles),
                  dst = zs::proxy<execspace_e::cuda>(
                      dst)] __device__(size_t ei) mutable {
                   dst[ei] = zseles.pack<3>("inds", ei).reinterpret_bits<int>();
                 });

        prim->tris.resize(numEle);
        auto &tris = prim->tris.values;
        copy(zs::mem_device, tris.data(), dst.data(),
             sizeof(zeno::vec3i) * numEle);
      } break;
      case ZenoParticles::tet: {
        zs::Vector<zs::vec<int, 4>> dst{numEle, memsrc_e::device, 0};
        cudaExec(zs::range(numEle),
                 [zseles = zs::proxy<execspace_e::cuda>({}, zseles),
                  dst = zs::proxy<execspace_e::cuda>(
                      dst)] __device__(size_t ei) mutable {
                   dst[ei] = zseles.pack<4>("inds", ei).reinterpret_bits<int>();
                 });

        prim->quads.resize(numEle);
        auto &quads = prim->quads.values;
        copy(zs::mem_device, quads.data(), dst.data(),
             sizeof(zeno::vec4i) * numEle);
      } break;
      default:
        break;
      };
    }
#endif
    fmt::print(fg(fmt::color::cyan), "done executing "
                                     "ZSParticlesToPrimitiveObject\n");
    set_output("prim", prim);
  }
};

ZENDEFNODE(ZSParticlesToPrimitiveObject, {
                                             {"ZSParticles"},
                                             {"prim"},
                                             {},
                                             {"MPM"},
                                         });

struct WriteZSParticles : zeno::INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing WriteZSParticles\n");
    auto &pars = get_input<ZenoParticles>("ZSParticles")->getParticles();
    auto path = get_param<std::string>("path");
    auto cudaExec = zs::cuda_exec().device(0);
    zs::Vector<zs::vec<float, 3>> pos{pars.size(), zs::memsrc_e::um, 0};
    zs::Vector<float> vms{pars.size(), zs::memsrc_e::um, 0};
    cudaExec(zs::range(pars.size()),
             [pos = zs::proxy<zs::execspace_e::cuda>(pos),
              vms = zs::proxy<zs::execspace_e::cuda>(vms),
              pars = zs::proxy<zs::execspace_e::cuda>(
                  {}, pars)] __device__(size_t pi) mutable {
               pos[pi] = pars.pack<3>("pos", pi);
               vms[pi] = pars("vms", pi);
             });
    std::vector<std::array<float, 3>> posOut(pars.size());
    std::vector<float> vmsOut(pars.size());
    copy(zs::mem_device, posOut.data(), pos.data(),
         sizeof(zeno::vec3f) * pars.size());
    copy(zs::mem_device, vmsOut.data(), vms.data(),
         sizeof(float) * pars.size());

    zs::write_partio_with_stress<float, 3>(path, posOut, vmsOut);
    fmt::print(fg(fmt::color::cyan), "done executing WriteZSParticles\n");
  }
};

ZENDEFNODE(WriteZSParticles, {
                                 {"ZSParticles"},
                                 {},
                                 {{"string", "path", ""}},
                                 {"MPM"},
                             });

struct ComputeVonMises : INode {
  template <typename Model>
  void computeVms(zs::CudaExecutionPolicy &cudaPol, const Model &model,
                  typename ZenoParticles::particles_t &pars, int option) {
    using namespace zs;
    cudaPol(range(pars.size()), [pars = proxy<execspace_e::cuda>({}, pars),
                                 model, option] __device__(size_t pi) mutable {
      auto F = pars.pack<3, 3>("F", pi);
      auto [U, S, V] = math::svd(F);
      auto cauchy = model.dpsi_dsigma(S) * S / S.prod();

      auto diff = cauchy;
      for (int d = 0; d != 3; ++d)
        diff(d) -= cauchy((d + 1) % 3);

      auto vms = ::sqrt(diff.l2NormSqr() * 0.5f);
      pars("vms", pi) = option ? ::log10(vms + 1) : vms;
    });
  }
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ComputeVonMises\n");
    auto zspars = get_input<ZenoParticles>("ZSParticles");
    auto &pars = zspars->getParticles();
    auto model = zspars->getModel();
    auto option = get_param<int>("by_log1p(base10)");

    auto cudaExec = zs::cuda_exec().device(0);
    zs::match([&](auto &elasticModel) {
      computeVms(cudaExec, elasticModel, pars, option);
    })(model.getElasticModel());

    set_output("ZSParticles", std::move(zspars));
    fmt::print(fg(fmt::color::cyan), "done executing ComputeVonMises\n");
  }
};

ZENDEFNODE(ComputeVonMises, {
                                {"ZSParticles"},
                                {"ZSParticles"},
                                {{"int", "by_log1p(base10)", "1"}},
                                {"MPM"},
                            });

} // namespace zeno