#include "hip/hip_runtime.h"
#include "../Structures.hpp"
#include "../Utils.hpp"

#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/io/ParticleIO.hpp"
#include "zensim/math/matrix/QRSVD.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/simulation/Utils.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

namespace zeno {

/// sparsity
struct ZSPartitionForZSParticles : INode {
  void apply() override {
    using namespace zs;
    fmt::print(fg(fmt::color::green),
               "begin executing ZSPartitionForZSParticles\n");
    auto table = get_input<ZenoPartition>("ZSPartition");
    auto &partition = table->get();
    auto zsgrid = get_input<ZenoGrid>("ZSGrid");
    auto &grid = zsgrid->get();
    auto cudaPol = cuda_exec().device(0);

    bool cached = get_param<std::string>("strategy") == "cache" ? true : false;
    if (!table->requestRebuild && cached && table->hasTags()) {
      zs::Vector<int> bRebuild{1, memsrc_e::device, 0};
      bRebuild.setVal(0);
      cudaPol(range(table->numBoundaryEntries()), // table->getTags(),
              [tags = proxy<execspace_e::cuda>(table->getTags()),
               flag = proxy<execspace_e::cuda>(
                   bRebuild)] __device__(auto i) mutable {
                auto tag = tags[i];
                if (tag == 1 && flag[0] == 0) {
                  // atomic_cas(exec_cuda, &flag[0], 0, 1);
                  flag[0] = 1;
                }
              });
      // no boundary entry touched yet, no need for rebuild
      if (bRebuild.getVal() == 0) {
        table->rebuilt = false;
        fmt::print(fg(fmt::color::cyan),
                   "done executing ZSPartitionForZSParticles (skipping full "
                   "rebuild)\n");
        set_output("ZSPartition", table);
        return;
      }
    }

    auto parObjPtrs = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");

    std::size_t cnt = 0;
    for (auto &&parObjPtr : parObjPtrs) {
      if (parObjPtr->category != ZenoParticles::bending)
        cnt += (std::size_t)std::ceil(parObjPtr->getParticles().size() /
                                      get_input2<float>("ppb"));
      if (parObjPtr->isMeshPrimitive())
        cnt += (std::size_t)std::ceil(parObjPtr->numElements() /
                                      get_input2<float>("ppb"));
    }
    if (partition.size() * 2 < cnt)
      partition.resize(cudaPol, cnt * 2);

    using Partition = typename ZenoPartition::table_t;
    // reset
    partition.reset(cudaPol, true);

    using grid_t = typename ZenoGrid::grid_t;
    static_assert(grid_traits<grid_t>::is_power_of_two,
                  "grid side_length should be power of two");

    for (auto &&parObjPtr : parObjPtrs) {
      auto &pars = parObjPtr->getParticles();
      cudaPol(range(pars.size()),
              [pars = proxy<execspace_e::cuda>({}, pars),
               table = proxy<execspace_e::cuda>(partition),
               dxinv = 1.f / grid.dx] __device__(size_t pi) mutable {
                auto x = pars.template pack<3>("pos", pi);
                auto c = (x * dxinv - 0.5);
                typename Partition::key_t coord{};
                for (int d = 0; d != 3; ++d)
                  coord[d] = lower_trunc(c[d]);
                table.insert(coord - (coord & (grid_t::side_length - 1)));
              });
      if (parObjPtr->isMeshPrimitive()) { // including tracker, but not bending
        auto &eles = parObjPtr->getQuadraturePoints();
        cudaPol(range(eles.size()),
                [eles = proxy<execspace_e::cuda>({}, eles),
                 table = proxy<execspace_e::cuda>(partition),
                 dxinv = 1.f / grid.dx] __device__(size_t ei) mutable {
                  auto x = eles.template pack<3>("pos", ei);
                  auto c = (x * dxinv - 0.5);
                  typename Partition::key_t coord{};
                  for (int d = 0; d != 3; ++d)
                    coord[d] = lower_trunc(c[d]);
                  table.insert(coord - (coord & (grid_t::side_length - 1)));
                });
      }
    }
    if (cached) {
      table->reserveTags();
      identify_boundary_indices(cudaPol, *table, wrapv<grid_t::side_length>{});
    }
    table->rebuilt = true;
    if (table->requestRebuild) // request processed
      table->requestRebuild = false;

    fmt::print("partition of [{}] blocks for {} particles\n", partition.size(),
               cnt);

    fmt::print(fg(fmt::color::cyan),
               "done executing ZSPartitionForZSParticles\n");
    set_output("ZSPartition", table);
  }
};

ZENDEFNODE(ZSPartitionForZSParticles,
           {
               {"ZSPartition", "ZSGrid", "ZSParticles", {"float", "ppb", "1"}},
               {"ZSPartition"},
               {{"enum force cache", "strategy", "force"}},
               {"MPM"},
           });

struct ExpandZSPartition : INode {
  using grid_t = typename ZenoGrid::grid_t;

  template <typename VecT,
            zs::enable_if_t<std::is_integral_v<typename VecT::value_type>> = 0>
  static constexpr int getDirIndex(const zs::VecInterface<VecT> &dir) noexcept {
    // dir [-1, 1]
    auto offset = dir + 1;
    return offset[0] * 9 + offset[1] * 3 + offset[2];
  }

  void registerNewBlockEntries(zs::CudaExecutionPolicy &policy,
                               typename ZenoPartition::table_t &table,
                               zs::Vector<zs::i32> &dirTags, std::size_t offset,
                               std::size_t numNewEntries) const {
    using namespace zs;
    policy(range(numNewEntries), [table = proxy<execspace_e::cuda>(table),
                                  dirTags = proxy<execspace_e::cuda>(dirTags),
                                  offset] __device__(int bi) mutable {
      using table_t = RM_CVREF_T(table);
      bi += offset;
      auto bcoord = table._activeKeys[bi];
      using key_t = typename table_t::key_t;
      for (auto ijk : ndrange<3>(3)) {
        auto dir = make_vec<int>(ijk) - 1; // current expanding direction
        if (auto neighborNo = table.query(
                bcoord + dir * (int)grid_traits<grid_t>::side_length);
            neighborNo != table_t::sentinel_v)
          atomic_or(exec_cuda, &dirTags[neighborNo],
                    (i32)(1 << getDirIndex(-dir)));
      }
    });
  }
  void apply() override {
    using namespace zs;
    fmt::print(fg(fmt::color::green), "begin executing ExpandZSPartition\n");
    auto table = get_input<ZenoPartition>("ZSPartition");
    auto &partition = table->get();
    auto offset = get_param<int>("offset");
    auto extent = get_param<int>("extent");

    auto lower = std::abs(offset);
    auto higher = std::abs(offset + extent);
    auto niters = std::min(lower, higher);

    if (niters == 0 || !table->rebuilt) { // only expand after a fresh rebuilt
      fmt::print(fg(fmt::color::cyan), "done executing ExpandZSPartition "
                                       "(skipping expansion due to caching)\n");
      set_output("ZSPartition", std::move(table));
      return;
    }

    auto prevCnt = partition.size();
    fmt::print(
        "expect {} iterations to complete partition expansion of {} entries.\n",
        niters, prevCnt);

    // at least 27 bits for 3d[-1, 1] range
    Vector<i32> dirs{partition.get_allocator(), (std::size_t)prevCnt};
    dirs.reset(0);
    auto cudaPol = cuda_exec().device(0);
    registerNewBlockEntries(cudaPol, partition, dirs, 0, prevCnt);

    static_assert(grid_traits<grid_t>::is_power_of_two,
                  "grid side_length should be power of two");

    for (int iter = 0; iter != niters; ++iter) {
      cudaPol(range(prevCnt), [dirs = proxy<execspace_e::cuda>(dirs),
                               table = proxy<execspace_e::cuda>(
                                   partition)] __device__(auto bi) mutable {
        using table_t = RM_CVREF_T(table);
        auto blockid = table._activeKeys[bi];
        for (auto ijk : ndrange<3>(3)) {
          auto dir = make_vec<int>(ijk) - 1; // current expanding direction
          auto dirNo = getDirIndex(dir);
          if (dirs[bi] & (1 << dirNo)) // check if this direction is necessary
            continue;
          table.insert(blockid + dir * (int)grid_traits<grid_t>::side_length);
        }
      });
      auto curCnt = partition.size();
      fmt::print("partition insertion iter [{}]: [{}] blocks -> [{}] blocks\n",
                 iter, prevCnt, curCnt);

      dirs.resize(curCnt, 0);
      fmt::print("done dirtag resize\n");
      registerNewBlockEntries(cudaPol, partition, dirs, prevCnt,
                              curCnt - prevCnt);

      prevCnt = curCnt;
    }
    if (table->hasTags()) {
      // identify_boundary_indices(cudaPol, *table,
      // wrapv<grid_t::side_length>{});
      using Ti = ZenoPartition::Ti;
      using indices_t = ZenoPartition::indices_t;
      indices_t marks{partition.get_allocator(), (std::size_t)prevCnt + 1},
          offsets{partition.get_allocator(), (std::size_t)prevCnt + 1};
      cudaPol(Collapse{prevCnt}, [dirs = proxy<execspace_e::cuda>(dirs),
                                  marks = proxy<execspace_e::cuda>(
                                      marks)] __device__(Ti bi) mutable {
        marks[bi] = dirs[bi] != (i32)27;
      });
      exclusive_scan(cudaPol, std::begin(marks), std::end(marks),
                     std::begin(offsets));
      auto bouCnt = offsets.getVal(prevCnt);

      auto &boundaryIndices = table->getBoundaryIndices();
      boundaryIndices.resize(bouCnt);
      cudaPol(range(prevCnt),
              [marks = proxy<execspace_e::cuda>(marks),
               boundaryIndices = proxy<execspace_e::cuda>(boundaryIndices),
               offsets = proxy<execspace_e::cuda>(
                   offsets)] __device__(Ti bi) mutable {
                if (marks[bi])
                  boundaryIndices[offsets[bi]] = bi;
              });

      auto &tags = table->getTags();
      tags.resize(bouCnt);
      tags.reset(0);
    }

    fmt::print(fg(fmt::color::cyan), "done executing ExpandZSPartition\n");

    set_output("ZSPartition", std::move(table));
  }
};

ZENDEFNODE(ExpandZSPartition,
           {
               {"ZSPartition"},
               {"ZSPartition"},
               {{"int", "offset", "0"}, {"int", "extent", "2"}},
               {"MPM"},
           });

/// grid
struct ZSGridFromZSPartition : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green),
               "begin executing ZSGridFromZSPartition\n");
    auto zspartition = get_input<ZenoPartition>("ZSPartition");
    auto &partition = zspartition->get();
    auto cnt = partition.size();

    auto zsgrid = get_input<ZenoGrid>("ZSGrid");
    zsgrid->partition = zspartition;
    auto &grid = zsgrid->get();
    grid.resize(cnt);

    using namespace zs;
    auto cudaPol = cuda_exec().device(0);
    // clear grid
    cudaPol(Collapse{cnt, ZenoGrid::grid_t::block_space()},
            [grid = proxy<execspace_e::cuda>({}, grid)] __device__(
                int bi, int ci) mutable {
              auto block = grid.block(bi);
              const auto nchns = grid.numChannels();
              for (int i = 0; i != nchns; ++i)
                block(i, ci) = 0;
            });

    fmt::print(fg(fmt::color::cyan), "done executing ZSGridFromZSPartition\n");
    set_output("ZSGrid", zsgrid);
  }
};

ZENDEFNODE(ZSGridFromZSPartition, {
                                      {"ZSPartition", "ZSGrid"},
                                      {"ZSGrid"},
                                      {},
                                      {"MPM"},
                                  });

struct UpdateZSGrid : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing UpdateZSGrid\n");
    // auto dt = get_input("dt")->as<zeno::NumericObject>()->get<float>();
    auto maxVelSqr = std::make_shared<NumericObject>();

    auto &partition = get_input<ZenoPartition>("ZSPartition")->get();
    auto zsgrid = get_input<ZenoGrid>("ZSGrid");
    auto &grid = zsgrid->get();
    auto stepDt = get_input<NumericObject>("dt")->get<float>();

    using namespace zs;
    auto gravity = get_input2<float>("gravity");
    auto accel = zs::vec<float, 3>::zeros();
    if (has_input("Accel")) {
      auto tmp = get_input<NumericObject>("Accel")->get<vec3f>();
      accel = zs::vec<float, 3>{tmp[0], tmp[1], tmp[2]};
    } else
      accel[1] = gravity;

    Vector<float> velSqr{1, zs::memsrc_e::um, 0};
    velSqr[0] = 0;
    auto cudaPol = cuda_exec().device(0);

    if (zsgrid->isPicStyle())
      cudaPol(Collapse{partition.size(), ZenoGrid::grid_t::block_space()},
              [grid = proxy<execspace_e::cuda>({}, grid),
               /*table = proxy<execspace_e::cuda>(partition), */ stepDt, accel,
               ptr = velSqr.data()] __device__(auto bi, auto ci) mutable {
                auto block = grid.block(bi);
                auto mass = block("m", ci);
                if (mass != 0.f) {
                  mass = 1.f / mass;
                  auto vel = block.pack<3>("v", ci) * mass;
#if 0
                  if (vel.norm() > 0.2) {
                    auto pos =
                        (table._activeKeys[bi] + grid.cellid_to_coord(ci)) *
                        grid.dx;
                    printf("(%f, %f, %f) vel: %f, %f, %f\n", pos[0], pos[1],
                           pos[2], vel[0], vel[1], vel[2]);
                  }
#endif
                  vel += accel * stepDt;
                  block.set("v", ci, vel);
                  /// cfl dt
                  auto velSqr = vel.l2NormSqr();
                  atomic_max(exec_cuda, ptr, velSqr);
                }
              });
    else if (zsgrid->isFlipStyle())
      cudaPol(Collapse{partition.size(), ZenoGrid::grid_t::block_space()},
              [grid = proxy<execspace_e::cuda>({}, grid), stepDt, accel,
               ptr = velSqr.data()] __device__(auto bi, auto ci) mutable {
                auto block = grid.block(bi);
                auto mass = block("m", ci);
                if (mass != 0.f) {
                  mass = 1.f / mass;

                  auto vel = block.pack<3>("v", ci) * mass;
                  // vel += accel * stepDt;
                  block.set("v", ci, vel);

                  auto vstar =
                      block.pack<3>("vstar", ci) * mass + vel + accel * stepDt;
                  block.set("vstar", ci, vstar);

                  /// cfl dt
                  auto velSqr = vstar.l2NormSqr();
                  atomic_max(exec_cuda, ptr, velSqr);
                }
              });
    else if (zsgrid->transferScheme == "boundary")
      cudaPol(Collapse{partition.size(), ZenoGrid::grid_t::block_space()},
              [grid = proxy<execspace_e::cuda>({}, grid)] __device__(
                  auto bi, auto ci) mutable {
                auto block = grid.block(bi);
                auto mass = block("m", ci);
                if (mass != 0.f) {
                  {
                    mass = 1.f / mass;
                    auto vel = block.pack<3>("v", ci) * mass;
                    block.set("v", ci, vel);
                  }
                  auto nrm = block.pack<3>("nrm", ci) * mass;
                  block.set("nrm", ci, nrm.normalized());
                }
              });

#if 0
    cudaPol(Collapse{partition.size(), ZenoGrid::grid_t::block_space()},
            [grid = proxy<execspace_e::cuda>({}, grid),
             table = proxy<execspace_e::cuda>(
                 partition)] __device__(auto bi, auto ci) mutable {
              auto block = grid.block(bi);
              auto mass = block("m", ci);
              if (mass != 0.f) {
                auto vel = block.pack<3>("v", ci);
#if 1
                if ((vel(1) < -5.1 || vel(1) > -4.9) && ci == 0) {
                  auto pos =
                      (table._activeKeys[bi] + grid.cellid_to_coord(ci)) *
                      grid.dx;
                  printf("(%f, %f, %f) mass: %f, vel: %f, %f, %f\n", pos[0],
                         pos[1], pos[2], mass, vel[0], vel[1], vel[2]);
                }
#endif
              }
            });
    puts("done gridupdate check");
    getchar();
#endif

    maxVelSqr->set<float>(velSqr[0]);
    fmt::print(fg(fmt::color::cyan), "done executing GridUpdate\n");
    set_output("ZSGrid", zsgrid);
    set_output("MaxVelSqr", maxVelSqr);
  }
};

ZENDEFNODE(
    UpdateZSGrid,
    {
        {{"float", "gravity", "-9.8"}, "ZSPartition", "ZSGrid", "dt", "Accel"},
        {"ZSGrid", "MaxVelSqr"},
        {},
        {"MPM"},
    });

struct ZSReturnMapping : INode {
  template <typename PM>
  void returnMapping(zs::CudaExecutionPolicy &cudaPol,
                     typename ZenoParticles::particles_t &pars,
                     const zs::StvkWithHencky<float> &elasticModel,
                     const PM &plasticModel) const {
    using namespace zs;
    cudaPol(range(pars.size()),
            [pars = proxy<execspace_e::cuda>({}, pars),
             elasticModel = elasticModel,
             plasticModel = plasticModel] __device__(size_t pi) mutable {
              auto FeHat = pars.pack<3, 3>("F", pi);
              if constexpr (is_same_v<zs::NonAssociativeCamClay<float>,
                                      RM_CVREF_T(plasticModel)>) {
                auto logJp = pars("logJp", pi);
                if (plasticModel.project_strain(FeHat, elasticModel, logJp)) {
                  pars("logJp", pi) = logJp;
                  pars.tuple<9>("F", pi) = FeHat;
                }
              } else { // vm, dp
                if (plasticModel.project_strain(FeHat, elasticModel))
                  pars.tuple<9>("F", pi) = FeHat;
              }
            });
  }
  void return_mapping_surface(zs::CudaExecutionPolicy &cudaPol,
                              typename ZenoParticles::particles_t &eles) const {
    using namespace zs;
    cudaPol(range(eles.size()), [eles = proxy<execspace_e::cuda>(
                                     {}, eles)] __device__(size_t pi) mutable {
      auto d = eles.pack<3, 3>("d", pi);
      // hard code ftm
      constexpr auto gamma = 0.f;
      constexpr auto k = 40000.f;
      constexpr auto friction_coeff = 0.f;
      // constexpr auto friction_coeff = 0.17f;
      auto [Q, R] = math::gram_schmidt(d);
      auto apply = [&, &Q = Q, &R = R]() {
        d = Q * R;
        eles.tuple<9>("d", pi) = d;
        eles.tuple<9>("F", pi) = d * eles.pack<3, 3>("DmInv", pi);
      };
      if (gamma == 0.f) {
        R(0, 2) = R(1, 2) = 0;
        R(2, 2) = zs::min(R(2, 2), 1.f);
        apply();
      } else if (R(2, 2) > 1) {
        R(0, 2) = R(1, 2) = 0;
        R(2, 2) = 1;
        apply();
      } else if (R(2, 2) <= 0) { // inversion
        R(0, 2) = R(1, 2) = 0;
        R(2, 2) = zs::max(R(2, 2), -1.f);
        apply();
      } else if (R(2, 2) < 1) {
        auto rr = R(0, 2) * R(0, 2) + R(1, 2) * R(1, 2);
        auto r33_m_1 = R(2, 2) - 1;
        auto gamma_over_k = gamma / k;
        auto zz = friction_coeff * r33_m_1 * r33_m_1; // normal traction
        if (gamma_over_k * gamma_over_k * rr - zz * zz > 0) {
          auto scale = zz / (gamma_over_k * zs::sqrt(rr));
          R(0, 2) *= scale;
          R(1, 2) *= scale;
          apply();
        }
      }
    });
  }
  void return_mapping_curve(zs::CudaExecutionPolicy &cudaPol,
                            const zs::StvkWithHencky<float> &stvkModel,
                            typename ZenoParticles::particles_t &eles) const {
    using namespace zs;
    bool materialParamOverride =
        eles.hasProperty("mu") && eles.hasProperty("lam");
    // drucker prager for stvk elastic model
    // ref: libwetcloth, Jiang 2017
    cudaPol(range(eles.size()),
            [eles = proxy<execspace_e::cuda>({}, eles), stvkModel = stvkModel,
             materialParamOverride] __device__(size_t pi) mutable {
              // hard code ftm
              constexpr auto gamma = 10.f;
              constexpr auto alpha = 0.f;
              constexpr auto beta = 0.f;
              constexpr auto alpha_tangent = 0.f;
              constexpr auto cohesion = 0.f; // no cohesion ftm
              bool projected = false;
              if (materialParamOverride) {
                stvkModel.mu = eles("mu", pi);
                stvkModel.lam = eles("lam", pi);
              }
              auto d = eles.pack<3, 3>("d", pi);
              auto [Q, R] = math::gram_schmidt(d);

              using vec2 = zs::vec<float, 2>;
              using mat2 = zs::vec<float, 2, 2>;

              mat2 R_hat{R(1, 1), R(1, 2), R(2, 1), R(2, 2)};
              auto [U, S, V] = math::qr_svd(R_hat);
              auto eps =
                  S.abs().max(limits<float>::epsilon() * 128).log() - cohesion;
              auto eps_trace = eps.sum() /*+ logJp*/;
              if (eps_trace < 0) {
                auto eps_hat = eps - 0.5f * eps_trace;
                auto eps_hat_norm = eps_hat.norm();
                auto dgp = eps_hat_norm + (stvkModel.mu + stvkModel.lam) /
                                              stvkModel.mu * eps_trace * alpha;
                if (eps_hat_norm < limits<float>::epsilon())
                  eps = eps.zeros() + cohesion;
                else
                  eps = eps - dgp / eps_hat_norm * eps_hat + cohesion;
              } else {
                eps = eps.zeros() + cohesion;
              }
              S = eps.exp();
              auto R2 = diag_mul(U, S) * V.transpose();
              R(1, 1) = R2(0, 0);
              R(1, 2) = R2(0, 1);
              R(2, 1) = R2(1, 0);
              R(2, 2) = R2(1, 1);

              auto tau_hat = stvkModel.first_piola(R2) * R2.transpose();
              auto p_cohesion =
                  (stvkModel.mu * 2 + stvkModel.lam * 2) * cohesion;
              auto p = zs::min(trace(tau_hat) / 2, p_cohesion);

              auto r = vec2{R(0, 1), R(0, 2)};
              auto gammaRr = gamma * (R2 * r).norm();
              auto f = gammaRr + alpha_tangent * p;

#if 1
              // Jiang
              if (gamma == 0.f) {
                R(0, 1) = R(0, 2) = 0;
              } else if (f > p_cohesion) {
                auto scale = (p_cohesion - alpha_tangent * p) / gammaRr;
                r *= scale;
                R(0, 1) = r(0);
                R(0, 2) = r(1);
              }
#else
              // Raymond
              const auto ff =
                  stvkModel.mu * zs::sqrt(sqr(R(0, 1)) + sqr(R(0, 2)));
              auto tmp = eps / S;
              const auto fn =
                  (2.0f * stvkModel.mu * tmp + stvkModel.lam * eps.sum() / S)
                      .norm() *
                  0.5f;
              if (ff > 0 && ff > fn * beta) {
                const auto scale = zs::min(1.f, beta * fn / ff);
                R(0, 1) *= scale;
                R(0, 2) *= scale;
              }
#endif

              d = Q * R;
              eles.tuple<9>("d", pi) = d;
              eles.tuple<9>("F", pi) = d * eles.pack<3, 3>("DmInv", pi);
            });
  }
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ZSReturnMapping\n");

    auto parObjPtrs = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");

    using namespace zs;
    auto cudaPol = cuda_exec().device(0);

    for (auto &&parObjPtr : parObjPtrs) {
      auto &pars = parObjPtr->getParticles();
      if (parObjPtr->category == ZenoParticles::mpm) {
        if (parObjPtr->getModel().hasPlasticity()) {
          match(
              [this, &cudaPol,
               &pars](const zs::StvkWithHencky<float> &elasticModel,
                      const auto &plasticModel)
                  -> std::enable_if_t<
                      !is_same_v<RM_CVREF_T(plasticModel), std::monostate>> {
                returnMapping(cudaPol, pars, elasticModel, plasticModel);
              },
              [](...) {
                throw std::runtime_error(
                    "unsupported elasto-plasticity models");
              })(parObjPtr->getModel().getElasticModel(),
                 parObjPtr->getModel().getPlasticModel());
        }
      } else if (parObjPtr->category == ZenoParticles::tracker) {
      } else {
        auto &eles = parObjPtr->getQuadraturePoints();
        if (parObjPtr->category == ZenoParticles::surface)
          return_mapping_surface(cudaPol, eles);
        else if (parObjPtr->category == ZenoParticles::curve) {
          const auto &models = parObjPtr->getModel();
          match(
              [this, &eles, &cudaPol](const StvkWithHencky<float> &stvkModel) {
                // use drucker prager plasticity for friction handling
                return_mapping_curve(cudaPol, stvkModel, eles);
              },
              [](...) {
                // do nothing
              })(models.getElasticModel());
        }
      }
    }

    fmt::print(fg(fmt::color::cyan), "done executing ZSReturnMapping\n");
    set_output("ZSParticles", get_input("ZSParticles"));
  }
};

ZENDEFNODE(ZSReturnMapping, {
                                {"ZSParticles"},
                                {"ZSParticles"},
                                {},
                                {"MPM"},
                            });

} // namespace zeno