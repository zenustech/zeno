#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/Logger.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/geometry/PoissonDisk.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

#include "../geometry/linear_system/mfcg.hpp"

#include "../geometry/kernel/calculate_facet_normal.hpp"
#include "../geometry/kernel/topology.hpp"
#include "../geometry/kernel/compute_characteristic_length.hpp"
// #include "../geometry/kernel/calculate_bisector_normal.hpp"

#include "../geometry/kernel/tiled_vector_ops.hpp"
#include "../geometry/kernel/geo_math.hpp"

#include "../geometry/kernel/calculate_edge_normal.hpp"

#include "zensim/container/Bvh.hpp"
#include "zensim/container/Bvs.hpp"
#include "zensim/container/Bvtt.hpp"

#include "collision_energy/vertex_face_sqrt_collision.hpp"
#include "collision_energy/vertex_face_collision.hpp"
// #include "collision_energy/edge_edge_sqrt_collision.hpp"
// #include "collision_energy/edge_edge_collision.hpp"

#include "collision_energy/evaluate_collision.hpp"
#include "../geometry/kernel/intersection.hpp"

#include "zensim/math/matrix/SparseMatrix.hpp"

namespace zeno {

#define MAX_FP_COLLISION_PAIRS 4

#define USE_SPARSE_MATRIX


template <typename SpmatT, typename VecTM, typename VecTI,
          zs::enable_if_all<VecTM::dim == 2, VecTM::template range_t<0>::value == VecTM::template range_t<1>::value,
                            VecTI::dim == 1, VecTI::extent * 3 == VecTM::template range_t<0>::value> = 0>
__forceinline__ __device__ void
update_hessian(cooperative_groups::thread_block_tile<8, cooperative_groups::thread_block> &tile, SpmatT &spmat,
               const VecTI &inds, const VecTM &hess) {
    using namespace zs;
    constexpr int codim = VecTI::extent;
    using mat3 = typename SpmatT::value_type;
    const auto nnz = spmat.nnz();
    const int cap = __popc(tile.ballot(1)); // assume active pattern 0...001111 [15, 14, ..., 0]
    auto laneId = tile.thread_rank();
#pragma unroll
    for (int i = 0; i != codim; ++i) {
        auto subOffsetI = i * 3;
        auto row = inds[i];
        // diagonal
        auto loc = spmat._ptrs[row];
        auto &mat = const_cast<mat3 &>(spmat._vals[loc]);

        for (int d = laneId; d < 9; d += cap) {
            atomic_add(exec_cuda, &mat(d / 3, d % 3), hess(subOffsetI + d / 3, subOffsetI + d % 3));
        }
        // non-diagonal
        for (int j = i + 1; j < codim; ++j) {
            auto subOffsetJ = j * 3;
            auto col = inds[j];
            if (row < col) {
                auto loc = spmat.locate(row, col, zs::true_c);
                auto &mat = const_cast<mat3 &>(spmat._vals[loc]);
                for (int d = laneId; d < 9; d += cap)
                    atomic_add(exec_cuda, &mat.val(d), hess(subOffsetI + d / 3, subOffsetJ + d % 3));
            } else {
                auto loc = spmat.locate(col, row, zs::true_c);
                auto &mat = const_cast<mat3 &>(spmat._vals[loc]);
                for (int d = laneId; d < 9; d += cap)
                    atomic_add(exec_cuda, &mat.val(d), hess(subOffsetI + d % 3, subOffsetJ + d / 3));
            }
        }
    }
}
template <typename T, zs::enable_if_t<std::is_fundamental_v<T>> = 0>
__forceinline__ __device__ T tile_shfl(cooperative_groups::thread_block_tile<8, cooperative_groups::thread_block> &tile,
                                       T var, int srcLane) {
    return tile.shfl(var, srcLane);
}
template <typename VecT, zs::enable_if_t<zs::is_vec<VecT>::value> = 0>
__forceinline__ __device__ VecT tile_shfl(
    cooperative_groups::thread_block_tile<8, cooperative_groups::thread_block> &tile, const VecT &var, int srcLane) {
    VecT ret{};
    for (typename VecT::index_type i = 0; i != VecT::extent; ++i)
        ret.val(i) = tile_shfl(tile, var.val(i), srcLane);
    return ret;
}
template <typename SpmatT, typename VecTM, typename VecTI,
          zs::enable_if_all<VecTM::dim == 2, VecTM::template range_t<0>::value == VecTM::template range_t<1>::value,
                            VecTI::dim == 1, VecTI::extent * 3 == VecTM::template range_t<0>::value> = 0>
__forceinline__ __device__ void update_hessian(SpmatT &spmat, const VecTI &inds, const VecTM &hess,
                                               bool has_work = true) {
    using namespace zs;
    // constexpr int codim = VecTI::extent;
    auto tile = cg::tiled_partition<8>(cg::this_thread_block());

    u32 work_queue = tile.ballot(has_work);
    while (work_queue) {
        auto cur_rank = __ffs(work_queue) - 1;
        auto cur_work = tile_shfl(tile, hess, cur_rank);
        auto cur_index = tile.shfl(inds, cur_rank); // gather index as well
        update_hessian(tile, spmat, cur_index, cur_work);

        if (tile.thread_rank() == cur_rank)
            has_work = false;
        work_queue = tile.ballot(has_work);
    }
    return;
}

struct FleshDynamicStepping : INode {

    using T = float;
    using Ti = int;
    using dtiles_t = zs::TileVector<T,32>;
    using tiles_t = typename ZenoParticles::particles_t;
    using vec2 = zs::vec<T,2>;
    using vec3 = zs::vec<T, 3>;
    using vec12 = zs::vec<T,12>;
    using mat3 = zs::vec<T, 3, 3>;
    using mat9 = zs::vec<T,9,9>;
    using mat12 = zs::vec<T,12,12>;

    using bvh_t = zs::LBvh<3,int,T>;
    using bv_t = zs::AABBBox<3, T>;

    using pair3_t = zs::vec<Ti,3>;
    using pair4_t = zs::vec<Ti,4>;

    using spmat_t = zs::SparseMatrix<mat3, true>;

    // currently only backward euler integrator is supported
    // topology evaluation should be called before applying this node
    struct FEMDynamicSteppingSystem {
        template <typename Model>
        void computeCollisionEnergy(zs::CudaExecutionPolicy& cudaPol,const Model& model,
                dtiles_t& vtemp,
                dtiles_t& etemp,
                dtiles_t& sttemp,
                dtiles_t& setemp,
                dtiles_t& ee_buffer,
                dtiles_t& fe_buffer) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            T lambda = model.lam;
            T mu = model.mu;
        }


        void findInversion(zs::CudaExecutionPolicy& cudaPol,dtiles_t& vtemp,dtiles_t& etemp) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            TILEVEC_OPS::fill(cudaPol,vtemp,"is_inverted",(T)0.0);  
            TILEVEC_OPS::fill(cudaPol,etemp,"is_inverted",(T)0.0);  
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                    auto DmInv = quads.template pack<3,3>("IB",ei);
                    auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                    vec3 x1[4] = {vtemp.template pack<3>("xn", inds[0]),
                            vtemp.template pack<3>("xn", inds[1]),
                            vtemp.template pack<3>("xn", inds[2]),
                            vtemp.template pack<3>("xn", inds[3])};   

                    mat3 F{};
                    {
                        auto x1x0 = x1[1] - x1[0];
                        auto x2x0 = x1[2] - x1[0];
                        auto x3x0 = x1[3] - x1[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                        x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        F = Ds * DmInv;
                    } 
                    if(zs::determinant(F) < 0.0){
                        // for(int i = 0;i < 4;++i)
                        //     vtemp("is_inverted",inds[i]) = reinterpret_bits<T>((int)1);   
                        etemp("is_inverted",ei) = (T)1.0;   
                    }
                    else {
                        etemp("is_inverted",ei) = (T)0.0;   
                    }               
            });
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                auto is_inverted = etemp("is_inverted",ei) > (T)0.5;  
                if(is_inverted)
                    for(int i = 0;i != 4;++i){
                        vtemp("is_inverted",inds[i]) = (T)1.0;     
                    }       
            });
        }


        void accumInversion(zs::CudaExecutionPolicy& cudaPol,dtiles_t& vtemp,dtiles_t& etemp) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                    auto DmInv = quads.template pack<3,3>("IB",ei);
                    auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                    vec3 x1[4] = {vtemp.template pack<3>("xn", inds[0]),
                            vtemp.template pack<3>("xn", inds[1]),
                            vtemp.template pack<3>("xn", inds[2]),
                            vtemp.template pack<3>("xn", inds[3])};   

                    mat3 F{};
                    {
                        auto x1x0 = x1[1] - x1[0];
                        auto x2x0 = x1[2] - x1[0];
                        auto x3x0 = x1[3] - x1[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                        x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        F = Ds * DmInv;
                    } 
                    if(zs::determinant(F) < 0.0){
                        // for(int i = 0;i < 4;++i)
                        //     vtemp("is_inverted",inds[i]) = reinterpret_bits<T>((int)1);   
                        etemp("is_inverted",ei) = (T)1.0;   
                    }
                    // else {
                    //     etemp("is_inverted",ei) = reinterpret_bits<T>((int)0);   
                    // }               
            });
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                auto is_inverted = etemp("is_inverted",ei) > (T)0.5;  
                if(is_inverted)
                    for(int i = 0;i != 4;++i){
                        vtemp("is_inverted",inds[i]) = (T)1.0;     
                    }       
            });
        }


        void computePositionConstraintGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
            dtiles_t& vtemp,
            const std::string& binderTag,
            const std::string& thicknessTag,
            const std::string& inversionTag,
            const dtiles_t& kverts,
            dtiles_t& gh_buffer) {
                using namespace zs;
                constexpr auto space = execspace_e::cuda;
                int max_nm_binders = tris.getChannelSize(binderTag);
                printf("max_nm_binders = %d\n",max_nm_binders);

                cudaPol(zs::range(tris.size()),
                    [vtemp = proxy<space>({},vtemp),
                        verts = proxy<space>({},verts),
                        eles = proxy<space>({},eles),
                        binderTag = zs::SmallString(binderTag),
                        thicknessTag = zs::SmallString(thicknessTag),
                        inversionTag = zs::SmallString(inversionTag),
                        tris = proxy<space>({},tris),
                        kverts = proxy<space>({},kverts),
                        binderStiffness = binderStiffness,
                        max_nm_binders = max_nm_binders,
                        gh_buffer = proxy<space>({},gh_buffer)] ZS_LAMBDA(int ti) mutable {
                    int nm_binders = 0;

                    for(int i = 0;i != max_nm_binders;++i){
                        auto idx = reinterpret_bits<int>(tris(binderTag,i,ti));
                        if(idx < 0)
                            break;
                        ++nm_binders;
                    }


                    if(nm_binders == 0)
                        return;
                    auto tri = tris.pack(dim_c<3>,"inds",ti).reinterpret_bits(int_c);
                    if(verts.hasProperty("binder_fail"))
                        for(int i = 0;i != 3;++i)
                            if(verts("binder_fail",tri[i]) > (T)0.5)
                                return;
                    auto binder_weakness_param = (T)1.0;
                    // for(int i = 0;i != 3;++i)
                    //     if(vtemp("is_inverted",tri[i]) > (T)0.5)
                    //         return;

                    auto ei = reinterpret_bits<int>(tris("ft_inds",ti));
                    if(ei < 0) {
                        // printf("impossible reaching here with negative ft_inds\n");
                        return;
                    }
                    auto tet = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);


                    auto mu = eles("mu",ei);
                    auto lam = eles("lam",ei);
                    // auto vole = tris("vol",ti);
                    vec3 cp[4] = {};

                    cp[1] = vtemp.pack(dim_c<3>,"xn",tri[0]);
                    cp[2] = vtemp.pack(dim_c<3>,"xn",tri[1]);
                    cp[3] = vtemp.pack(dim_c<3>,"xn",tri[2]);


                    auto inds_reorder = zs::vec<int,3>::zeros();
                    for(int i = 0;i != 3;++i){
                        auto idx = tri[i];
                        for(int j = 0;j != 4;++j)
                            if(idx == tet[j])
                                inds_reorder[i] = j;
                    }

                    for(int i = 0;i != nm_binders;++i) {
                        auto idx = reinterpret_bits<int>(tris(binderTag,i,ti));
                        
                        auto ceps = tris(thicknessTag,i,ti);
                        auto from_inside = tris(inversionTag,i,ti) > (T)0.0;
                        if(kverts.hasProperty("b_fail"))
                            if(kverts("b_fail",idx) > (T)0.5)
                                continue;

                        if(idx >= kverts.size()){
                            printf("kverts buffer overflow %d >= %d\n",idx,kverts.size());
                        }

                        cp[0] = kverts.pack(dim_c<3>,"x",idx);
                        auto kstiffness = (T)1.0;
                        if(kverts.hasProperty("binderStiffness"))
                            kstiffness *= kverts("binderStiffness",idx);
                        auto average_vert_stiffness = (T)1.0;
                        if(verts.hasProperty("binderStiffness")){
                            average_vert_stiffness = (T)0.0;
                            for(int j = 0;j != 3;++j)
                                average_vert_stiffness += verts("binderStiffness",tri[j]);
                        }
                        kstiffness *= average_vert_stiffness;
                        auto alpha = binderStiffness * binder_weakness_param * kstiffness;
                        auto beta = (T)1.0/(T)nm_binders;
                        auto cgrad = -alpha * beta * VERTEX_FACE_SQRT_COLLISION::gradient(cp,mu,lam,ceps,from_inside);
                        auto cH = alpha * beta * VERTEX_FACE_SQRT_COLLISION::hessian(cp,mu,lam,ceps,from_inside);

                        if(isnan(cH.norm()) || isnan(cgrad.norm())) {
                            printf("nan cH and cgrad detected at [%d] [%d] : %f %f\n",ti,idx,(float)cH.norm(),(float)cgrad.norm());
                        }

                        for(int i = 3;i != 12;++i){
                            int d0 = i % 3;
                            int row = inds_reorder[i/3 - 1]*3 + d0;
                            atomic_add(exec_cuda,&gh_buffer("grad",row,ei),cgrad[i]);
                            for(int j = 3;j != 12;++j){
                                int d1 = j % 3;
                                int col = inds_reorder[j/3 - 1]*3 + d1;
                                if(row >= 12 || col >= 12){
                                    printf("invalid row = %d and col = %d %d %d detected %d %d %d\n",row,col,i/3,j/3,
                                        inds_reorder[0],
                                        inds_reorder[1],
                                        inds_reorder[2]);
                                }
                                atomic_add(exec_cuda,&gh_buffer("H",row*12 + col,ei),cH(i,j));
                            }                    
                        }                        
                    }
                });
        }


        void computePlaneConstraintGradientAndHessian2(zs::CudaExecutionPolicy& cudaPol,
                            const dtiles_t& vtemp,
                            dtiles_t& sttemp,
                            const dtiles_t& kverts,
                            const dtiles_t& ktris,
                            const std::string& planeConsBaryTag,
                            const std::string& planeConsIDTag,
                            dtiles_t& nodal_gh_buffer,
                            dtiles_t& tris_gh_buffer,
                            T cnorm,bool use_sticky_condition) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            cudaPol(zs::range(verts.size()),[
                    verts = proxy<space>({},verts),
                    vtemp = proxy<space>({},vtemp),
                    kverts = proxy<space>({},kverts),
                    ktris = proxy<space>({},ktris),
                    planeConsBaryTag = zs::SmallString(planeConsBaryTag),
                    planeConsIDTag = zs::SmallString(planeConsIDTag),
                    kine_out_collisionEps = kine_out_collisionEps,
                    plane_constraint_stiffness = plane_constraint_stiffness,
                    use_sticky_condition = use_sticky_condition,
                    nodal_gh_buffer = proxy<space>({},nodal_gh_buffer)] ZS_LAMBDA(int vi) mutable {
                // return;
                auto idx = reinterpret_bits<int>(verts(planeConsIDTag,vi));
                if(idx < 0)
                    return;      
                auto ktri = ktris.pack(dim_c<3>,"inds",idx).reinterpret_bits(int_c);

                auto is_inverted_vert = vtemp("is_inverted",vi) > (T)0.5;
                if(is_inverted_vert)
                    return;

#if 1
                auto plane_root = kverts.pack(dim_c<3>,"x",ktri[0]);
                auto plane_nrm = ktris.pack(dim_c<3>,"nrm",idx);

                auto mu = verts("mu",vi);
                auto lam = verts("lam",vi);
                    // if(distance > collisionEps)
                auto eps = kine_out_collisionEps;
                auto p = vtemp.pack(dim_c<3>,"xn",vi);
                auto seg = p - plane_root;

                auto fc = vec3::zeros();
                auto Hc = mat3::zeros();
                auto dist = seg.dot(plane_nrm) - eps;
                if(dist < (T)0 || use_sticky_condition){
                    fc = -dist * mu * plane_constraint_stiffness * plane_nrm;
                    Hc = mu * plane_constraint_stiffness * dyadic_prod(plane_nrm,plane_nrm);
                }
#else
                vec3 cv[4] = {};
                // auto ktri = ktris.pack(dim_c<3>,"inds",idx,int_c);
                cv[0] = vtemp.pack(dim_c<3>,"xn",vi);
                for(int i = 0;i != 3;++i)
                    cv[i + 1] = kverts.pack(dim_c<3>,"x",ktri[i]);
                
                auto mu = verts("mu",vi);
                auto lam = verts("lam",vi);
                auto eps = kine_out_collisionEps;
                auto cforce = -mu * plane_constraint_stiffness * VERTEX_FACE_SQRT_COLLISION::gradient(cv,mu,lam,eps);
                auto cK = mu * plane_constraint_stiffness * VERTEX_FACE_SQRT_COLLISION::hessian(cv,mu,lam,eps);

                vec3 fc{};
                mat3 Hc{};

                for(int i = 0;i != 3;++i)
                    fc[i] = cforce[i];
                for(int i = 0;i != 3;++i)
                    for(int j = 0;j != 3;++j)
                        Hc(i,j) = cK(i,j);

#endif

                // printf("apply plane constraint with force : %f %f\n",(float)dist,(float)fc.norm());

                nodal_gh_buffer.tuple(dim_c<3>,"grad",vi) = fc;
                nodal_gh_buffer.tuple(dim_c<3,3>,"H",vi) = Hc;

            });


            cudaPol(zs::range(kverts.size()),[
                vtemp = proxy<space>({},vtemp),
                tris = proxy<space>({},tris),
                kverts = proxy<space>({},kverts),
                verts = proxy<space>({},verts),
                sttemp = proxy<space>({},sttemp),
                planeConsIDTag = zs::SmallString(planeConsIDTag),
                kine_out_collisionEps = kine_out_collisionEps,
                plane_constraint_stiffness = plane_constraint_stiffness,
                use_sticky_condition = use_sticky_condition] ZS_LAMBDA(int kvi) mutable {
                    auto ti = reinterpret_bits<int>(kverts(planeConsIDTag,kvi));
                    if(ti < 0)
                        return;
                    auto kp = kverts.pack(dim_c<3>,"x",kvi);
                    auto tri = tris.pack(dim_c<3>,"inds",ti,int_c);
                    
                    auto plane_root = vtemp.pack(dim_c<3>,"xn",tri[0]);
                    vec3 tvs[3] = {};
                    for(int i = 0;i != 3;++i)
                        tvs[i] = vtemp.pack(dim_c<3>,"xn",tri[i]);
                    auto plane_nrm = LSL_GEO::facet_normal(tvs[0],tvs[1],tvs[2]);
                    // auto plane_nrm = tris.pack(dim_c<3>,"nrm",ti);

                    auto mu = verts("mu",tri[0]);
                    auto lam = verts("lam",tri[0]);

                    auto eps = kine_out_collisionEps;
                    auto seg = kp - plane_root;

                    auto fc = vec12::zeros();
                    auto Hc = mat12::zeros();
                    auto dist = seg.dot(plane_nrm) + eps;
                    if(dist > (T)0 || use_sticky_condition) {
                        // fc = -dist * mu * plane_constraint_stiffness * plane_nrm;
                        // Hc = mu * plane_constraint_stiffness * dyadic_prod(plane_nrm,plane_nrm);
                        // printf("detected kv2t pairs : %f %f\n",(T)fc.norm(),(T)Hc.norm());
                        vec3 vs[4] = {};
                        for(int i = 0;i != 3;++i)
                            vs[i + 1] = tvs[i];
                        vs[0] = kp;
                        fc = -VERTEX_FACE_COLLISION::gradient(vs,mu,lam,-eps);
                        Hc = VERTEX_FACE_COLLISION::hessian(vs,mu,lam,-eps);
                    }

                    

                    for(int i = 0;i != 9;++i)
                        atomic_add(exec_cuda,&sttemp("grad",i,ti),fc[i + 3]);
                    for(int i = 0;i != 9;++i)
                        for(int j = 0;j != 9;++j)
                            atomic_add(exec_cuda,&sttemp("H",i * 9 + j,ti),Hc(i + 3,j + 3));
            });
        }

        void  computePlaneConstraintGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
                            const dtiles_t& vtemp,
                            const std::string& planeConsPosTag,
                            const std::string& planeConsNrmTag,
                            const std::string& planeConsIDTag,
                            dtiles_t& nodal_gh_buffer) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            cudaPol(zs::range(verts.size()),[
                    verts = proxy<space>({},verts),
                    vtemp = proxy<space>({},vtemp),
                    planeConsPosTag = zs::SmallString(planeConsPosTag),
                    planeConsNrmTag = zs::SmallString(planeConsNrmTag),
                    planeConsIDTag = zs::SmallString(planeConsIDTag),
                    kine_out_collisionEps = kine_out_collisionEps,
                    plane_constraint_stiffness = plane_constraint_stiffness,
                    nodal_gh_buffer = proxy<space>({},nodal_gh_buffer)] ZS_LAMBDA(int vi) mutable {
                auto idx = reinterpret_bits<int>(verts(planeConsIDTag,vi));
                if(idx < 0)
                    return;

                // if(kverts.hasProperty("k_fail"))
                // if(verts("is_inverted",vi) > (T)0.5)
                //     return;


                auto mu = verts("mu",vi);
                auto lam = verts("lam",vi);

                auto eps = kine_out_collisionEps;
                auto plane_nrm = verts.pack(dim_c<3>,planeConsNrmTag,vi);
                auto plane_root = verts.pack(dim_c<3>,planeConsPosTag,vi);

                auto p = vtemp.pack(dim_c<3>,"xn",vi);
                auto seg = p - plane_root;

                auto fc = vec3::zeros();
                auto Hc = mat3::zeros();
                auto dist = seg.dot(plane_nrm) - eps;
                if(dist < (T)0){
                    fc = -dist * mu * plane_constraint_stiffness * plane_nrm;
                    Hc = mu * plane_constraint_stiffness * dyadic_prod(plane_nrm,plane_nrm);
                }

                // printf("apply plane constraint with force : %f %f\n",(float)dist,(float)fc.norm());

                nodal_gh_buffer.tuple(dim_c<3>,"grad",vi) = fc;
                nodal_gh_buffer.tuple(dim_c<3,3>,"H",vi) = Hc;
            });
        }

        template <typename ElasticModel,typename AnisoElasticModel>
        void computeGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
                            const ElasticModel& model,
                            const AnisoElasticModel& amodel,
                            const dtiles_t& vtemp,
                            const dtiles_t& etemp,
                            dtiles_t& gh_buffer,
                            bool use_kinematic_potential,
                            T kd_alpha = (T)0.0,
                            T kd_beta = (T)0.0,
                            bool use_anisotropic_jiggling = false) {        
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            int offset = 0;
            TILEVEC_OPS::copy<4>(cudaPol,eles,"inds",gh_buffer,"inds",offset);   
            // eval the inertia term gradient
            if(use_kinematic_potential) {
                cudaPol(zs::range(eles.size()),[dt2 = dt2,
                            verts = proxy<space>({},verts),
                            use_anisotropic_jiggling = use_anisotropic_jiggling,
                            eles = proxy<space>({},eles),
                            etemp = proxy<space>({},etemp),
                            vtemp = proxy<space>({},vtemp),
                            kd_alpha = kd_alpha,
                            gh_buffer = proxy<space>({},gh_buffer),
                            dt = dt,offset = offset] ZS_LAMBDA(int ei) mutable {
                    auto m = eles("m",ei)/(T)4.0;
                    auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                    // auto pgrad = zs::vec<T,12>::zeros();
                    // auto H  = zs::vec<T,12,12>::zeros();
                    // if(eles.hasProperty("dt")) {
                    //     dt2 = eles("dt",ei) * eles("dt",ei);
                    // }

                    auto A = mat3::identity();
                    if(etemp.hasProperty("dfiber") && use_anisotropic_jiggling) {
                        auto f = etemp.pack(dim_c<3>,"dfiber",ei);
                        A = A - dyadic_prod(f,f) * 0.99;
                    }

                    auto inertia = (T)1.0;
                    if(eles.hasProperty("inertia"))
                        inertia = eles("inertia",ei);
                    for(int i = 0;i != 4;++i){
                        auto x1 = vtemp.pack(dim_c<3>,"xn",inds[i]);
                        auto x0 = vtemp.pack(dim_c<3>,"xp",inds[i]);
                        auto v1 = (x1 - x0) / dt;
                        auto v0 = vtemp.pack(dim_c<3>,"vp",inds[i]);

                        auto alpha = (inertia * m / dt2) * A;
                        auto nodal_pgrad = -alpha * (x1 - x0 - v0 * dt + v1 * dt * kd_alpha);

                        if(isnan(nodal_pgrad.norm())) {
                            printf("nan nodal pgrad detected : %f %f %f %f\n",(float)alpha.norm(),(float)x1.norm(),(float)x0.norm(),(float)v0.norm());
                        }

                        for(int d = 0;d != 3;++d){
                            auto idx = i * 3 + d;
                            gh_buffer("grad",idx,ei + offset) = nodal_pgrad[d];
                            // gh_buffer("H",idx*12 + idx,ei + offset) = alpha;
                        }
                        for(int d = 0;d != 9;++d){
                            auto idx = (i * 3 + (d / 3)) * 12 + (i * 3 + d % 3);
                            gh_buffer("H",idx,ei + offset) = alpha(d / 3,d % 3) * ((float)1.0 + kd_alpha);
                        }
                        
                    }
                    // gh_buffer.tuple(dim_c<12>,"grad",ei + offset) = pgrad;
                    // gh_buffer.template tuple<12*12>("H",ei + offset) = H;
                });
            }

            // auto gradn_after_inertia = TILEVEC_OPS::dot<12>(cudaPol,gh_buffer,"grad","grad");
            // std::cout << "gradn_after_inertia : " << gradn_after_inertia << std::endl;

            cudaPol(zs::range(eles.size()), [dt = dt,dt2 = dt2,aniso_strength = aniso_strength,
                            verts = proxy<space>({},verts),
                            vtemp = proxy<space>({}, vtemp),
                            etemp = proxy<space>({}, etemp),
                            gh_buffer = proxy<space>({},gh_buffer),
                            eles = proxy<space>({}, eles),
                            kd_alpha = kd_alpha,kd_beta = kd_beta,
                            // dt = dt,
                            model = model,amodel = amodel, volf = volf,offset = offset] ZS_LAMBDA (int ei) mutable {
                auto DmInv = eles.pack(dim_c<3,3>,"IB",ei);
                auto dFdX = dFdXMatrix(DmInv);
                auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                vec3 x1[4] = {vtemp.pack(dim_c<3>,"xn", inds[0]),
                                vtemp.pack(dim_c<3>,"xn", inds[1]),
                                vtemp.pack(dim_c<3>,"xn", inds[2]),
                                vtemp.pack(dim_c<3>,"xn", inds[3])};

                vec3 x0[4] = {vtemp.pack(dim_c<3>,"xp", inds[0]),
                                vtemp.pack(dim_c<3>,"xp", inds[1]),
                                vtemp.pack(dim_c<3>,"xp", inds[2]),
                                vtemp.pack(dim_c<3>,"xp", inds[3])};

                vec3 v[4] = {};
                for(int i = 0;i != 4;++i)
                    v[i] = (x1[i] - x0[i]) / dt;

                mat3 FAct{};
                mat3 F{};
                {
                    auto x1x0 = x1[1] - x1[0];
                    auto x2x0 = x1[2] - x1[0];
                    auto x3x0 = x1[3] - x1[0];
                    auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                    x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                    F = Ds * DmInv;
                    FAct = F * etemp.template pack<3,3>("ActInv",ei);
                } 
                auto dFActdF = dFAdF(etemp.template pack<3,3>("ActInv",ei));

                // add the force term in gradient
                if(eles.hasProperty("mu") && eles.hasProperty("lam")) {
                    model.mu = eles("mu",ei);
                    model.lam = eles("lam",ei);
                }

                auto inversion_strength = (T)1.0;
                // for(int i = 0;i != 4;++i)
                //     if(vtemp("is_inverted",inds[i]) < (T)0.5)
                //         inversion_strength = (T)1.0;

                auto P = model.first_piola(FAct) * inversion_strength;
                auto vole = eles("vol", ei);
                auto vecP = flatten(P);
                vecP = dFActdF.transpose() * vecP;
                auto dFdXT = dFdX.transpose();
                auto vf = -vole * (dFdXT * vecP);     


                auto mg = volf * vole / (T)4.0;
                for(int i = 0;i != 4;++i)
                    for(int d = 0;d !=3 ;++d){
                        vf[i*3 + d] += mg[d];
                    }


                // assemble element-wise hessian matrix
                auto Hq = model.first_piola_derivative(FAct, true_c) * inversion_strength;
                auto dFdAct_dFdX = dFActdF * dFdX; 
                // add inertia hessian term
                auto H = dFdAct_dFdX.transpose() * Hq * dFdAct_dFdX * vole;

                if(eles.hasProperty("Muscle_ID") && (int)eles("Muscle_ID",ei) >= 0) {
                    auto fiber = eles.pack(dim_c<3>,"fiber",ei);
                    if(zs::abs(fiber.norm() - 1.0) < 1e-3) {
                        fiber /= fiber.norm();
                        if(eles.hasProperty("mu") && eles.hasProperty("lam")) {
                            amodel.mu = eles("mu",ei);
                            // amodel.lam = eles("lam",ei);
                            
                        }
                        // COMMIT FIND A ANISOTROPIC BUG HERE
                        auto aP = amodel.do_first_piola(FAct,fiber);
                        auto vecAP = flatten(aP);
                        vecAP = dFActdF.transpose() * vecAP;
                        vf -= vole  * dFdXT * vecAP * aniso_strength;

                        auto aHq = amodel.do_first_piola_derivative(FAct,fiber);
                        // make_pd(aHq);
                        H += dFdAct_dFdX.transpose() * aHq * dFdAct_dFdX * vole * aniso_strength;
                        
                        // if((int)eles("Muscle_ID",ei) == 0){
                        //     printf("fiber : %f %f %f,Fa = %f,aP = %f,aHq = %f,H = %f\n",fiber[0],fiber[1],fiber[2],(float)FAct.norm(),(float)aP.norm(),(float)aHq.norm(),(float)H.norm());
                        // }

                        if(isnan(vf.norm())) {
                            printf("nan nodal aniso_vf detected : %f %f %f %f\n",(float)vecP.norm(),(float)volf.norm(),(float)vecAP.norm(),(float)aHq.norm());
                        }
                    }
                }

                // if(isnan(vf.norm())) {
                //     printf("nan nodal vf detected : %f %f %f %f\n",(float)vecP.norm(),(float)volf.norm(),(float)P.norm(),(float)FAct.norm());
                // }

                zs::vec<T,12> rdamping{};
                for(int i = 0;i != 4;++i) {
                    for(int d = 0;d != 3;++d)
                        rdamping[i * 3 + d] = v[i][d];
                }
                rdamping = -kd_beta * H * rdamping;

                gh_buffer.tuple(dim_c<12>,"grad",ei + offset) = gh_buffer.pack(dim_c<12>,"grad",ei + offset) + vf + rdamping; 
                // gh_buffer.tuple(dim_c<12>,"grad",ei + offset) = gh_buffer.pack(dim_c<12>,"grad",ei + offset) - rdamping; 
                H += kd_beta * H / dt;

                if(isnan(H.norm())) {
                    printf("nan elastic hessian detected[%d] with Hq = %f\n",ei,(float)Hq.norm());
                }

                gh_buffer.template tuple<12*12>("H",ei + offset) = gh_buffer.template pack<12,12>("H",ei + offset) + H/* + C/dt*/;
            });
        // Bone Driven Potential Energy
            // T lambda = model.lam;
            // T mu = model.mu;


            // auto gradn_after_elastic = TILEVEC_OPS::dot<12>(cudaPol,gh_buffer,"grad","grad");
            // std::cout << "gradn_after_elastic : " << gradn_after_elastic << std::endl;

            auto nmEmbedVerts = b_verts.size();

            cudaPol(zs::range(nmEmbedVerts), [
                    gh_buffer = proxy<space>({},gh_buffer),model = model,
                    kd_beta = kd_beta,
                    dt = dt,
                    bcws = proxy<space>({},b_bcws),b_verts = proxy<space>({},b_verts),vtemp = proxy<space>({},vtemp),etemp = proxy<space>({},etemp),
                    eles = proxy<space>({},eles),bone_driven_weight = bone_driven_weight,offset = offset] ZS_LAMBDA(int vi) mutable {
                        auto ei = reinterpret_bits<int>(bcws("inds",vi));
 
                        if(ei < 0){

                            return;
                        }

                        if(b_verts.hasProperty("intersect"))
                            if(b_verts("intersect",vi) > (T)0.5){
                                // printf("skip bverts[%d] due to intersection\n",vi);
                                return;
                            }


                        auto lambda = model.lam;
                        auto mu = model.mu;
                        // if(eles.hasProperty("mu") && eles.hasProperty("lam")) {
                        //     mu = eles("mu",ei);
                        //     lambda = eles("lam",ei);
                        // }

                        auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                        // gh_buffer.tuple(dim_c<4>,"inds",vi + offset + eles.size()) = eles.pack(dim_c<4>,"inds",ei);
                        auto w = bcws.pack(dim_c<4>,"w",vi);
                        // if(w[0] < 1e-4 || w[1] < 1e-4 || w[2] < 1e-4 || w[3] < 1e-4){
                        //     // if(vi == 0)
                        //     //     printf("boundary tet\n");
                        //     return;
                        // }
                        auto tpos = vec3::zeros();
                        for(int i = 0;i != 4;++i)
                            tpos += w[i] * vtemp.pack(dim_c<3>,"xn",inds[i]);

                        auto tvel = vec3::zeros();
                        for(int i = 0;i != 4;++i)
                            tvel += w[i] * vtemp.pack(dim_c<3>,"xp",inds[i]);
                        tvel = tpos - tvel;
                        tvel /= dt;

                        auto pdiff = tpos - b_verts.pack<3>("x",vi);
                        // auto pdiff = tpos - b_verts[vi];

                        T stiffness = (2.0066 * mu + 1.0122 * lambda) * b_verts("strength",vi);


                        auto area = (T)1.0;
                        if(b_verts.hasProperty("area"))
                            area = b_verts("area",vi);
                        auto alpha = area * stiffness * bone_driven_weight * bcws("strength",vi) * bcws("cnorm",vi) * eles("vol",ei) * eles("bdw",ei);

                        for(size_t i = 0;i != 4;++i){
                            auto tmp = -pdiff * alpha * w[i]; 

                            tmp -= tvel * alpha * w[i] * kd_beta;
                            // if(vi == 0 && i == 0) {
                                // printf("check: %f %f %f\n",(float)tmp[0],(float)tmp[1],(float)tmp[2]);
                            // }
                            for(size_t d = 0;d != 3;++d){
                                atomic_add(exec_cuda,&gh_buffer("grad",i*3 + d,ei),tmp[d]);
                                // elm_grad[i*3 + d] = tmp[d];
                                // atomic_add(exec_cuda,&gh_buffer("grad",i * 3 + d,ei),tmp[d]);
                            }
                        }
                        for(int i = 0;i != 4;++i)
                            for(int j = 0;j != 4;++j){
                                T beta = alpha * w[i] * w[j] * (1 + kd_beta / dt);
                                if(isnan(beta))
                                    printf("nan H detected at driver : %d %f %f\n",vi,(float)b_verts("strength",vi),(float)alpha);
                                for(int d = 0;d != 3;++d){
                                    atomic_add(exec_cuda,&gh_buffer("H",(i*3 + d)*12 + j*3 + d,ei),beta);
                                }
                            }

            });

            // auto gradn_after_driver = TILEVEC_OPS::dot<12>(cudaPol,gh_buffer,"grad","grad");
            // std::cout << "gradn_after_driver : " << gradn_after_driver << std::endl;

        }

        template <typename ElasticModel>
        void computeElasticBonesEnergy(zs::CudaExecutionPolicy& cudaPol,
                            const ElasticModel& model,    
                            const dtiles_t& vtemp,
                            const dtiles_t& etemp,
                            T& res) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            Vector<T> psi{vtemp.get_allocator(), 1};
            psi.setVal((T)0);
            cudaPol(zs::range(eles.size()), [
                            verts = proxy<space>({},verts),
                            vtemp = proxy<space>({}, vtemp),
                            etemp = proxy<space>({}, etemp),
                            psi = proxy<space>(psi),
                            eles = proxy<space>({}, eles),
                            model = model] ZS_LAMBDA (int ei) mutable {
                auto DmInv = eles.pack(dim_c<3,3>,"IB",ei);
                auto dFdX = dFdXMatrix(DmInv);
                auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                vec3 x1[4] = {vtemp.pack(dim_c<3>,"xn", inds[0]),
                                vtemp.pack(dim_c<3>,"xn", inds[1]),
                                vtemp.pack(dim_c<3>,"xn", inds[2]),
                                vtemp.pack(dim_c<3>,"xn", inds[3])};   
                mat3 FAct{};
                {
                    auto x1x0 = x1[1] - x1[0];
                    auto x2x0 = x1[2] - x1[0];
                    auto x3x0 = x1[3] - x1[0];
                    auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                    x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                    FAct = Ds * DmInv;
                    FAct = FAct * etemp.template pack<3,3>("ActInv",ei);
                } 
                auto inversion_strength = (T)1.0;
                // for(int i = 0;i != 4;++i)
                //     if(vtemp("is_inverted",inds[i]) < (T)0.5)
                //         inversion_strength = (T)1.0;
                auto vole = eles("vol", ei);
                auto epsi = vole * model.psi(FAct) * inversion_strength;                

                atomic_add(exec_cuda,&psi[0],epsi);
            });

            auto nmEmbedVerts = b_verts.size();
            cudaPol(zs::range(nmEmbedVerts), [
                    model = model,psi = proxy<space>(psi),
                    bcws = proxy<space>({},b_bcws),b_verts = proxy<space>({},b_verts),vtemp = proxy<space>({},vtemp),etemp = proxy<space>({},etemp),
                    eles = proxy<space>({},eles),bone_driven_weight = bone_driven_weight] ZS_LAMBDA(int vi) mutable {
                        auto ei = reinterpret_bits<int>(bcws("inds",vi));
 
                        if(ei < 0){
                            return;
                        }

                        auto lambda = model.lam;
                        auto mu = model.mu;

                        auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                        // gh_buffer.tuple(dim_c<4>,"inds",vi + offset + eles.size()) = eles.pack(dim_c<4>,"inds",ei);
                        auto w = bcws.pack(dim_c<4>,"w",vi);

                        auto tpos = vec3::zeros();
                        for(int i = 0;i != 4;++i)
                            tpos += w[i] * vtemp.pack(dim_c<3>,"xn",inds[i]);
                        auto pdiff = tpos - b_verts.pack<3>("x",vi);
                        // auto pdiff = tpos - b_verts[vi];

                        T stiffness = (2.0066 * mu + 1.0122 * lambda) * b_verts("strength",vi);

                        auto alpha = stiffness * bone_driven_weight * bcws("strength",vi) * bcws("cnorm",vi) * eles("vol",ei) * eles("bdw",ei);
                        T bpsi = (T)0.5 * pdiff.l2NormSqr() * alpha; 

                        atomic_add(exec_cuda,&psi[0],bpsi);

            });            


            res = psi.getVal();
        }

        void computeKinematicCollisionGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
            const std::vector<ZenoParticles*>& kinematics,
            dtiles_t& verts_buffer,
            dtiles_t& tri_buffer,
            const ZenoParticles::particles_t& halfedges,
            dtiles_t& vtemp,dtiles_t& sttemp,bool colliding_from_inside) {
                using namespace zs;
                constexpr auto space = RM_CVREF_T(cudaPol)::exec_tag::value;
                constexpr auto exec_tag = wrapv<space>{};

                topological_sample(cudaPol,points,vtemp,"xn",verts_buffer,"x");
                
                auto thickness = 3 * compute_average_edge_length(cudaPol,verts_buffer,"x",tri_buffer);
                auto tetBvh = bvh_t{};
                auto tetBvs = retrieve_bounding_volumes(cudaPol,vtemp,eles,wrapv<4>{},(T)0,"xn");
                tetBvh.build(cudaPol,tetBvs);

                auto triBvh = bvh_t{};
                auto triBvs = retrieve_bounding_volumes(cudaPol,vtemp,tris,wrapv<3>{},(T)0,"xn");
                triBvh.build(cudaPol,triBvs);

                auto spBvh = bvh_t{};
                auto spBvs = retrieve_bounding_volumes(cudaPol,vtemp,points,wrapv<1>{},(T)thickness,"xn");
                spBvh.build(cudaPol,spBvs);

                for(auto kinematic : kinematics) {
                    TILEVEC_OPS::fill(cudaPol,verts_buffer,"ring_mask",zs::reinterpret_bits<T>((int)0));
                    TILEVEC_OPS::fill(cudaPol,tri_buffer,"ring_mask",zs::reinterpret_bits<T>((int)0));

                    const auto& kverts = kinematic->getParticles();
                    const auto& ktris = kinematic->getQuadraturePoints();
                    const auto& khalfedges = (*kinematic)[ZenoParticles::s_surfHalfEdgeTag];
                    // do gia analysis
                    auto kthickness = 3 * compute_average_edge_length(cudaPol,kverts,"x",ktris);
                    auto bvh_thickness = thickness > kthickness ? thickness : kthickness;

                    dtiles_t gia_res{verts_buffer.get_allocator(),{
                        {"ring_mask",1}
                    },verts_buffer.size() + kverts.size()};

                    dtiles_t tris_gia_res(tri_buffer.get_allocator(),{
                        {"ring_mask",1}
                    },tri_buffer.size() + ktris.size());
                    dtiles_t kverts_buffer{kverts.get_allocator(),{
                        {"x",3},
                        // {"GIA_must_exclude",1}
                    },kverts.size()};
                    TILEVEC_OPS::copy(cudaPol,kverts,"x",kverts_buffer,"x");

                    int nm_rings = 0;
                    // auto nm_rings = do_global_intersection_analysis(cudaPol,
                    //     verts_buffer,"x",tri_buffer,halfedges,kverts_buffer,"x",ktris,khalfedges,
                    //     gia_res,tris_gia_res);
                    
                    // finding the process_vertex_kface_collision_pairs 
                    // zs::Vector<int> nm_csPT{points.get_allocator(),1};
                    // nm_csPT.setVal(0);
                    zs::bht<int,2,int> p2k_csPT{points.get_allocator(),10000};
                    p2k_csPT.reset(cudaPol,true);
                    cudaPol(zs::range(ktris.size()),[
                        csPT = proxy<space>(p2k_csPT),
                        colFromInside = !colliding_from_inside,
                        verts_buffer = proxy<space>({},verts_buffer),
                        gia_res = proxy<space>({},gia_res),
                        tris_gia_res = proxy<space>({},tris_gia_res),
                        offset = tri_buffer.size(),
                        thickness = thickness,
                        in_collisionEps = in_collisionEps,
                        out_collisionEps = out_collisionEps,
                        // vtemp = proxy<space>({},vtemp),
                        spBvh = proxy<space>(spBvh),
                        khalfedges = proxy<space>({},khalfedges),
                        kverts_buffer = proxy<space>({},kverts_buffer),
                        ktris = proxy<space>({},ktris)] ZS_LAMBDA(int kti) mutable {
                            auto tri = ktris.pack(dim_c<3>,"inds",kti,int_c);
                            vec3 tvs[3] = {};
                            for(int i = 0;i != 3;++i)
                                tvs[i] = kverts_buffer.pack(dim_c<3>,"x",tri[i]);
                            auto cp = vec3::zeros();
                            for(int i = 0;i != 3;++i)
                                cp += tvs[i] / (T)3.0;
                            auto bv = bv_t{get_bounding_box(cp - thickness,cp + thickness)};

                            auto tnrm = LSL_GEO::facet_normal(tvs[0],tvs[1],tvs[2]);
                            auto hi = zs::reinterpret_bits<int>(ktris("he_inds",kti));
                            vec3 bnrms[3] = {};
                            for(int i = 0;i != 3;++i) {
                                auto edge_normal = tnrm;
                                auto opposite_he = zs::reinterpret_bits<int>(khalfedges("opposite_he",hi));
                                if(opposite_he >= 0) {
                                    auto nti = zs::reinterpret_bits<int>(khalfedges("to_face",opposite_he));
                                    auto ntri = ktris.pack(dim_c<3>,"inds",nti,int_c);
                                    auto ntnrm = LSL_GEO::facet_normal(
                                        kverts_buffer.pack(dim_c<3>,"x",ntri[0]),
                                        kverts_buffer.pack(dim_c<3>,"x",ntri[1]),
                                        kverts_buffer.pack(dim_c<3>,"x",ntri[2]));
                                    edge_normal = tnrm + ntnrm;
                                    edge_normal = edge_normal/(edge_normal.norm() + (T)1e-6);
                                }
                                auto e01 = tvs[(i + 1) % 3] - tvs[(i + 0) % 3];
                                bnrms[i] = edge_normal.cross(e01).normalized();
                                hi = zs::reinterpret_bits<int>(khalfedges("next_he",hi));
                            }

                            T min_penertration_distance = (T)1e8;
                            int min_spI = -1;

                            auto process_vertex_face_collision_pairs = [&](int spI) {
                                auto p = verts_buffer.pack(dim_c<3>,"x",spI);
                                auto seg = p - tvs[0];
                                auto dist = seg.dot(tnrm);
                                if(colFromInside)
                                    dist = -dist;

                                auto collisionEps = dist > 0 ?  out_collisionEps : in_collisionEps;
                                auto barySum = (T)1.0;
                                T distance = LSL_GEO::get_vertex_triangle_distance(tvs[0],tvs[1],tvs[2],p,barySum);

                                if(distance > collisionEps)
                                    return;

                                if(barySum > (T)(1.0 + 1e-6)) {
                                    for(int i = 0;i != 3;++i){
                                        seg = p - tvs[i];
                                        if(bnrms[i].dot(seg) < 0)
                                            return;
                                    }
                                }

                                if(dist < 0 && distance < min_penertration_distance) {
                                    // do gia intersection test
                                    int RING_MASK = zs::reinterpret_bits<int>(gia_res("ring_mask",spI)) & zs::reinterpret_bits<int>(tris_gia_res("ring_mask",kti + offset));
                                    // if(RING_MASK == 0)
                                    //     return;

                                    min_penertration_distance = distance;
                                    min_spI = spI;
                                }
                                if(dist > 0)
                                    csPT.insert(zs::vec<int,2>{spI,kti});
                            };
                            spBvh.iter_neighbors(bv,process_vertex_face_collision_pairs);
                            if(min_spI >= 0) {
                                csPT.insert(zs::vec<int,2>{min_spI,kti});
                            }
                    });
                    std::cout << "nm_p2k_intersections : " << p2k_csPT.size() << std::endl;


                    zs::bht<int,2,int> k2p_csPT{points.get_allocator(),10000};
                    k2p_csPT.reset(cudaPol,true);     
                    cudaPol(zs::range(kverts_buffer.size()),[
                        kverts_buffer = proxy<space>({},kverts_buffer),
                        csPT = proxy<space>(k2p_csPT),
                        colliding_from_inside = colliding_from_inside,
                        verts_buffer = proxy<space>({},verts_buffer),
                        gia_res = proxy<space>({},gia_res),
                        // tris = proxy<space>({},tris),
                        tri_buffer = proxy<space>({},tri_buffer),
                        tris_gia_res = proxy<space>({},tris_gia_res),
                        offset = verts_buffer.size(),
                        thickness = thickness,
                        in_collisionEps = in_collisionEps,
                        out_collisionEps = out_collisionEps,
                        // vtemp = proxy<space>({},vtemp),
                        triBvh = proxy<space>(triBvh),
                        halfedges = proxy<space>({},halfedges)] ZS_LAMBDA(int kvi) mutable {
                            auto kp = kverts_buffer.pack(dim_c<3>,"x",kvi);
                            auto bv = bv_t{get_bounding_box(kp - thickness,kp + thickness)};

                            T min_penertration_distance = (T)1e8;
                            int min_ti = -1;

                            auto process_vertex_face_collision_pairs = [&](int ti) {
                                auto tri = tri_buffer.pack(dim_c<3>,"inds",ti,int_c);
                                vec3 tvs[3] = {};
                                for(int i = 0;i != 3;++i)
                                    tvs[i] = verts_buffer.pack(dim_c<3>,"x",tri[i]);
                                auto tnrm = LSL_GEO::facet_normal(tvs[0],tvs[1],tvs[2]);

                                auto seg = kp - tvs[0];
                                auto dist = tnrm.dot(seg);
                                if(colliding_from_inside)
                                    dist = -dist;
                                
                                auto collisionEps = dist > 0 ? out_collisionEps : in_collisionEps;
                                auto barySum = (T)1.0;
                                T distance = LSL_GEO::get_vertex_triangle_distance(tvs[0],tvs[1],tvs[2],kp,barySum);

                                if(distance > collisionEps)
                                    return;       

                                auto hi = zs::reinterpret_bits<int>(tri_buffer("he_inds",ti));
                                vec3 bnrms[3] = {};
                                for(int i = 0;i != 3;++i) {
                                    auto edge_normal = tnrm;
                                    auto opposite_he = zs::reinterpret_bits<int>(halfedges("opposite_he",hi));
                                    if(opposite_he >= 0) {
                                        auto nti = zs::reinterpret_bits<int>(halfedges("to_face",opposite_he));
                                        auto ntri = tri_buffer.pack(dim_c<3>,"inds",nti,int_c);
                                        auto ntnrm = LSL_GEO::facet_normal(
                                            verts_buffer.pack(dim_c<3>,"x",ntri[0]),
                                            verts_buffer.pack(dim_c<3>,"x",ntri[1]),
                                            verts_buffer.pack(dim_c<3>,"x",ntri[2]));
                                        edge_normal = tnrm + ntnrm;
                                        edge_normal = edge_normal/(edge_normal.norm() + (T)1e-6);
                                    }
                                    auto e01 = tvs[(i + 1) % 3] - tvs[(i + 0) % 3];
                                    bnrms[i] = edge_normal.cross(e01).normalized();
                                    hi = zs::reinterpret_bits<int>(halfedges("next_he",hi));
                                }

                                if(barySum > (T)(1.0 + 1e-6)) {
                                    for(int i = 0;i != 3;++i){
                                        seg = kp - tvs[i];
                                        if(bnrms[i].dot(seg) < 0)
                                            return;
                                    }
                                }

                                if(dist < 0 && distance < min_penertration_distance) {
                                    int RING_MASK = zs::reinterpret_bits<int>(gia_res("ring_mask",kvi + offset)) & zs::reinterpret_bits<int>(tris_gia_res("ring_mask",ti));
                                    // if(RING_MASK == 0)
                                    //     return;
                                    min_penertration_distance = distance;
                                    min_ti = ti;
                                }

                                if(dist > 0)
                                    csPT.insert(zs::vec<int,2>{kvi,ti});
                            };
                            triBvh.iter_neighbors(bv,process_vertex_face_collision_pairs);
                            if(min_ti >= 0)
                                csPT.insert(zs::vec<int,2>{kvi,min_ti});
                    }); 
                    std::cout << "nm_k2p_intersections : " << k2p_csPT.size() << std::endl;

                }
        }


        FEMDynamicSteppingSystem(const tiles_t &verts, const tiles_t &eles,
                const tiles_t& points,const tiles_t& tris,
                T in_collisionEps,T out_collisionEps,
                const tiles_t &b_bcws, const tiles_t& b_verts,T bone_driven_weight,
                const vec3& volf,const T& _dt,const T& collisionStiffness,
                const T& kine_in_collisionEps,const T& kine_out_collisionEps,
                const T& kineCollisionStiffness,const T& aniso_strength,const T& binderStiffness,const T& plane_constraint_stiffness)
            : verts{verts}, eles{eles},points{points}, tris{tris},
                    in_collisionEps{in_collisionEps},out_collisionEps{out_collisionEps},
                    b_bcws{b_bcws}, b_verts{b_verts}, bone_driven_weight{bone_driven_weight},
                    volf{volf},binderStiffness{binderStiffness},plane_constraint_stiffness{plane_constraint_stiffness},
                    kine_in_collisionEps{kine_in_collisionEps},kine_out_collisionEps{kine_out_collisionEps},
                    kineCollisionStiffness{kineCollisionStiffness},aniso_strength{aniso_strength},
                    dt{_dt}, dt2{_dt * _dt},collisionStiffness{collisionStiffness},use_edge_edge_collision{true}, use_vertex_facet_collision{true} {}

        const tiles_t &verts;
        const tiles_t &eles;
        const tiles_t &points;
        const tiles_t &tris;
        const tiles_t &b_bcws;  // the barycentric interpolation of embeded bones 
        const tiles_t &b_verts; // the position of embeded bones

        T bone_driven_weight;
        vec3 volf;
        T dt;
        T dt2;
        T in_collisionEps;
        T out_collisionEps;

        T collisionStiffness;

        bool bvh_initialized;
        bool use_edge_edge_collision;
        bool use_vertex_facet_collision;

        T kine_in_collisionEps;
        T kine_out_collisionEps;
        T kineCollisionStiffness;

        T aniso_strength;


        T binderStiffness;
        T plane_constraint_stiffness;
    };




    void apply() override {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        auto cudaPol = cuda_exec();

        auto zsparticles = get_input<ZenoParticles>("ZSParticles");
        auto gravity = zeno::vec<3,T>(0);
        if(has_input("gravity"))
            gravity = get_input2<zeno::vec<3,T>>("gravity");
        T armijo = (T)1e-4;
        T wolfe = (T)0.9;
        // T cg_res = (T)0.01;
        // T cg_res = (T)0.0001;
        // T cg_res = get_param<float>("cg_res");
        T cg_res = get_input2<float>("cg_res");
        T btl_res = (T)0.1;
        auto models = zsparticles->getModel();
        auto& verts = zsparticles->getParticles();
        auto& eles = zsparticles->getQuadraturePoints();

        // zs::Vector<vec3>(MAX_VERTS)
        // TileVec("pos","tag","deleted","")

        if(eles.getChannelSize("inds") != 4)
            throw std::runtime_error("the input zsparticles is not a tetrahedra mesh");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfTriTag))
            throw std::runtime_error("the input zsparticles has no surface tris");
        // if(!zsparticles->hasAuxData(ZenoParticles::s_surfEdgeTag))
        //     throw std::runtime_error("the input zsparticles has no surface lines");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfVertTag)) 
            throw std::runtime_error("the input zsparticles has no surface points");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfHalfEdgeTag))
            throw std::runtime_error("the input zsparticles has no half edge structures");

        auto& tris  = (*zsparticles)[ZenoParticles::s_surfTriTag];
        // auto& lines = (*zsparticles)[ZenoParticles::s_surfEdgeTag];
        auto& points = (*zsparticles)[ZenoParticles::s_surfVertTag];
        auto& halfedges = (*zsparticles)[ZenoParticles::s_surfHalfEdgeTag];
        auto& halffacets = (*zsparticles)[ZenoParticles::s_tetHalfFacetTag];

        auto muscle_id_tag = get_input2<std::string>("muscle_id_tag");


        auto record_newton_step = get_input2<bool>("preserve_newton_step");
        auto nm_record_newton_step = get_input2<int>("nm_record_newton_step");
        if(record_newton_step) {
            for(int i = 0;i != nm_record_newton_step;++i) {
                auto step_name = std::string("x_") + std::to_string(i);
                if(!verts.hasProperty(step_name)) 
                    verts.append_channels(cudaPol,{{step_name,3}});
            }
        }

        auto planeConsPosTag = get_param<std::string>("planeConsPosTag");
        auto planeConsNrmTag = get_param<std::string>("planeConsNrmTag");
        auto planeConsIDTag = get_param<std::string>("planeConsIDTag");
        auto planeConsBaryTag = get_param<std::string>("planeConsBaryTag");

        // auto bone_driven_weight = (T)0.02;

        auto newton_res = get_input2<float>("newton_res");

        auto dt = get_input2<float>("dt");

        auto volf = vec3::from_array(gravity * models.density);

        std::vector<zeno::vec2f> act_;    
        std::size_t nm_acts = 0;

        if(has_input("Acts")) {
            act_ = get_input<zeno::ListObject>("Acts")->getLiterial<zeno::vec2f>();
            nm_acts = act_.size();
        }

        // std::cout << "nmActs:" << nm_acts << std::endl;

        constexpr auto host_space = zs::execspace_e::openmp;
        auto ompExec = zs::omp_exec();
        auto act_buffer = dtiles_t{{{"act",2}},nm_acts,zs::memsrc_e::host};
        ompExec(zs::range(act_buffer.size()),
            [act_buffer = proxy<host_space>({},act_buffer),act_] (int i) mutable {
                act_buffer.tuple(dim_c<2>,"act",i) = vec2(act_[i][0],act_[i][1]);
        });

        act_buffer = act_buffer.clone({zs::memsrc_e::device});

        auto driven_tag = get_input2<std::string>("driven_tag");
        auto bone_driven_weight = get_input2<float>("driven_weight");

        auto bbw = typename ZenoParticles::particles_t({
            {"X",3},
            {"inds",1},
            {"w",4},
            {"strength",1},
            {"cnorm",1}},0,zs::memsrc_e::device);

        auto bverts = typename ZenoParticles::particles_t({
            {"x",3},
            {"intersect",1},
            {"area",1},
            {"strength",1}},0,zs::memsrc_e::device);
        if(has_input<ZenoParticles>("driven_boudary") && zsparticles->hasAuxData(driven_tag)){
            auto zsbones = get_input<ZenoParticles>("driven_boudary");
            const auto& zsbones_verts = zsbones->getParticles();
            bverts.resize(zsbones_verts.size());


            TILEVEC_OPS::copy(cudaPol,zsbones_verts,"x",bverts,"x");
            if(zsbones_verts.hasProperty("strength"))
                TILEVEC_OPS::copy(cudaPol,zsbones_verts,"strength",bverts,"strength");
            else   
                TILEVEC_OPS::fill(cudaPol,bverts,"strength",(T)1.0);

            if(zsbones_verts.hasProperty("intersect"))
                TILEVEC_OPS::copy(cudaPol,zsbones_verts,"intersect",bverts,"intersect");
            else
               TILEVEC_OPS::fill(cudaPol,bverts,"intersect",(T)0.0); 

            if(zsbones_verts.hasProperty("area")) {
                TILEVEC_OPS::copy(cudaPol,zsbones_verts,"area",bverts,"area");
                std::cout << "use dynamic area driven weight" << std::endl;
            } else
                TILEVEC_OPS::fill(cudaPol,bverts,"area",(T)1.0);

            const auto& inbbw = (*zsparticles)[driven_tag];
            bbw.resize(inbbw.size());
            TILEVEC_OPS::copy(cudaPol,inbbw,"X",bbw,"X");
            TILEVEC_OPS::copy(cudaPol,inbbw,"inds",bbw,"inds");
            TILEVEC_OPS::copy(cudaPol,inbbw,"w",bbw,"w");
            TILEVEC_OPS::copy(cudaPol,inbbw,"strength",bbw,"strength");
            TILEVEC_OPS::copy(cudaPol,inbbw,"cnorm",bbw,"cnorm");
        }
        // bverts = bverts.clone({zs::memsrc_e::device});
        // std::cout << "bverts.size() = " << bverts.size() << std::endl;

        auto kverts = typename ZenoParticles::particles_t({
                {"x",3},
                {"xp",3},
                {"b_fail",1},
                {"binderStiffness",1},
                {planeConsIDTag,1},
                {"nrm",3},
                {"area",1}},0,zs::memsrc_e::device);
        auto ktris = typename ZenoParticles::particles_t({
                {"inds",3},
                {"nrm",3}},0,zs::memsrc_e::device);

        dtiles_t gia_res{points.get_allocator(),{
            {"ring_mask",1},
            {"type_mask",1},
            {"color_mask",1},
            {"is_loop_vertex",1}
        },points.size()};
        dtiles_t tri_gia_res{tris.get_allocator(),{
            {"ring_mask",1},
            {"type_mask",1},
            {"color_mask",1}           
        },tris.size()};
        // dtiles_t tris_gia_res{points.get_allocator(),{
        //     ring_mask
        // }}
        // zs::Vector<zs::vec<int,2>> instBuffer{surf_verts_buffer.get_allocator(),surf_verts_buffer.size() * 8};
        dtiles_t inst_buffer_info{tris.get_allocator(),{
            {"pair",2},
            {"type",1},
            {"its_edge_mark",6},
            {"int_points",6}
        },tris.size() * 2};



        bool use_plane_constraint = get_input2<bool>("use_plane_constraint");
        bool use_binder_constraint = get_input2<bool>("use_binder_constraint");
        bool use_kinematic_potential = get_input2<bool>("with_kinematic_potential");

        if(has_input<ZenoParticles>("kinematic_boundary")){
            auto kinematic_boundary = get_input<ZenoParticles>("kinematic_boundary");
            // if (kinematic_boundary.empty())

            // const auto& prim_kverts = kinematic_boundary.verts;
            // auto& prim_kverts_area = kinematic_boundary.attr<float>("area");
            auto& kb_verts = kinematic_boundary->getParticles();
            auto& kb_tris = kinematic_boundary->getQuadraturePoints();

            // auto& kb_tris = kinematic_boundary->getQuadraturePoints();
            // if(kb_tris.getPropertySize("inds") != 3){
            //     fmt::print(fg(fmt::color::red),"the kinematic boundary is not a surface triangulate mesh\n");
            //     throw std::runtime_error("the kinematic boundary is not a surface triangulate mesh");
            // }
            // if(!kb_tris.hasProperty("area")){
            //     fmt::print(fg(fmt::color::red),"the kinematic boundary has no 'area' channel\n");
            //     throw std::runtime_error("the kinematic boundary has no 'area' channel");
            // }     
            kverts.resize(kb_verts.size());
            TILEVEC_OPS::copy<3>(cudaPol,kb_verts,"x",kverts,"x");
            TILEVEC_OPS::copy<3>(cudaPol,kb_verts,"x",kverts,"xp");
            TILEVEC_OPS::copy<3>(cudaPol,kb_verts,"nrm",kverts,"nrm");
            TILEVEC_OPS::fill(cudaPol,kverts,"area",(T)1.0);
            if(kb_verts.hasProperty("b_fail"))
                TILEVEC_OPS::copy(cudaPol,kb_verts,"b_fail",kverts,"b_fail");
            else 
                TILEVEC_OPS::fill(cudaPol,kverts,"b_fail",(T)0.0);
            if(kb_verts.hasProperty("binderStiffness"))
                TILEVEC_OPS::copy(cudaPol,kb_verts,"binderStiffness",kverts,"binderStiffness");
            else 
                TILEVEC_OPS::fill(cudaPol,kverts,"binderStiffness",(T)1.0);
            
            if(kb_verts.hasProperty(planeConsIDTag))
                TILEVEC_OPS::copy(cudaPol,kb_verts,planeConsIDTag,kverts,planeConsIDTag);
            else  
                TILEVEC_OPS::fill(cudaPol,kverts,planeConsIDTag,zs::reinterpret_bits<T>((int)-1));

            ktris.resize(kb_tris.size());
            TILEVEC_OPS::copy<3>(cudaPol,kb_tris,"nrm",ktris,"nrm");
            TILEVEC_OPS::copy<3>(cudaPol,kb_tris,"inds",ktris,"inds");            
        }
        // std::cout << "nm_kb_tris : " << kb_tris.size() << " nm_kb_verts : " << kb_verts.size() << std::endl;
        // cudaPol(zs::range(kb_tris.size()),
        //     [kb_verts = proxy<space>({},kb_verts),kb_tris = proxy<space>({},kb_tris),kverts = proxy<space>({},kverts)] ZS_LAMBDA(int ti) mutable {
        //         auto tri = kb_tris.pack(dim_c<3>,"inds",ti).reinterpret_bits(int_c);
        //         for(int i = 0;i != 3;++i)
        //             atomic_add(exec_cuda,&kverts("area",tri[i]),(T)kb_tris("area",ti)/(T)3.0);
        //         if(ti == 0)
        //             printf("tri[0] area : %f\n",(float)kb_tris("area",ti));
        // });

        // the temp buffer only store the data that will change every iterations or every frame
        dtiles_t vtemp{verts.get_allocator(),
                            {
                                {"grad", 3},
                                {"P", 9},
                                {"bou_tag",1},
                                {"dir", 3},
                                {"xn", 3},
                                {"xp",3},
                                {"vn",3},
                                {"vp",3},
                                {"is_inverted",1},
                                {"active",1},
                                {"k_active",1},
                                {"ring_mask",1},
                                {"color_mask",1},
                                {"type_mask",1},
                                {"embed_tet_id",1},
                                {"grad",3},
                                {"H",9},
                                {"inds",1},
                                {"area",1},
                                {"mu",1},
                                {"lam",1}
                            },verts.size()};
        TILEVEC_OPS::copy(cudaPol,verts,"area",vtemp,"area");
        TILEVEC_OPS::copy(cudaPol,verts,"mu",vtemp,"mu");
        TILEVEC_OPS::copy(cudaPol,verts,"lam",vtemp,"lam");
        

        // auto max_collision_pairs = tris.size() / 10; 
        dtiles_t etemp(eles.get_allocator(), {
                // {"H", 12 * 12},
                    {"ActInv",3*3},
                    {"dfiber",3},
                    // {"muscle_ID",1},
                    {"is_inverted",1},
                    {"fiberStretch",1}
                }, eles.size()
        );

                // {{tags}, cnt, memsrc_e::um, 0}
        dtiles_t sttemp(tris.get_allocator(),
            {
                {"nrm",3},
                {"inds",3},
                {"grad",9},
                {"H",9 * 9},
                {"ring_mask",1}
            },tris.size()
        );
        TILEVEC_OPS::copy(cudaPol,tris,"inds",sttemp,"inds");
        // dtiles_t setemp(lines.get_allocator(),
        //     {
        //         {"nrm",3}
        //     },lines.size()
        // );

        // std::cout << "sttemp.size() << " << sttemp.size() << std::endl;
        // std::cout << "setemp.size() << " << setemp.size() << std::endl;

        bool turn_on_self_collision = get_input2<bool>("use_self_collision");

        // int fp_buffer_size = turn_on_self_collision ? points.size() * MAX_FP_COLLISION_PAIRS : 0;


        dtiles_t self_collision_fp_buffer(points.get_allocator(),{
            {"inds",4},
            {"grad",12},
            {"H",12 * 12},
        },points.size());

        dtiles_t ktris_vert_collision_buffer(points.get_allocator(),{
            {"inds",1},
            {"grad",3},
            {"H",3 * 3}
        },points.size());


        // static dtiles_t ee_buffer(lines.get_allocator(),{
        //     {"inds",4},
        //     {"area",1},
        //     {"inverted",1},
        //     {"abary",2},
        //     {"bbary",2},
        //     {"bary",4}
        // },lines.size());

        // int ee_buffer_size = ee_buffer.size();
        int ee_buffer_size = 0;


        int kc_buffer_size = kverts.size() * MAX_FP_COLLISION_PAIRS;
        // kc_buffer_size = 0;

        dtiles_t kc_buffer(points.get_allocator(),{
            {"inds",2},
            {"area",1},
            {"inverted",1},
        },kc_buffer_size);

        // int kc_buffer_size = kc_buffer.size();
        // int kc_buffer_size = 0;

        dtiles_t gh_buffer(eles.get_allocator(),{
            {"inds",4},
            {"H",12*12},
            {"grad",12}
        },eles.size());


        // dtiles_t tri_gh_buffer(tris.size(),{
        //     {"inds",3},
        //     {"H",9 * 9},
        //     {"grad",9}
        // },tris.size());


        // TILEVEC_OPS::fill<4>(cudaPol,etemp,"inds",zs::vec<int,4>::uniform(-1).template reinterpret_bits<T>())
        // TILEVEC_OPS::copy<4>(cudaPol,eles,"inds",etemp,"inds");
        TILEVEC_OPS::fill<9>(cudaPol,etemp,"ActInv",zs::vec<T,9>{1.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,1.0});
        TILEVEC_OPS::fill<3>(cudaPol,etemp,"dfiber",zs::vec<T,3>::zeros());
        // TILEVEC_OPS::fill(cudaPol,vtemp,"inertia",(T)1.0);
        // if(verts.hasProperty("inertia"))
        //     TILEVEC_OPS::copy(cudaPol,verts,"inertia",vtemp,"inertia");
        if(verts.hasProperty("is_inverted"))
            TILEVEC_OPS::copy(cudaPol,verts,"is_inverted",vtemp,"is_inverted");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"is_inverted",(T)0.0);
        cudaPol(zs::range(vtemp.size()),
            [vtemp = proxy<space>({},vtemp)] ZS_LAMBDA(int vi) mutable {
                vtemp("inds",vi) = reinterpret_bits<T>(vi);
        });
        // apply muscle activation

        if(!eles.hasProperty("Act"))
            eles.append_channels(cudaPol,{{"Act",1}});


        if(!eles.hasProperty("fiberStretch"))
            eles.append_channels(cudaPol,{{"fiberStretch",1}});

        TILEVEC_OPS::fill(cudaPol,eles,"fiberStretch",1.f);
        
        if(!eles.hasProperty("fiber"))
            fmt::print(fg(fmt::color::red),"the quadrature has no \"fiber\"\n");
        if(!verts.hasProperty(muscle_id_tag))
            fmt::print(fg(fmt::color::red),"the verts has no \"{}\" muscle_id_tag\n",muscle_id_tag);
        if(nm_acts == 0)
            fmt::print(fg(fmt::color::red),"no activation input\n");

        auto use_anisotropic_jiggling = get_input2<bool>("use_anisotropic_jiggling");

        cudaPol(zs::range(eles.size()),
            [etemp = proxy<space>({},etemp),eles = proxy<space>({},eles),verts = proxy<space>({},verts),use_anisotropic_jiggling = use_anisotropic_jiggling,
                act_buffer = proxy<space>({},act_buffer),muscle_id_tag = SmallString(muscle_id_tag),nm_acts] ZS_LAMBDA(int ei) mutable {
                // auto act = eles.template pack<3>("act",ei);
                // auto fiber = etemp.template pack<3>("fiber",ei);

                vec3 act{1.0,1.0,1.0};
                vec3 fiber{};
                // float a = 1.0f;
                if(eles.hasProperty("fiber") && verts.hasProperty(muscle_id_tag) && nm_acts > 0 && act_buffer.size() > 0 && fabs(eles.template pack<3>("fiber",ei).norm() - 1.0) < 0.001){
                    // printf("apply muscle activation\n");
                    const auto& inds = eles.template pack<4>("inds",ei).template reinterpret_bits<int>();
                    fiber = eles.template pack<3>("fiber",ei);
                    int ID = (int)eles(muscle_id_tag,ei);
                    // if(verts.hasProperty(muscle_id_tag)) {
                    auto tet = eles.pack(dim_c<4>,"inds",ei,int_c);
                    auto nm_ids = zs::vec<int,4>::uniform(0);
                    for(int i = 0;i != 4;++i){
                        auto idx0 = (int)verts(muscle_id_tag,tet[i]);
                        if(idx0 < 0)
                            continue;
                        for(int j = i+1;j != 4;++j){
                            auto idx1 = (int)verts(muscle_id_tag,tet[j]);
                            if(idx1 < 0)
                                continue;
                            if(idx0 == idx1){
                                nm_ids[i] = nm_ids[i] + 1;
                                nm_ids[j] = nm_ids[j] + 1;
                            }
                        }
                    }
                    int max_ID = -1;
                    int max_IDs = -1;
                    for(int i = 0;i != 4;++i) {
                        auto idx = (int)verts(muscle_id_tag,tet[i]);
                        if(idx < 0)
                            continue;
                        if(nm_ids[i] > max_IDs) {
                            max_IDs = nm_ids[i];
                            max_ID = (int)verts(muscle_id_tag,tet[i]);
                        }
                    }
                    if(max_ID < 0)
                        return;
                    ID = max_ID;
                    // }
                    
                    auto a = 1. - act_buffer("act",0,ID);
                    auto b = 1. - act_buffer("act",1,ID);
                    // act = vec3{zs::sqrt(a),zs::sqrt(1./a),zs::sqrt(1./a)};
                    // auto aclamp = 
                    // act = vec3{a < 0.7 ? 0.7 : a,zs::sqrt(1./a),zs::sqrt(1./a)};
                    act = vec3{a,zs::sqrt(1./b),zs::sqrt(1./b)};
                    eles("Act",ei) = act_buffer("act",0,ID) + 1e-6;
                    if(use_anisotropic_jiggling){
                        zs::vec<T,3,3> F{};
                        F = LSL_GEO::deformation_gradient(
                            verts.template pack<3>("x",inds[0]),
                            verts.template pack<3>("x",inds[1]),
                            verts.template pack<3>("x",inds[2]),
                            verts.template pack<3>("x",inds[3]),
                            eles.template pack<3,3>("IB",ei));
                        auto dfiber = F * fiber;
                        auto dfiberN = dfiber.norm();
                        auto fiberN = fiber.norm();
                        dfiber = dfiber / dfiber.norm();
                        etemp.tuple(dim_c<3>,"dfiber",ei) = dfiber;
                        eles("fiberStretch",ei) = dfiberN / fiberN;
                    }
                }else{
                    fiber = zs::vec<T,3>(1.0,0.0,0.0);
                    act = vec3{1,1,1};
                    eles("Act",ei) = (T)0.0;
                    
                }
                if(fabs(fiber.norm() - 1.0) > 0.1) {
                    printf("invalid fiber[%d] detected : %f %f %f\n",(int)ei,
                        (float)fiber[0],(float)fiber[1],(float)fiber[2]);
                }

                vec3 dir[3];
                dir[0] = fiber;
                auto tmp = vec3{0.0,1.0,0.0};
                dir[1] = dir[0].cross(tmp);
                if(dir[1].length() < 1e-3) {
                    tmp = vec3{0.0,0.0,1.0};
                    dir[1] = dir[0].cross(tmp);
                }

                dir[1] = dir[1] / dir[1].length();
                dir[2] = dir[0].cross(dir[1]);
                dir[2] = dir[2] / dir[2].length();

                auto R = mat3{};
                for(int i = 0;i < 3;++i)
                    for(int j = 0;j < 3;++j)
                        R(i,j) = dir[j][i];

                auto Act = mat3::zeros();
                Act(0,0) = act[0];
                Act(1,1) = act[1];
                Act(2,2) = act[2];

                Act = R * Act * R.transpose();
                etemp.template tuple<9>("ActInv",ei) = zs::inverse(Act);
                // if(a < 1.0f) {
                //     auto ActInv = etemp.template pack<3,3>("ActInv",ei);
                //     printf("ActInv[%d] : \n%f %f %f\n%f %f %f\n%f %f %f\n",ei,
                //         (float)ActInv(0,0),(float)ActInv(0,1),(float)ActInv(0,2),
                //         (float)ActInv(1,0),(float)ActInv(1,1),(float)ActInv(1,2),
                //         (float)ActInv(2,0),(float)ActInv(2,1),(float)ActInv(2,2));
                // }
        });
        auto collisionStiffness = get_input2<float>("cstiffness");
        auto kineCollisionStiffness = get_input2<float>("kineCstiffness");


        // auto inset_ratio = get_input2<float>("collision_inset");
        // auto outset_ratio = get_input2<float>("collision_outset");    

        auto in_collisionEps = get_input2<float>("in_collisionEps");
        auto out_collisionEps = get_input2<float>("out_collisionEps");

        auto kine_in_collisionEps = get_input2<float>("kine_inCollisionEps");
        auto kine_out_collisionEps = get_input2<float>("kine_outCollisionEps");

        auto aniso_strength = get_input2<float>("aniso_strength");

        auto binderStiffness = get_input2<float>("binderStiffness");
        auto binderTag = get_param<std::string>("binderTag");
        auto binderThicknessTag = get_param<std::string>("binderThicknessTag");
        auto binderInversionTag = get_param<std::string>("binderInversionTag");

        auto planeConsStiffness = get_input2<float>("planeConsStiffness");

        FEMDynamicSteppingSystem A{
            verts,eles,
            points,tris,
            (T)in_collisionEps,(T)out_collisionEps,
            bbw,bverts,bone_driven_weight,
            volf,dt,collisionStiffness,
            (T)kine_in_collisionEps,(T)kine_out_collisionEps,
            (T)kineCollisionStiffness,(T)aniso_strength,(T)binderStiffness,(T)planeConsStiffness};

        // std::cout << "set initial guess" << std::endl;
        // setup initial guess
        // if(verts.hasProperty("dt")) {
        //     std::cout << "verts has property 'dt'" << std::endl;
        // }

        TILEVEC_OPS::copy<3>(cudaPol,verts,"x",vtemp,"xp");
        TILEVEC_OPS::copy<3>(cudaPol,verts,"v",vtemp,"vp");
        if(verts.hasProperty("active"))
            TILEVEC_OPS::copy(cudaPol,verts,"active",vtemp,"active");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"active",(T)1.0);

        if(verts.hasProperty("k_active"))
            TILEVEC_OPS::copy(cudaPol,verts,"k_active",vtemp,"k_active");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"k_active",(T)1.0);

        // if there is no init_x as guess, then use the baraff witkin approach
        // if(verts.hasProperty("init_x"))
        //     TILEVEC_OPS::copy<3>(cudaPol,verts,"init_x",vtemp,"xn");   
        // else {
            // TILEVEC_OPS::add<3>(cudaPol,vtemp,"xp",1.0,"vp",dt,"xn");
        TILEVEC_OPS::copy(cudaPol,verts,"v",vtemp,"vn");  
        TILEVEC_OPS::copy(cudaPol,verts,"x",vtemp,"xn");
            // TILEVEC_OPS::add<3>(cudaPol,verts,"x",1.0,"vp",(T)0.0,"xn");  
        // }
        if(verts.hasProperty("bou_tag") && verts.getPropertySize("bou_tag") == 1)
            TILEVEC_OPS::copy(cudaPol,verts,"bou_tag",vtemp,"bou_tag");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"bou_tag",(T)0.0);

        int max_newton_iterations = get_input2<int>("max_newton_iters");
        int nm_iters = 0;
        // make sure, at least one baraf simi-implicit step will be taken
        auto res0 = 1e10;

        auto kd_alpha = get_input2<float>("kd_alpha");
        auto kd_beta = get_input2<float>("kd_beta");
        auto kd_theta = get_input2<float>("kd_theta");

        auto max_cg_iters = get_param<int>("max_cg_iters");


        bool use_line_search = get_param<bool>("use_line_search");

        zs::CppTimer timer;

        #ifdef USE_SPARSE_MATRIX

        timer.tick();


        spmat_t spmat{};
        zs::Vector<int> is{verts.get_allocator(),verts.size()};
        zs::Vector<int> js{verts.get_allocator(),verts.size()};
        // init diagonal entries
        // cudaPol(zs::range(verts.size()),
        //         [is = proxy<space>(is),js = proxy<space>(js)] ZS_LAMBDA(int vi) mutable {
        //     is[vi] = js[vi] = vi;
        // });
        cudaPol(enumerate(is, js), [] ZS_LAMBDA(int no, int &i, int &j) mutable { i = j = no; });
        auto reserveStorage = [&is, &js](std::size_t n) {
            auto size = is.size();
            is.resize(size + n);
            js.resize(size + n);
            return size;
        };

        // init tet incidents' entries, off-diagonal
        auto tets_entry_offset = reserveStorage(eles.size() * 6);
        cudaPol(zs::range(eles.size()),[offset = tets_entry_offset,
                stride = eles.size(),
                is = proxy<space>(is),
                js = proxy<space>(js),
                eles = proxy<space>({},eles)] ZS_LAMBDA(int ei) mutable {
            auto inds = eles.pack(dim_c<4>,"inds",ei,int_c);
            for (int d = 1; d < 4; ++d)
                for (int k = 0; k < 4 - d; ++k)
                    if (inds[k] > inds[k + 1]) {
                        auto t = inds[k];
                        inds[k] = inds[k + 1];
                        inds[k + 1] = t;
                    }

            // <0, 1>, <0, 2>, <0, 3>, <1, 2>, <1, 3>, <2, 3>
            is[offset + ei] = inds[0];
            is[offset + stride + ei] = inds[0];
            is[offset + stride * 2 + ei] = inds[0];
            is[offset + stride * 3 + ei] = inds[1];
            is[offset + stride * 4 + ei] = inds[1];
            is[offset + stride * 5 + ei] = inds[2];

            js[offset + ei] = inds[1];
            js[offset + stride + ei] = inds[2];
            js[offset + stride * 2 + ei] = inds[3];
            js[offset + stride * 3 + ei] = inds[2];
            js[offset + stride * 4 + ei] = inds[3];
            js[offset + stride * 5 + ei] = inds[3];

            // js[offset + stride * 6 + ei] = inds[0];
            // js[offset + stride * 7 + ei] = inds[0];
            // js[offset + stride * 8 + ei] = inds[0];
            // js[offset + stride * 9 + ei] = inds[1];
            // js[offset + stride * 10 + ei] = inds[1];
            // js[offset + stride * 11 + ei] = inds[2];

            // is[offset + stride * 6 + ei] = inds[1];
            // is[offset + stride * 7 + ei] = inds[2];
            // is[offset + stride * 8 + ei] = inds[3];
            // is[offset + stride * 9 + ei] = inds[2];
            // is[offset + stride * 10 + ei] = inds[3];
            // is[offset + stride * 11 + ei] = inds[3];

        });

        spmat = spmat_t{verts.get_allocator(),(int)verts.size(),(int)verts.size()};
        spmat.build(cudaPol,(int)verts.size(),(int)verts.size(),zs::range(is),zs::range(js),zs::false_c);
        spmat.localOrdering(cudaPol, zs::false_c);
        spmat._vals.resize(spmat.nnz());
        spmat._vals.reset(0);   

        timer.tock("setup spmat");

        #endif

        auto cnorm = compute_average_edge_length(cudaPol,kverts,"x",ktris);

        auto use_sticky_condition = get_input2<bool>("use_sticky_condition");

        zs::Vector<zs::vec<int,4>> csPT{points.get_allocator(),points.size()};
        int nm_csPT = 0;

        // for kinematic colllision
        auto kinematics = RETRIEVE_OBJECT_PTRS(ZenoParticles,"kinematics");
        // auto kinematic = get_input<ZenoParticles>("kinematic");
        auto use_kinematics_collision = get_input2<bool>("use_kinematics_collision");
        // dtiles_t verts_buffer{points.get_allocator(),{
        //     {"inds",1},
        //     {"x",3},
        //     {"he_inds",1},
        //     {"ring_mask",1},
        // },points.size()};
        // TILEVEC_OPS::copy(cudaPol,points,"inds",verts_buffer,"inds");
        // TILEVEC_OPS::copy(cudaPol,points,"he_inds",verts_buffer,"he_inds");
        // topological_sample(cudaPol,points,verts,"x",verts_buffer);
        // TILEVEC_OPS::fill(cudaPol,verts_buffer,"ring_mask",zs::reinterpret_bits<T>((int)0));
        
        // dtiles_t tri_buffer{tris.get_allocator(),{
        //     {"inds",3},
        //     {"he_inds",1},
        //     {"ring_mask",1}
        // },tris.size()};
        // TILEVEC_OPS::copy(cudaPol,tris,"inds",tri_buffer,"inds");
        // TILEVEC_OPS::copy(cudaPol,tris,"he_inds",tri_buffer,"he_inds");
        // reorder_topology(cudaPol,points,tri_buffer);
        // TILEVEC_OPS::fill(cudaPol,tri_buffer,"ring_mask",zs::reinterpret_bits<T>((int)0));

        while(nm_iters < max_newton_iterations) {
            if(record_newton_step && nm_iters < nm_record_newton_step) {
                auto record_attr_name = std::string("x_") + std::to_string(nm_iters);
                TILEVEC_OPS::copy(cudaPol,vtemp,"xn",verts,record_attr_name);
            }

            // break;
            T e0 = (T)0;
            if(use_line_search){
                match([&](auto &elasticModel){
                    A.computeElasticBonesEnergy(cudaPol, elasticModel,vtemp,etemp,e0);
                },[](...) {
                    throw std::runtime_error("unsupported anisotropic elasticity model");
                })(models.getElasticModel());      
            }      

            TILEVEC_OPS::fill(cudaPol,gh_buffer,"grad",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,gh_buffer,"H",(T)0.0);  
            TILEVEC_OPS::fill<4>(cudaPol,gh_buffer,"inds",zs::vec<int,4>::uniform(-1).reinterpret_bits(float_c)); 
            TILEVEC_OPS::fill(cudaPol,vtemp,"grad",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,vtemp,"H",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,sttemp,"grad",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,sttemp,"H",(T)0.0);

            A.findInversion(cudaPol,vtemp,etemp);  

            // match([&](auto &elasticModel,auto &anisoModel) -> std::enable_if_t<zs::is_same_v<RM_CVREF_T(anisoModel),zs::AnisotropicArap<float>>> {...},[](...) {
            //     A.computeGradientAndHessian(cudaPol, elasticModel,vtemp,etemp,gh_buffer,kd_alpha,kd_beta);
            // })(models.getElasticModel(),models.getAnisoElasticModel());
            timer.tick();
            match([&](auto &elasticModel,zs::AnisotropicArap<float> &anisoModel){
                A.computeGradientAndHessian(cudaPol, elasticModel,anisoModel,vtemp,etemp,gh_buffer,use_kinematic_potential,kd_alpha,kd_beta,use_anisotropic_jiggling);
            },[](...) {
                throw std::runtime_error("unsupported anisotropic elasticity model");
            })(models.getElasticModel(),models.getAnisoElasticModel());

            // {
            //     auto gradn = TILEVEC_OPS::dot<12>(cudaPol,gh_buffer,"grad","grad");
            //     std::cout << "gradn after elastic and inertia : " << gradn << std::endl;
            //     if(std::isnan(gradn)) {
            //         printf("nan gradn = %f detected after compute computeGradientAndHessian\n",gradn);
            //         // printf("Hn = ")
            //         throw std::runtime_error("nan gradn detected after compute computeGradientAndHessian");
            //     }
            // }

            // std::cout << "computePositionConstraintGradientAndHessian : " << kverts.size() << std::endl;
            // the binder constraint gradient and hessian
            if(use_binder_constraint) {
                std::cout << "apply binder constraint " << std::endl;
                A.computePositionConstraintGradientAndHessian(cudaPol,
                    vtemp,
                    binderTag,
                    binderThicknessTag,
                    binderInversionTag,
                    kverts,
                    gh_buffer);
                std::cout << "finish apply position constraint" << std::endl;
            }else {
                std::cout << "apply no binder constraint" << std::endl;
            }
            if(verts.hasProperty(planeConsIDTag) && use_plane_constraint){
                std::cout << "apply plane constraint" << std::endl;
                // A.computePlaneConstraintGradientAndHessian(cudaPol,
                
                A.computePlaneConstraintGradientAndHessian2(cudaPol,
                    vtemp,
                    sttemp,
                    kverts,
                    ktris,
                    planeConsBaryTag,
                    planeConsIDTag,
                    vtemp,
                    sttemp,cnorm,use_sticky_condition);

                    auto v2kt_force = TILEVEC_OPS::dot<3>(cudaPol,vtemp,"grad","grad");
                    auto kv2t_force = TILEVEC_OPS::dot<9>(cudaPol,sttemp,"grad","grad");
                    std::cout << "v2kt_force : " << v2kt_force << std::endl;
                    std::cout << "kv2t_force : " << kv2t_force << std::endl;
            }
            else{
                std::cout << "apply no plane constraint : " << 
                    verts.hasProperty(planeConsPosTag) << "\t" << 
                    verts.hasProperty(planeConsNrmTag) << "\t" << 
                    verts.hasProperty(planeConsIDTag) << "\t" << use_plane_constraint << std::endl;
            }
            if(use_kinematics_collision && kinematics.size() > 0) {
                std::cout << "apply kinematic constraint " << std::endl;
                zs::bht<int,2,int> csPT{vtemp.get_allocator(),10000};
                csPT.reset(cudaPol,true);
            #if 0
                // auto nm_csPT = COLLISION_UTILS::do_tetrahedra_surface_mesh_and_kinematic_boundary_collision_detection(cudaPol,
                //     kinematics[0],
                //     vtemp,"xn",
                //     eles,
                //     points,tris,
                //     halfedges,
                //     out_collisionEps,
                //     in_collisionEps,
                //     csPT,false);
                
                COLLISION_UTILS::evaluate_tri_kvert_collision_gradient_and_hessian(cudaPol,
                    kinematics,
                    vtemp,"xn","area",
                    tris,"area",
                    csPT,
                    sttemp,collisionStiffness,out_collisionEps,true);
                auto cforce = TILEVEC_OPS::dot<9>(cudaPol,sttemp,"grad","grad");
                std::cout << "nm_csPT = " << nm_csPT << "\tkin_cforce : " << cforce << std::endl;
            #else
                std::cout << "apply kinematic collision" << std::endl;
                // auto nm_csPT = COLLISION_UTILS::do_tetrahedra_surface_points_and_kinematic_boundary_collision_detection(cudaPol,
                //     kinematics[0],
                //     vtemp,"xn",
                //     eles,
                //     points,tris,
                //     halfedges,
                //     out_collisionEps,
                //     in_collisionEps,
                //     csPT,false,false);

                COLLISION_UTILS::evaluate_ktri_vert_collision_gradient_and_hessian(cudaPol,
                    kinematics[0],
                    vtemp,"xn","area",
                    tris,"area",
                    points,
                    csPT,
                    ktris_vert_collision_buffer,collisionStiffness,out_collisionEps,false);
                auto cforce = TILEVEC_OPS::dot<3>(cudaPol,ktris_vert_collision_buffer,"grad","grad");
                std::cout << "nm_csPT = " << nm_csPT << "\tkin_cforce : " << cforce << std::endl;
            #endif
            }


            if(turn_on_self_collision) {
                zs::bht<int,2,int> csPT{gh_buffer.get_allocator(),10000};
                csPT.reset(cudaPol,true);
                COLLISION_UTILS::do_tetrahedra_surface_tris_and_points_self_collision_detection(
                    cudaPol,vtemp,"xn",
                    eles,
                    points,tris,
                    halfedges,halffacets,
                    out_collisionEps,
                    in_collisionEps,
                    csPT);

                COLLISION_UTILS::evaluate_fp_self_collision_gradient_and_hessian(cudaPol,
                    vtemp,"xn","area",
                    points,
                    tris,"area",
                    csPT,self_collision_fp_buffer,collisionStiffness,out_collisionEps);

                auto cforce = TILEVEC_OPS::dot<12>(cudaPol,self_collision_fp_buffer,"grad","grad");
                std::cout << "cforce " << cforce << "\tnmcsPT : " << csPT.size() << std::endl;
            }

            timer.tock("eval hessian and gradient");
            timer.tick();
            // TILEVEC_OPS::fill(cudaPol,vtemp,"grad",(T)0.0); 
            TILEVEC_OPS::assemble(cudaPol,gh_buffer,"grad","inds",vtemp,"grad");
            TILEVEC_OPS::assemble(cudaPol,sttemp,"grad","inds",vtemp,"grad");

            if(turn_on_self_collision)
                TILEVEC_OPS::assemble(cudaPol,self_collision_fp_buffer,"grad","inds",vtemp,"grad");
            if(use_kinematics_collision && kinematics.size() > 0) 
                TILEVEC_OPS::assemble(cudaPol,ktris_vert_collision_buffer,"grad","inds",vtemp,"grad");

            TILEVEC_OPS::fill(cudaPol,vtemp,"P",(T)0.0);
            PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",gh_buffer,"P",vtemp,false,true);
            if(turn_on_self_collision)
                PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",self_collision_fp_buffer,"P",vtemp,false,true);
            if(use_kinematics_collision && kinematics.size() > 0) 
                PCG::prepare_block_diagonal_preconditioner<1,3>(cudaPol,"H",ktris_vert_collision_buffer,"P",vtemp,false,true);
            PCG::prepare_block_diagonal_preconditioner<3,3>(cudaPol,"H",sttemp,"P",vtemp,false,true);
            PCG::prepare_block_diagonal_preconditioner<1,3>(cudaPol,"H",vtemp,"P",vtemp,true,true);
            timer.tock("precondition and assemble setup");

            // eval sparse matrix
            #ifdef USE_SPARSE_MATRIX
            timer.tick();
            spmat._vals.reset(0);  


            cudaPol(zs::range(eles.size()),
                [gh_buffer = proxy<space>({},gh_buffer),
                        spmat = view<space>(spmat),
                        vsize = verts.size(),
                        verts = proxy<space>({},verts)] ZS_LAMBDA(int ei) mutable {
                    auto inds = gh_buffer.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                    for(int i = 0;i != 4;++i)
                        if(inds[i] < 0 || inds[i] >= vsize)
                            printf("negative sttemp inds : %d %d %d, %d\n",inds[0],inds[1],inds[2],inds[3]);
                    auto H = gh_buffer.pack(dim_c<12,12>,"H",ei);

                    // if(isnan(H.norm())) {
                    //     printf("nan ele[%d %d %d %d] detected\n",inds[0],inds[1],inds[2],inds[3]);
                    // }
                    update_hessian(spmat,inds,H,true);
            });

            // std::cout << "update sttemp spmat" << std::endl;

             cudaPol(zs::range(sttemp.size()),
                [sttemp = proxy<space>({},sttemp),vsize = verts.size(),spmat = proxy<space>(spmat)] ZS_LAMBDA(int vi) mutable {
                    auto inds = sttemp.pack(dim_c<3>,"inds",vi,int_c);
                    for(int i = 0;i != 3;++i)
                        if(inds[i] < 0 || inds[i] >= vsize)
                            printf("negative sttemp inds : %d %d %d\n",inds[0],inds[1],inds[2]);
                    auto H = sttemp.pack(dim_c<9,9>,"H",vi);
                    if(isnan(H.norm())) {
                        printf("nan ele[%d %d %d] detected\n",inds[0],inds[1],inds[2]);
                    }
                    update_hessian(spmat,inds,H,true);
            });

            // std::cout << "update vtemp spmat" << std::endl;

            cudaPol(zs::range(vtemp.size()),
                [vtemp = proxy<space>({},vtemp),vsize = verts.size(),spmat = proxy<space>(spmat)] ZS_LAMBDA(int vi) mutable {
                    auto inds = vtemp.pack(dim_c<1>,"inds",vi,int_c);
                    if(inds[0] < 0 || inds[0] >= vsize)
                        printf("negative vtemp_inds : %d\n",inds[0]);
                    auto H = vtemp.pack(dim_c<3,3>,"H",vi);
                    if(isnan(H.norm())) {
                        printf("nan ele[%d] detected\n",vi);
                    }
                    update_hessian(spmat,inds,H,true);
            });

            // std::cout << "update kinematics spmat" << std::endl;

            if(use_kinematics_collision && kinematics.size() > 0)
                cudaPol(zs::range(ktris_vert_collision_buffer.size()),[
                            ktris_vert_collision_buffer = proxy<space>({},ktris_vert_collision_buffer),
                            spmat = proxy<space>(spmat)] ZS_LAMBDA(int ci) mutable {
                        auto inds = ktris_vert_collision_buffer.pack(dim_c<1>,"inds",ci,int_c);
                        auto H = ktris_vert_collision_buffer.pack(dim_c<3,3>,"H",ci);
                        update_hessian(spmat,inds,H,true);
                });

            // std::cout << "finish upate hessian" << std::endl;

            timer.tock("spmat evaluation");
            #endif
            // PCG::precondition<3>(cudaPol,vtemp,"P","grad","q");
            // T res = TILEVEC_OPS::inf_norm<3>(cudaPol, vtemp, "q");
            // if(res < newton_res){
            //     fmt::print(fg(fmt::color::cyan),"reach desire newton res {} : {}\n",newton_res,res);
            //     break;
            // }
            // auto nP = TILEVEC_OPS::inf_norm<9>(cudaPol,vtemp,"P");
            // std::cout << "nP : " << nP << std::endl;
            // PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",etemp,"P",vtemp);
            // if the grad is too small, return the result
            // Solve equation using PCG
            timer.tick();
            TILEVEC_OPS::fill(cudaPol,vtemp,"dir",(T)0.0);
            // std::cout << "solve using pcg" << std::endl;
            // auto Hn = TILEVEC_OPS::dot<12 * 12>(cudaPol,gh_buffer,"H","H");
            // std::cout << "Hn : " << Hn << std::endl;
            int nm_CG_iters = 0;
            #ifdef USE_SPARSE_MATRIX
                // auto gradn = TILEVEC_OPS::dot<3>(cudaPol,vtemp,"grad","grad");
                // if(std::isnan(gradn)) {
                //     printf("nan gradn = %f detected\n",gradn);
                //     // printf("Hn = ")
                //     throw std::runtime_error("nan gradn detected");
                // }

                if(turn_on_self_collision)
                    nm_CG_iters = PCG::pcg_with_fixed_sol_solve<3>(cudaPol,vtemp,spmat,self_collision_fp_buffer,"dir","bou_tag","grad","P","inds","H",(T)cg_res,max_cg_iters,100);
                else
                    nm_CG_iters = PCG::pcg_with_fixed_sol_solve<3>(cudaPol,vtemp,spmat,"dir","bou_tag","grad","P","inds","H",(T)cg_res,max_cg_iters,100);

            #else
                nm_CG_iters = PCG::pcg_with_fixed_sol_solve<3,4>(cudaPol,vtemp,gh_buffer,"dir","bou_tag","grad","P","inds","H",(T)cg_res,max_cg_iters,100);
            #endif
            timer.tock("CG SOLVER");
            fmt::print(fg(fmt::color::cyan),"nm_cg_iters : {}\n",nm_CG_iters);
            // T alpha = 1.;

            // auto nxn = TILEVEC_OPS::inf_norm<3>(cudaPol,vtemp,"xn");
            // auto ndir = TILEVEC_OPS::dot<3>(cudaPol,vtemp,"dir","dir");
            // auto nP = TILEVEC_OPS::dot<9>(cudaPol,vtemp,"P","P");

            // std::cout << "vtemp's xn : " << nxn << std::endl;
            // std::cout << "vtemp's dir : " << ndir << std::endl;
            // std::cout << "vtemp's P : " << nP << std::endl;

            if(use_line_search) {
                int search_idx = 0;     
                T alpha = (T)2.0;
                T beta = (T)0.5;
                T c1 = (T)0.0001;

                auto eg0 = (T)-1.0 * TILEVEC_OPS::dot<3>(cudaPol,vtemp,"grad","dir");
                if(eg0 > 0)
                    throw std::runtime_error("invalid searching direction");
                double armijo_condition;
                int max_line_search = 5;


                do {
                    if(search_idx != 0){
                        TILEVEC_OPS::add<3>(cudaPol,vtemp,"xn",(T)1.0,"dir",-alpha,"xn"); 
                    }
                    alpha *= beta;
                    TILEVEC_OPS::add<3>(cudaPol,vtemp,"xn",(T)1.0,"dir",alpha,"xn"); 
                    T e1;
                    match([&](auto &elasticModel){
                        A.computeElasticBonesEnergy(cudaPol, elasticModel,vtemp,etemp,e1);
                    },[](...) {
                        throw std::runtime_error("unsupported anisotropic elasticity model");
                    })(models.getElasticModel());

                    ++search_idx;            

                    armijo_condition = double(e1) - double(e0) - double(c1) * double(alpha) * double(eg0);   
                }while(armijo_condition > 0.0);

            }else{
                cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),dt] __device__(int i) mutable {
                    vtemp.template tuple<3>("xn", i) =
                        vtemp.template pack<3>("xn", i) + vtemp.template pack<3>("dir", i);
                    vtemp.template tuple<3>("vn",i) = 
                        (vtemp.template pack<3>("xn",i) - vtemp.template pack<3>("xp",i))/dt; 
                });
            }
            nm_iters++;
        }


        cudaPol(zs::range(verts.size()),
                [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),dt = dt] __device__(int vi) mutable {
                    // auto newX = vtemp.pack(dim_c<3>,"xn", vi);
                    verts.tuple<3>("x", vi) = vtemp.pack(dim_c<3>,"xn", vi);
                    // if(verts.hasProperty("dt"))
                    //     dt = verts("dt",vi);
                    verts.tuple<3>("v",vi) = vtemp.pack<3>("vn",vi);
                });

        // auto do_gia_analysis_at_end_of_step = get_input2<bool>("record_gia_analysis");

        set_output("ZSParticles", zsparticles);
    }
};

ZENDEFNODE(FleshDynamicStepping, {{"ZSParticles","kinematic_boundary",
                                    "gravity","Acts",
                                    "driven_boudary",
                                    "kinematics",
                                    {"int","max_newton_iters","5"},
                                    {"float","cg_res","0.0001"},
                                    {"string","driven_tag","bone_bw"},
                                    {"float","driven_weight","0.02"},
                                    {"string","muscle_id_tag","ms_id_tag"},
                                    {"float","cstiffness","0.0"},
                                    {"float","in_collisionEps","0.01"},
                                    {"float","out_collisionEps","0.01"},
                                    {"float","kineCstiffness","1"},
                                    {"float","kine_inCollisionEps","0.01"},
                                    {"float","kine_outCollisionEps","0.02"},
                                    {"float","dt","0.5"},
                                    {"float","newton_res","0.001"},
                                    {"float","kd_alpha","0.01"},
                                    {"float","kd_beta","0.01"},
                                    {"float","kd_theta","0.01"},
                                    {"float","aniso_strength","1.0"},
                                    {"float","binderStiffness","1.0"},
                                    {"float","planeConsStiffness","0.01"},
                                    {"bool","use_plane_constraint","0"},
                                    {"bool","use_binder_constraint","0"},
                                    {"bool","use_self_collision","0"},
                                    {"bool","use_sticky_condition","0"},
                                    {"bool","with_kinematic_potential","0"},
                                    {"bool","use_kinematics_collision","0"},
                                    {"bool","use_anisotropic_jiggling","0"},
                                    {"bool","preserve_newton_step","0"},
                                    {"int","nm_record_newton_step","1"}
                                    },
                                  {"ZSParticles"},
                                  {
                                    {"int","max_cg_iters","1000"}, 
                                    {"string","binderTag","binder_tag"},
                                    {"string","binderThicknessTag","binder_thickness"},
                                    {"string","binderInversionTag","binder_inversion"},
                                    {"string","planeConsPosTag","planeConsPosTag"},
                                    {"string","planeConsNrmTag","planeConsNrmTag"},
                                    {"string","planeConsIDTag","planeConsIDTag"},
                                    {"string","planeConsBaryTag","planeConsBaryTag"},
                                    {"bool","use_line_search","0"}
                                  },
                                  {"FEM"}});
};