#include "hip/hip_runtime.h"
#include "kernel/bary_centric_weights.hpp"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

#include "zensim/container/Bcht.hpp"
#include "kernel/tiled_vector_ops.hpp"

#include <iostream>

namespace zeno{

using T = float;
using vec3 = zs::vec<T,3>;
using vec4 = zs::vec<T,4>;
using mat3 = zs::vec<T,3,3>;
using mat4 = zs::vec<T,4,4>;


// 给定一个四面网格与一组点，计算每个点在四面体网格单元中的质心坐标
struct ZSComputeBaryCentricWeights2 : INode {
    void apply() override {
        using namespace zs;


    }
};



struct ZSComputeBaryCentricWeights : INode {
    void apply() override {
        using namespace zs;

        // fmt::print("ENTERING NODES\n");
        // std::cout << "ENTERING NODES" << std::endl;

        auto zsvolume = get_input<ZenoParticles>("zsvolume");
        auto zssurf = get_input<ZenoParticles>("zssurf");
        auto mark_embed_elm = get_input2<int>("mark_elm");
        // the bvh of zstets
        // auto lbvh = get_input<zeno::LBvh>("lbvh");
        auto thickness = get_param<float>("bvh_thickness");
        auto fitting_in = get_param<int>("fitting_in");

        auto bvh_channel = get_param<std::string>("bvh_channel");
        auto tag = get_input2<std::string>("tag");

        auto& verts = zsvolume->getParticles();
        auto& eles = zsvolume->getQuadraturePoints();

        const auto& everts = zssurf->getParticles();
        // const auto& e_eles = zssurf->getQuadraturePoints();

        auto &bcw = (*zsvolume)[tag];

        bcw = typename ZenoParticles::particles_t({
            {"X",3},
            {"inds",1},
            {"w",4},
            {"strength",1},
            {"cnorm",1}},everts.size(),zs::memsrc_e::device,0);
        



        // auto topo_tag = tag + std::string("_topo");
        // auto &bcw_topo = (*zsvolume)[topo_tag];

        // auto e_dim = e_eles.getPropertySize("inds");
        // bcw_topo = typename ZenoParticles::particles_t({{"inds",e_dim}},e_eles.size(),zs::memsrc_e::device,0);


        auto cudaExec = zs::cuda_exec();
        const auto numFEMVerts = verts.size();
        const auto numFEMEles = eles.size();
        const auto numEmbedVerts = bcw.size();
        // const auto numEmbedEles = e_eles.size();
        constexpr auto space = zs::execspace_e::cuda;

        TILEVEC_OPS::copy<3>(cudaExec,everts,"x",bcw,"X");

        compute_barycentric_weights(cudaExec,verts,eles,everts,"x",bcw,"inds","w",thickness,fitting_in);

        cudaExec(zs::range(numEmbedVerts),
            [bcw = proxy<space>({},bcw),fitting_in] ZS_LAMBDA(int vi) mutable {
                auto idx = reinterpret_bits<int>(bcw("inds",vi));
                if(fitting_in && idx < 0)
                    printf("Unbind vert %d under fitting-in mode\n",vi);
            }
        );


        // cudaExec(zs::range(e_eles.size()),[e_dim = e_dim,
        //     e_eles = proxy<space>({},e_eles),bcw_topo = proxy<space>({},bcw_topo)] ZS_LAMBDA(int ei) mutable {
        //         for(int i = 0;i != e_dim;++i)
        //             bcw_topo("inds",i,ei) = e_eles("inds",i,ei);
        // });


        cudaExec(zs::range(numEmbedVerts),
            [bcw = proxy<space>({},bcw)] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(bcw)::value_type;
                bcw("cnorm",vi) = (T)0.;
        });

        zs::Vector<T> nmEmbedVerts(eles.get_allocator(),eles.size());
        cudaExec(zs::range(eles.size()),[nmEmbedVerts = proxy<space>(nmEmbedVerts)]
            ZS_LAMBDA(int ei) mutable{
                using T = typename RM_CVREF_T(bcw)::value_type;
                nmEmbedVerts[ei] = (T)0.;
        });

        // if(e_dim !=3 && e_dim !=4) {
        //     throw std::runtime_error("INVALID EMBEDDED PRIM TOPO");
        // }  

        if(mark_embed_elm && everts.hasProperty("tag")){
            eles.append_channels(cudaExec,{{"nmBones",1},{"bdw",1}});

            cudaExec(zs::range(eles.size()),
                [eles = proxy<space>({},eles)] ZS_LAMBDA(int elm_id) mutable{
                    eles("nmBones",elm_id) = (T)0.0;
                    eles("bdw",elm_id) = (T)1.0;
            });  


            auto nmBones = get_input2<int>("nmCpns");
            using vec2i = zs::vec<int,2>;
            using vec3i = zs::vec<int,3>;
            bcht<vec2i, int, true, universal_hash<vec2i>, 32> ebtab{eles.get_allocator(), eles.size() * nmBones};
            cudaExec(zs::range(bcw.size()),
                [bcw = proxy<space>({},bcw),ebtab = proxy<space>(ebtab),everts = proxy<space>({},everts)] 
                    ZS_LAMBDA(int vi) mutable{
                        auto ei = reinterpret_bits<int>(bcw("inds",vi));
                        if(ei < 0)
                            return;
                        else{
                            int tag = (int)everts("tag",vi);
                            ebtab.insert(vec2i{ei,tag});
                        }
            });

            cudaExec(zs::range(eles.size()),
                [eles = proxy<space>({},eles),ebtab = proxy<space>(ebtab),nmBones] ZS_LAMBDA(int ei) mutable {
                    for(int i = 0;i != nmBones;++i) {
                        auto res = ebtab.query(vec2i{ei,i});
                        if(res < 0)
                            continue;
                        eles("nmBones",ei) += (T)1.0;
                    }
                    // if(eles("nmBones",ei) > 0)
                        // printf("nmEmbedCmps[%d] : [%d]\n",ei,(int)eles("nmBones",ei));
            });
        }else {
            eles.append_channels(cudaExec,{{"nmBones",1},{"bdw",1}});
            cudaExec(zs::range(eles.size()),[
                eles = proxy<space>({},eles)] ZS_LAMBDA(int ei) mutable {
                    eles("bdw",ei) = (T)1.0;
                    eles("nmBones",ei) = (T)1.0;
            });
        }

        cudaExec(zs::range(bcw.size()),
            [everts = proxy<space>({},everts),
                    bcw = proxy<space>({},bcw),
                    execTag = wrapv<space>{},
                    nmEmbedVerts = proxy<space>(nmEmbedVerts),
                    eles = proxy<space>({},eles),
                    verts = proxy<space>({},verts)]
                ZS_LAMBDA (int vi) mutable {
                    using T = typename RM_CVREF_T(bcw)::value_type;
                    auto ei = reinterpret_bits<int>(bcw("inds",vi));
                    if(ei < 0)
                        return;
                    auto tet = eles.pack(dim_c<3>,"inds",ei).reinterpret_bits(int_c);
                    atomic_add(execTag,&nmEmbedVerts[ei],(T)1.0);                  
        });

        cudaExec(zs::range(bcw.size()),
            [bcw = proxy<space>({},bcw),nmEmbedVerts = proxy<space>(nmEmbedVerts),eles = proxy<space>({},eles),everts = proxy<space>({},everts)] 
                ZS_LAMBDA(int vi) mutable{
                    auto ei = reinterpret_bits<int>(bcw("inds",vi));
                    if(everts.hasProperty("strength"))
                        bcw("strength",vi) = everts("strength",vi);
                    else
                        bcw("strength",vi) = (T)1.0;
                    if(ei >= 0){
                        auto alpha = (T)1.0/(T)nmEmbedVerts[ei];
                        bcw("cnorm",vi) = (T)alpha;
                        if(eles("nmBones",ei) > (T)1.5)
                            eles("bdw",ei) = (T)0.0;
                    }

                    // if(ei < 0 || eles("nmBones",ei) > (T)1.5){
                    //     // bcw("strength",vi) = (T)0.0;
                    //     bcw("cnorm",vi) = (T)0.0;
                    //     if(ei >= 0)
                    //         eles("bdw",ei) = (T)0.0;
                    // }
                    // else{

                    //     // bcw("cnorm",vi) = (T)1.0;
                    // }
        });

        
        // we might also do some smoothing on cnorm

        set_output("zsvolume", zsvolume);
    }
};

ZENDEFNODE(ZSComputeBaryCentricWeights, {{{"interpolator","zsvolume"}, {"embed surf", "zssurf"},{"int","mark_elm","0"},{"int","nmCpns","1"},{"string","tag","skin"}},
                            {{"interpolator on gpu", "zsvolume"}},
                            {{"float","bvh_thickness","0"},{"int","fitting_in","1"},{"string","bvh_channel","x"}},
                            {"ZSGeometry"}});

struct VisualizeInterpolator : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zsvolume = get_input<ZenoParticles>("zsvolume");
        auto tag = get_input2<std::string>("interpolator_name");
        const auto& bcw = (*zsvolume)[tag].clone({zs::memsrc_e::host});
        auto topo_tag = tag + std::string("_topo");
        const auto &bcw_topo = (*zsvolume)[topo_tag].clone({zs::memsrc_e::host});

        auto bcw_vis = std::make_shared<zeno::PrimitiveObject>();
        bcw_vis->resize(bcw.size());
        auto& bcw_X = bcw_vis->verts;
        auto& bcw_cnorm = bcw_vis->add_attr<float>("cnorm");
        auto& bcw_strength = bcw_vis->add_attr<float>("strength");

        auto ompPol = omp_exec();  
        constexpr auto omp_space = execspace_e::openmp;        
        ompPol(zs::range(bcw.size()),
            [&bcw_X,&bcw_cnorm,&bcw_strength,bcw = proxy<omp_space>({},bcw)] (int vi) mutable {
                bcw_X[vi] = bcw.pack(dim_c<3>,"X",vi).to_array();
                bcw_cnorm[vi] = bcw("cnorm",vi);
                bcw_strength[vi] = bcw("strength",vi);
        });

        set_output("bcw_vis",std::move(bcw_vis));
    }
};

ZENDEFNODE(VisualizeInterpolator, {{{"interpolator","zsvolume"},{"string","interpolator_name","skin"}},
                            {{"visual bcw", "bcw_vis"}},
                            {},
                            {"ZSGeometry"}});

struct ZSSampleEmbedVectorField : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zstets = get_input<ZenoParticles>("volume");
        auto sampler = get_input<ZenoParticles>("vec_field");

        auto tag = get_param<std::string>("bcw_channel");
        auto sample_attr = get_param<std::string>("sampleAttr");
        auto out_attr = get_param<std::string>("outAttr");
        auto tag_type = get_param<std::string>("type");

        auto cudaExec = zs::cuda_exec();

        auto& verts = zstets->getParticles();
        if(!verts.hasProperty(out_attr))
            verts.append_channels(cudaExec,{{out_attr,3}});

        const auto& sample_verts = sampler->getParticles();
        const auto& sample_eles = sampler->getQuadraturePoints();
  
        if(!sampler->hasAuxData(tag)){
            fmt::print("no specified bcw channel detected, create a new one...\n");
            auto& sample_bcw = (*sampler)[tag];
            sample_bcw = typename ZenoParticles::particles_t({{"inds",1},{"w",4}},verts.size(),zs::memsrc_e::device,0);
        }
        const auto& sample_bcw = (*sampler)[tag];

        if(sample_bcw.size() != verts.size())
            throw std::runtime_error("SMAPLE_BCW SIZE UNEQUAL VOLUME SIZE");
        
        constexpr auto space = zs::execspace_e::cuda;

        auto default_val = vec3::from_array(get_input<zeno::NumericObject>("default")->get<zeno::vec3f>());
        bool on_elm = tag_type == "element";

        cudaExec(zs::range(sample_bcw.size()),
            [sample_bcw = proxy<space>({},sample_bcw),verts = proxy<space>({},verts),sample_eles = proxy<space>({},sample_eles),sample_verts = proxy<space>({},sample_verts),
                sample_attr = zs::SmallString(sample_attr),out_attr = zs::SmallString(out_attr),default_val,on_elm] ZS_LAMBDA(int vi) mutable {
                    auto ei = reinterpret_bits<int>(sample_bcw("inds",vi));
                    if(ei < 0){
                        verts.template tuple<3>(out_attr,vi) = default_val;
                        return;
                    }
                    if(on_elm){
                        verts.template tuple<3>(out_attr,vi) = sample_eles.template pack<3>(sample_attr,ei);
                        return;
                    }

                    const auto& w = sample_bcw.pack<4>("w",vi);
                    verts.template tuple<3>(out_attr,vi) = vec3::zeros();
                    for(int i = 0;i < 4;++i){
                        auto idx = sample_eles.template pack<4>("inds",ei).template reinterpret_bits<int>()[i];
                        verts.template tuple<3>(out_attr,vi) = verts.template pack<3>(out_attr,vi) + w[i] * sample_verts.template pack<3>(sample_attr,idx);
                    }
        });


        set_output("volume",zstets);
    }

};

ZENDEFNODE(ZSSampleEmbedVectorField, {{{"volume"}, {"embed vec field", "vec_field"},{"default value","default"}},
                            {{"out volume", "volume"}},
                            {{"string","bcw_channel","bcw"},{"string","sampleAttr","vec_field"},{"string","outAttr"," vec_field"},{"enum element vert","type","element"}},
                            {"ZSGeometry"}});

struct ZSSampleEmbedTagField : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zstets = get_input<ZenoParticles>("volume");
        auto sampler = get_input<ZenoParticles>("tag_field");

        auto tag = get_param<std::string>("bcw_channel");
        auto sample_attr = get_param<std::string>("tagAttr");
        auto out_attr = get_param<std::string>("outAttr");
        auto tag_type = get_param<std::string>("type");

        auto default_tag_value = get_param<int>("default");

        auto cudaExec = zs::cuda_exec();

        auto& verts = zstets->getParticles();
        if(!verts.hasProperty(out_attr))
            verts.append_channels(cudaExec,{{out_attr,1}});

        const auto& sample_verts = sampler->getParticles();
        const auto& sample_eles = sampler->getQuadraturePoints();

        if(!sampler->hasAuxData(tag)){
            fmt::print("no specified bcw channel detected, create a new one...\n");
            auto& sample_bcw = (*sampler)[tag];
            sample_bcw = typename ZenoParticles::particles_t({{"inds",1},{"w",4}},verts.size(),zs::memsrc_e::device,0);
        }
        const auto& sample_bcw = (*sampler)[tag];

        if(sample_bcw.size() != verts.size())
            throw std::runtime_error("SMAPLE_BCW SIZE UNEQUAL VOLUME SIZE");
        
        constexpr auto space = zs::execspace_e::cuda;

        bool on_elm = tag_type == "element";

        cudaExec(zs::range(sample_bcw.size()),
            [sample_bcw = proxy<space>({},sample_bcw),verts = proxy<space>({},verts),sample_eles = proxy<space>({},sample_eles),sample_verts = proxy<space>({},sample_verts),
                sample_attr = zs::SmallString(sample_attr),out_attr = zs::SmallString(out_attr),default_tag_value,on_elm] ZS_LAMBDA(int vi) mutable {
                     auto ei = reinterpret_bits<int>(sample_bcw("inds",vi));
                     if(ei < 0){
                         verts(out_attr,vi) = reinterpret_bits<float>(default_tag_value);
                         return;
                     }

                    if(on_elm)
                        verts(out_attr,vi) = sample_eles(sample_attr,ei);
                    else{
                        auto idx = sample_eles.pack<4>("inds",ei).reinterpret_bits<int>()[0];
                        verts(out_attr,vi) = sample_verts(sample_attr,idx);
                    }
        });


        set_output("volume",zstets);
    }

};

ZENDEFNODE(ZSSampleEmbedTagField, {{{"volume"}, {"embed tag field", "tag_field"},{"default value","default"}},
                            {{"out volume", "volume"}},
                            {{"string","interpolate_tag","bws"},{"string","sampleAttr","vec_field"},{"string","outAttr"," vec_field"},{"enum element vert","type","element"}},
                            {"ZSGeometry"}});


struct ZSInterpolateEmbedAttr : zeno::INode {
    template<int DIM,typename SRC_TILEVEC,typename DST_TILEVEC,typename TOPO_TIELVEC,typename BCW_TILEVEC>
    void interpolate_p2p_imp(const std::string& srcAttr,const std::string& dstAttr,
            const SRC_TILEVEC& src_tilevec,DST_TILEVEC& dst_tilevec,const TOPO_TIELVEC& src_topo,const BCW_TILEVEC& bcw) {
        auto cudaExec = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        if(!dst_tilevec.hasProperty(dstAttr))
            dst_tilevec.append_channels(cudaExec, {{dstAttr, DIM}});

        cudaExec(zs::range(dst_tilevec.size()),
            [srcAttr = zs::SmallString{srcAttr},dstAttr = zs::SmallString{dstAttr},
                    src_tilevec = zs::proxy<space>({},src_tilevec), bcw = zs::proxy<space>({},bcw),
                    dst_tilevec = zs::proxy<space>({},dst_tilevec),
                    src_topo = zs::proxy<space>({},src_topo)] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(dst_tilevec)::value_type;
                const auto& ei = bcw.template pack<1>("inds",vi).template reinterpret_bits<int>()[0];
                if(ei < 0)
                    return;
                const auto& inds = src_topo.template pack<4>("inds",ei).template reinterpret_bits<int>();

                const auto& w = bcw.template pack<4>("w",vi);
                dst_tilevec.template tuple<DIM>(dstAttr,vi) = zs::vec<T,DIM>::zeros();
                for(size_t i = 0;i < 4;++i){
                    auto idx = inds[i];
                    dst_tilevec.template tuple<DIM>(dstAttr,vi) = dst_tilevec.template pack<DIM>(dstAttr,vi) + w[i] * src_tilevec.template pack<DIM>(srcAttr, idx);
                }

        });
    }

    template<int DIM,typename QUAD_TILEVEC,typename POINT_TILEVEC,typename BCW_TILEVEC>
    void interpolate_q2p_imp(const std::string& quadAttr,const std::string& pointAttr,
            const QUAD_TILEVEC& quad_tilevec,POINT_TILEVEC& point_tilevec,const BCW_TILEVEC& bcw) {
        auto cudaExec = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        if(!point_tilevec.hasProperty(pointAttr))
            point_tilevec.append_channels(cudaExec, {{pointAttr, DIM}});   
             
        cudaExec(zs::range(point_tilevec.size()),
            [pointAttr = zs::SmallString{pointAttr},quadAttr = zs::SmallString{quadAttr},
                point_tilevec = zs::proxy<space>({},point_tilevec),bcw = zs::proxy<space>({},bcw),
                quad_tilevec = zs::proxy<space>({},quad_tilevec)] ZS_LAMBDA (int vi) mutable {
            using T = typename RM_CVREF_T(point_tilevec)::value_type;
            const auto& ei = bcw.template pack<1>("inds",vi).template reinterpret_bits<int>()[0];
            if(ei < 0)
                return;
            point_tilevec.template tuple<DIM>(pointAttr,vi) = quad_tilevec.template pack<DIM>(quadAttr,ei);
        });
    }


    void apply() override {
        using namespace zs;
        auto source = get_input<ZenoParticles>("source");
        auto dest = get_input<ZenoParticles>("dest");

        auto srcAttr = get_param<std::string>("srcAttr");
        auto dstAttr = get_param<std::string>("dstAttr");
        auto bcw_tag = get_input2<std::string>("bcw_tag");
        auto strategy = get_param<std::string>("strategy");
        const auto& bcw = (*source)[bcw_tag];
        auto& dest_pars = dest->getParticles();

        if(bcw.size() != dest_pars.size()) {
            fmt::print("the dest and bcw's size not match\n");
            throw std::runtime_error("the dest and bcw's size not match");
        }

        
        if(strategy == "p2p") {
            const auto& source_pars = source->getParticles();
            const auto& topo = source->getQuadraturePoints();
            if(!source_pars.hasProperty(srcAttr)) {
                fmt::print("the source have no {} channel\n",srcAttr);
                throw std::runtime_error("the source have no specified channel");
            }           
            if(topo.getChannelSize("inds") != 4) {
                fmt::print("only support tetrahedra mesh as source\n");
                throw std::runtime_error("only support tetrahedra mesh as source");
            }
            if(dest_pars.hasProperty(dstAttr) && dest_pars.getChannelSize(dstAttr) != source_pars.getChannelSize(srcAttr)){
                fmt::print("the dest attr_{} and source attr_{} not match in size\n",dstAttr,srcAttr);
                throw std::runtime_error("the dest attr and source attr not match in size");
            }

            if(source_pars.getChannelSize(srcAttr) == 1)
                interpolate_p2p_imp<1>(srcAttr,dstAttr,source_pars,dest_pars,topo,bcw);
            if(source_pars.getChannelSize(srcAttr) == 2)
                interpolate_p2p_imp<2>(srcAttr,dstAttr,source_pars,dest_pars,topo,bcw);
            if(source_pars.getChannelSize(srcAttr) == 3)
                interpolate_p2p_imp<3>(srcAttr,dstAttr,source_pars,dest_pars,topo,bcw);
        }else if(strategy == "q2p") {
            const auto& source_quads = source->getQuadraturePoints();
            if(!source_quads.hasProperty(srcAttr)) {
                fmt::print("the source have no {} channel\n",srcAttr);
                throw std::runtime_error("the source have no specified channel");
            }    
            if(dest_pars.hasProperty(dstAttr) && dest_pars.getChannelSize(dstAttr) != source_quads.getChannelSize(srcAttr)){
                fmt::print("the dest attr_{} and source attr_{} not match in size\n",dstAttr,srcAttr);
                throw std::runtime_error("the dest attr and source attr not match in size");
            }

            if(source_quads.getChannelSize(srcAttr) == 1)
                interpolate_q2p_imp<1>(srcAttr,dstAttr,source_quads,dest_pars,bcw);
            if(source_quads.getChannelSize(srcAttr) == 2)
                interpolate_q2p_imp<2>(srcAttr,dstAttr,source_quads,dest_pars,bcw);
            if(source_quads.getChannelSize(srcAttr) == 3)
                interpolate_q2p_imp<3>(srcAttr,dstAttr,source_quads,dest_pars,bcw);
        }
        set_output("dest",dest);
    }
};


ZENDEFNODE(ZSInterpolateEmbedAttr, {{{"source"}, {"dest"},{"string","bcw_tag","skin_bw"}},
                            {{"dest"}},
                            {
                                {"string","srcAttr","x"},
                                {"string","dstAttr","x"},
                                {"enum p2p q2p","strategy","p2p"}

                            },
                            {"ZSGeometry"}});

// deprecated
struct ZSInterpolateEmbedPrim : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zstets = get_input<ZenoParticles>("zsvolume");
        auto zssurf = get_input<ZenoParticles>("zssurf");

        auto tag = get_param<std::string>("tag");
        auto inAttr = get_param<std::string>("inAttr");
        auto outAttr = get_param<std::string>("outAttr");
        // auto refAttr = get_param<std::string>("refAttr");

        // auto useDispMap = get_param<int>("useDispMap");
        // auto refDispMapTag = get_param<std::string>("refDispMapTag");
        // auto outDispMapTag = get_param<std::string>("outDispMapTag");

        // auto use_xform = get_param<int>("use_xform");

        auto &everts = zssurf->getParticles();
    
        const auto& verts = zstets->getParticles();
        const auto& eles = zstets->getQuadraturePoints();
        const auto& bcw = (*zstets)[tag];

        // if(useDispMap && (!everts.hasProperty(refDispMapTag) || !everts.hasProperty(outDispMapTag))) {
        //     fmt::print("the input everts have no {} or {} dispMap when useDispMap is on\n",refDispMapTag,outDispMapTag);
        //     throw std::runtime_error("the input everts have no specified dispMap when useDispMap is on");
        // }


        // if(use_xform && !everts.hasProperty(refAttr)) {
        //     fmt::print("the input everts have no {} channel when use_xform is on\n",refAttr);
        //     throw std::runtime_error("the input everts have no refAttr channel when use_xform is on");
        // }
        // if(use_xform && !verts.hasProperty(refAttr)) {
        //     fmt::print("the input verts have no {} channel when use_xform is on\n",refAttr);
        //     throw std::runtime_error("the input verts have no refAttr channel when use_xform is on");
        // }

        const auto nmEmbedVerts = bcw.size();
        if(everts.size() != nmEmbedVerts)
            throw std::runtime_error("INPUT SURF SIZE AND BCWS SIZE DOES NOT MATCH");


        auto cudaExec = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        cudaExec(zs::range(nmEmbedVerts),
            [inAttr = zs::SmallString{inAttr},outAttr = zs::SmallString{outAttr},
                    verts = proxy<space>({},verts),eles = proxy<space>({},eles),
                    bcw = proxy<space>({},bcw),everts = proxy<space>({},everts)
                    // use_xform,refAttr = zs::SmallString{refAttr},
                    // useDispMap,
                    // refDispMapTag = zs::SmallString{refDispMapTag},
                    // outDispMapTag = zs::SmallString{outDispMapTag}
                    ] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(verts)::value_type;
                const auto& ei = bcw.pack<1>("inds",vi).reinterpret_bits<int>()[0];
                if(ei < 0)
                    return;
                const auto& inds = eles.template pack<4>("inds",ei).template reinterpret_bits<int>();
                // if(use_xform || useDispMap) {
                //     zs::vec<T,3,3> F{};
                //     zs::vec<T,3> b{};

                //     LSL_GEO::deformation_xform(
                //         verts.template pack<3>(inAttr,inds[0]),
                //         verts.template pack<3>(inAttr,inds[1]),
                //         verts.template pack<3>(inAttr,inds[2]),
                //         verts.template pack<3>(inAttr,inds[3]),
                //         verts.template pack<3>(refAttr,inds[0]),
                //         eles.template pack<3,3>("IB",ei),F,b);
                    
                //     everts.template tuple<3>(outAttr,vi) = F * everts.template pack<3>(refAttr,vi) + b;

                //     // if(vi == 0){
                //     //     printf("F : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
                //     //         (float)F(0,0),(float)F(0,1),(float)F(0,2),
                //     //         (float)F(1,0),(float)F(1,1),(float)F(1,2),
                //     //         (float)F(2,0),(float)F(2,1),(float)F(2,2));
                //     //     printf("b : %f %f %f\n",(float)b[0],(float)b[1],(float)b[2]);
                //     // }

                //     if(useDispMap) {
                //         everts.template tuple<3>(outDispMapTag,vi) = F * everts.template pack<3>(refDispMapTag,vi);
                //     }
                // }else{
                    const auto& w = bcw.pack<4>("w",vi);
                    everts.tuple<3>(outAttr,vi) = vec3::zeros();
                    for(size_t i = 0;i < 4;++i){
                        // const auto& idx = eles.pack<4>("inds",ei).reinterpret_bits<int>()[i];
                        // const auto idx = reinterpret_bits<int>(eles("inds", i, ei));
                        auto idx = inds[i];
                        everts.tuple<3>(outAttr,vi) = everts.pack<3>(outAttr,vi) + w[i] * verts.pack<3>(inAttr, idx);
                    }
// #if 0
//                     if(vi == 100){
//                         auto vert = everts.pack<3>(outAttr,vi);
//                         printf("V<%d>->E<%d>(%f,%f,%f,%f) :\t%f\t%f\t%f\n",vi,ei,w[0],w[1],w[2],w[3],vert[0],vert[1],vert[2]);
//                     }
// #endif

                // }
        });
        set_output("zssurf",zssurf);
    }
};

ZENDEFNODE(ZSInterpolateEmbedPrim, {{{"zsvolume"}, {"embed primitive", "zssurf"}},
                            {{"embed primitive", "zssurf"}},
                            {
                                {"string","inAttr","x"},
                                {"string","outAttr","x"},
                                // {"string","refAttr","X"},
                                {"string","tag","skin_bw"}
                                // {"int","use_xform","0"},
                                // {"int","useDispMap","0"},
                                // {"string","refDispMapTag","dX"},
                                // {"string","outDispMapTag","dx"}
                                },
                            {"ZSGeometry"}});


struct ZSDeformEmbedPrim : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zsvolume = get_input<ZenoParticles>("zsvolume");
        auto zssurf = get_input<ZenoParticles>("zssurf");

        auto tag = get_param<std::string>("tag");
        auto inAttr = get_param<std::string>("inAttr");
        auto outAttr = get_param<std::string>("outAttr");

        auto deformField = get_param<std::string>("deformField");

        auto &everts = zssurf->getParticles();

        auto cudaExec = zs::cuda_exec();

        if(!everts.hasProperty(inAttr)) {
            fmt::print("the embed prim has no {} attribute as input\n",inAttr);
            throw std::runtime_error("the embed prim has no attribute as input");
        }
        if(!everts.hasProperty(outAttr))
            everts.append_channels(cudaExec, {{outAttr, 3}});

        
        const auto& verts = zsvolume->getParticles();
        const auto& eles = zsvolume->getQuadraturePoints();
        const auto& bcw = (*zsvolume)[tag];

        if(!eles.hasProperty(deformField)) {
            fmt::print("the embed prim has no {} deformField\n",deformField);
            throw std::runtime_error("the embed prim has no deformField");
        }

        const auto nmEmbedVerts = bcw.size();

        if(everts.size() != nmEmbedVerts)
            throw std::runtime_error("INPUT SURF SIZE AND BCWS SIZE DOES NOT MATCH");


        constexpr auto space = zs::execspace_e::cuda;

        cudaExec(zs::range(nmEmbedVerts),
            [inAttr = zs::SmallString{inAttr},outAttr = zs::SmallString{outAttr},
                    everts = proxy<space>({},everts),eles = proxy<space>({},eles),
                    bcw = proxy<space>({},bcw),
                    deformField = zs::SmallString{deformField}] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(verts)::value_type;
                const auto& ei = bcw.pack<1>("inds",vi).reinterpret_bits<int>()[0];
                if(ei < 0)
                    return;
                everts.template tuple<3>(outAttr,vi) = eles.template pack<3,3>(deformField,ei) * everts.template pack<3>(inAttr,vi);
                // if(vi == 114754){
                //     auto dx = everts.template pack<3>(outAttr,vi);
                //     auto dX = everts.template pack<3>(inAttr,vi);
                //     auto F = eles.template pack<3,3>(deformField,ei);
                //     printf("F : %f %f %f\n%f %f %f\n%f %f %f\n",
                //         (float)F(0,0),(float)F(0,1),(float)F(0,2),
                //         (float)F(1,0),(float)F(1,1),(float)F(1,2),
                //         (float)F(2,0),(float)F(2,1),(float)F(2,2)
                //     );
                //     printf("Fdet : %f\n",(float)zs::determinant(F));
                //     printf("dX : %f %f %f with length %f\n",(float)dX[0],(float)dX[1],(float)dX[2],(float)dX.norm());
                //     printf("dx : %f %f %f with length %f\n",(float)dx[0],(float)dx[1],(float)dx[2],(float)dx.norm());
                // }

        });
        set_output("zssurf",zssurf);
        set_output("zsvolume",zsvolume);
    }
};

ZENDEFNODE(ZSDeformEmbedPrim, {{{"zsvolume"}, {"embed primitive", "zssurf"}},
                            {{"embed primitive", "zssurf"},{"zsvolume"}},
                            {
                                {"string","inAttr","V"},
                                {"string","outAttr","v"},
                                {"string","tag","skin_bw"},
                                {"string","deformField","F"}
                                },
                            {"ZSGeometry"}});

} // namespace zeno