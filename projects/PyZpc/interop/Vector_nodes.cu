#include "hip/hip_runtime.h"
#include "Vector.hpp"
#include "zensim/ZpcFunctional.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include <fmt/core.h>
#include <tuple>
#include <variant>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/zeno.h>

namespace zeno {
struct MakeZsVector : INode {
  void apply() override {
    // TODO
    auto input_size = get_input2<int>("size");
    auto input_memsrc = get_input2<std::string>("memsrc");
    auto intput_devid = get_input2<int>("dev_id");
    // auto input_virtual = get_input2<bool>("virtual");
    auto intput_elem_type = get_input2<std::string>("elem_type");

    zs::memsrc_e memsrc;
    if (input_memsrc == "host")
      memsrc = zs::memsrc_e::host;
    else if (input_memsrc == "device")
      memsrc = zs::memsrc_e::device;
    else
      memsrc = zs::memsrc_e::um;

#define MAKE_VECTOR_OBJ_T(T)                                                   \
  if (intput_elem_type == #T) {                                                \
    auto allocator =                                                           \
        zs::get_memory_source(memsrc, static_cast<zs::ProcID>(intput_devid));  \
    vectorObj->set(zs::Vector<T, zs::ZSPmrAllocator<false>>{allocator, 0});    \
  }

    auto vectorObj = std::make_shared<ZsVectorObject>();
    MAKE_VECTOR_OBJ_T(int)
    MAKE_VECTOR_OBJ_T(float)
    MAKE_VECTOR_OBJ_T(double)
    std::visit([input_size](auto &vec) { vec.resize(input_size); },
               vectorObj->value);

    set_output("ZsVector", std::move(vectorObj));
  }
};

//  memsrc, size, elem_type, dev_id, virtual
ZENDEFNODE(MakeZsVector, {
                             {{"int", "size", "0"},
                              {"enum host device um", "memsrc", "device"},
                              {"int", "dev_id", "0"},
                              //   {"bool", "virtual", "false"},
                              {"enum float int double", "elem_type", "float"}},
                             {"ZsVector"},
                             {},
                             {"PyZFX"},
                         });

struct ReduceZsVector : INode {
  void apply() override {
    auto vectorObj = get_input<ZsVectorObject>("ZsVector");
    auto opStr = get_input2<std::string>("op");
    auto &vector = vectorObj->value;

    float result;
    std::visit(
        [&result, &opStr](auto &vector) {
          auto pol = zs::cuda_exec();
          using vector_t = RM_CVREF_T(vector);
          using val_t = typename vector_t::value_type;
          zs::Vector<val_t> res{1, zs::memsrc_e::device};
          if (opStr == "add")
            zs::reduce(pol, std::begin(vector), std::end(vector),
                       std::begin(res), static_cast<val_t>(0),
                       zs::plus<val_t>{});
          else if (opStr == "max")
            zs::reduce(pol, std::begin(vector), std::end(vector),
                       std::begin(res), zs::detail::deduce_numeric_min<val_t>(),
                       zs::getmax<val_t>{});
          else
            zs::reduce(pol, std::begin(vector), std::end(vector),
                       std::begin(res), zs::detail::deduce_numeric_max<val_t>(),
                       zs::getmin<val_t>{});
          result = static_cast<float>(res.getVal());
        },
        vector);
    set_output2("result", result);
  }
};

ZENDEFNODE(ReduceZsVector, {
                               {"ZsVector", {"enum add max min", "op", "add"}},
                               {"result"},
                               {},
                               {"PyZFX"},
                           });

template <typename T>
struct _is_float : std::bool_constant<zs::is_floating_point_v<T>> {};
template <size_t N, typename T>
struct _is_float<zeno::vec<N, T>>
    : std::bool_constant<zs::is_floating_point_v<T>> {};

struct CopyZsVectorTo : INode {
  void apply() override {
    auto vectorObj = get_input<ZsVectorObject>("ZsVector");
    auto prim = get_input<PrimitiveObject>("prim");
    auto attr = get_input2<std::string>("attr");
    auto &vector = vectorObj->value;

    std::visit(
        [&prim, &attr](auto &vector) {
          using vector_t = RM_CVREF_T(vector);
          using val_t = typename vector_t::value_type;
          if constexpr (zs::is_same_v<val_t, float> ||
                        zs::is_same_v<val_t, int>) {

            auto process = [&prim = prim, &vector = vector,
                            &attr](auto &primAttr) {
              using T = RM_CVREF_T(primAttr[0]);
              constexpr bool sameType =
                  _is_float<T>::value == zs::is_same_v<val_t, float>;

              constexpr auto nbytes = sizeof(T);
              if (prim->size() * (nbytes / sizeof(float)) < vector.size()) {
                fmt::print("BEWARE! copy sizes mismatch! resize to match.\n");
                if (vector.size() % (nbytes / sizeof(float)) != 0) {
                  throw std::runtime_error(fmt::format(
                      "vector of type {} copied to primattr [{}] "
                      "containing {} "
                      "elements of type {}, yet vector size is {}\n",
                      zs::get_type_str<val_t>().asChars(), attr, prim->size(),
                      zs::get_type_str<T>().asChars(), vector.size()));
                }
                /// @note this does not invalidate primAttr
                prim->resize(vector.size() / (nbytes / sizeof(float)));
              }
              if constexpr (sameType)
                zs::Resource::copy(
                    zs::MemoryEntity{zs::MemoryLocation{zs::memsrc_e::host, -1},
                                     (void *)primAttr.data()},
                    zs::MemoryEntity{vector.memoryLocation(),
                                     (void *)vector.data()},
                    sizeof(val_t) * vector.size());
              else {
                if constexpr (zs::is_same_v<val_t, float>) {
                  // float -> int
                  zs::omp_exec()(
                      zs::range(vector.size()),
                      [&vector, primAttrAddr = (int *)primAttr.data()](
                          size_t i) { primAttrAddr[i] = (int)vector[i]; });
                } else {
                  // int -> float
                  zs::omp_exec()(
                      zs::range(vector.size()),
                      [&vector, primAttrAddr = (float *)primAttr.data()](
                          size_t i) { primAttrAddr[i] = (float)vector[i]; });
                }
              }
            };
            if (attr == "pos")
              process(prim->verts.values);
            else
              std::visit(process, prim->attr(attr));

          } else if constexpr (zs::is_same_v<val_t, double>) {
            auto process = [&prim = prim, &vector = vector,
                            &attr](auto &primAttr) {
              using T = RM_CVREF_T(primAttr[0]);
              constexpr auto nbytes = sizeof(T);
              if (prim->size() * (nbytes / sizeof(float)) < vector.size()) {
                fmt::print("BEWARE! copy sizes mismatch! resize to match.\n");
                if (vector.size() % (nbytes / sizeof(float)) != 0) {
                  throw std::runtime_error(fmt::format(
                      "vector of type {} copied to primattr [{}] "
                      "containing {} "
                      "elements of type {}, yet vector size is {}\n",
                      zs::get_type_str<val_t>().asChars(), attr, prim->size(),
                      zs::get_type_str<T>().asChars(), vector.size()));
                }
                /// @note this does not invalidate primAttr
                prim->resize(vector.size() / (nbytes / sizeof(float)));
              }

              if constexpr (!_is_float<T>::value) {
                // double -> int
                zs::omp_exec()(
                    zs::range(vector.size()),
                    [&vector, primAttrAddr = (int *)primAttr.data()](size_t i) {
                      primAttrAddr[i] = (int)vector[i];
                    });
              } else {
                // double -> float
                zs::omp_exec()(
                    zs::range(vector.size()),
                    [&vector, primAttrAddr = (float *)primAttr.data()](
                        size_t i) { primAttrAddr[i] = (float)vector[i]; });
              }
            };

            if (attr == "pos")
              process(prim->verts.values);
            else
              std::visit(process, prim->attr(attr));
          }
        },
        vector);

    set_output2("prim", prim);
  }
};

ZENDEFNODE(CopyZsVectorTo, {
                               {"ZsVector", "prim", {"string", "attr", "clr"}},
                               {"prim"},
                               {},
                               {"PyZFX"},
                           });

struct CopyZsVectorFrom : INode {
  void apply() override {
    auto vectorObj = get_input<ZsVectorObject>("ZsVector");
    auto prim = get_input<PrimitiveObject>("prim");
    auto attr = get_input2<std::string>("attr");
    auto &vector = vectorObj->value;

    std::visit(
        [&prim, &attr](auto &vector) {
          using vector_t = RM_CVREF_T(vector);
          using val_t = typename vector_t::value_type;
          if constexpr (zs::is_same_v<val_t, float> ||
                        zs::is_same_v<val_t, int>) {

            auto process = [&prim = prim, &vector = vector,
                            &attr](auto &primAttr) {
              using T = RM_CVREF_T(primAttr[0]);
              constexpr bool sameType =
                  _is_float<T>::value == zs::is_same_v<val_t, float>;

              constexpr auto nbytes = sizeof(T);
              if (prim->size() * (nbytes / sizeof(float)) > vector.size()) {
                fmt::print("BEWARE! copy sizes mismatch! resize to match.\n");
                vector.resize(prim->size() * (nbytes / sizeof(float)));
              }
              if constexpr (sameType)
                zs::Resource::copy(
                    zs::MemoryEntity{vector.memoryLocation(),
                                     (void *)vector.data()},
                    zs::MemoryEntity{zs::MemoryLocation{zs::memsrc_e::host, -1},
                                     (void *)primAttr.data()},
                    nbytes * prim->size());
              else {
                if constexpr (zs::is_same_v<val_t, float>) {
                  // float <- int
                  zs::omp_exec()(
                      zs::range(prim->size() * (nbytes / sizeof(float))),
                      [&vector, primAttrAddr = (int *)primAttr.data()](
                          size_t i) { vector[i] = (float)primAttrAddr[i]; });
                } else {
                  // int <- float
                  zs::omp_exec()(
                      zs::range(prim->size() * (nbytes / sizeof(float))),
                      [&vector, primAttrAddr = (float *)primAttr.data()](
                          size_t i) { vector[i] = (int)primAttrAddr[i]; });
                }
              }
            };

            if (attr == "pos")
              process(prim->verts.values);
            else
              std::visit(process, prim->attr(attr));

          } else if constexpr (zs::is_same_v<val_t, double>) {
            auto process = [&prim = prim, &vector = vector,
                            &attr](auto &primAttr) {
              using T = RM_CVREF_T(primAttr[0]);
              constexpr auto nbytes = sizeof(T);
              if (prim->size() * (nbytes / sizeof(float)) > vector.size()) {
                fmt::print("BEWARE! copy sizes mismatch! resize to match.\n");
                vector.resize(prim->size() * (nbytes / sizeof(float)));
              }

              if constexpr (!_is_float<T>::value) {
                // double <- int
                zs::omp_exec()(
                    zs::range(prim->size() * (nbytes / sizeof(float))),
                    [&vector, primAttrAddr = (int *)primAttr.data()](size_t i) {
                      vector[i] = (double)primAttrAddr[i];
                    });
              } else {
                // double <- float
                zs::omp_exec()(
                    zs::range(prim->size() * (nbytes / sizeof(float))),
                    [&vector, primAttrAddr = (float *)primAttr.data()](
                        size_t i) { vector[i] = (double)primAttrAddr[i]; });
              }
            };

            if (attr == "pos")
              process(prim->verts.values);
            else
              std::visit(process, prim->attr(attr));
          }
        },
        vector);

    set_output2("ZsVector", vectorObj);
  }
};

ZENDEFNODE(CopyZsVectorFrom,
           {
               {"ZsVector", "prim", {"string", "attr", "clr"}},
               {"ZsVector"},
               {},
               {"PyZFX"},
           });
} // namespace zeno