#include "hip/hip_runtime.h"
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/Ray.h>
#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include "volume.h"
#include "TraceStuff.h"
#include "zxxglslvec.h"
#include "hip/hip_math_constants.h"

// #include <hip/hip_fp16.h>
// #include "nvfunctional"

enum struct VolumeEmissionScaleType {
    Raw, Density, Absorption
};

//PLACEHOLDER
using DataTypeNVDB0 = nanovdb::Fp32;
using GridTypeNVDB0 = nanovdb::NanoGrid<DataTypeNVDB0>;
#define VolumeEmissionScale VolumeEmissionScaleType::Raw
//PLACEHOLDER

#define _USING_NANOVDB_ true

//COMMON_CODE

/* w0, w1, w2, and w3 are the four cubic B-spline basis functions. */
inline __device__ float cubic_w0(float a)
{
  return (1.0f / 6.0f) * (a * (a * (-a + 3.0f) - 3.0f) + 1.0f);
}
inline __device__ float cubic_w1(float a)
{
  return (1.0f / 6.0f) * (a * a * (3.0f * a - 6.0f) + 4.0f);
}
inline __device__ float cubic_w2(float a)
{
  return (1.0f / 6.0f) * (a * (a * (-3.0f * a + 3.0f) + 3.0f) + 1.0f);
}
inline __device__ float cubic_w3(float a)
{
  return (1.0f / 6.0f) * (a * a * a);
}

/* g0 and g1 are the two amplitude functions. */
inline __device__ float cubic_g0(float a)
{
  return cubic_w0(a) + cubic_w1(a);
}
inline __device__ float cubic_g1(float a)
{
  return cubic_w2(a) + cubic_w3(a);
}

/* h0 and h1 are the two offset functions */
inline __device__ float cubic_h0(float a)
{
  return (cubic_w1(a) / cubic_g0(a)) - 1.0f;
}
inline __device__ float cubic_h1(float a)
{
  return (cubic_w3(a) / cubic_g1(a)) + 1.0f;
}

template<typename S>
inline __device__ float interp_tricubic_nanovdb(S &s, float x, float y, float z)
{
  float px = floorf(x);
  float py = floorf(y);
  float pz = floorf(z);
  float fx = x - px;
  float fy = y - py;
  float fz = z - pz;

  float g0x = cubic_g0(fx);
  float g1x = cubic_g1(fx);
  float g0y = cubic_g0(fy);
  float g1y = cubic_g1(fy);
  float g0z = cubic_g0(fz);
  float g1z = cubic_g1(fz);

  float x0 = px + cubic_h0(fx);
  float x1 = px + cubic_h1(fx);
  float y0 = py + cubic_h0(fy);
  float y1 = py + cubic_h1(fy);
  float z0 = pz + cubic_h0(fz);
  float z1 = pz + cubic_h1(fz);

  using namespace nanovdb;

  return g0z * (g0y * (g0x * s(Vec3f(x0, y0, z0)) + g1x * s(Vec3f(x1, y0, z0))) +
                g1y * (g0x * s(Vec3f(x0, y1, z0)) + g1x * s(Vec3f(x1, y1, z0)))) +
         g1z * (g0y * (g0x * s(Vec3f(x0, y0, z1)) + g1x * s(Vec3f(x1, y0, z1))) +
                g1y * (g0x * s(Vec3f(x0, y1, z1)) + g1x * s(Vec3f(x1, y1, z1))));
}

inline __device__ float _LERP_(float t, float s1, float s2)
{
    //return (1 - t) * s1 + t * s2;
    return fma(t, s2, fma(-t, s1, s1));
}

struct VolumeIn2 {
    float3 pos_world;
    float3 pos_view;

    bool isShadowRay;

	float sigma_t;
	uint32_t* seed;

    
    void* sbt_ptr;
    float* world2object;

	inline float rndf() const {
		return rnd(*seed);
	}

    vec3 _local_pos_ = vec3(HIP_NAN_F);
    vec3 _uniform_pos_ = vec3(HIP_NAN_F);

    __device__ vec3 localPosLazy() {
		if (isfinite(_local_pos_.x)) return _local_pos_;

        if (world2object != nullptr) {
            mat4* _w2o = reinterpret_cast<mat4*>(world2object);
            vec4 tmp = (*_w2o) * vec4(pos_view.x, pos_view.y, pos_view.z, 1.0f);
            
            _local_pos_ = *(vec3*)&tmp;
        }
        return _local_pos_;
    };

    __device__ vec3 uniformPosLazy() {
		if (isfinite(_uniform_pos_.x)) return _uniform_pos_;

        using GridTypeNVDB = GridTypeNVDB0;
        const HitGroupData* sbt_data = reinterpret_cast<HitGroupData*>( sbt_ptr );

        assert(sbt_data != nullptr);

        const auto grid_ptr = sbt_data->vdb_grids[0];
        const auto* _grid = reinterpret_cast<const GridTypeNVDB*>(grid_ptr);

        if (_grid == nullptr) {
            auto local_pos = localPosLazy();
            _uniform_pos_ = local_pos + 0.5f;
            return _uniform_pos_;
        }

        auto bbox = _grid->indexBBox();

        nanovdb::Coord boundsMin( bbox.min() );
        nanovdb::Coord boundsMax( bbox.max() + nanovdb::Coord( 1 ) ); // extend by one unit

        vec3 min = { 
            static_cast<float>( boundsMin[0] ), 
            static_cast<float>( boundsMin[1] ), 
            static_cast<float>( boundsMin[2] )};
        vec3 max = {
            static_cast<float>( boundsMax[0] ),
            static_cast<float>( boundsMax[1] ),
            static_cast<float>( boundsMax[2] )};

        auto local_pos = localPosLazy();

        auto _uniform_pos_ = (local_pos - min) / (max - min);
        _uniform_pos_ = clamp(_uniform_pos_, vec3(0.0f), vec3(1.0f));

        // assert(_uniform_pos_.x >= 0);
        // assert(_uniform_pos_.y >= 0);
        // assert(_uniform_pos_.z >= 0);
        return _uniform_pos_;
    };
};

template <typename Acc, typename DataTypeNVDB, uint8_t Order>
inline __device__ float nanoSampling(Acc& acc, nanovdb::Vec3f& point_indexd, const VolumeIn2& volin) {
    
    using GridTypeNVDB = nanovdb::NanoGrid<DataTypeNVDB>;

    if constexpr(3 > Order) {
        using Sampler = nanovdb::SampleFromVoxels<typename GridTypeNVDB::AccessorType, Order, true>;
        return Sampler(acc)(point_indexd);
    }

    if constexpr(3 == Order) {
        nanovdb::SampleFromVoxels<typename GridTypeNVDB::AccessorType, 1, true> s(acc);
        return interp_tricubic_nanovdb(s, point_indexd[0], point_indexd[1], point_indexd[2]);
    } 
    
    if constexpr(4 == Order) {

        auto uuu = nanovdb::Vec3f(volin.rndf(), volin.rndf(), volin.rndf());
             uuu -= nanovdb::Vec3f(0.5f);
        auto pick = nanovdb::RoundDown<nanovdb::Vec3f>(point_indexd + uuu);
        auto coord = nanovdb::Coord(pick[0], pick[1], pick[2]);
        return acc.getValue(coord);
    }

    return 0.0f;
}

template <uint8_t Order, bool WorldSpace, typename DataTypeNVDB>
static __inline__ __device__ vec2 samplingVDB(const unsigned long long grid_ptr, vec3 att_pos, VolumeIn2& volin, bool cihou) {
    using GridTypeNVDB = nanovdb::NanoGrid<DataTypeNVDB>;

    const auto* _grid = reinterpret_cast<const GridTypeNVDB*>(grid_ptr);
    const auto& _acc = _grid->tree().getAccessor();

    if (_grid == nullptr) { return {}; }

    auto pos_indexed = reinterpret_cast<const nanovdb::Vec3f&>(att_pos);

    if constexpr(WorldSpace) 
    {
        if (cihou) {
            pos_indexed = volin.localPosLazy();
        } else {
            pos_indexed = _grid->worldToIndexF(pos_indexed);
        }
    } //_grid->tree().root().maximum();

    return vec2 { nanoSampling<decltype(_acc), DataTypeNVDB, Order>(_acc, pos_indexed, volin), _grid->tree().root().maximum() };
}

extern "C" __device__ VolumeOut __direct_callable__evalmat(const float4* uniforms, VolumeIn2& attrs) {

    auto& prd = attrs;

    vec3& att_pos = reinterpret_cast<vec3&>(attrs.pos_world);
    auto att_clr = vec3(0);
    auto att_uv = vec3(0);
    auto att_nrm = vec3(0);
    auto att_tang = vec3(0);
	
    HitGroupData* sbt_data = reinterpret_cast<HitGroupData*>(attrs.sbt_ptr);
    auto zenotex = sbt_data->textures;
    auto vdb_grids = sbt_data->vdb_grids;
    auto vdb_max_v = sbt_data->vdb_max_v;

    auto att_isShadowRay = attrs.isShadowRay ? 1.0f:0.0f;

#ifndef _FALLBACK_

    //GENERATED_BEGIN_MARK 
    auto anisotropy = 0.0f;
    auto density = 0.0f;

    vec3 emission = vec3(0.0f);
    vec3 albedo = vec3(0.5f);
    auto extinction = vec3(1.0f);
    //GENERATED_END_MARK
#else
	auto anisotropy = 0.0f;
    auto density = 0.1f;

	vec3 tmp = { 1, 0, 1 };

    vec3 emission = tmp / 50.f;
    vec3 albedo = tmp;
    auto extinction = vec3(1.0f);
#endif // _FALLBACK_

VolumeOut output;

#if _USING_NANOVDB_

    output.albedo = clamp(albedo, 0.0f, 1.0f);
    output.anisotropy = clamp(anisotropy, -1.0f, 1.0f);
    output.extinction = extinction;

    output.density = fmaxf(density, 0.0f);
    output.emission = fmaxf(emission, vec3(0.0f));

	if constexpr(VolumeEmissionScale == VolumeEmissionScaleType::Raw) {
		//output.emission = output.emission; 
	} else if constexpr(VolumeEmissionScale == VolumeEmissionScaleType::Density) {
		output.emission = output.density * output.emission;
	} else if constexpr(VolumeEmissionScale == VolumeEmissionScaleType::Absorption) {

		auto sigma_t = attrs.sigma_t;

		float sigma_a = sigma_t * output.density * average(1.0f - output.albedo);
		sigma_a = fmaxf(sigma_a, 0.0f);
		auto tmp = output.emission * sigma_a;
		output.step_scale = 1.0f / fmaxf(sigma_t, average(tmp)); 
		output.emission = tmp / sigma_t;
	}
    
#else
    //USING 3D ARRAY
    //USING 3D Noise 
#endif
	return output;
}