#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <cuda/helpers.h>

#include <sutil/vec_math.h>
#include "optixPathTracer.h"

#include "TraceStuff.h"
#include "zxxglslvec.h"

#include "IOMat.h"
#include "Light.h"

#include "DisneyBRDF.h"
#include "DisneyBSDF.h"

#include <OptiXToolkit/ShaderUtil/SelfIntersectionAvoidance.h>

#ifndef __CUDACC_RTC__
#define _P_TYPE_ 0
#endif

#if (_P_TYPE_==2)
#include "Curves.h"
#endif

static __inline__ __device__ bool isBadVector(const vec3& vector) {

    for (size_t i=0; i<3; ++i) {
        if(!isfinite(vector[i])) {
            return true;
        }
    }
    return dot(vector, vector) <= 0.0f;
}

static __inline__ __device__ bool isBadVector(const float3& vector) {
    return isBadVector(reinterpret_cast<const vec3&>(vector));
}

__inline__ __device__ void cihouSphereInstanceAux(MatInput& attrs) {

    if (params.sphereInstAuxLutBuffer != 0 && optixGetInstanceId() < params.firstSoloSphereOffset) {

        auto lut = reinterpret_cast<unsigned long long*>(params.sphereInstAuxLutBuffer);
        assert(lut != nullptr);

        auto tmp = lut[optixGetInstanceId()];
        auto auxBuffer = reinterpret_cast<float3*>(tmp);
        assert(auxBuffer != nullptr);

        attrs.clr = {};
        attrs.tang = {};
        attrs.instPos = {}; //rt_data->instPos[inst_idx2];
        attrs.instNrm = {}; //rt_data->instNrm[inst_idx2];
        attrs.instUv = {}; //rt_data->instUv[inst_idx2];
        attrs.instClr = auxBuffer[optixGetPrimitiveIndex()];
        attrs.instTang = {}; //rt_data->instTang[inst_idx2];
    }
}

extern "C" __global__ void __anyhit__shadow_cutout()
{

    const OptixTraversableHandle gas = optixGetGASTraversableHandle();
    const uint           sbtGASIndex = optixGetSbtGASIndex();
    const uint               primIdx = optixGetPrimitiveIndex();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float3 P = ray_orig + optixGetRayTmax() * ray_dir;

    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    ShadowPRD* prd = getPRD<ShadowPRD>();
    MatInput attrs{};

    auto pType = optixGetPrimitiveType();
    if (pType != OPTIX_PRIMITIVE_TYPE_SPHERE && pType != OPTIX_PRIMITIVE_TYPE_TRIANGLE) {
        
        prd->attanuation = vec3(0);
        optixTerminateRay();
        return;
    }

    bool sphere_external_ray = false;

#if (_P_TYPE_==2)
    float3 N = {};
    printf("Should not reach here\n");
    return;
#elif (_P_TYPE_==1)

    float4 q;
    // sphere center (q.x, q.y, q.z), sphere radius q.w
    optixGetSphereData( gas, primIdx, sbtGASIndex, 0.f, &q );

    float3 _pos_world_      = P;
    float3 _pos_object_     = optixTransformPointFromWorldToObjectSpace( _pos_world_ );

    float3& _center_object_ = *(float3*)&q; 

    float3 _normal_object_  = ( _pos_object_ - _center_object_ ) / q.w;
    float3 _normal_world_   = normalize( optixTransformNormalFromObjectToWorldSpace( _normal_object_ ) );

    auto _origin_object_ = optixGetObjectRayOrigin();
    sphere_external_ray = length(_origin_object_ - _center_object_) > q.w;

    float3 N = _normal_world_;
    N = faceforward( N, -ray_dir, N );

    attrs.pos = P;
    attrs.nrm = N;
    attrs.uv = sphereUV(_normal_object_, false);

    cihouSphereInstanceAux(attrs);

#else

    size_t inst_idx = optixGetInstanceId();
    
    auto instToMesh = reinterpret_cast<uint*>(params.instToMesh);
    auto meshID = instToMesh[inst_idx];

    auto aux_ptr = reinterpret_cast<void**>(params.meshAux);
    aux_ptr = aux_ptr + (meshID*5);

    auto idx_ptr = reinterpret_cast<uint3*>(aux_ptr[0]);

    auto uv_ptr  = reinterpret_cast<ushort2*>(aux_ptr[1]);
    auto clr_ptr = reinterpret_cast<ushort3*>(aux_ptr[2]);
    auto nrm_ptr = reinterpret_cast<ushort3*>(aux_ptr[3]);
    auto tan_ptr = reinterpret_cast<ushort3*>(aux_ptr[4]);

    auto vertex_idx = idx_ptr[primIdx];

    float3 _vertices_[3];
    optixGetTriangleVertexData( gas, primIdx, sbtGASIndex, 0, _vertices_);

    const float3& v0 = _vertices_[0];
    const float3& v1 = _vertices_[1];
    const float3& v2 = _vertices_[2];

    float3 N_Local = normalize( cross( normalize(v1-v0), normalize(v2-v1) ) );
    
    /* MODMA */
    float2       barys    = optixGetTriangleBarycentrics();

    float3 n0 = normalize( decodeHalf(nrm_ptr[vertex_idx.x]) );
    n0 = dot(n0, N_Local)>0.8f?n0:N_Local;
    float3 n1 = normalize( decodeHalf(nrm_ptr[vertex_idx.y]) );
    n1 = dot(n1, N_Local)>0.8f?n1:N_Local;
    float3 n2 = normalize( decodeHalf(nrm_ptr[vertex_idx.z]) );
    n2 = dot(n2, N_Local)>0.8f?n2:N_Local;

    N_Local = normalize(interp(barys, n0, n1, n2));
    float3 N_World = optixTransformNormalFromObjectToWorldSpace(N_Local);

    if (isBadVector(N_World)) 
    {  
        N_World = DisneyBSDF::SampleScatterDirection(prd->seed);
    }

    float3 N = faceforward( N_World, -ray_dir, N_World );
    
    attrs.pos = P;
    attrs.nrm = N;

    auto uv0  = decodeHalf( uv_ptr[ vertex_idx.x ] );
    auto uv1  = decodeHalf( uv_ptr[ vertex_idx.y ] );
    auto uv2  = decodeHalf( uv_ptr[ vertex_idx.z ] );
    auto clr0 = decodeHalf( clr_ptr[ vertex_idx.x ] );
    auto clr1 = decodeHalf( clr_ptr[ vertex_idx.y ] );
    auto clr2 = decodeHalf( clr_ptr[ vertex_idx.z ] );
    auto tan0 = decodeHalf( tan_ptr[ vertex_idx.x ] );
    auto tan1 = decodeHalf( tan_ptr[ vertex_idx.y ] );
    auto tan2 = decodeHalf( tan_ptr[ vertex_idx.z ] );

    auto _uv_ = interp(barys, uv0, uv1, uv2);
    attrs.uv = vec3{ _uv_.x, _uv_.y, 0 };
    attrs.clr = interp(barys, clr0, clr1, clr2);
    attrs.tang = interp(barys, tan0, tan1, tan2);
    attrs.tang = optixTransformVectorFromObjectToWorldSpace(attrs.tang);
    attrs.rayLength = optixGetRayTmax();

    attrs.instPos  = decodeHalf( rt_data->instPos[inst_idx] );
    attrs.instNrm  = decodeHalf( rt_data->instNrm[inst_idx] );
    attrs.instUv   = decodeHalf( rt_data->instUv[inst_idx]  );
    attrs.instClr  = decodeHalf( rt_data->instClr[inst_idx] );
    attrs.instTang = decodeHalf( rt_data->instTang[inst_idx]);

#endif

    attrs.pos = attrs.pos + vec3(params.cam.eye);
    attrs.isShadowRay = true;
    //MatOutput mats = evalMaterial(rt_data->textures, rt_data->uniforms, attrs);
    MatOutput mats = optixDirectCall<MatOutput, hipTextureObject_t[], float4*, const MatInput&>( rt_data->dc_index, rt_data->textures, rt_data->uniforms, attrs );

    if(length(attrs.tang)>0)
    {
        vec3 b = cross(attrs.tang, attrs.nrm);
        attrs.tang = cross(attrs.nrm, b);
        N = mats.nrm.x * attrs.tang + mats.nrm.y * b + mats.nrm.z * attrs.nrm;
    }

    //end of material computation
    //mats.metallic = clamp(mats.metallic,0.01, 0.99);
    mats.roughness = clamp(mats.roughness, 0.01f,0.99f);

    /* MODME */
    auto basecolor = mats.basecolor;
    auto metallic = mats.metallic;
    auto roughness = mats.roughness;
    auto subsurface = mats.subsurface;
    auto specular = mats.specular;
    auto specularTint = mats.specularTint;
    auto anisotropic = mats.anisotropic;
    auto sheen = mats.sheen;
    auto sheenTint = mats.sheenTint;
    auto clearcoat = mats.clearcoat;
    auto opacity = mats.opacity;
    auto flatness = mats.flatness;
    auto specTrans = mats.specTrans;
    auto scatterDistance = mats.scatterDistance;
    auto ior = mats.ior;
    auto thin = mats.thin;
    auto doubleSide = mats.doubleSide;
    auto sssParam = mats.sssParam;
    auto scatterStep = mats.scatterStep;


    if(params.simpleRender==true)
        opacity = 0;
    //opacity = clamp(opacity, 0.0f, 0.99f);
    // Stochastic alpha test to get an alpha blend effect.
    if (opacity >0.99f) // No need to calculate an expensive random number if the test is going to fail anyway.
    {
        optixIgnoreIntersection();
        return;
    }
    else
    {
        //roll a dice
        float p = rnd(prd->seed);

        float skip = opacity;
        #if (_P_TYPE_==1)
            if (sphere_external_ray) {
                skip *= opacity;
            }
        #endif

        if (p < skip){
            optixIgnoreIntersection();
            return;
        }else{
          if(mats.isHair>0.5f)
          {
             vec3 extinction = exp( - DisneyBSDF::CalculateExtinction(mats.sssParam,1.0f) );
             if(p<min(min(extinction.x, extinction.y), extinction.z))
             {
               optixIgnoreIntersection();
               return;
             }
          }

            if(length(prd->attanuation) < 0.01f){
                prd->attanuation = vec3(0.0f);
                optixTerminateRay();
                return;
            }

            if(specTrans==0.0f){
                prd->attanuation = vec3(0.0f);
                optixTerminateRay();
                return;
            }
            
            if(specTrans > 0.0f){

                if(thin == 0.0f && ior>=1.0f)
                {
                    prd->nonThinTransHit++;
                }
                if(rnd(prd->seed)<(1-specTrans)||prd->nonThinTransHit>1)
                {
                    prd->attanuation = vec3(0,0,0);
                    optixTerminateRay();
                    return;
                }

                float nDi = fabs(dot(N,normalize(ray_dir)));
                vec3 fakeTrans = vec3(1)-BRDFBasics::fresnelSchlick(vec3(1) - mats.transColor,nDi);
                prd->attanuation = prd->attanuation * fakeTrans;

                #if (_P_TYPE_==1)
                    if (sphere_external_ray) {
                        prd->attanuation *= vec3(1, 0, 0);
                        if (nDi < (1.0f-_FLT_EPL_)) {
                            prd->attanuation = {};
                            optixTerminateRay(); return;
                        } else {
                            prd->attanuation *= fakeTrans;
                        }
                    }
                #endif
                optixIgnoreIntersection();
                return;
            }
        }

        prd->attanuation = vec3(0);
        optixTerminateRay();
        return;
    }
}

static __inline__ __device__
vec3 projectedBarycentricCoord(vec3 p, vec3 q, vec3 u, vec3 v)
{
    vec3 n = cross(u,v);
    float a = 1.0f / dot(n,n);
    vec3 w = p - q;
    vec3 o;
    o.z = dot(cross(u,w),n) * a;
    o.y = dot(cross(w,v),n) * a;
    o.x = 1.0 - o.y - o.z;
    return o;
}

extern "C" __global__ void __closesthit__radiance()
{
    RadiancePRD* prd = getPRD();

    const OptixTraversableHandle gas = optixGetGASTraversableHandle();
    const uint           sbtGASIndex = optixGetSbtGASIndex();
    const uint               primIdx = optixGetPrimitiveIndex();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    float3 P = ray_orig + optixGetRayTmax() * ray_dir;

    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    MatInput attrs{};
    float estimation = 0;

#if (_P_TYPE_==2)

    float3 N = {}; 

    auto pType = optixGetPrimitiveType();
    if (pType == OPTIX_PRIMITIVE_TYPE_SPHERE || pType == OPTIX_PRIMITIVE_TYPE_TRIANGLE) {
        prd->done = true;
        return;
    }

    float3 normal = computeCurveNormal( optixGetPrimitiveType(), primIdx );

    if (dot(normal, -ray_dir) < 0) {
        normal = -normal;
    }

    N = normal;
        
    float3 wldPos = P; 
    float3 wldNorm = normal;
    float wldOffset = 0.0f;

    prd->geometryNormal = N;

    attrs.pos = P;
    attrs.nrm = N;

    auto hair_idx = optixGetInstanceId() - params.hairInstOffset;
    auto hairAux = reinterpret_cast<CurveGroupAux*>(params.hairAux);

    auto& aux = hairAux[hair_idx];

    uint strandIndex = aux.strand_i[primIdx];

    float  segmentU   = optixGetCurveParameter();
    float2 strand_u = aux.strand_u[primIdx];
    float u = strand_u.x + segmentU * strand_u.y;

    attrs.uv = {u, (float)strandIndex/ aux.strand_info.count, 0};

#elif (_P_TYPE_==1)

    float4 q;
    // sphere center (q.x, q.y, q.z), sphere radius q.w
    optixGetSphereData( gas, primIdx, sbtGASIndex, 0.0f, &q );

    float3& sphere_center = *(float3*)&q;

    float3 objPos   = optixTransformPointFromWorldToObjectSpace(P);
    float3 objNorm  = normalize( ( objPos - sphere_center ) / q.w );

    objPos = sphere_center + objNorm * q.w;

    const float c0 = 5.9604644775390625E-8f;
    const float c1 = 1.788139769587360206060111522674560546875E-7f;
    const float c2 = 1.19209317972490680404007434844970703125E-7f;

    auto fma = [](auto a, auto b, auto c) -> auto {
        return a * b + c;
    };

    vec3 objErr = fma( vec3( c0 ), abs( sphere_center ), vec3( c1 * q.w * 2.0f ) );
    float objOffset = dot( objErr, abs( objNorm ) );

    float3 wldPos, wldNorm; float wldOffset;
    SelfIntersectionAvoidance::transformSafeSpawnOffset( wldPos, wldNorm, wldOffset, objPos, objNorm, objOffset );

    P = wldPos;
    float3 N = wldNorm;

    prd->geometryNormal = N;

    attrs.pos = P;
    attrs.nrm = N;
    attrs.uv = sphereUV(objNorm, false);

    cihouSphereInstanceAux(attrs);

#else

    float3 _vertices_[3];
    optixGetTriangleVertexData( gas, primIdx, sbtGASIndex, 0, _vertices_);

    const float3& v0 = _vertices_[0];
    const float3& v1 = _vertices_[1];
    const float3& v2 = _vertices_[2];

    float3 objPos, objNorm; float objOffset; 
    //SelfIntersectionAvoidance::getSafeTriangleSpawnOffset( objPos, objNorm, objOffset );
    float2 barys = optixGetTriangleBarycentrics();
    SelfIntersectionAvoidance::getSafeTriangleSpawnOffset( objPos, objNorm, objOffset, v0, v1, v2, barys );

    float3 wldPos, wldNorm; float wldOffset;
    SelfIntersectionAvoidance::transformSafeSpawnOffset( wldPos, wldNorm, wldOffset, objPos, objNorm, objOffset );

    /* MODMA */
    P = wldPos;
    attrs.pos = P;

    const float3& N_Local = objNorm;
    float3 N = wldNorm;

    if (isBadVector(N)) 
    {  
        N = normalize(DisneyBSDF::SampleScatterDirection(prd->seed));
        N = faceforward( N, -ray_dir, N );
    }
    prd->geometryNormal = N;

    attrs.nrm = N;

    size_t inst_idx = optixGetInstanceId();
    
    // { d_uv.handle, d_clr.handle, d_nrm.handle, d_tan.handle };
    auto instToMesh = reinterpret_cast<uint*>(params.instToMesh);
    auto meshID = instToMesh[inst_idx];

    auto aux_ptr = reinterpret_cast<void**>(params.meshAux);
    aux_ptr = aux_ptr + (meshID*5);

    auto idx_ptr = reinterpret_cast<uint3*>(aux_ptr[0]);

    auto uv_ptr  = reinterpret_cast<ushort2*>(aux_ptr[1]);
    auto clr_ptr = reinterpret_cast<ushort3*>(aux_ptr[2]);
    auto nrm_ptr = reinterpret_cast<ushort3*>(aux_ptr[3]);
    auto tan_ptr = reinterpret_cast<ushort3*>(aux_ptr[4]);

    auto vertex_idx = idx_ptr[primIdx];
    
    auto uv0  = decodeHalf( uv_ptr[ vertex_idx.x ] );
    auto uv1  = decodeHalf( uv_ptr[ vertex_idx.y ] );
    auto uv2  = decodeHalf( uv_ptr[ vertex_idx.z ] );

    auto clr0 = decodeHalf( clr_ptr[ vertex_idx.x ] );
    auto clr1 = decodeHalf( clr_ptr[ vertex_idx.y ] );
    auto clr2 = decodeHalf( clr_ptr[ vertex_idx.z ] );
    auto tan0 = decodeHalf( tan_ptr[ vertex_idx.x ] );
    auto tan1 = decodeHalf( tan_ptr[ vertex_idx.y ] );
    auto tan2 = decodeHalf( tan_ptr[ vertex_idx.z ] );

    float tri_area = length(cross(_vertices_[1]-_vertices_[0], _vertices_[2]-_vertices_[1]));
    
    auto _uv_ = interp(barys, uv0, uv1, uv2);
    attrs.uv = vec3{ _uv_.x, _uv_.y, 0 };
    attrs.clr = interp(barys, clr0, clr1, clr2);
    attrs.tang = normalize(interp(barys, tan0, tan1, tan2));
    attrs.tang = optixTransformNormalFromObjectToWorldSpace(attrs.tang);

    attrs.instPos  = decodeHalf( rt_data->instPos[inst_idx] );
    attrs.instNrm  = decodeHalf( rt_data->instNrm[inst_idx] );
    attrs.instUv   = decodeHalf( rt_data->instUv[inst_idx]  );
    attrs.instClr  = decodeHalf( rt_data->instClr[inst_idx] );
    attrs.instTang = decodeHalf( rt_data->instTang[inst_idx]);

    attrs.rayLength = optixGetRayTmax();

    float3 n0 = normalize( decodeHalf(nrm_ptr[ vertex_idx.x ]) );
    float3 n1 = normalize( decodeHalf(nrm_ptr[ vertex_idx.y ]) );
    float3 n2 = normalize( decodeHalf(nrm_ptr[ vertex_idx.z ]) );

    auto N_smooth = normalize(interp(barys, n0, n1, n2));
    attrs.N = optixTransformNormalFromObjectToWorldSpace(N_smooth);

#endif

    attrs.pos = attrs.pos + vec3(params.cam.eye);
    if(! (length(attrs.tang)>0.0f) )
    {
      Onb a(attrs.N);
      attrs.T = a.m_tangent;
    }
    else
    {
      attrs.T = attrs.tang;
    }
    attrs.V = -(ray_dir);
    attrs.isShadowRay = false;
    //MatOutput mats = evalMaterial(rt_data->textures, rt_data->uniforms, attrs);
    MatOutput mats = optixDirectCall<MatOutput, hipTextureObject_t[], float4*, const MatInput&>( rt_data->dc_index, rt_data->textures, rt_data->uniforms, attrs );
    prd->mask_value = mats.mask_value;
    prd->click_pos = P;

    if (prd->test_distance) {
    
        if(mats.opacity>0.99f) { // it's actually transparency not opacity
            prd->_tmin_ = optixGetRayTmax();
        } else if(rnd(prd->seed)<mats.opacity) {
            prd->_tmin_ = optixGetRayTmax();
        } else {
            prd->test_distance = false;
            prd->maxDistance = optixGetRayTmax();
        }
        return;
    }

#if (_P_TYPE_==2)
    if(mats.doubleSide>0.5f||mats.thin>0.5f){
        N = faceforward( N, -ray_dir, N );
        prd->geometryNormal = N;
    }
#elif (_P_TYPE_==1)

    if(mats.doubleSide>0.5f||mats.thin>0.5f){
        N = faceforward( N, -ray_dir, N );
        prd->geometryNormal = N;
    }

#else

    n0 = dot(n0, N_Local)>(1-mats.smoothness)?n0:N_Local;
    n1 = dot(n1, N_Local)>(1-mats.smoothness)?n1:N_Local;
    n2 = dot(n2, N_Local)>(1-mats.smoothness)?n2:N_Local;

    N_smooth = normalize(interp(barys, n0, n1, n2));
    N = optixTransformNormalFromObjectToWorldSpace(N_smooth);

    if(mats.doubleSide>0.5f||mats.thin>0.5f){
        N = faceforward( N, -ray_dir, N );
        prd->geometryNormal = faceforward( prd->geometryNormal, -ray_dir, prd->geometryNormal );
    }
#endif

    attrs.nrm = N;
  
    mats.roughness = clamp(mats.roughness, 0.01f,0.99f);
    if(length(attrs.tang)>0)
    {
        vec3 b = cross(attrs.tang, attrs.nrm);
        attrs.tang = cross(attrs.nrm, b);
        N = mats.nrm.x * attrs.tang + mats.nrm.y * b + mats.nrm.z * attrs.nrm;
    }
//    if(dot(vec3(ray_dir), vec3(N)) * dot(vec3(ray_dir), vec3(prd->geometryNormal))<0)
//    {
//      N = prd->geometryNormal;
//    }

    if (prd->trace_denoise_albedo) {

        if(0.0f == mats.roughness) {
            prd->tmp_albedo = make_float3(1.0f);
        } else {
            prd->tmp_albedo = mats.basecolor;
        }
    }

    if (prd->trace_denoise_normal) {
        prd->tmp_normal = N;
    }

    bool next_ray_is_going_inside = false;
    mats.sssParam = mats.subsurface>0 ? mats.subsurface*mats.sssParam : mats.sssParam;
    mats.subsurface = mats.subsurface>0 ? 1 : 0;

    /* MODME */

    if(prd->diffDepth>=1)
        mats.roughness = clamp(mats.roughness, 0.2,0.99);
    if(prd->diffDepth>=2)
        mats.roughness = clamp(mats.roughness, 0.3,0.99);
    if(prd->diffDepth>=3)
        mats.roughness = clamp(mats.roughness, 0.5,0.99);

    
    if(prd->isSS == true) {
        mats.basecolor = vec3(1.0f);
        mats.roughness = 1.0f;
        mats.anisotropic = 0.0f;
        mats.sheen = 0.0f;
        mats.clearcoat = 0.0f;
        mats.specTrans = 0.0f;
        mats.ior = 1.0f;
        if(mats.subsurface==0.0f){
            prd->passed = true;
            prd->samplePdf = 1.0f;
            prd->radiance = make_float3(0.0f, 0.0f, 0.0f);
            prd->readMat(prd->sigma_t, prd->ss_alpha);
            auto trans = DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
            prd->attenuation2 *= trans;
            prd->attenuation *= trans;
            //prd->origin = P;
            prd->direction = ray_dir;
            //auto n = prd->geometryNormal;
            //n = faceforward(n, -ray_dir, n);
            prd->_tmin_ = optixGetRayTmax();
            return;
        }
        if(mats.subsurface>0.0f && dot(normalize(ray_dir),N)<0.0f){
            prd->attenuation2 = make_float3(0.0f,0.0f,0.0f);
            prd->attenuation = make_float3(0.0f,0.0f,0.0f);
            prd->radiance = make_float3(0.0f,0.0f,0.0f);
            prd->done = true;
            return;
        }
    }

    prd->attenuation2 = prd->attenuation;
    prd->countEmitted = false;
    prd->prob2 = prd->prob;
    prd->passed = false;

    if(mats.opacity > 0.99f || rnd(prd->seed)<mats.opacity)
    {
        if (prd->curMatIdx > 0) {
          vec3 sigma_t, ss_alpha;
          //vec3 sigma_t, ss_alpha;
          prd->readMat(sigma_t, ss_alpha);
          if (ss_alpha.x < 0.0f) { // is inside Glass
            prd->attenuation *= DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
          } else {
            prd->attenuation *= DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
          }
        }
        prd->attenuation2 = prd->attenuation;
        prd->passed = true;
        //you shall pass!
        prd->radiance = make_float3(0.0f);
        prd->_tmin_ = optixGetRayTmax();
        prd->alphaHit = true;

        prd->prob *= 1;
        prd->countEmitted = false;
        return;
    }

    if(prd->depth==0&&mats.flatness>0.5)
    {
        prd->radiance = make_float3(0.0f);
        prd->done = true;
        return;
    }
    
    float is_refl;
    float3 inDir = ray_dir;
    vec3 wi = vec3(0.0f);
    float pdf = 0.0f;
    float rPdf = 0.0f;
    float fPdf = 0.0f;
    float rrPdf = 0.0f;

    float3 T = attrs.tang;
    float3 B;
    if(length(T)>0)
    {
        B = cross(N, T);
    } else
    {
        Onb a(N);
        T = a.m_tangent;
        B = a.m_binormal;
    }

    DisneyBSDF::SurfaceEventFlags flag;
    DisneyBSDF::PhaseFunctions phaseFuncion;
    vec3 extinction;
    vec3 reflectance = vec3(0.0f);
    bool isDiff = false;
    bool isSS = false;
    bool isTrans = false;
    flag = DisneyBSDF::scatterEvent;

    //sssColor = mix(basecolor, sssColor, subsurface);

    while(DisneyBSDF::SampleDisney2(
                prd->seed,
                prd->eventseed,
                mats,
                T,
                B,
                N,
                prd->geometryNormal,
                -normalize(ray_dir),
                mats.thin>0.5f,
                next_ray_is_going_inside,
                wi,
                reflectance,
                rPdf,
                fPdf,
                flag,
                prd->medium,
                extinction,
                isDiff,
                isSS,
                isTrans,
                prd->minSpecRough
                )  == false)
        {
            isSS = false;
            isDiff = false;
            prd->samplePdf = fPdf;
            reflectance = fPdf>0?reflectance/fPdf:vec3(0.0f);
            prd->done = fPdf>0?true:prd->done;
            flag = DisneyBSDF::scatterEvent;
        }
        
    prd->samplePdf = fPdf;
    reflectance = fPdf>0?reflectance/fPdf:vec3(0.0f);
    prd->done = fPdf>0?prd->done:true;
    prd->isSS = isSS;
    pdf = 1.0;
    if(isDiff || prd->diffDepth>0){
        prd->diffDepth++;
    }


    prd->passed = false;
    bool inToOut = false;
    bool outToIn = false;

    bool istransmission = dot(vec3(prd->geometryNormal), vec3(wi)) * dot(vec3(prd->geometryNormal), vec3(-normalize(ray_dir)))<0;
    //istransmission = (istransmission && thin<0.5 && mats.doubleSide==false);
    if(istransmission || flag == DisneyBSDF::diracEvent) {
    //if(flag == DisneyBSDF::transmissionEvent || flag == DisneyBSDF::diracEvent) {
        next_ray_is_going_inside = dot(vec3(prd->geometryNormal),vec3(wi))<=0;
    }
    prd->max_depth = ((prd->depth==0 && isSS) || (prd->depth>0 && (mats.specTrans>0||mats.isHair>0)) )?12:prd->max_depth;
    if(mats.thin>0.5f || mats.doubleSide>0.5f)
    {
        if (prd->curMatIdx > 0) {
            vec3 sigma_t, ss_alpha;
            prd->readMat(sigma_t, ss_alpha);

            vec3 trans;
            if (ss_alpha.x<0.0f) { // is inside Glass
                trans = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
            } else {
                trans = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
            }
            prd->attenuation *= trans;
            prd->attenuation2 *= trans;
        }

        next_ray_is_going_inside = false;

    }else{
    
        //if(flag == DisneyBSDF::transmissionEvent || flag == DisneyBSDF::diracEvent) {
        if(istransmission || flag == DisneyBSDF::diracEvent) {
            if(next_ray_is_going_inside){

                    outToIn = true;
                    inToOut = false;

                    prd->medium = DisneyBSDF::PhaseFunctions::isotropic;

                    if (prd->curMatIdx > 0) {
                        vec3 sigma_t, ss_alpha;
                        //vec3 sigma_t, ss_alpha;0
                        prd->readMat(sigma_t, ss_alpha);
                        if (ss_alpha.x < 0.0f) { // is inside Glass
                            prd->attenuation *= DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
                        } else {
                            prd->attenuation *= DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                        }
                    }
                    prd->channelPDF = vec3(1.0f/3.0f);
                    if (isTrans) {
                        vec3 channelPDF = vec3(1.0f/3.0f);
                        prd->pushMat(extinction);
                        prd->isSS = false;
                        prd->scatterDistance = mats.scatterDistance;
                        prd->maxDistance = mats.scatterStep>0.5f? DisneyBSDF::SampleDistance(prd->seed, prd->scatterDistance) : 1e16f;
                    } else {

                        prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * prd->ss_alpha, prd->sigma_t, prd->channelPDF);
                        //here is the place caused inf ray:fixed
                        auto min_sg = fmax(fmin(fmin(prd->sigma_t.x, prd->sigma_t.y), prd->sigma_t.z), 1e-8f);
                        //what should be the right value???
                        //prd->maxDistance = max(prd->maxDistance, 10/min_sg);
                        //printf("maxdist:%f\n",prd->maxDistance);
                        
                        // already calculated in BxDF
                        prd->pushMat(prd->sigma_t, prd->ss_alpha);
                        prd->isSS = true;
                        prd->scatterDistance = mats.scatterDistance;
                    }


                    prd->scatterStep = mats.scatterStep;
            }
            else{
                outToIn = false;
                inToOut = true;

                float3 trans;
                vec3 sigma_t, ss_alpha;
                prd->readMat(sigma_t, ss_alpha);
                if(prd->curMatIdx==0)
                { 
                    trans = vec3(1.0f); 
                }
                else if (ss_alpha.x<0.0f) { // Glass
                    trans = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
                } else {
                    trans = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                }

                prd->attenuation *= trans;
                prd->attenuation2 *= trans;
                
                prd->popMat(sigma_t, ss_alpha);

                prd->medium = (prd->curMatIdx==0)? DisneyBSDF::PhaseFunctions::vacuum : DisneyBSDF::PhaseFunctions::isotropic;

                if(ss_alpha.x < 0.0f) 
                {
                    prd->isSS = false;
                    prd->maxDistance = 1e16;
                }
                else //next ray in 3s object
                {
                    prd->isSS = true;
                    prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * ss_alpha, sigma_t, prd->channelPDF);
                }
            }
        }else{
            if(prd->medium == DisneyBSDF::PhaseFunctions::isotropic){
                    vec3 trans = vec3(1.0f);
                    vec3 sigma_t, ss_alpha;
                    prd->readMat(sigma_t, ss_alpha);
                    prd->isSS = false;
                    if(prd->curMatIdx==0)
                    {
                        prd->maxDistance = 1e16f;
                    }
                    else if (ss_alpha.x<0.0f) { // Glass
                        trans = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
                        vec3 channelPDF = vec3(1.0f/3.0f);
                        prd->maxDistance = mats.scatterStep>0.5f? DisneyBSDF::SampleDistance2(prd->seed, sigma_t, sigma_t, channelPDF) : 1e16f;
                    } else { // SSS
                        trans = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                        prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * ss_alpha, sigma_t, prd->channelPDF);
                        prd->isSS = true;
                    }

                    prd->attenuation2 *= trans;
                    prd->attenuation *= trans;
            }
            else
            {
                prd->isSS = false;
                prd->medium = DisneyBSDF::PhaseFunctions::vacuum;
                prd->channelPDF = vec3(1.0f/3.0f);
                prd->maxDistance = 1e16f;
            }
        }
    }

    prd->medium = next_ray_is_going_inside?DisneyBSDF::PhaseFunctions::isotropic : prd->curMatIdx==0?DisneyBSDF::PhaseFunctions::vacuum : DisneyBSDF::PhaseFunctions::isotropic;
 

//    if(mats.thin>0.5f){
//        vec3 H = normalize(vec3(normalize(wi)) + vec3(-normalize(ray_dir)));
//        attrs.N = N;
//        attrs.T = cross(B,N);
//        attrs.L = vec3(normalize(wi));
//        attrs.V = vec3(-normalize(ray_dir));
//        attrs.H = normalize(H);
//        attrs.reflectance = reflectance;
//        attrs.fresnel = DisneyBSDF::DisneyFresnel(mats.basecolor, mats.metallic, mats.ior, mats.specularTint, dot(attrs.H, attrs.V), dot(attrs.H, attrs.L), false);
//        MatOutput mat2 = evalReflectance(zenotex, rt_data->uniforms, attrs);
//        reflectance = mat2.reflectance;
//    }


    prd->countEmitted = false;
    prd->attenuation *= reflectance;
    if(mats.subsurface>0 && (mats.thin>0.5 || mats.doubleSide>0.5) && istransmission){
      prd->attenuation2 *= reflectance;
    }
    prd->depth++;

    if(prd->depth>=3)
        mats.roughness = clamp(mats.roughness, 0.5f,0.99f);

    auto evalBxDF = [&](const float3& _wi_, const float3& _wo_, float& thisPDF) -> float3 {

        const auto& L = _wi_; // pre-normalized
        const vec3& V = _wo_; // pre-normalized
        vec3 rd, rs, rt; // captured by lambda

        float3 lbrdf = DisneyBSDF::EvaluateDisney2(vec3(1.0f), mats, L, V, T, B, N,prd->geometryNormal,
            mats.thin > 0.5f, flag == DisneyBSDF::transmissionEvent ? inToOut : next_ray_is_going_inside, thisPDF, rrPdf,
            dot(N, L), rd, rs, rt);

        prd->radiance_d = rd;
        prd->radiance_s = rs;
        prd->radiance_t = rt;
//        MatOutput mat2;
//        if(mats.thin>0.5f){
//            vec3 H = normalize(vec3(normalize(L)) + V);
//            attrs.N = N;
//            attrs.T = cross(B,N);
//            attrs.L = vec3(normalize(L));
//            attrs.V = V;
//            attrs.H = normalize(H);
//            attrs.reflectance = lbrdf;
//            attrs.fresnel = DisneyBSDF::DisneyFresnel( mats.basecolor, mats.metallic, mats.ior, mats.specularTint, dot(attrs.H, attrs.V), dot(attrs.H, attrs.L), false);
//            mat2 = evalReflectance(zenotex, rt_data->uniforms, attrs);
//        }

        return lbrdf;

    };

    auto taskAux = [&](const vec3& radiance) {
        prd->radiance_d *= radiance;
        prd->radiance_s *= radiance;
        prd->radiance_t *= radiance;
    };

    ShadowPRD shadowPRD {};
    shadowPRD.seed = prd->seed;
    shadowPRD.attanuation = make_float3(1.0f, 1.0f, 1.0f);
    shadowPRD.nonThinTransHit = (mats.thin < 0.5f && mats.specTrans > 0) ? 1 : 0;

    float3 frontPos, backPos;
    if (wldOffset > 0) {
        SelfIntersectionAvoidance::offsetSpawnPoint( frontPos, backPos, wldPos, prd->geometryNormal, wldOffset );
    } else {
        frontPos = wldPos;
        backPos = wldPos;
    }

    shadowPRD.origin = dot(wi, vec3(prd->geometryNormal)) > 0 ? frontPos : backPos;
    //auto shadingP = rtgems::offset_ray(shadowPRD.origin + params.cam.eye,  prd->geometryNormal); // world space
    
    //shadowPRD.origin = frontPos;
    //if(mats.subsurface>0 && (mats.thin>0.5 || mats.doubleSide>0.5) && istransmission){
        //shadowPRD.origin = backPos; //rtgems::offset_ray(P,  -prd->geometryNormal);
    //}
    
    auto shadingP = rtgems::offset_ray(P + params.cam.eye, dot(wi, vec3(prd->geometryNormal)) > 0 ? prd->geometryNormal:-prd->geometryNormal); // world space
    //if(mats.subsurface>0 && (mats.thin>0.5 || mats.doubleSide>0.5) && istransmission){
        //shadingP = rtgems::offset_ray(P + params.cam.eye,  -prd->geometryNormal);
    //}

    prd->radiance = {};
    prd->direction = normalize(wi);
    prd->origin = dot(prd->direction, wldNorm) > 0 ? frontPos : backPos;


    float3 radianceNoShadow = {};
    float3* dummy_prt = nullptr;
    if (mats.shadowReceiver > 0.5f) {
        dummy_prt = &radianceNoShadow;
    }

    prd->lightmask = DefaultMatMask;
    shadowPRD.ShadowNormal = dot(wi, vec3(prd->geometryNormal)) > 0 ? prd->geometryNormal:-prd->geometryNormal;
    DirectLighting<true>(prd, shadowPRD, shadingP, ray_dir, evalBxDF, &taskAux, dummy_prt);
    if(mats.shadowReceiver > 0.5f)
    {
      auto radiance = length(prd->radiance);
      prd->radiance.x = radiance;//the light contribution received with shadow attenuation
      prd->radiance.y = length(radianceNoShadow);
      prd->radiance.z = 0;
      prd->done = true;
    }

    prd->direction = normalize(wi);

    prd->origin = dot(prd->direction, prd->geometryNormal) < 0.0f ? backPos : frontPos;

    if (prd->medium != DisneyBSDF::vacuum) {
        prd->_mask_ = (uint8_t)(EverythingMask ^ VolumeMatMask);
    } else {
        prd->_mask_ = EverythingMask;
    }

    prd->radiance += mats.emission;
    if(length(mats.emission)>0)
    {
      prd->done = true;
    }
}

extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion( true );
}
