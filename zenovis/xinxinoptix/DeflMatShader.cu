#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <cuda/helpers.h>

#include <sutil/vec_math.h>
#include "optixPathTracer.h"

#include "TraceStuff.h"
#include "zxxglslvec.h"

#include "IOMat.h"
#include "Light.h"

#include "DisneyBRDF.h"
#include "DisneyBSDF.h"

#include <OptiXToolkit/ShaderUtil/SelfIntersectionAvoidance.h>

static __inline__ __device__ bool isBadVector(const vec3& vector) {

    for (size_t i=0; i<3; ++i) {
        if(!isfinite(vector[i])) {
            return true;
        }
    }
    return dot(vector, vector) <= 0.0f;
}

static __inline__ __device__ bool isBadVector(const float3& vector) {
    return isBadVector(reinterpret_cast<const vec3&>(vector));
}

__inline__ __device__ void cihouSphereInstanceAux(MatInput& attrs) {

    if (params.sphereInstAuxLutBuffer != 0 && optixGetInstanceId() < params.firstSoloSphereOffset) {

        auto lut = reinterpret_cast<unsigned long long*>(params.sphereInstAuxLutBuffer);
        assert(lut != nullptr);

        auto tmp = lut[optixGetInstanceId()];
        auto auxBuffer = reinterpret_cast<float3*>(tmp);
        assert(auxBuffer != nullptr);

        attrs.clr = {};
        attrs.tang = {};
        attrs.instPos = {}; //rt_data->instPos[inst_idx2];
        attrs.instNrm = {}; //rt_data->instNrm[inst_idx2];
        attrs.instUv = {}; //rt_data->instUv[inst_idx2];
        attrs.instClr = auxBuffer[optixGetPrimitiveIndex()];
        attrs.instTang = {}; //rt_data->instTang[inst_idx2];
    }
}

extern "C" __global__ void __anyhit__shadow_cutout()
{

    const OptixTraversableHandle gas = optixGetGASTraversableHandle();
    const uint           sbtGASIndex = optixGetSbtGASIndex();
    const uint               primIdx = optixGetPrimitiveIndex();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float3 P = ray_orig + optixGetRayTmax() * ray_dir;

    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    ShadowPRD* prd = getPRD<ShadowPRD>();
    MatInput attrs{};

    bool sphere_external_ray = false;

#if (_SPHERE_)

    float4 q;
    // sphere center (q.x, q.y, q.z), sphere radius q.w
    optixGetSphereData( gas, primIdx, sbtGASIndex, 0.f, &q );

    float3 _pos_world_      = P;
    float3 _pos_object_     = optixTransformPointFromWorldToObjectSpace( _pos_world_ );

    float3& _center_object_ = *(float3*)&q; 

    float3 _normal_object_  = ( _pos_object_ - _center_object_ ) / q.w;
    float3 _normal_world_   = normalize( optixTransformNormalFromObjectToWorldSpace( _normal_object_ ) );

    auto _origin_object_ = optixGetObjectRayOrigin();
    sphere_external_ray = length(_origin_object_ - _center_object_) > q.w;

    float3 N = _normal_world_;
    N = faceforward( N, -ray_dir, N );

    attrs.pos = P;
    attrs.nrm = N;
    attrs.uv = sphereUV(_normal_object_, false);

    cihouSphereInstanceAux(attrs);

#else
    size_t inst_idx = optixGetInstanceIndex();
    size_t vert_aux_offset = rt_data->auxOffset[inst_idx];
    size_t vert_idx_offset = vert_aux_offset + primIdx*3;

    float3 _vertices_[3];
    optixGetTriangleVertexData( gas, primIdx, sbtGASIndex, 0, _vertices_);

    const float3& v0 = _vertices_[0];
    const float3& v1 = _vertices_[1];
    const float3& v2 = _vertices_[2];

    float3 N_Local = normalize( cross( normalize(v1-v0), normalize(v2-v1) ) );
    
    /* MODMA */
    float2       barys    = optixGetTriangleBarycentrics();

    float3 n0 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+0 ]) );
    n0 = dot(n0, N_Local)>0.8f?n0:N_Local;
    float3 n1 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+1 ]) );
    n1 = dot(n1, N_Local)>0.8f?n1:N_Local;
    float3 n2 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+2 ]) );
    n2 = dot(n2, N_Local)>0.8f?n2:N_Local;

    N_Local = normalize(interp(barys, n0, n1, n2));
    float3 N_World = optixTransformNormalFromObjectToWorldSpace(N_Local);

    if (isBadVector(N_World)) 
    {  
        N_World = DisneyBSDF::SampleScatterDirection(prd->seed);
    }

    float3 N = faceforward( N_World, -ray_dir, N_World );
    
    attrs.pos = P;
    attrs.nrm = N;

    const float3& uv0  = decodeColor( rt_data->uv[ vert_idx_offset+0 ]   );
    const float3& uv1  = decodeColor( rt_data->uv[ vert_idx_offset+1 ]   );
    const float3& uv2  = decodeColor( rt_data->uv[ vert_idx_offset+2 ]   );
    const float3& clr0 = decodeColor( rt_data->clr[ vert_idx_offset+0 ]  );
    const float3& clr1 = decodeColor( rt_data->clr[ vert_idx_offset+1 ]  );
    const float3& clr2 = decodeColor( rt_data->clr[ vert_idx_offset+2 ]  );
    const float3& tan0 = decodeNormal( rt_data->tan[ vert_idx_offset+0 ] );
    const float3& tan1 = decodeNormal( rt_data->tan[ vert_idx_offset+1 ] );
    const float3& tan2 = decodeNormal( rt_data->tan[ vert_idx_offset+2 ] );

    attrs.uv = interp(barys, uv0, uv1, uv2);//todo later
    attrs.clr = interp(barys, clr0, clr1, clr2);
    attrs.tang = interp(barys, tan0, tan1, tan2);
    attrs.tang = optixTransformVectorFromObjectToWorldSpace(attrs.tang);
    attrs.rayLength = optixGetRayTmax();

    attrs.instPos =  decodeColor( rt_data->instPos[inst_idx] );
    attrs.instNrm =  decodeColor( rt_data->instNrm[inst_idx] );
    attrs.instUv =   decodeColor( rt_data->instUv[inst_idx]  );
    attrs.instClr =  decodeColor( rt_data->instClr[inst_idx] );
    attrs.instTang = decodeColor( rt_data->instTang[inst_idx]);

#endif

    attrs.pos = attrs.pos + vec3(params.cam.eye);
    //MatOutput mats = evalMaterial(rt_data->textures, rt_data->uniforms, attrs);
    MatOutput mats = optixDirectCall<MatOutput, hipTextureObject_t[], float4*, const MatInput&>( rt_data->dc_index, rt_data->textures, rt_data->uniforms, attrs );

    if(length(attrs.tang)>0)
    {
        vec3 b = cross(attrs.tang, attrs.nrm);
        attrs.tang = cross(attrs.nrm, b);
        N = mats.nrm.x * attrs.tang + mats.nrm.y * b + mats.nrm.z * attrs.nrm;
    }

    //end of material computation
    //mats.metallic = clamp(mats.metallic,0.01, 0.99);
    mats.roughness = clamp(mats.roughness, 0.01f,0.99f);

    /* MODME */
    auto basecolor = mats.basecolor;
    auto metallic = mats.metallic;
    auto roughness = mats.roughness;
    auto subsurface = mats.subsurface;
    auto specular = mats.specular;
    auto specularTint = mats.specularTint;
    auto anisotropic = mats.anisotropic;
    auto sheen = mats.sheen;
    auto sheenTint = mats.sheenTint;
    auto clearcoat = mats.clearcoat;
    auto opacity = mats.opacity;
    auto flatness = mats.flatness;
    auto specTrans = mats.specTrans;
    auto scatterDistance = mats.scatterDistance;
    auto ior = mats.ior;
    auto thin = mats.thin;
    auto doubleSide = mats.doubleSide;
    auto sssParam = mats.sssParam;
    auto scatterStep = mats.scatterStep;


    if(params.simpleRender==true)
        opacity = 0;
    //opacity = clamp(opacity, 0.0f, 0.99f);
    // Stochastic alpha test to get an alpha blend effect.
    if (opacity >0.99f) // No need to calculate an expensive random number if the test is going to fail anyway.
    {
        optixIgnoreIntersection();
        return;
    }
    else
    {
        //roll a dice
        float p = rnd(prd->seed);

        float skip = opacity;
        #if (_SPHERE_)
            if (sphere_external_ray) {
                skip *= opacity;
            }
        #endif

        if (p < skip){
            optixIgnoreIntersection();
            return;
        }else{
          if(mats.isHair>0.5f)
          {
             vec3 extinction = exp( - DisneyBSDF::CalculateExtinction(mats.sssParam,1.0f) );
             if(p<min(min(extinction.x, extinction.y), extinction.z))
             {
               optixIgnoreIntersection();
               return;
             }
          }

            if(length(prd->attanuation) < 0.01f){
                prd->attanuation = vec3(0.0f);
                optixTerminateRay();
                return;
            }

            if(specTrans==0.0f){
                prd->attanuation = vec3(0.0f);
                optixTerminateRay();
                return;
            }
            
            if(specTrans > 0.0f){

                if(thin == 0.0f && ior>=1.0f)
                {
                    prd->nonThinTransHit++;
                }
                if(rnd(prd->seed)<(1-specTrans)||prd->nonThinTransHit>1)
                {
                    prd->attanuation = vec3(0,0,0);
                    optixTerminateRay();
                    return;
                }

                float nDi = fabs(dot(N,normalize(ray_dir)));
                vec3 fakeTrans = vec3(1)-BRDFBasics::fresnelSchlick(vec3(1) - mats.transColor,nDi);
                prd->attanuation = prd->attanuation * fakeTrans;

                #if (_SPHERE_)
                    if (sphere_external_ray) {
                        prd->attanuation *= vec3(1, 0, 0);
                        if (nDi < (1.0f-_FLT_EPL_)) {
                            prd->attanuation = {};
                            optixTerminateRay(); return;
                        } else {
                            prd->attanuation *= fakeTrans;
                        }
                    }
                #endif
                optixIgnoreIntersection();
                return;
            }
        }

        prd->attanuation = vec3(0);
        optixTerminateRay();
        return;
    }
}

static __inline__ __device__
vec3 projectedBarycentricCoord(vec3 p, vec3 q, vec3 u, vec3 v)
{
    vec3 n = cross(u,v);
    float a = 1.0f / dot(n,n);
    vec3 w = p - q;
    vec3 o;
    o.z = dot(cross(u,w),n) * a;
    o.y = dot(cross(w,v),n) * a;
    o.x = 1.0 - o.y - o.z;
    return o;
}

extern "C" __global__ void __closesthit__radiance()
{
    RadiancePRD* prd = getPRD();

    const OptixTraversableHandle gas = optixGetGASTraversableHandle();
    const uint           sbtGASIndex = optixGetSbtGASIndex();
    const uint               primIdx = optixGetPrimitiveIndex();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    float3 P = ray_orig + optixGetRayTmax() * ray_dir;

    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    MatInput attrs{};
    float estimation = 0;

#if (_SPHERE_)

    float4 q;
    // sphere center (q.x, q.y, q.z), sphere radius q.w
    optixGetSphereData( gas, primIdx, sbtGASIndex, 0.0f, &q );

    float3& sphere_center = *(float3*)&q;

    float3 objPos   = optixTransformPointFromWorldToObjectSpace(P);
    float3 objNorm  = normalize( ( objPos - sphere_center ) / q.w );

    objPos = sphere_center + objNorm * q.w;

    const float c0 = 5.9604644775390625E-8f;
    const float c1 = 1.788139769587360206060111522674560546875E-7f;
    const float c2 = 1.19209317972490680404007434844970703125E-7f;

    auto fma = [](auto a, auto b, auto c) -> auto {
        return a * b + c;
    };

    vec3 objErr = fma( vec3( c0 ), abs( sphere_center ), vec3( c1 * q.w * 2.0f ) );
    float objOffset = dot( objErr, abs( objNorm ) );

    float3 wldPos, wldNorm; float wldOffset;
    SelfIntersectionAvoidance::transformSafeSpawnOffset( wldPos, wldNorm, wldOffset, objPos, objNorm, objOffset );

    P = wldPos;
    float3 N = wldNorm;

    prd->geometryNormal = N;

    attrs.pos = P;
    attrs.nrm = N;
    attrs.uv = sphereUV(objNorm, false);

    cihouSphereInstanceAux(attrs);

#else

    size_t inst_idx = optixGetInstanceIndex();
    size_t vert_aux_offset = rt_data->auxOffset[inst_idx];
    size_t vert_idx_offset = vert_aux_offset + primIdx*3;
    //size_t tri_aux_offset = rt_data->auxTriOffset[inst_idx];
    //size_t tri_idx_offset = tri_aux_offset + primIdx;
    //size_t vidx0 = rt_data->vidx[tri_idx_offset * 3 + 0];
    //size_t vidx1 = rt_data->vidx[tri_idx_offset * 3 + 1];
    //size_t vidx2 = rt_data->vidx[tri_idx_offset * 3 + 2];

    float3 _vertices_[3];
    optixGetTriangleVertexData( gas, primIdx, sbtGASIndex, 0, _vertices_);
    
    const float3& v0 = _vertices_[0];
    const float3& v1 = _vertices_[1];
    const float3& v2 = _vertices_[2];

    float3 objPos, objNorm; float objOffset; 
    //SelfIntersectionAvoidance::getSafeTriangleSpawnOffset( objPos, objNorm, objOffset );
    float2 barys = optixGetTriangleBarycentrics();
    SelfIntersectionAvoidance::getSafeTriangleSpawnOffset( objPos, objNorm, objOffset, v0, v1, v2, barys );

    float3 wldPos, wldNorm; float wldOffset;
    SelfIntersectionAvoidance::transformSafeSpawnOffset( wldPos, wldNorm, wldOffset, objPos, objNorm, objOffset );

    /* MODMA */
    P = wldPos;
    attrs.pos = P;

    const float3& N_Local = objNorm;
    float3 N = wldNorm;

    if (isBadVector(N)) 
    {  
        N = normalize(DisneyBSDF::SampleScatterDirection(prd->seed));
        N = faceforward( N, -ray_dir, N );
    }
    prd->geometryNormal = N;

    attrs.nrm = N;

    const float3& uv0  = decodeColor( rt_data->uv[ vert_idx_offset+0 ] );
    const float3& uv1  = decodeColor( rt_data->uv[ vert_idx_offset+1 ] );
    const float3& uv2  = decodeColor( rt_data->uv[ vert_idx_offset+2 ] );
    const float3& clr0 = decodeColor( rt_data->clr[ vert_idx_offset+0 ] );
    const float3& clr1 = decodeColor( rt_data->clr[ vert_idx_offset+1 ] );
    const float3& clr2 = decodeColor( rt_data->clr[ vert_idx_offset+2 ] );
    const float3& tan0 = decodeNormal( rt_data->tan[ vert_idx_offset+0 ] );
    const float3& tan1 = decodeNormal( rt_data->tan[ vert_idx_offset+1 ] );
    const float3& tan2 = decodeNormal( rt_data->tan[ vert_idx_offset+2 ] );
    float tri_area = length(cross(_vertices_[1]-_vertices_[0], _vertices_[2]-_vertices_[1]));
    float uv_area = length(cross(uv1 - uv0, uv2-uv0));
    estimation = uv_area * 4096.0f*4096.0f / (tri_area + 1e-6);
        attrs.uv = interp(barys, uv0, uv1, uv2);//todo later
    attrs.clr = interp(barys, clr0, clr1, clr2);
    attrs.tang = normalize(interp(barys, tan0, tan1, tan2));
    attrs.tang = optixTransformVectorFromObjectToWorldSpace(attrs.tang);

    attrs.instPos =  decodeColor( rt_data->instPos[inst_idx] );
    attrs.instNrm =  decodeColor( rt_data->instNrm[inst_idx] );
    attrs.instUv =   decodeColor( rt_data->instUv[inst_idx]  );
    attrs.instClr =  decodeColor( rt_data->instClr[inst_idx] );
    attrs.instTang = decodeColor( rt_data->instTang[inst_idx]);
    attrs.rayLength = optixGetRayTmax();

    float3 n0 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+0 ]) );
    n0 = n0;

    float3 n1 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+1 ]) );
    n1 = n1;

    float3 n2 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+2 ]) );
    n2 = n2;

    auto N_smooth = normalize(interp(barys, n0, n1, n2));
    attrs.N = optixTransformNormalFromObjectToWorldSpace(N_smooth);


#endif

    attrs.pos = attrs.pos + vec3(params.cam.eye);
    if(! (length(attrs.tang)>0.0f) )
    {
      Onb a(attrs.N);
      attrs.T = a.m_tangent;
    }
    else
    {
      attrs.T = attrs.tang;
    }
    attrs.V = -(ray_dir);
    //MatOutput mats = evalMaterial(rt_data->textures, rt_data->uniforms, attrs);
    MatOutput mats = optixDirectCall<MatOutput, hipTextureObject_t[], float4*, const MatInput&>( rt_data->dc_index, rt_data->textures, rt_data->uniforms, attrs );
    prd->mask_value = mats.mask_value;
    prd->click_pos = P;

    if (prd->test_distance) {
    
        if(mats.opacity>0.99f) { // it's actually transparency not opacity
            prd->_tmin_ = optixGetRayTmax();
        } else if(rnd(prd->seed)<mats.opacity) {
            prd->_tmin_ = optixGetRayTmax();
        } else {
            prd->test_distance = false;
            prd->maxDistance = optixGetRayTmax();
        }
        return;
    }

#if _SPHERE_

    if(mats.doubleSide>0.5f||mats.thin>0.5f){
        N = faceforward( N, -ray_dir, N );
        prd->geometryNormal = N;
    }

#else
    n0 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+0 ]) );
    n0 = dot(n0, N_Local)>(1-mats.smoothness)?n0:N_Local;

    n1 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+1 ]) );
    n1 = dot(n1, N_Local)>(1-mats.smoothness)?n1:N_Local;

    n2 = normalize( decodeNormal(rt_data->nrm[ vert_idx_offset+2 ]) );
    n2 = dot(n2, N_Local)>(1-mats.smoothness)?n2:N_Local;

    N_smooth = normalize(interp(barys, n0, n1, n2));
    N = optixTransformNormalFromObjectToWorldSpace(N_smooth);

    if(mats.doubleSide>0.5f||mats.thin>0.5f){
        N = faceforward( N, -ray_dir, N );
        prd->geometryNormal = faceforward( prd->geometryNormal, -ray_dir, prd->geometryNormal );
    }
#endif

    attrs.nrm = N;
    float term = log2(optixGetRayTmax()*prd->pixel_area*sqrt(estimation))/4.0f;
//    printf("rayDist:%f, tex_per_area:%f, term:%f, pixel_area:%f\n", optixGetRayTmax(),
//           sqrt(estimation), term, prd->pixel_area);
    //mats.nrm = normalize(mix(mats.nrm, vec3(0,0,1), clamp(term,0.0f,1.0f)));
    //end of material computation
    //mats.metallic = clamp(mats.metallic,0.01, 0.99);
    mats.roughness = clamp(mats.roughness, 0.01f,0.99f);
    if(length(attrs.tang)>0)
    {
        vec3 b = cross(attrs.tang, attrs.nrm);
        attrs.tang = cross(attrs.nrm, b);
        N = mats.nrm.x * attrs.tang + mats.nrm.y * b + mats.nrm.z * attrs.nrm;
    }
//    if(dot(vec3(ray_dir), vec3(N)) * dot(vec3(ray_dir), vec3(prd->geometryNormal))<0)
//    {
//      N = prd->geometryNormal;
//    }

    if (prd->trace_denoise_albedo) {

        if(0.0f == mats.roughness) {
            prd->tmp_albedo = make_float3(1.0f);
        } else {
            prd->tmp_albedo = mats.basecolor;
        }
    }

    if (prd->trace_denoise_normal) {
        prd->tmp_normal = N;
    }

    bool next_ray_is_going_inside = false;
    mats.sssParam = mats.subsurface>0 ? mats.subsurface*mats.sssParam : mats.sssParam;
    mats.subsurface = mats.subsurface>0 ? 1 : 0;

    /* MODME */

    if(prd->diffDepth>=1)
        mats.roughness = clamp(mats.roughness, 0.2,0.99);
    if(prd->diffDepth>=2)
        mats.roughness = clamp(mats.roughness, 0.3,0.99);
    if(prd->diffDepth>=3)
        mats.roughness = clamp(mats.roughness, 0.5,0.99);

    
    if(prd->isSS == true) {
        mats.basecolor = vec3(1.0f);
        mats.roughness = 1.0f;
        mats.anisotropic = 0.0f;
        mats.sheen = 0.0f;
        mats.clearcoat = 0.0f;
        mats.specTrans = 0.0f;
        mats.ior = 1.0f;
        if(mats.subsurface==0.0f){
            prd->passed = true;
            prd->samplePdf = 1.0f;
            prd->radiance = make_float3(0.0f, 0.0f, 0.0f);
            prd->readMat(prd->sigma_t, prd->ss_alpha);
            auto trans = DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
            prd->attenuation2 *= trans;
            prd->attenuation *= trans;
            //prd->origin = P;
            prd->direction = ray_dir;
            //auto n = prd->geometryNormal;
            //n = faceforward(n, -ray_dir, n);
            prd->_tmin_ = optixGetRayTmax();
            return;
        }
        if(mats.subsurface>0.0f && dot(normalize(ray_dir),N)<0.0f){
            prd->attenuation2 = make_float3(0.0f,0.0f,0.0f);
            prd->attenuation = make_float3(0.0f,0.0f,0.0f);
            prd->radiance = make_float3(0.0f,0.0f,0.0f);
            prd->done = true;
            return;
        }
    }

    prd->attenuation2 = prd->attenuation;
    prd->countEmitted = false;
    prd->prob2 = prd->prob;
    prd->passed = false;

    if(mats.opacity > 0.99f || rnd(prd->seed)<mats.opacity)
    {
        if (prd->curMatIdx > 0) {
          vec3 sigma_t, ss_alpha;
          //vec3 sigma_t, ss_alpha;
          prd->readMat(sigma_t, ss_alpha);
          if (ss_alpha.x < 0.0f) { // is inside Glass
            prd->attenuation *= DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
          } else {
            prd->attenuation *= DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
          }
        }
        prd->attenuation2 = prd->attenuation;
        prd->passed = true;
        prd->adepth++;
        //prd->samplePdf = 0.0f;
        //you shall pass!
        prd->radiance = make_float3(0.0f);
        prd->_tmin_ = optixGetRayTmax();
        prd->alphaHit = true;

        prd->prob *= 1;
        prd->countEmitted = false;
        return;
    }

    if(prd->depth==0&&mats.flatness>0.5)
    {
        prd->radiance = make_float3(0.0f);
        prd->done = true;
        return;
    }
    
    float is_refl;
    float3 inDir = ray_dir;
    vec3 wi = vec3(0.0f);
    float pdf = 0.0f;
    float rPdf = 0.0f;
    float fPdf = 0.0f;
    float rrPdf = 0.0f;

    float3 T = attrs.tang;
    float3 B;
    if(length(T)>0)
    {
        B = cross(N, T);
    } else
    {
        Onb a(N);
        T = a.m_tangent;
        B = a.m_binormal;
    }

    DisneyBSDF::SurfaceEventFlags flag;
    DisneyBSDF::PhaseFunctions phaseFuncion;
    vec3 extinction;
    vec3 reflectance = vec3(0.0f);
    bool isDiff = false;
    bool isSS = false;
    bool isTrans = false;
    flag = DisneyBSDF::scatterEvent;

    //sssColor = mix(basecolor, sssColor, subsurface);

    while(DisneyBSDF::SampleDisney2(
                prd->seed,
                prd->eventseed,
                mats,
                T,
                B,
                N,
                prd->geometryNormal,
                -normalize(ray_dir),
                mats.thin>0.5f,
                next_ray_is_going_inside,
                wi,
                reflectance,
                rPdf,
                fPdf,
                flag,
                prd->medium,
                extinction,
                isDiff,
                isSS,
                isTrans,
                prd->minSpecRough
                )  == false)
        {
            isSS = false;
            isDiff = false;
            prd->samplePdf = fPdf;
            reflectance = fPdf>0?reflectance/fPdf:vec3(0.0f);
            prd->done = fPdf>0?true:prd->done;
            flag = DisneyBSDF::scatterEvent;
        }
        
    prd->samplePdf = fPdf;
    reflectance = fPdf>0?reflectance/fPdf:vec3(0.0f);
    prd->done = fPdf>0?prd->done:true;
    prd->isSS = isSS;
    pdf = 1.0;
    if(isDiff || prd->diffDepth>0){
        prd->diffDepth++;
    }


    prd->passed = false;
    bool inToOut = false;
    bool outToIn = false;

    bool istransmission = dot(vec3(prd->geometryNormal), vec3(wi)) * dot(vec3(prd->geometryNormal), vec3(-normalize(ray_dir)))<0;
    //istransmission = (istransmission && thin<0.5 && mats.doubleSide==false);
    if(istransmission || flag == DisneyBSDF::diracEvent) {
    //if(flag == DisneyBSDF::transmissionEvent || flag == DisneyBSDF::diracEvent) {
        next_ray_is_going_inside = dot(vec3(prd->geometryNormal),vec3(wi))<=0;
    }
    prd->max_depth = ((prd->depth==0 && isSS) || (prd->depth>0 && (mats.specTrans>0||mats.isHair>0)) )?16:prd->max_depth;
    if(mats.thin>0.5f || mats.doubleSide>0.5f)
    {
        if (prd->curMatIdx > 0) {
            vec3 sigma_t, ss_alpha;
            prd->readMat(sigma_t, ss_alpha);

            vec3 trans;
            if (ss_alpha.x<0.0f) { // is inside Glass
                trans = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
            } else {
                trans = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
            }
            prd->attenuation *= trans;
            prd->attenuation2 *= trans;
        }

        next_ray_is_going_inside = false;

    }else{
    
        //if(flag == DisneyBSDF::transmissionEvent || flag == DisneyBSDF::diracEvent) {
        if(istransmission || flag == DisneyBSDF::diracEvent) {
            if(next_ray_is_going_inside){

                    outToIn = true;
                    inToOut = false;

                    prd->medium = DisneyBSDF::PhaseFunctions::isotropic;

                    if (prd->curMatIdx > 0) {
                        vec3 sigma_t, ss_alpha;
                        //vec3 sigma_t, ss_alpha;0
                        prd->readMat(sigma_t, ss_alpha);
                        if (ss_alpha.x < 0.0f) { // is inside Glass
                            prd->attenuation *= DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
                        } else {
                            prd->attenuation *= DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                        }
                    }
                    prd->channelPDF = vec3(1.0f/3.0f);
                    if (isTrans) {
                        vec3 channelPDF = vec3(1.0f/3.0f);
                        prd->pushMat(extinction);
                        prd->isSS = false;
                        prd->scatterDistance = mats.scatterDistance;
                        prd->maxDistance = mats.scatterStep>0.5f? DisneyBSDF::SampleDistance(prd->seed, prd->scatterDistance) : 1e16f;
                    } else {

                        prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * prd->ss_alpha, prd->sigma_t, prd->channelPDF);
                        //here is the place caused inf ray:fixed
                        auto min_sg = fmax(fmin(fmin(prd->sigma_t.x, prd->sigma_t.y), prd->sigma_t.z), 1e-8f);
                        //what should be the right value???
                        //prd->maxDistance = max(prd->maxDistance, 10/min_sg);
                        //printf("maxdist:%f\n",prd->maxDistance);
                        
                        // already calculated in BxDF
                        prd->pushMat(prd->sigma_t, prd->ss_alpha);
                        prd->isSS = true;
                        prd->scatterDistance = mats.scatterDistance;
                    }


                    prd->scatterStep = mats.scatterStep;
            }
            else{
                outToIn = false;
                inToOut = true;

                float3 trans;
                vec3 sigma_t, ss_alpha;
                prd->readMat(sigma_t, ss_alpha);
                if(prd->curMatIdx==0)
                { 
                    trans = vec3(1.0f); 
                }
                else if (ss_alpha.x<0.0f) { // Glass
                    trans = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
                } else {
                    trans = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                }

                prd->attenuation *= trans;
                prd->attenuation2 *= trans;
                
                prd->popMat(sigma_t, ss_alpha);

                prd->medium = (prd->curMatIdx==0)? DisneyBSDF::PhaseFunctions::vacuum : DisneyBSDF::PhaseFunctions::isotropic;

                if(ss_alpha.x < 0.0f) 
                {
                    prd->isSS = false;
                    prd->maxDistance = 1e16;
                }
                else //next ray in 3s object
                {
                    prd->isSS = true;
                    prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * ss_alpha, sigma_t, prd->channelPDF);
                }
            }
        }else{
            if(prd->medium == DisneyBSDF::PhaseFunctions::isotropic){
                    vec3 trans = vec3(1.0f);
                    vec3 sigma_t, ss_alpha;
                    prd->readMat(sigma_t, ss_alpha);
                    prd->isSS = false;
                    if(prd->curMatIdx==0)
                    {
                        prd->maxDistance = 1e16f;
                    }
                    else if (ss_alpha.x<0.0f) { // Glass
                        trans = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
                        vec3 channelPDF = vec3(1.0f/3.0f);
                        prd->maxDistance = mats.scatterStep>0.5f? DisneyBSDF::SampleDistance2(prd->seed, sigma_t, sigma_t, channelPDF) : 1e16f;
                    } else { // SSS
                        trans = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                        prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * ss_alpha, sigma_t, prd->channelPDF);
                        prd->isSS = true;
                    }

                    prd->attenuation2 *= trans;
                    prd->attenuation *= trans;
            }
            else
            {
                prd->isSS = false;
                prd->medium = DisneyBSDF::PhaseFunctions::vacuum;
                prd->channelPDF = vec3(1.0f/3.0f);
                prd->maxDistance = 1e16f;
            }
        }
    }

    prd->medium = next_ray_is_going_inside?DisneyBSDF::PhaseFunctions::isotropic : prd->curMatIdx==0?DisneyBSDF::PhaseFunctions::vacuum : DisneyBSDF::PhaseFunctions::isotropic;
 

//    if(mats.thin>0.5f){
//        vec3 H = normalize(vec3(normalize(wi)) + vec3(-normalize(ray_dir)));
//        attrs.N = N;
//        attrs.T = cross(B,N);
//        attrs.L = vec3(normalize(wi));
//        attrs.V = vec3(-normalize(ray_dir));
//        attrs.H = normalize(H);
//        attrs.reflectance = reflectance;
//        attrs.fresnel = DisneyBSDF::DisneyFresnel(mats.basecolor, mats.metallic, mats.ior, mats.specularTint, dot(attrs.H, attrs.V), dot(attrs.H, attrs.L), false);
//        MatOutput mat2 = evalReflectance(zenotex, rt_data->uniforms, attrs);
//        reflectance = mat2.reflectance;
//    }


    prd->countEmitted = false;
    prd->attenuation *= reflectance;
    if(mats.subsurface>0 && (mats.thin>0.5 || mats.doubleSide>0.5) && istransmission){
      prd->attenuation2 *= reflectance;
    }
    prd->depth++;

    if(prd->depth>=3)
        mats.roughness = clamp(mats.roughness, 0.5f,0.99f);

    auto evalBxDF = [&](const float3& _wi_, const float3& _wo_, float& thisPDF) -> float3 {

        const auto& L = _wi_; // pre-normalized
        const vec3& V = _wo_; // pre-normalized
        vec3 rd, rs, rt; // captured by lambda

        float3 lbrdf = DisneyBSDF::EvaluateDisney2(vec3(1.0f), mats, L, V, T, B, N,prd->geometryNormal,
            mats.thin > 0.5f, flag == DisneyBSDF::transmissionEvent ? inToOut : next_ray_is_going_inside, thisPDF, rrPdf,
            dot(N, L), rd, rs, rt);

        prd->radiance_d = rd;
        prd->radiance_s = rs;
        prd->radiance_t = rt;
//        MatOutput mat2;
//        if(mats.thin>0.5f){
//            vec3 H = normalize(vec3(normalize(L)) + V);
//            attrs.N = N;
//            attrs.T = cross(B,N);
//            attrs.L = vec3(normalize(L));
//            attrs.V = V;
//            attrs.H = normalize(H);
//            attrs.reflectance = lbrdf;
//            attrs.fresnel = DisneyBSDF::DisneyFresnel( mats.basecolor, mats.metallic, mats.ior, mats.specularTint, dot(attrs.H, attrs.V), dot(attrs.H, attrs.L), false);
//            mat2 = evalReflectance(zenotex, rt_data->uniforms, attrs);
//        }

        return lbrdf;

    };

    auto taskAux = [&](const vec3& radiance) {
        prd->radiance_d *= radiance;
        prd->radiance_s *= radiance;
        prd->radiance_t *= radiance;
    };

    ShadowPRD shadowPRD {};
    shadowPRD.seed = prd->seed;
    shadowPRD.attanuation = make_float3(1.0f, 1.0f, 1.0f);
    shadowPRD.nonThinTransHit = (mats.thin < 0.5f && mats.specTrans > 0) ? 1 : 0;

    float3 frontPos, backPos;
    SelfIntersectionAvoidance::offsetSpawnPoint( frontPos, backPos, wldPos, prd->geometryNormal, wldOffset );

    shadowPRD.origin = dot(-ray_dir, wldNorm) > 0 ? frontPos : backPos;
    //auto shadingP = rtgems::offset_ray(shadowPRD.origin + params.cam.eye,  prd->geometryNormal); // world space
    
    shadowPRD.origin = frontPos;
    if(mats.subsurface>0 && (mats.thin>0.5 || mats.doubleSide>0.5) && istransmission){
        shadowPRD.origin = backPos; //rtgems::offset_ray(P,  -prd->geometryNormal);
    }
    
    auto shadingP = rtgems::offset_ray(P + params.cam.eye,  prd->geometryNormal); // world space
    if(mats.subsurface>0 && (mats.thin>0.5 || mats.doubleSide>0.5) && istransmission){
        shadingP = rtgems::offset_ray(P + params.cam.eye,  -prd->geometryNormal);
    }

    prd->radiance = {};
    prd->direction = normalize(wi);
    prd->origin = dot(prd->direction, wldNorm) > 0 ? frontPos : backPos;


    float3 radianceNoShadow = {};
    float3* dummy_prt = nullptr;
    if (mats.shadowReceiver > 0.5f) {
        dummy_prt = &radianceNoShadow;
    }

    prd->lightmask = DefaultMatMask;
    DirectLighting<true>(prd, shadowPRD, shadingP, ray_dir, evalBxDF, &taskAux, dummy_prt);
    if(mats.shadowReceiver > 0.5f)
    {
      auto radiance = length(prd->radiance);
      prd->radiance.x = radiance;//the light contribution received with shadow attenuation
      prd->radiance.y = length(radianceNoShadow);
      prd->radiance.z = 0;
      prd->done = true;
    }

    prd->direction = normalize(wi);

    if(mats.thin<0.5f && mats.doubleSide<0.5f){
        //auto p_prim = vec3(prd->origin) + optixGetRayTmax() * vec3(prd->direction);
        //float3 p = p_prim;
        prd->origin = next_ray_is_going_inside? backPos : frontPos;
    }
    else {
        //auto p_prim = vec3(prd->origin) + optixGetRayTmax() * vec3(prd->direction);
        //float3 p = p_prim;
        prd->origin = dot(prd->direction, prd->geometryNormal) < 0? backPos : frontPos;
    }

    if (prd->medium != DisneyBSDF::vacuum) {
        prd->_mask_ = (uint8_t)(EverythingMask ^ VolumeMatMask);
    } else {
        prd->_mask_ = EverythingMask;
    }

    prd->radiance += mats.emission;
    if(length(mats.emission)>0)
    {
      prd->done = true;
    }
}

extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion( true );
}
