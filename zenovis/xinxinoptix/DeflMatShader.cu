#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <sutil/vec_math.h>
#include <cuda/helpers.h>
#include "optixPathTracer.h"
#include "TraceStuff.h"
#include "zxxglslvec.h"
#include "DisneyBRDF.h"
#include "DisneyBSDF.h"
#include "IOMat.h"

//COMMON_CODE

template<bool isDisplacement>
static __inline__ __device__ MatOutput evalMat(hipTextureObject_t zenotex[], float4* uniforms, MatInput const &attrs) {

    /* MODMA */
    auto att_pos = attrs.pos;
    auto att_clr = attrs.clr;
    auto att_uv = attrs.uv;
    auto att_nrm = attrs.nrm;
    auto att_tang = attrs.tang;
    auto att_instPos = attrs.instPos;
    auto att_instNrm = attrs.instNrm;
    auto att_instUv = attrs.instUv;
    auto att_instClr = attrs.instClr;
    auto att_instTang = attrs.instTang;
    /** generated code here beg **/
    //GENERATED_BEGIN_MARK
    /* MODME */
    float mat_base = 1.0;
    vec3 mat_basecolor = vec3(1.0, 1.0, 1.0);
    float mat_metallic = 0.0;
    float mat_roughness = 0.5;
    float mat_subsurface = 0.0;
    float mat_specular = 0;
    float mat_specularTint = 0.0;
    float mat_anisotropic = 0.0;
    float mat_anisoRotation = 0.0;
    float mat_sheen = 0.0;
    float mat_sheenTint = 0.0;
    float mat_clearcoat = 0.0;
    float mat_clearcoatGloss = 0.0;
    float mat_clearcoatRoughness = 0.0;
    float mat_clearcoatIOR = 1.5;
    float mat_opacity = 0.0;
    float mat_specTrans = 0.0;
    float mat_ior = 1.0;
    float mat_scatterDistance = 0.0;
    float mat_flatness = 0.0;
    float mat_thin = 0.0;
    float mat_doubleSide= 0.0;
    float mat_scatterStep = 0.0f;
    float mat_smoothness = 0.0f;
    vec3  mat_sssColor = vec3(0.0f,0.0f,0.0f);
    vec3  mat_sssParam = vec3(0.0f,0.0f,0.0f);
    vec3  mat_normal = vec3(0.0f, 0.0f, 1.0f);
    float mat_emissionIntensity = float(0);
    vec3 mat_emission = vec3(1.0f, 1.0f, 1.0f);
    float mat_displacement = 0.0f;
    //GENERATED_END_MARK
    /** generated code here end **/
    MatOutput mats;
    if constexpr(isDisplacement)
    {
        mats.displacement = mat_displacement;
        return mats;
    }else {
        /* MODME */
        mats.basecolor = mat_base * mat_basecolor;
        mats.metallic = clamp(mat_metallic, 0.0f, 1.0f);
        mats.roughness = clamp(mat_roughness, 0.01, 0.99);
        mats.subsurface = mat_subsurface;
        mats.specular = mat_specular;
        mats.specularTint = mat_specularTint;
        mats.anisotropic = clamp(mat_anisotropic, 0.0f, 1.0f);
        mats.anisoRotation = clamp(mat_anisoRotation, 0.0f, 1.0f);
        mats.sheen = mat_sheen;
        mats.sheenTint = mat_sheenTint;
        mats.clearcoat = clamp(mat_clearcoat, 0.0f, 1.0f);
        mats.clearcoatGloss = mat_clearcoatGloss;
        mats.clearcoatRoughness = clamp(mat_clearcoatRoughness, 0.01, 0.99);
        mats.clearcoatIOR = mat_clearcoatIOR;
        mats.opacity = mat_opacity;
        mats.nrm = mat_normal;
        mats.emission = mat_emissionIntensity * mat_emission;
        mats.specTrans = clamp(mat_specTrans, 0.0f, 1.0f);
        mats.ior = mat_ior;
        mats.scatterDistance = mat_scatterDistance;
        mats.flatness = mat_flatness;
        mats.thin = mat_thin;
        mats.doubleSide = mat_doubleSide;
        mats.sssColor = mat_sssColor;
        mats.sssParam = mat_sssParam;
        mats.scatterStep = mat_scatterStep;
        mats.smoothness = mat_smoothness;
        return mats;
    }
}

static __inline__ __device__ MatOutput evalMaterial(hipTextureObject_t zenotex[], float4* uniforms, MatInput const &attrs)
{
    return evalMat<false>(zenotex, uniforms, attrs);
}

static __inline__ __device__ MatOutput evalGeometry(hipTextureObject_t zenotex[], float4* uniforms, MatInput const &attrs)
{
    return evalMat<true>(zenotex, uniforms, attrs);
}


__forceinline__ __device__ float3 interp(float2 barys, float3 a, float3 b, float3 c)
{
    float w0 = 1 - barys.x - barys.y;
    float w1 = barys.x;
    float w2 = barys.y;
    return w0*a + w1*b + w2*c;
}

extern "C" __global__ void __anyhit__shadow_cutout()
{
    RadiancePRD* prd = getPRD();
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    const int    prim_idx        = optixGetPrimitiveIndex();
    const float3 ray_dir         = optixGetWorldRayDirection();

    int inst_idx2 = optixGetInstanceIndex();
    int inst_idx = rt_data->meshIdxs[inst_idx2];
    int vert_idx_offset = (inst_idx * 1024 + prim_idx)*3;

    float* meshMats = rt_data->meshMats;
    mat4 meshMat = mat4(
        meshMats[16 * inst_idx2 + 0], meshMats[16 * inst_idx2 + 1], meshMats[16 * inst_idx2 + 2], meshMats[16 * inst_idx2 + 3],
        meshMats[16 * inst_idx2 + 4], meshMats[16 * inst_idx2 + 5], meshMats[16 * inst_idx2 + 6], meshMats[16 * inst_idx2 + 7],
        meshMats[16 * inst_idx2 + 8], meshMats[16 * inst_idx2 + 9], meshMats[16 * inst_idx2 + 10], meshMats[16 * inst_idx2 + 11],
        meshMats[16 * inst_idx2 + 12], meshMats[16 * inst_idx2 + 13], meshMats[16 * inst_idx2 + 14], meshMats[16 * inst_idx2 + 15]);
    float3 av0 = make_float3(rt_data->vertices[vert_idx_offset + 0]);
    float3 av1 = make_float3(rt_data->vertices[vert_idx_offset + 1]);
    float3 av2 = make_float3(rt_data->vertices[vert_idx_offset + 2]);
    vec4 bv0 = vec4(av0.x, av0.y, av0.z, 1);
    vec4 bv1 = vec4(av1.x, av1.y, av1.z, 1);
    vec4 bv2 = vec4(av2.x, av2.y, av2.z, 1);
    bv0 = meshMat * bv0;
    bv1 = meshMat * bv1;
    bv2 = meshMat * bv2;
    float3 v0 = make_float3(bv0.x, bv0.y, bv0.z);
    float3 v1 = make_float3(bv1.x, bv1.y, bv1.z);
    float3 v2 = make_float3(bv2.x, bv2.y, bv2.z);

    float3 N_0  = normalize( cross( v1-v0, v2-v0 ) );
    
    const float3 P    = optixGetWorldRayOrigin() + optixGetRayTmax()*ray_dir;

    float w = rt_data->vertices[ vert_idx_offset+0 ].w;
    
    auto zenotex = rt_data->textures;

    MatInput attrs;
    /* MODMA */
    float2       barys    = optixGetTriangleBarycentrics();
    
    mat3 meshMat3x3(meshMat);
    float3 an0 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+0 ] ));
    vec3 bn0(an0);
    bn0 = meshMat3x3 * bn0;
    float3 n0 = make_float3(bn0.x, bn0.y, bn0.z);
    n0 = dot(n0, N_0)>0.8?n0:N_0;

    float3 an1 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+1 ] ));
    vec3 bn1(an1);
    bn1 = meshMat3x3 * bn1;
    float3 n1 = make_float3(bn1.x, bn1.y, bn1.z);
    n1 = dot(n1, N_0)>0.8?n1:N_0;

    float3 an2 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+2 ] ));
    vec3 bn2(an2);
    bn2 = meshMat3x3 * bn2;
    float3 n2 = make_float3(bn2.x, bn2.y, bn2.z);
    n2 = dot(n2, N_0)>0.8?n2:N_0;
    float3 uv0 = make_float3(rt_data->uv[ vert_idx_offset+0 ] );
    float3 uv1 = make_float3(rt_data->uv[ vert_idx_offset+1 ] );
    float3 uv2 = make_float3(rt_data->uv[ vert_idx_offset+2 ] );
    float3 clr0 = make_float3(rt_data->clr[ vert_idx_offset+0 ] );
    float3 clr1 = make_float3(rt_data->clr[ vert_idx_offset+1 ] );
    float3 clr2 = make_float3(rt_data->clr[ vert_idx_offset+2 ] );
    float3 atan0 = make_float3(rt_data->tan[ vert_idx_offset+0 ] );
    float3 atan1 = make_float3(rt_data->tan[ vert_idx_offset+1 ] );
    float3 atan2 = make_float3(rt_data->tan[ vert_idx_offset+2 ] );
    vec3 btan0(atan0);
    vec3 btan1(atan1);
    vec3 btan2(atan2);
    btan0 = meshMat3x3 * btan0;
    btan1 = meshMat3x3 * btan1;
    btan2 = meshMat3x3 * btan2;
    float3 tan0 = make_float3(btan0.x, btan0.y, btan0.z);
    float3 tan1 = make_float3(btan1.x, btan1.y, btan1.z);
    float3 tan2 = make_float3(btan2.x, btan2.y, btan2.z);
    
    N_0 = normalize(interp(barys, n0, n1, n2));
    float3 N = faceforward( N_0, -ray_dir, N_0 );

    attrs.pos = vec3(P.x, P.y, P.z);
    attrs.nrm = N;
    attrs.uv = interp(barys, uv0, uv1, uv2);//todo later
    //attrs.clr = rt_data->face_attrib_clr[vert_idx_offset];
    attrs.clr = interp(barys, clr0, clr1, clr2);
    attrs.tang = interp(barys, tan0, tan1, tan2);
    attrs.instPos = rt_data->instPos[inst_idx2];
    attrs.instNrm = rt_data->instNrm[inst_idx2];
    attrs.instUv = rt_data->instUv[inst_idx2];
    attrs.instClr = rt_data->instClr[inst_idx2];
    attrs.instTang = rt_data->instTang[inst_idx2];
    MatOutput mats = evalMaterial(zenotex, rt_data->uniforms, attrs);

    if(length(attrs.tang)>0)
    {
        vec3 b = cross(attrs.tang, attrs.nrm);
        attrs.tang = cross(attrs.nrm, b);
        N = mats.nrm.x * attrs.tang + mats.nrm.y * b + mats.nrm.z * attrs.nrm;
    }
    //end of material computation
    //mats.metallic = clamp(mats.metallic,0.01, 0.99);
    mats.roughness = clamp(mats.roughness, 0.01,0.99);

    /* MODME */
    auto basecolor = mats.basecolor;
    auto metallic = mats.metallic;
    auto roughness = mats.roughness;
    auto subsurface = mats.subsurface;
    auto specular = mats.specular;
    auto specularTint = mats.specularTint;
    auto anisotropic = mats.anisotropic;
    auto sheen = mats.sheen;
    auto sheenTint = mats.sheenTint;
    auto clearcoat = mats.clearcoat;
    auto clearcoatGloss = mats.clearcoatGloss;
    auto opacity = mats.opacity;
    auto flatness = mats.flatness;
    auto specTrans = mats.specTrans;
    auto scatterDistance = mats.scatterDistance;
    auto ior = mats.ior;
    auto thin = mats.thin;
    auto doubleSide = mats.doubleSide;
    auto sssParam = mats.sssParam;
    auto scatterStep = mats.scatterStep;
    unsigned short isLight = rt_data->lightMark[inst_idx * 1024 + prim_idx];
    if(params.simpleRender==true)
        opacity = 0;
    //opacity = clamp(opacity, 0.0f, 0.99f);
    // Stochastic alpha test to get an alpha blend effect.
    if (opacity >0.99 || isLight == 1) // No need to calculate an expensive random number if the test is going to fail anyway.
    {
        optixIgnoreIntersection();
    }
    else
    {

        //roll a dice
        float p = rnd(prd->seed);
        if (p < opacity){
            optixIgnoreIntersection();
        }else{
            if(length(prd->shadowAttanuation) < 0.01){
                prd->shadowAttanuation = vec3(0.0f);
                optixTerminateRay();
                return;
            }
            if(specTrans==0.0f){
                prd->shadowAttanuation = vec3(0.0f);
                optixTerminateRay();
                return;
            }
            //prd->shadowAttanuation = vec3(0,0,0);
            //optixTerminateRay();
            
            if(specTrans > 0.0f){
                if(thin == 0.0f && ior>1.0f)
                {
                    prd->nonThinTransHit++;
                }
                if(rnd(prd->seed)<(1-specTrans)||prd->nonThinTransHit>1)
                {
                    prd->shadowAttanuation = vec3(0,0,0);
                    optixTerminateRay();
                    return;
                }
                float nDi = fabs(dot(N,ray_dir));
                vec3 tmp = prd->shadowAttanuation;
                tmp = tmp * (vec3(1)-BRDFBasics::fresnelSchlick(vec3(1)-basecolor,nDi));
                prd->shadowAttanuation = tmp;

                optixIgnoreIntersection();
            }
        }

        prd->shadowAttanuation = vec3(0);
        optixTerminateRay();
        return;
    }
}

static __inline__ __device__
int GetLightIndex(float p, ParallelogramLight* lightP, int n)
{
    int s = 0, e = n-1;
    while( s < e )
    {
        int j = (s+e)/2;
        float pc = lightP[j].cdf/lightP[n-1].cdf;
        if(pc<p)
        {
            s = j+1;
        }
        else
        {
            e = j;
        }
    }
    return e;
}
static __inline__ __device__
vec3 projectedBarycentricCoord(vec3 p, vec3 q, vec3 u, vec3 v)
{
    vec3 n = cross(u,v);
    float a = 1.0 / dot(n,n);
    vec3 w = p - q;
    vec3 o;
    o.z = dot(cross(u,w),n) * a;
    o.y = dot(cross(w,v),n) * a;
    o.x = 1.0 - o.y - o.z;
    return o;
}

extern "C" __global__ void __closesthit__radiance()
{
    RadiancePRD* prd = getPRD();

    if(prd->test_distance)
    {
        prd->vol_t1 = optixGetRayTmax();
        return;
    }
    prd->test_distance = false;

    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    int    prim_idx        = optixGetPrimitiveIndex();
    float3 ray_dir         = optixGetWorldRayDirection();

    int inst_idx2 = optixGetInstanceIndex();
    int inst_idx = rt_data->meshIdxs[inst_idx2];
    int vert_idx_offset = (inst_idx * 1024 + prim_idx)*3;

    float* meshMats = rt_data->meshMats;
    mat4 meshMat = mat4(
        meshMats[16 * inst_idx2 + 0], meshMats[16 * inst_idx2 + 1], meshMats[16 * inst_idx2 + 2], meshMats[16 * inst_idx2 + 3],
        meshMats[16 * inst_idx2 + 4], meshMats[16 * inst_idx2 + 5], meshMats[16 * inst_idx2 + 6], meshMats[16 * inst_idx2 + 7],
        meshMats[16 * inst_idx2 + 8], meshMats[16 * inst_idx2 + 9], meshMats[16 * inst_idx2 + 10], meshMats[16 * inst_idx2 + 11],
        meshMats[16 * inst_idx2 + 12], meshMats[16 * inst_idx2 + 13], meshMats[16 * inst_idx2 + 14], meshMats[16 * inst_idx2 + 15]);
    float3 av0 = make_float3(rt_data->vertices[vert_idx_offset + 0]);
    float3 av1 = make_float3(rt_data->vertices[vert_idx_offset + 1]);
    float3 av2 = make_float3(rt_data->vertices[vert_idx_offset + 2]);
    vec4 bv0 = vec4(av0.x, av0.y, av0.z, 1);
    vec4 bv1 = vec4(av1.x, av1.y, av1.z, 1);
    vec4 bv2 = vec4(av2.x, av2.y, av2.z, 1);
    bv0 = meshMat * bv0;
    bv1 = meshMat * bv1;
    bv2 = meshMat * bv2;
    float3 v0 = make_float3(bv0.x, bv0.y, bv0.z);
    float3 v1 = make_float3(bv1.x, bv1.y, bv1.z);
    float3 v2 = make_float3(bv2.x, bv2.y, bv2.z);

    float3 N_0  = normalize( cross( v1-v0, v2-v1 ) );
        prd->geometryNormal = N_0;

    float3 P    = optixGetWorldRayOrigin() + optixGetRayTmax()*ray_dir;
    unsigned short isLight = rt_data->lightMark[inst_idx * 1024 + prim_idx];
    float w = rt_data->vertices[ vert_idx_offset+0 ].w;

    auto zenotex = rt_data->textures;

    MatInput attrs;
    /* MODMA */
    float2       barys    = optixGetTriangleBarycentrics();
    
//    float3 n0 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+0 ] ));
//
//    float3 n1 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+1 ] ));
//
//    float3 n2 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+2 ] ));

    float3 uv0 = make_float3(rt_data->uv[ vert_idx_offset+0 ] );
    float3 uv1 = make_float3(rt_data->uv[ vert_idx_offset+1 ] );
    float3 uv2 = make_float3(rt_data->uv[ vert_idx_offset+2 ] );
    float3 clr0 = make_float3(rt_data->clr[ vert_idx_offset+0 ] );
    float3 clr1 = make_float3(rt_data->clr[ vert_idx_offset+1 ] );
    float3 clr2 = make_float3(rt_data->clr[ vert_idx_offset+2 ] );
    float3 atan0 = make_float3(rt_data->tan[ vert_idx_offset+0 ] );
    float3 atan1 = make_float3(rt_data->tan[ vert_idx_offset+1 ] );
    float3 atan2 = make_float3(rt_data->tan[ vert_idx_offset+2 ] );
    mat3 meshMat3x3(meshMat);
    vec3 btan0(atan0);
    vec3 btan1(atan1);
    vec3 btan2(atan2);
    btan0 = meshMat3x3 * btan0;
    btan1 = meshMat3x3 * btan1;
    btan2 = meshMat3x3 * btan2;
    float3 tan0 = make_float3(btan0.x, btan0.y, btan0.z);
    float3 tan1 = make_float3(btan1.x, btan1.y, btan1.z);
    float3 tan2 = make_float3(btan2.x, btan2.y, btan2.z);
    
    //N_0 = normalize(interp(barys, n0, n1, n2));
    float3 N = N_0;//faceforward( N_0, -ray_dir, N_0 );
    P = interp(barys, v0, v1, v2);
    attrs.pos = vec3(P.x, P.y, P.z);
    attrs.nrm = N;
    attrs.uv = interp(barys, uv0, uv1, uv2);//todo later
    //attrs.clr = rt_data->face_attrib_clr[vert_idx_offset];
    attrs.clr = interp(barys, clr0, clr1, clr2);
    attrs.tang = normalize(interp(barys, tan0, tan1, tan2));
    attrs.instPos = rt_data->instPos[inst_idx2];
    attrs.instNrm = rt_data->instNrm[inst_idx2];
    attrs.instUv = rt_data->instUv[inst_idx2];
    attrs.instClr = rt_data->instClr[inst_idx2];
    attrs.instTang = rt_data->instTang[inst_idx2];

    MatOutput mats = evalMaterial(zenotex, rt_data->uniforms, attrs);
    
    float3 an0 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+0 ] ));
    vec3 bn0(an0);
    bn0 = meshMat3x3 * bn0;
    float3 n0 = make_float3(bn0.x, bn0.y, bn0.z);
    n0 = dot(n0, N_0)>(1-mats.smoothness)?n0:N_0;

    float3 an1 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+1 ] ));
    vec3 bn1(an1);
    bn1 = meshMat3x3 * bn1;
    float3 n1 = make_float3(bn1.x, bn1.y, bn1.z);
    n1 = dot(n1, N_0)>(1-mats.smoothness)?n1:N_0;

    float3 an2 = normalize(make_float3(rt_data->nrm[ vert_idx_offset+2 ] ));
    vec3 bn2(an2);
    bn2 = meshMat3x3 * bn2;
    float3 n2 = make_float3(bn2.x, bn2.y, bn2.z);
    n2 = dot(n2, N_0)>(1-mats.smoothness)?n2:N_0;
    N_0 = normalize(interp(barys, n0, n1, n2));
    N = N_0;
    if(mats.doubleSide>0.5||mats.thin>0.5){
        N = faceforward( N_0, -ray_dir, N_0 );
        prd->geometryNormal = faceforward( prd->geometryNormal, -ray_dir, prd->geometryNormal );
    }
    attrs.nrm = N;
    //end of material computation
    //mats.metallic = clamp(mats.metallic,0.01, 0.99);
    mats.roughness = clamp(mats.roughness, 0.01,0.99);
    auto N2 = N;
    if(length(attrs.tang)>0)
    {
        vec3 b = cross(attrs.tang, attrs.nrm);
        attrs.tang = cross(attrs.nrm, b);
        N = mats.nrm.x * attrs.tang + mats.nrm.y * b + mats.nrm.z * attrs.nrm;
    }


    /* MODME */
    auto basecolor = mats.basecolor;
    auto metallic = mats.metallic;
    auto roughness = mats.roughness;
    if(prd->diffDepth>=1)
        roughness = clamp(roughness, 0.2,0.99);
    if(prd->diffDepth>=2)
        roughness = clamp(roughness, 0.3,0.99);
    if(prd->diffDepth>=3)
        roughness = clamp(roughness, 0.5,0.99);


    auto subsurface = mats.subsurface;
    auto specular = mats.specular;
    auto specularTint = mats.specularTint;
    auto anisotropic = mats.anisotropic;
    auto anisoRotation = mats.anisoRotation;
    auto sheen = mats.sheen;
    auto sheenTint = mats.sheenTint;
    auto clearcoat = mats.clearcoat;
    auto clearcoatGloss = mats.clearcoatGloss;
    auto ccRough = mats.clearcoatRoughness;
    auto ccIor = mats.clearcoatIOR;
    auto opacity = mats.opacity;
    auto flatness = mats.flatness;
    auto specTrans = mats.specTrans;
    auto scatterDistance = mats.scatterDistance;
    auto ior = mats.ior;
    auto thin = mats.thin;

    auto sssColor = mats.sssColor;
    auto sssParam = mats.sssParam;

    auto scatterStep = mats.scatterStep;
    //discard fully opacity pixels
    //opacity = clamp(opacity, 0.0f, 0.99f);
    prd->opacity = opacity;
    if(prd->isSS == true) {
        basecolor = vec3(1.0f);
        roughness = 1.0;
        anisotropic = 0;
        sheen = 0;
        clearcoat = 0;
        specTrans = 0;
        ior = 1;
    }

    if(prd->isSS == true  && subsurface==0 )
    {
        prd->passed = true;
        prd->radiance = make_float3(0.0f, 0.0f, 0.0f);
        prd->opacity = 0;
        prd->readMat(prd->sigma_t, prd->ss_alpha);
        auto trans = DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
        prd->attenuation2 *= trans;
        prd->attenuation *= trans;
        //prd->origin = P + 1e-5 * ray_dir; 
        if(prd->maxDistance>optixGetRayTmax())
            prd->maxDistance-=optixGetRayTmax();
        prd->offsetUpdateRay(P, ray_dir); 
        return;
    }

    prd->attenuation2 = prd->attenuation;
    prd->countEmitted = false;
    if(isLight==1)
    {
        prd->countEmitted = true;
        //hit light, emit
//        float dist = length(P - optixGetWorldRayOrigin()) + 1e-5;
//        float3 lv1 = v1-v0;
//        float3 lv2 = v2-v0;
//        float A = 0.5 * length(cross(lv1, lv2));
//        float3 lnrm = normalize(cross(normalize(lv1), normalize(lv2)));
//        float3 L     = normalize(P - optixGetWorldRayOrigin());
//        float  LnDl  = clamp(-dot( lnrm, L ), 0.0f, 1.0f);
//        float weight = LnDl * A / (M_PIf * dist);
//        prd->radiance = attrs.clr * weight;
        prd->offsetUpdateRay(P, ray_dir); 
        return;
    }
    prd->prob2 = prd->prob;
    prd->passed = false;
    if(opacity>0.99)
    {
        prd->passed = true;
        prd->radiance = make_float3(0.0f);
        //prd->origin = P + 1e-5 * ray_dir; 
        prd->offsetUpdateRay(P, ray_dir);
        return;
    }

    
    float is_refl;
    float3 inDir = ray_dir;
    vec3 wi = vec3(0.0f);
    float pdf = 0.0f;
    float rPdf = 0.0f;
    float fPdf = 0.0f;
    float rrPdf = 0.0f;
    float ffPdf = 0.0f;
    float3 T = attrs.tang;
    float3 B;
    if(length(T)>0)
    {
        B = cross(N, T);
    } else
    {
        Onb a(N);
        T = a.m_tangent;
        B = a.m_binormal;
    }

    DisneyBSDF::SurfaceEventFlags flag;
    DisneyBSDF::PhaseFunctions phaseFuncion;
    vec3 extinction;
    vec3 reflectance = vec3(0.0f);
    bool isDiff = false;
    bool isSS = false;
    bool isTrans = false;
    flag = DisneyBSDF::scatterEvent;

    //sssColor = mix(basecolor, sssColor, subsurface);

    while(DisneyBSDF::SampleDisney(
                prd->seed,
                basecolor,
                sssParam,
                sssColor,
                metallic,
                subsurface,
                specular,
                roughness,
                specularTint,
                anisotropic,
                anisoRotation,
                sheen,
                sheenTint,
                clearcoat,
                clearcoatGloss,
                ccRough,
                ccIor,
                flatness,
                specTrans,
                scatterDistance,
                ior,
                T,
                B,
                N,
                prd->geometryNormal,
                -normalize(ray_dir),
                thin>0.5f,
                prd->next_ray_is_going_inside,
                wi,
                reflectance,
                rPdf,
                fPdf,
                flag,
                prd->medium,
                extinction,
                isDiff,
                isSS,
                isTrans
                )  == false)
        {
            isSS = false;
            isDiff = false;
            rPdf = 0.0f;
            fPdf = 0.0f;
            reflectance = vec3(0.0f);
            flag = DisneyBSDF::scatterEvent;
        }
    prd->isSS = isSS;
    pdf = fPdf;
    if(isDiff || prd->diffDepth>0){
        prd->diffDepth++;
    }
    
    if(opacity<=0.99)
    {
        //we have some simple transparent thing
        //roll a dice to see if just pass
        if(rnd(prd->seed)<opacity)
        {
            prd->passed = true;
            //you shall pass!
            prd->radiance = make_float3(0.0f);

            prd->origin = P;
            prd->direction = ray_dir;
            prd->offsetUpdateRay(P, ray_dir); 

            prd->prob *= 1;
            prd->countEmitted = false;
            prd->attenuation *= 1;
            return;
        }
    }

    prd->passed = false;
    bool inToOut = false;
    bool outToIn = false;

    bool istransmission = dot(vec3(prd->geometryNormal), vec3(wi)) * dot(vec3(prd->geometryNormal), vec3(-normalize(ray_dir)))<0;
    //istransmission = (istransmission && thin<0.5 && mats.doubleSide==false);
    if(istransmission || flag == DisneyBSDF::diracEvent) {
    //if(flag == DisneyBSDF::transmissionEvent || flag == DisneyBSDF::diracEvent) {
        prd->next_ray_is_going_inside = dot(vec3(prd->geometryNormal),vec3(wi))<=0;
    }

    if(thin>0.5 || mats.doubleSide>0.5)
    {
        if (prd->curMatIdx > 0) {
            //vec3 sigma_t, ss_alpha;
            prd->readMat(prd->sigma_t, prd->ss_alpha);
            if (isTrans) { // Glass
                prd->attenuation *= DisneyBSDF::Transmission(prd->sigma_t, optixGetRayTmax());
                prd->attenuation2 *= DisneyBSDF::Transmission(prd->sigma_t, optixGetRayTmax());
            } else {
                prd->attenuation *= DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                prd->attenuation2 *= DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
            }
        }else {
            prd->attenuation *= 1;
        }
        prd->next_ray_is_going_inside = false;
    }else{
    
        //if(flag == DisneyBSDF::transmissionEvent || flag == DisneyBSDF::diracEvent) {
        if(istransmission || flag == DisneyBSDF::diracEvent) {
            if(prd->next_ray_is_going_inside){
                if(thin < 0.5 && mats.doubleSide < 0.5 ) 
                {
                    outToIn = true;
                    inToOut = false;

                    prd->medium = DisneyBSDF::PhaseFunctions::isotropic;

                    if (prd->curMatIdx > 0) {
                        //vec3 sigma_t, ss_alpha;
                        prd->readMat(prd->sigma_t, prd->ss_alpha);
                        if (isTrans) { // Glass
                            prd->attenuation *= DisneyBSDF::Transmission(prd->sigma_t, optixGetRayTmax());
                        } else {
                            prd->attenuation *= DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                        }
                    }

                    if (isTrans) {
                        prd->maxDistance = 1e16;
                        //printf("sigma_t: %f, %f, %f\n", extinction.x, extinction.y, extinction.z);
                        prd->pushMat(extinction);
                    } else {

                        vec3 channelPDF = vec3(1.0/3.0);
                        prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * prd->ss_alpha, prd->sigma_t, channelPDF);
                        //here is the place caused inf ray:fixed
                        auto min_sg = max(min(min(prd->sigma_t.x, prd->sigma_t.y), prd->sigma_t.z), 1e-8);
                        //what should be the right value???
                        //prd->maxDistance = max(prd->maxDistance, 10/min_sg);
                        //printf("maxdist:%f\n",prd->maxDistance);
                        prd->channelPDF = channelPDF;
                        // already calculated in BxDF
                        prd->pushMat(prd->sigma_t, prd->ss_alpha);
                    }

                    prd->scatterDistance = scatterDistance;
                    prd->scatterStep = scatterStep;
                }
                
            }
            else{
                outToIn = false;
                inToOut = true;

                float3 trans;
                prd->readMat(prd->sigma_t, prd->ss_alpha);
                if (isTrans) { // Glass
                
                    trans = DisneyBSDF::Transmission(prd->sigma_t, optixGetRayTmax());
                } else {
                    trans = DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                }

                prd->attenuation2 *= trans;
                prd->attenuation *= trans;

                prd->popMat(prd->sigma_t, prd->ss_alpha);

                prd->medium = (prd->curMatIdx==0)? DisneyBSDF::PhaseFunctions::vacuum : DisneyBSDF::PhaseFunctions::isotropic;

                // if (prd->medium != DisneyBSDF::PhaseFunctions::vacuum) {

                //     prd->bad = true;
                    
                //     printf("%f %f %f %f %f %f %f %f \n matIdx = %d isotropic = %d \n", prd->sigma_t_queue[0].x, prd->sigma_t_queue[1].x, prd->sigma_t_queue[2].x, prd->sigma_t_queue[3].x, prd->sigma_t_queue[4].x, prd->sigma_t_queue[5].x, prd->sigma_t_queue[6].x, prd->sigma_t_queue[7].x,
                //         prd->curMatIdx, prd->medium);
                //     printf("matIdx = %d isotropic = %d \n\n", prd->curMatIdx, prd->medium);
                // }
            }
        }else{
            if(prd->medium == DisneyBSDF::PhaseFunctions::isotropic){
                    vec3 trans;
                    prd->readMat(prd->sigma_t, prd->ss_alpha);
                    if (isTrans) { // Glass
                        trans = DisneyBSDF::Transmission(prd->sigma_t, optixGetRayTmax());
                        prd->maxDistance = 1e16;
                    } else { // SSS
                        trans = DisneyBSDF::Transmission2(prd->sigma_s(), prd->sigma_t, prd->channelPDF, optixGetRayTmax(), true);
                        prd->channelPDF = vec3(1.0/3.0);
                        prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * prd->ss_alpha, prd->sigma_t, prd->channelPDF);
                    }

                    prd->attenuation2 *= trans;
                    prd->attenuation *= trans;
            }
                else
                {
                    prd->medium = DisneyBSDF::PhaseFunctions::vacuum;
                    prd->channelPDF = vec3(1.0f/3.0f);
                    prd->maxDistance = 1e16f;
                }
        }
    }
    prd->medium = prd->next_ray_is_going_inside?DisneyBSDF::PhaseFunctions::isotropic : prd->curMatIdx==0?DisneyBSDF::PhaseFunctions::vacuum : DisneyBSDF::PhaseFunctions::isotropic;
 
    prd->countEmitted = false;
    prd->attenuation *= reflectance;
    prd->depth++;

    auto P_OLD = P;
    P = rtgems::offset_ray(P,  prd->geometryNormal);

    prd->radiance = make_float3(0.0f,0.0f,0.0f);
    float3 light_attenuation = make_float3(1.0f,1.0f,1.0f);
    float pl = rnd(prd->seed);
    int lidx = GetLightIndex(pl, params.lights, params.num_lights);
    float sum = 0.0f;
    for(int lidx=0;lidx<params.num_lights;lidx++)
    {
            ParallelogramLight light = params.lights[lidx];
            float3 light_pos = light.corner + light.v1 * 0.5 + light.v2 * 0.5;

            // Calculate properties of light sample (for area based pdf)
            float Ldist = length(light_pos - P);
            float3 L = normalize(light_pos - P);
            float nDl = 1.0f;//clamp(dot(N, L), 0.0f, 1.0f);
            float LnDl = clamp(-dot(light.normal, L), 0.000001f, 1.0f);
            float A = length(cross(params.lights[lidx].v1, params.lights[lidx].v2));
            sum += length(light.emission)  * nDl * LnDl * A / (M_PIf * Ldist * Ldist );

    }
    if(prd->depth>=3)
        roughness = clamp(roughness, 0.5,0.99);

    RadiancePRD shadow_prd {};
    shadow_prd.seed = prd->seed;
    shadow_prd.shadowAttanuation = make_float3(1.0f, 1.0f, 1.0f);
    shadow_prd.nonThinTransHit = (thin == false && specTrans > 0) ? 1 : 0;

    if(rnd(prd->seed)<=0.5) {
        bool computed = false;
        float ppl = 0;
        for (int lidx = 0; lidx < params.num_lights && computed == false; lidx++) {
            ParallelogramLight light = params.lights[lidx];
            float2 z = {rnd(prd->seed), rnd(prd->seed)};
            const float z1 = z.x;
            const float z2 = z.y;
            float3 light_tpos = light.corner + light.v1 * 0.5 + light.v2 * 0.5;
            float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

            // Calculate properties of light sample (for area based pdf)
            float tLdist = length(light_tpos - P);
            float3 tL = normalize(light_tpos - P);
            float tnDl = 1.0f; //clamp(dot(N, tL), 0.0f, 1.0f);
            float tLnDl = clamp(-dot(light.normal, tL), 0.000001f, 1.0f);
            float tA = length(cross(params.lights[lidx].v1, params.lights[lidx].v2));
            ppl += length(light.emission) * tnDl * tLnDl * tA / (M_PIf * tLdist * tLdist) / sum;
            if (ppl > pl) {
                float Ldist = length(light_pos - P) + 1e-6;
                float3 L = normalize(light_pos - P);
                float nDl = 1.0f; //clamp(dot(N, L), 0.0f, 1.0f);
                float LnDl = clamp(-dot(light.normal, L), 0.0f, 1.0f);
                float A = length(cross(params.lights[lidx].v1, params.lights[lidx].v2));
                float weight = 0.0f;
                if (nDl > 0.0f && LnDl > 0.0f) {

                    traceOcclusion(params.handle, P, L,
                                   1e-5f,         // tmin
                                   Ldist - 1e-5f, // tmax,
                                   &shadow_prd);

                    light_attenuation = shadow_prd.shadowAttanuation;
                    if (fmaxf(light_attenuation) > 0.0f) {

                        weight = sum * nDl / tnDl * LnDl / tLnDl * (tLdist * tLdist) / (Ldist  * Ldist) /
                                 (length(light.emission)+1e-6f) ;
                    }
                }
                prd->LP = P;
                prd->Ldir = L;
                prd->nonThinTransHit = (thin == false && specTrans > 0) ? 1 : 0;
                prd->Lweight = weight;

                float3 lbrdf = DisneyBSDF::EvaluateDisney(
                    basecolor, metallic, subsurface, specular, roughness, specularTint, anisotropic, anisoRotation, sheen, sheenTint,
                    clearcoat, clearcoatGloss, ccRough, ccIor, specTrans, scatterDistance, ior, flatness, L, -normalize(inDir), T, B, N,
                    thin > 0.5f, flag == DisneyBSDF::transmissionEvent ? inToOut : prd->next_ray_is_going_inside, ffPdf, rrPdf,
                    dot(N, L));

                prd->radiance = light_attenuation * weight * 2.0 * light.emission * lbrdf;
                computed = true;
            }
        }
    } else {
    
        float3 lbrdf {};
        bool inside = false;

        vec3 sunLightDir = vec3(params.sunLightDirX, params.sunLightDirY, params.sunLightDirZ);
        auto sun_dir = BRDFBasics::halfPlaneSample(prd->seed, sunLightDir,
                                                   params.sunSoftness * 0.2); //perturb the sun to have some softness
        sun_dir = normalize(sun_dir);
        prd->LP = P;
        prd->Ldir = sun_dir;
        prd->nonThinTransHit = (thin == false && specTrans > 0) ? 1 : 0;
        prd->Lweight = 1.0;

        traceOcclusion(params.handle, P, sun_dir,
                       1e-5f, // tmin
                       1e16f, // tmax,
                       &shadow_prd);
        lbrdf = DisneyBSDF::EvaluateDisney(
            basecolor, metallic, subsurface, specular, roughness, specularTint, anisotropic, anisoRotation, sheen, sheenTint,
            clearcoat, clearcoatGloss, ccRough, ccIor, specTrans, scatterDistance, ior, flatness, sun_dir, -normalize(inDir), T, B, N,
            thin > 0.5f, flag == DisneyBSDF::transmissionEvent ? inToOut : prd->next_ray_is_going_inside, ffPdf, rrPdf,
            dot(N, float3(sun_dir)));
        light_attenuation = shadow_prd.shadowAttanuation;
        //if (fmaxf(light_attenuation) > 0.0f) {
            auto sky = float3(envSky(sun_dir, sunLightDir, make_float3(0., 0., 1.),
                                          10, // be careful
                                          .45, 15., 1.030725 * 0.3, params.elapsedTime));

            prd->radiance = light_attenuation * params.sunLightIntensity * 2.0 * sky * lbrdf;
    }

    P = P_OLD;
    prd->direction = normalize(wi);
    if(thin<0.5 && mats.doubleSide<0.5){
        prd->origin = rtgems::offset_ray(P, (prd->next_ray_is_going_inside)? -prd->geometryNormal : prd->geometryNormal);
    }
    else {
        prd->origin = rtgems::offset_ray(P, ( dot(prd->direction, prd->geometryNormal) <0 )? -prd->geometryNormal : prd->geometryNormal);
    }

    

    prd->radiance +=  float3(mats.emission);
    prd->CH = 1.0;
}

extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion( true );
}
