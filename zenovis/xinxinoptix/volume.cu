#include "hip/hip_runtime.h"
#include "volume.h"

#include "TraceStuff.h"

#include "DisneyBRDF.h"
#include "DisneyBSDF.h"

// #include <hip/hip_fp16.h>
// #include "nvfunctional"

#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/Ray.h>
#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/SampleFromVoxels.h>

//PLACEHOLDER
static const int   _vol_depth = 99;
static const float _vol_extinction = 1.0f;
//PLACEHOLDER

//COMMON_CODE

inline __device__ float _LERP_(float t, float s1, float s2)
{
    //return (1 - t) * s1 + t * s2;
    return fma(t, s2, fma(-t, s1, s1));
}

template <typename Acc, int Order>
inline __device__ float linearSampling(Acc& acc, nanovdb::Vec3f& point_indexd) {

        using Sampler = nanovdb::SampleFromVoxels<nanovdb::FloatGrid::AccessorType, Order, true>;
        return Sampler(acc)(point_indexd);

        //nanovdb::BaseStencil<typename DerivedType, int SIZE, typename GridT>
        //auto bs = nanovdb::BoxStencil<nanovdb::FloatGrid*>(grid);

    // auto point_floor = nanovdb::RoundDown<nanovdb::Vec3f>(point_indexd); 
    // auto point_a = nanovdb::Coord(point_floor[0], point_floor[1], point_floor[2]);

    //     auto value_000 = acc.getValue(point_a);
    //     auto value_100 = acc.getValue(point_a + nanovdb::Coord(1, 0, 0));
    //     auto value_010 = acc.getValue(point_a + nanovdb::Coord(0, 1, 0));
    //     auto value_110 = acc.getValue(point_a + nanovdb::Coord(1, 1, 0));
    //     auto value_001 = acc.getValue(point_a + nanovdb::Coord(0, 0, 1));
    //     auto value_101 = acc.getValue(point_a + nanovdb::Coord(1, 0, 1));
    //     auto value_011 = acc.getValue(point_a + nanovdb::Coord(0, 1, 1));
    //     auto value_111 = acc.getValue(point_a + nanovdb::Coord(1, 1, 1));

    // auto delta = point_indexd - point_floor; 

    //     auto value_00 = _LERP_(delta[0], value_000, value_100);
    //     auto value_10 = _LERP_(delta[0], value_010, value_110);
    //     auto value_01 = _LERP_(delta[0], value_001, value_101);
    //     auto value_11 = _LERP_(delta[0], value_011, value_111);
        
    //     auto value_0 = _LERP_(delta[1], value_00, value_10);
    //     auto value_1 = _LERP_(delta[1], value_01, value_11);

    // return _LERP_(delta[2], value_0, value_1);
}

struct VolumeIn {
    vec3 pos;

    vec3 _local_pos_ = vec3(HIP_NAN_F);
    __inline__ __device__ vec3 localPosLazy() {
        if ( isnan(_local_pos_.x) ) {

            const HitGroupData* sbt_data = reinterpret_cast<HitGroupData*>( optixGetSbtDataPointer() );

            const auto grid_ptr = sbt_data->vdb_grids[0];
            const auto* _grid = reinterpret_cast<const nanovdb::FloatGrid*>(grid_ptr);
            //const auto& _acc = _grid->tree().getAccessor();
            auto pos_indexed = reinterpret_cast<const nanovdb::Vec3f&>(pos);
            pos_indexed = _grid->worldToIndexF(pos_indexed);

            _local_pos_ = reinterpret_cast<vec3&>(pos_indexed);
        }
        return _local_pos_;
    }

    vec3 _uniform_pos_ = vec3(HIP_NAN_F);
    __inline__ __device__ vec3 uniformPosLazy() {
        if ( isnan(_uniform_pos_.x) ) {

            const HitGroupData* sbt_data = reinterpret_cast<HitGroupData*>( optixGetSbtDataPointer() );

            const auto grid_ptr = sbt_data->vdb_grids[0];
            const auto* _grid = reinterpret_cast<const nanovdb::FloatGrid*>(grid_ptr);

            auto bbox = _grid->indexBBox();

            nanovdb::Coord boundsMin( bbox.min() );
            nanovdb::Coord boundsMax( bbox.max() + nanovdb::Coord( 1 ) ); // extend by one unit

            vec3 min = { 
                static_cast<float>( boundsMin[0] ), 
                static_cast<float>( boundsMin[1] ), 
                static_cast<float>( boundsMin[2] )};
            vec3 max = {
                static_cast<float>( boundsMax[0] ),
                static_cast<float>( boundsMax[1] ),
                static_cast<float>( boundsMax[2] )};

            auto local_pos = localPosLazy();

            _uniform_pos_ = (local_pos - min) / (max - min);
        }
        return _uniform_pos_;
    }
};

struct VolumeOut {
    float max_density;
    float density;

    float anisotropy;
    vec3 emission;
    vec3 albedo;
};

#define USING_VDB 1

template <int Order, bool WorldSpace>
static __inline__ __device__ vec2 samplingVDB(const unsigned long long grid_ptr, vec3 att_pos) {

    const auto* _grid = reinterpret_cast<const nanovdb::FloatGrid*>(grid_ptr);
    const auto& _acc = _grid->tree().getAccessor();

    auto pos_indexed = reinterpret_cast<const nanovdb::Vec3f&>(att_pos);

    if constexpr(WorldSpace)
    {
        pos_indexed = _grid->worldToIndexF(pos_indexed);
    } //_grid->tree().root().maximum();

    return vec2 { linearSampling<decltype(_acc), Order>(_acc, pos_indexed), _grid->tree().root().maximum() };
}

static __inline__ __device__ VolumeOut evalVolume(float4* uniforms, VolumeIn &attrs) {

    auto att_pos = attrs.pos;
    auto att_clr = vec3(0);
    auto att_uv = vec3(0);
    auto att_nrm = vec3(0);
    auto att_tang = vec3(0);

    HitGroupData* sbt_data = (HitGroupData*)optixGetSbtDataPointer();
    auto zenotex = sbt_data->textures;
    auto vdb_grids = sbt_data->vdb_grids;
    auto vdb_max_v = sbt_data->vdb_max_v;

    //GENERATED_BEGIN_MARK   
        auto vol_sample_anisotropy = 0.0f;
        auto vol_sample_density = 0.0f;

        vec3 vol_sample_emission = vec3(0.0f);
        vec3 vol_sample_albedo = vec3(0.5f);
    //GENERATED_END_MARK

#if USING_VDB

    VolumeOut output;

    output.albedo = clamp(vol_sample_albedo, 0.0f, 1.0f);
    output.anisotropy = clamp(vol_sample_anisotropy, -0.99, 0.99);

    output.density = clamp(vol_sample_density, 0.0f, 1.0f);
    output.emission = vol_sample_emission;
    
    return output;
#else
    //USING 3D ARRAY
    //USING 3D Noise 
#endif
}

// ----------------------------------------------------------------------------
// Volume programs
// ----------------------------------------------------------------------------

inline __device__ void confine( const nanovdb::BBox<nanovdb::Coord> &bbox, nanovdb::Vec3f &iVec )
{
    // NanoVDB's voxels and tiles are formed from half-open intervals, i.e.
    // voxel[0, 0, 0] spans the set [0, 1) x [0, 1) x [0, 1). To find a point's voxel,
    // its coordinates are simply truncated to integer. Ray-box intersections yield
    // pairs of points that, because of numerical errors, fall randomly on either side
    // of the voxel boundaries.
    // This confine method, given a point and a (integer-based/Coord-based) bounding
    // box, moves points outside the bbox into it. That means coordinates at lower
    // boundaries are snapped to the integer boundary, and in case of the point being
    // close to an upper boundary, it is move one EPS below that bound and into the volume.

    // get the tighter box around active values
    auto iMin = nanovdb::Vec3f( bbox.min() );
    auto iMax = nanovdb::Vec3f( bbox.max() ) + nanovdb::Vec3f( 1.0f );

    // move the start and end points into the bbox
    float eps = 1e-7f;
    if( iVec[0] < iMin[0] ) iVec[0] = iMin[0];
    if( iVec[1] < iMin[1] ) iVec[1] = iMin[1];
    if( iVec[2] < iMin[2] ) iVec[2] = iMin[2];
    if( iVec[0] >= iMax[0] ) iVec[0] = iMax[0] - fmaxf( 1.0f, fabsf( iVec[0] ) ) * eps;
    if( iVec[1] >= iMax[1] ) iVec[1] = iMax[1] - fmaxf( 1.0f, fabsf( iVec[1] ) ) * eps;
    if( iVec[2] >= iMax[2] ) iVec[2] = iMax[2] - fmaxf( 1.0f, fabsf( iVec[2] ) ) * eps;
}

inline __hostdev__ void confine( const nanovdb::BBox<nanovdb::Coord> &bbox, nanovdb::Vec3f &iStart, nanovdb::Vec3f &iEnd )
{
    confine( bbox, iStart );
    confine( bbox, iEnd );
}

template<typename AccT>
inline __device__ float transmittanceHDDA(
    const nanovdb::Vec3f& start,
    const nanovdb::Vec3f& end,
    AccT& acc, const float opacity )
{

    // transmittance along a ray through the volume is computed by
    // taking the negative exponential of volume's density integrated
    // along the ray.
    float transmittance = 1.f;
    auto dir = end - start;
    auto len = dir.length();
    nanovdb::Ray<float> ray( start, dir / len, 0.0f, len );
    nanovdb::Coord ijk = nanovdb::RoundDown<nanovdb::Coord>( ray.start() ); // first hit of bbox

    // Use NanoVDB's HDDA line digitization for fast integration.
    // This algorithm (http://www.museth.org/Ken/Publications_files/Museth_SIG14.pdf)
    // can skip over sparse parts of the data structure.
    //
    nanovdb::HDDA<nanovdb::Ray<float> > hdda( ray, acc.getDim( ijk, ray ) );

    float t = 0.0f;
    float density = acc.getValue( ijk ) * opacity;
    while( hdda.step())
    {
        float dt = hdda.time() - t; // compute length of ray-segment intersecting current voxel/tile
        transmittance *= expf( -density * dt );
        t = hdda.time();
        ijk = hdda.voxel();

        density = acc.getValue( ijk ) * opacity;
        hdda.update( ray, acc.getDim( ijk, ray ) ); // if necessary adjust DDA step size
    }

    return transmittance;
}


extern "C" __global__ void __intersection__volume()
{
    RadiancePRD* prd = getPRD();
    // if (prd->test_distance) { return; }
    // auto mask = optixGetRayVisibilityMask();
    {
        const auto* sbt_data = reinterpret_cast<const HitGroupData*>( optixGetSbtDataPointer() );
        const auto* grid = reinterpret_cast<const nanovdb::FloatGrid*>( sbt_data->vdb_grids[0] );
        assert( grid );

        // compute intersection points with the volume's bounds in index (object) space.
        const float3 ray_orig = optixGetWorldRayOrigin(); //optixGetObjectRayOrigin();
        const float3 ray_dir  = optixGetWorldRayDirection(); //optixGetObjectRayDirection();

        auto bbox = grid->worldBBox(); //grid->indexBBox();
        float t0 = optixGetRayTmin();
        float t1 = _FLT_MAX_; //optixGetRayTmax();

        auto iRay = nanovdb::Ray<float>( reinterpret_cast<const nanovdb::Vec3f&>( ray_orig ),
            reinterpret_cast<const nanovdb::Vec3f&>( ray_dir ), t0, t1 );
        
        if( iRay.intersects( bbox, t0, t1 )) // t0 >= 0
        {
            // report the entry-point as hit-point
            //auto kind = optixGetHitKind();
            t0 = fmaxf(t0, optixGetRayTmin());

            prd->vol_t0 = t0;
            prd->origin_inside_vdb = (t0 == 0);

            prd->vol_t1 = t1; //min(optixGetRayTmax(), t1);
            prd->surface_inside_vdb = (optixGetRayTmax() < t1); // In case triangles were visited before vdb

            if (optixGetRayTmax() > 0) {
                optixReportIntersection( t0, 0);
            }
        }
    } 
}

extern "C" __global__ void __closesthit__radiance_volume()
{
    RadiancePRD* prd = getPRD();
    if(prd->test_distance) { return; }
    
    prd->countEmitted = false;
    prd->radiance = make_float3(0);

    prd->trace_tmin = 0;
    prd->_mask_ = EverythingMask;

    const HitGroupData* sbt_data = reinterpret_cast<HitGroupData*>( optixGetSbtDataPointer() );

    float3 ray_orig = optixGetWorldRayOrigin();
    float3 ray_dir  = optixGetWorldRayDirection();

          float t0 = prd->vol_t0; // world space
          float t1 = prd->vol_t1; // world space

    RadiancePRD testPRD {};
    testPRD.vol_t1 = _FLT_MAX_;
    testPRD.test_distance = true;
    testPRD.isSS = false;
    testPRD.opacity = 0.0f;
    traceRadianceMasked(
        params.handle,
        ray_orig,
        ray_dir,
        0,
        _FLT_MAX_,
        DefaultMatMask,
        &testPRD);

    if(testPRD.vol_t1 < t1)
    {
        t1 = testPRD.vol_t1;
        prd->surface_inside_vdb = true;
    }

    const float t_max = max(0.f, t1 - t0); // world space
    float t_ele = 0;

    auto test_point = ray_orig; 
    float3 emitting = make_float3(0.0);
    float3 scattering = make_float3(1.0);
   
    float sigma_t = _vol_extinction;
    float v_density = 0.0;

    VolumeOut vol_out;

#if (!_DELTA_TRACKING_) 

    test_point = ray_orig + test_t * ray_dir;
    auto test_point_indexd = grid->worldToIndexF(reinterpret_cast<const nanovdb::Vec3f&>(test_point));
    v_density = linearSampling(acc, test_point_indexd);
    
    vec3 new_dir = ray_dir;
    
    if(v_density > 0){
        const auto ray = nanovdb::Ray<float>( reinterpret_cast<const nanovdb::Vec3f&>( ray_orig ),
                                              reinterpret_cast<const nanovdb::Vec3f&>( ray_dir ) );
        auto start = grid->worldToIndexF( ray( t0 ) );
        auto end   = grid->worldToIndexF( ray( test_t ) );

        auto bbox = grid->indexBBox();
        confine( bbox, start, end );

        const float opacity = sbt_data->opacityHDDA;
        
        //scattering *= transmittanceHDDA( start, end, acc, 0.01 );;
        new_dir = DisneyBSDF::SampleScatterDirection(prd->seed);

        pbrt::HenyeyGreenstein hg {sbt_data->greenstein};
        float3 new_dir; float2 uu = {rnd(prd->seed), rnd(prd->seed)};
        auto pdf = hg.Sample_p(-ray_dir,             new_dir, uu);
        // //scattering *= pdf;

        scattering *= sbt_data->colorVDB;        
        ray_dir = (prd->volumeHitSurface )? ray_dir : float3(new_dir);
    }

#else

    auto level = _vol_depth;
    while(--level > 0) {
        auto prob = rnd(prd->seed);
        t_ele -= log(prob) / (sigma_t);

        if (t_ele >= t_max) {

            if (prd->surface_inside_vdb) { // Hit other material

                prd->_mask_ = DefaultMatMask;
                prd->trace_tmin = 0;

                test_point = ray_orig;

            } else { // Volume edge

                prd->_mask_ = EverythingMask;
                prd->trace_tmin = 1e-5;

                test_point = ray_orig + t1 * ray_dir;
                test_point = rtgems::offset_ray(test_point, ray_dir);
            }

            v_density = 0;
            break;
        } // over shoot, outside of volume

        test_point = ray_orig + (t0+t_ele) * ray_dir;

        VolumeIn vol_in { test_point };
        
        vol_out = evalVolume(nullptr, vol_in);
        v_density = vol_out.density;

        //prd->vol_tr *= exp(-sigma_t * t_ele);
        
        float s_prob = vol_out.density;
        float n_prob = 1.0f - s_prob;
        
        float3 s_prob_rgb = vol_out.albedo;
        float3 a_prob_rgb = 1.0f - s_prob_rgb;

        float3 n_prob_rgb = ( n_prob/s_prob ) * s_prob_rgb;
        float3 _prob_rgb = n_prob_rgb + a_prob_rgb + s_prob_rgb;

        float3 le = vol_out.emission;

        if ( length(le) > 0.0f ) {
            //le *= exp(-sigma_t * t_ele);
        
            float3 emission_prob = a_prob_rgb / _prob_rgb; // scale by emission prob

            le *= emission_prob; 
            emitting += le;
        }

        if (rnd(prd->seed) < v_density) {

            float3 new_dir; 
            pbrt::HenyeyGreenstein hg { vol_out.anisotropy };
                
            float2 uu = {rnd(prd->seed), rnd(prd->seed)};
            auto prob = hg.Sample_p(-ray_dir, new_dir, uu);              
            //auto relative_prob = prob * (HIP_PI_F * 4);
            new_dir = normalize(new_dir);

            scattering = s_prob_rgb * (_prob_rgb - a_prob_rgb) / _prob_rgb; // scattering prob
                
            ray_dir = new_dir;
            break;
        } else {
            v_density = 0; 
        } 
    }

#endif // _DELTA_TRACKING_

    prd->updateAttenuation(scattering);

    ray_orig = test_point;
    prd->origin = ray_orig;
    prd->direction = ray_dir;

    prd->emission = emitting;

    if (v_density == 0) {
        prd->CH = 0.0;
        //prd->depth += 0;
        prd->radiance += prd->emission;
        return;
    }

    float3 light_attenuation = make_float3(1.0f);
    float pl = rnd(prd->seed);
    float sum = 0.0f;
    for(int lidx=0;lidx<params.num_lights;lidx++)
    {
            ParallelogramLight light = params.lights[lidx];
            float3 light_pos = light.corner + light.v1 * 0.5 + light.v2 * 0.5;

            // Calculate properties of light sample (for area based pdf)
            float Ldist = length(light_pos - test_point);
            float3 L = normalize(light_pos - test_point);
            float nDl = 1.0f;//clamp(dot(N, L), 0.0f, 1.0f);
            float LnDl = clamp(-dot(light.normal, L), 0.000001f, 1.0f);
            float A = length(cross(params.lights[lidx].v1, params.lights[lidx].v2));
            sum += length(light.emission)  * nDl * LnDl * A / (M_PIf * Ldist * Ldist);
    }

    RadiancePRD shadow_prd {};
    shadow_prd.seed = prd->seed;

    shadow_prd.nonThinTransHit = 0;
    shadow_prd.shadowAttanuation = make_float3(1.0f);

    scattering *= vol_out.albedo;
    
    if(rnd(prd->seed)<=0.5f) {
        bool computed = false;
        float ppl = 0;
        for (int lidx = 0; lidx < params.num_lights && computed == false; lidx++) {
            ParallelogramLight light = params.lights[lidx];
            float2 z = {rnd(prd->seed), rnd(prd->seed) };
            const float z1 = z.x;
            const float z2 = z.y;
            float3 light_tpos = light.corner + light.v1 * 0.5 + light.v2 * 0.5;
            float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

            // Calculate properties of light sample (for area based pdf)
            float tLdist = length(light_tpos - test_point);
            float3 tL = normalize(light_tpos - test_point);
            float tnDl = 1.0f; //clamp(dot(N, tL), 0.0f, 1.0f);
            float tLnDl = clamp(-dot(light.normal, tL), 0.000001f, 1.0f);
            float tA = length(cross(params.lights[lidx].v1, params.lights[lidx].v2));
            ppl += length(light.emission) * tnDl * tLnDl * tA / (M_PIf * tLdist * tLdist) / sum;
            if (ppl > pl) {
                float Ldist = length(light_pos - test_point) + 1e-6;
                float3 L = normalize(light_pos - test_point);
                float nDl = 1.0f; //clamp(dot(N, L), 0.0f, 1.0f);
                float LnDl = clamp(-dot(light.normal, L), 0.0f, 1.0f);
                float A = length(cross(params.lights[lidx].v1, params.lights[lidx].v2));
                float weight = 0.0f;
                if (nDl > 0.0f && LnDl > 0.0f) {
                    
                    traceOcclusion(params.handle, test_point, L,
                                   0,         // tmin
                                   Ldist - 1e-5f, // tmax,
                                   &shadow_prd);

                    light_attenuation = shadow_prd.shadowAttanuation;

                    weight = sum * nDl / tnDl * LnDl / tLnDl * (tLdist * tLdist) / (Ldist * Ldist) /
                                (length(light.emission)+1e-6f);
                }
                // prd->LP = test_point;
                // prd->Ldir = L;
                // prd->Lweight = weight;
                // prd->nonThinTransHit = 0;
                
                pbrt::HenyeyGreenstein hg { vol_out.anisotropy };
                float ray_prob = hg.p(-ray_dir, L);
                float3 lbrdf = scattering * ray_prob;

                prd->radiance = light_attenuation * weight * 2.0 * light.emission * lbrdf;
                computed = true;
            }
        }
    } else {

        vec3 sunLightDir = vec3(params.sunLightDirX, params.sunLightDirY, params.sunLightDirZ);
        auto sun_dir = BRDFBasics::halfPlaneSample(prd->seed, sunLightDir,
                                                   params.sunSoftness * 0.2); //perturb the sun to have some softness
        sun_dir = normalize(sun_dir);
        // prd->LP = test_point;
        // prd->Ldir = sun_dir;
        // prd->Lweight = 1.0;
        // prd->nonThinTransHit = 1;
        traceOcclusion(params.handle, test_point, sun_dir,
                       0, // tmin
                       1e16f, // tmax,
                       &shadow_prd);

        light_attenuation = shadow_prd.shadowAttanuation;

        pbrt::HenyeyGreenstein hg { vol_out.anisotropy };
        float ray_prob = hg.p(-ray_dir, sun_dir);
        float3 lbrdf = scattering * clamp(ray_prob, 0.0f, 1.0f);

        prd->radiance = light_attenuation * params.sunLightIntensity * 2.0f * lbrdf *
                        float3(envSky(sun_dir, sunLightDir, make_float3(0., 0., 1.),
                                       10, // be careful
                                       .45, 15., 1.030725 * 0.3, params.elapsedTime));
    }

    prd->CH = 1.0;
    prd->depth += 1;
    prd->radiance += prd->emission;

    return;
}

extern "C" __global__ void __anyhit__occlusion_volume()
{
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();

    RadiancePRD* prd = getPRD();

    const float t0 = prd->vol_t0;
    const float t1 = prd->vol_t1;

    const float t_max = t1 - t0; // world space
          float t_ele = 0;

    float3 test_point = ray_orig; 
    float3 transmittance = make_float3(1.0f);

    const float sigma_t = _vol_extinction;

#if (!_DELTA_TRACKING_) 

    const auto ray = nanovdb::Ray<float>( reinterpret_cast<const nanovdb::Vec3f&>( ray_orig ),
                                              reinterpret_cast<const nanovdb::Vec3f&>( ray_dir ) );
    auto start = grid->worldToIndexF( ray( t0 ) );
    auto end   = grid->worldToIndexF( ray( t1 ) );

    auto bbox = grid->indexBBox();
    confine( bbox, start, end );

    const float opacity = sbt_data->opacityHDDA;
    float transHDDA = transmittanceHDDA( start, end, acc, sbt_data->opacityHDDA );
    if (transHDDA < 1.0) {
        transmittance *= transHDDA;
        transmittance *= sbt_data->colorVDB;
    }

#else
    auto level = _vol_depth;
    while(--level > 0) {

        auto prob = rnd(prd->seed);
        t_ele -= log(prob) / (sigma_t);

        test_point = ray_orig + (t0+t_ele) * ray_dir;

        if (t_ele >= t_max) {
            break;
        } // over shoot, outside of volume

        VolumeIn vol_in { test_point };
        VolumeOut vol_out = evalVolume(nullptr, vol_in);

        const auto v_density = vol_out.density;

        transmittance *= 1 - clamp(v_density, 0.0f, 1.0f);
        auto avg = dot(transmittance, make_float3(1.0f/3.0f));
        if (avg < 0.1) {
            float q = max(0.05, 1 - avg);
            if (rnd(prd->seed) < q) { 
                transmittance = vec3(0);
                break; 
            } else {
                transmittance /= 1-q;
            }
        }
        if (v_density > 0) {
            transmittance *= vol_out.albedo;
        }
    }
#endif

    prd->shadowAttanuation *= transmittance;
    optixIgnoreIntersection();
    //prd->origin = ray_orig;
    //prd->direction = ray_dir;
    return;
}