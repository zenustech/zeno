#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <cuda/helpers.h>
#include <sutil/vec_math.h>

#include "optixPathTracer.h"
#include "TraceStuff.h"
#include "DisneyBSDF.h"
#include "zxxglslvec.h"

#include "volume.h"
#include <hip/hip_fp16.h>

extern "C" {
__constant__ Params params;

}
//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------
static __inline__ __device__
vec3 RRTAndODTFit(vec3 v)
{
    vec3 a = v * (v + 0.0245786f) - 0.000090537f;
    vec3 b = v * (0.983729f * v + 0.4329510f) + 0.238081f;
    return a / b;
}
static __inline__ __device__
vec3 ACESFitted(vec3 color, float gamma)
{
//    const mat3x3 ACESInputMat = mat3x3
//        (
//            0.59719, 0.35458, 0.04823,
//            0.07600, 0.90834, 0.01566,
//            0.02840, 0.13383, 0.83777
//        );
//    mat3x3 ACESOutputMat = mat3x3
//    (
//        1.60475, -0.53108, -0.07367,
//        -0.10208,  1.10813, -0.00605,
//        -0.00327, -0.07276,  1.07602
//    );
    vec3 v1 = vec3(0.59719, 0.35458, 0.04823);
    vec3 v2 = vec3(0.07600, 0.90834, 0.01566);
    vec3 v3 = vec3(0.02840, 0.13383, 0.83777);
    color = vec3(dot(color, v1), dot(color, v2), dot(color, v3));
    // Apply RRT and ODT
    color = RRTAndODTFit(color);

    v1 = vec3(1.60475, -0.53108, -0.07367);
    v2 = vec3(-0.10208,  1.10813, -0.00605);
    v3 = vec3(-0.00327, -0.07276,  1.07602);
    color = vec3(dot(color, v1), dot(color, v2), dot(color, v3));

    // Clamp to [0, 1]
    color = clamp(color, 0.0f, 1.0f);

    color = pow(color, vec3(1.0f / gamma));

    return color;
}

extern "C" __global__ void __raygen__rg()
{
    const int    w   = params.windowSpace.x;
    const int    h   = params.windowSpace.y;
    //const float3 eye = params.eye;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframe_index;
    const CameraInfo cam = params.cam;

    unsigned int seed = tea<4>( idx.y*w + idx.x, subframe_index );
    float focalPlaneDistance = cam.focalPlaneDistance>0.01f? cam.focalPlaneDistance : 0.01f;
    float aperture = clamp(cam.aperture,0.0f,100.0f);
    aperture/=10;

    float3 result = make_float3( 0.0f );
    float3 result_d = make_float3( 0.0f );
    float3 result_s = make_float3( 0.0f );
    float3 result_t = make_float3( 0.0f );
    float3 result_b = make_float3( 0.0f );
    int i = params.samples_per_launch;

    float3 tmp_albedo{};
    float3 tmp_normal{};

    do
    {
        // The center of each pixel is at fraction (0.5,0.5)
        float2 subpixel_jitter = {
            rnd(seed),
            rnd(seed)
        };

        float2 d = 2.0f * make_float2(
                ( static_cast<float>( idx.x + params.windowCrop_min.x ) + subpixel_jitter.x ) / static_cast<float>( w ),
                ( static_cast<float>( idx.y + params.windowCrop_min.y ) + subpixel_jitter.y ) / static_cast<float>( h )
                ) - 1.0f;
        //float3 ray_direction = normalize(cam.right * d.x + cam.up * d.y + cam.front);
        float2 r01 = {
            rnd(seed),
            rnd(seed)
        };

        float r0 = r01.x * 2.0f * M_PIf;
        float r1 = r01.y * aperture * aperture;
        r1 = sqrt(r1);

        // float3 ray_origin    = cam.eye + r1 * ( cosf(r0)* cam.right + sinf(r0)* cam.up);
        // float3 ray_direction = cam.eye + focalPlaneDistance *(cam.right * d.x + cam.up * d.y + cam.front) - ray_origin;
   
        float3 eye_shake     = r1 * ( cosf(r0)* normalize(cam.right) + sinf(r0)* normalize(cam.up)); // Camera local space

        float3 ray_origin    = cam.eye + eye_shake;
        float3 ray_direction = focalPlaneDistance *(cam.right * d.x + cam.up * d.y + cam.front) - eye_shake; // Camera local space
               ray_direction = normalize(ray_direction);

        RadiancePRD prd; 
        prd.emission     = make_float3(0.f);
        prd.radiance     = make_float3(0.f);
        prd.attenuation  = make_float3(1.f);
        prd.attenuation2 = make_float3(1.f);
        prd.prob         = 1.0f;
        prd.prob2        = 1.0f;
        prd.countEmitted = true;
        prd.done         = false;
        prd.seed         = seed;
        prd.opacity      = 0;
        prd.flags        = 0;
        prd.next_ray_is_going_inside    = false;
        prd.maxDistance  = 1e16f;
        prd.medium       = DisneyBSDF::PhaseFunctions::vacuum;

        prd.depth = 0;
        prd.diffDepth = 0;
        prd.isSS = false;
        prd.direction = ray_direction;
        prd.curMatIdx = 0;
        prd.test_distance = false;
        prd.sigma_t_queue[0] = vec3(0.0f);
        prd.ss_alpha_queue[0] = vec3(-1.0f);
        prd.minSpecRough = 0.01;
        prd.samplePdf = 1.0f;
        prd.first_hit_type = 0;
        prd.hitEnv = false;
        auto tmin = prd.trace_tmin;
        auto ray_mask = prd._mask_;

        // prd.channelPDF= vec3(1.0f/3.0f);
        // prd.ss_alpha = vec3(0.0f);
        // prd.sigma_t = vec3(0.0f);



        //if constexpr(params.denoise) 
        if (params.denoise) 
        {
            prd.trace_denoise_albedo = true;
            prd.trace_denoise_normal = true;
        }

        // Primary Ray
        traceRadianceMasked(params.handle, ray_origin, ray_direction, tmin, prd.maxDistance, ray_mask, &prd);

        tmp_albedo = prd.tmp_albedo;
        tmp_normal = prd.tmp_normal;

        prd.trace_denoise_albedo = false;
        prd.trace_denoise_normal = false;

        for(;;)
        {
            prd.radiance_d = make_float3(0);
            prd.radiance_s = make_float3(0);
            prd.radiance_t = make_float3(0);
            tmin = prd.trace_tmin;
            prd.trace_tmin = 0;

            ray_mask = prd._mask_; 
            prd._mask_ = EverythingMask;

            if(prd.countEmitted==false || prd.depth>0) {
                auto temp_radiance = prd.radiance * prd.attenuation2;

                //float upperBound = prd.fromDiff?1.0f:1.0f;
                float3 clampped = clamp(vec3(temp_radiance), vec3(0), vec3(10));

                result += prd.depth>1?clampped:temp_radiance;
                if(prd.depth==1 && prd.hitEnv == false)
                {
                  result_d += prd.radiance_d * prd.attenuation2;
                  result_s += prd.radiance_s * prd.attenuation2;
                  result_t += prd.radiance_t * prd.attenuation2;
                }
                if(prd.depth>1 || (prd.depth==1 && prd.hitEnv == true)) {
                  result_d +=
                      prd.first_hit_type == 1 ? clampped : make_float3(0, 0, 0);
                  result_s +=
                      prd.first_hit_type == 2 ? clampped : make_float3(0, 0, 0);
                  result_t +=
                      prd.first_hit_type == 3 ? clampped : make_float3(0, 0, 0);
                }

                // fire without smoke requires this line to work.
            }

            prd.radiance = make_float3(0);
            prd.emission = make_float3(0);

            if (ray_mask != EverythingMask && ray_mask != NothingMask) {
                //ray_origin = prd.origin;
                //ray_direction = prd.direction;
                traceRadianceMasked(params.handle, ray_origin, ray_direction, tmin, prd.maxDistance, ray_mask, &prd);
                continue; // trace again with same parameters but different mask
            }

            if(prd.countEmitted==true && prd.depth>0){
                prd.done = true;
            }

            if( prd.done || params.simpleRender==true){
                break;
            }

            if(prd.depth>16){
                //float RRprob = clamp(length(prd.attenuation)/1.732f,0.01f,0.9f);
                float RRprob = clamp(length(prd.attenuation),0.1f, 0.95f);
                if(rnd(prd.seed) > RRprob || prd.depth > 16){
                    prd.done=true;
                } else {
                    prd.attenuation = prd.attenuation / RRprob;
                }
            }
            if(prd.countEmitted == true)
                prd.passed = true;

            ray_origin    = prd.origin;
            ray_direction = prd.direction;
            
//            result_d = make_float3(0,0,0);
//            result_s = make_float3(0,0,0);
//            result_t = make_float3(0,0,0);

            traceRadianceMasked(params.handle, ray_origin, ray_direction, tmin, prd.maxDistance, ray_mask, &prd);
        }
        result_b += prd.first_hit_type == 0 ? make_float3(0, 0, 0)
                                            : make_float3(1, 1, 1);
        seed = prd.seed;
    }
    while( --i );

    float3         accum_color  = result / static_cast<float>( params.samples_per_launch );
    float3         accum_color_d  = result_d / static_cast<float>( params.samples_per_launch );
    float3         accum_color_s  = result_s / static_cast<float>( params.samples_per_launch );
    float3         accum_color_t  = result_t / static_cast<float>( params.samples_per_launch );
    float3         accum_color_b  = result_b / static_cast<float>( params.samples_per_launch );
    const uint3    launch_index = optixGetLaunchIndex();
    const unsigned int image_index  = launch_index.y * params.width + launch_index.x;

    if( subframe_index > 0 )
    {
        const float                 a = 1.0f / static_cast<float>( subframe_index+1 );
        const float3 accum_color_prev = make_float3( params.accum_buffer[ image_index ]);
        const float3 accum_color_prev_d = make_float3( params.accum_buffer_D[ image_index ]);
        const float3 accum_color_prev_s = make_float3( params.accum_buffer_S[ image_index ]);
        const float3 accum_color_prev_t = make_float3( params.accum_buffer_T[ image_index ]);
        const float3 accum_color_prev_b = make_float3( params.accum_buffer_B[ image_index ]);
        accum_color   = lerp( accum_color_prev, accum_color, a );
        accum_color_d = lerp( accum_color_prev_d, accum_color_d, a );
        accum_color_s = lerp( accum_color_prev_s, accum_color_s, a );
        accum_color_t = lerp( accum_color_prev_t, accum_color_t, a );
        accum_color_b = lerp( accum_color_prev_b, accum_color_b, a );

        if (params.denoise) {

            const float3 accum_albedo_prev = params.albedo_buffer[ image_index ];
            tmp_albedo = lerp(accum_albedo_prev, tmp_albedo, a);

            const float3 accum_normal_prev = params.normal_buffer[ image_index ];
            tmp_normal = lerp(accum_normal_prev, tmp_normal, a); 
        }
    }

    /*if (launch_index.x == 0) {*/
        /*printf("%p\n", params.accum_buffer);*/
        /*printf("%p\n", params.frame_buffer);*/
    /*}*/
    params.accum_buffer[ image_index ] = make_float4( accum_color, 1.0f);
    params.accum_buffer_D[ image_index ] = make_float4( accum_color_d, 1.0f);
    params.accum_buffer_S[ image_index ] = make_float4( accum_color_s, 1.0f);
    params.accum_buffer_T[ image_index ] = make_float4( accum_color_t, 1.0f);
    params.accum_buffer_B[ image_index ] = make_float4( accum_color_b, 1.0f);
    //vec3 aecs_fitted = ACESFitted(vec3(accum_color), 2.2);
    float3 out_color = accum_color;
    float3 out_color_d = accum_color_d;
    float3 out_color_s = accum_color_s;
    float3 out_color_t = accum_color_t;
    float3 out_color_b = accum_color_b;
    params.frame_buffer[ image_index ] = make_color ( out_color );
    params.frame_buffer_D[ image_index ] = make_color ( out_color_d );
    params.frame_buffer_S[ image_index ] = make_color ( out_color_s );
    params.frame_buffer_T[ image_index ] = make_color ( out_color_t );
    params.frame_buffer_B[ image_index ] = make_color ( out_color_b );
    
    if (params.denoise) {
        params.albedo_buffer[ image_index ] = tmp_albedo;
        params.normal_buffer[ image_index ] = tmp_normal;
    }
}

extern "C" __global__ void __miss__radiance()
{
    vec3 sunLightDir = vec3(
            params.sunLightDirX,
            params.sunLightDirY,
            params.sunLightDirZ
            );
    MissData* rt_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    RadiancePRD* prd = getPRD();
    prd->attenuation2 = prd->attenuation;
    prd->passed = false;
    prd->countEmitted = false;
    prd->CH = 0.0;
    if(prd->medium != DisneyBSDF::PhaseFunctions::isotropic){
        float upperBound = 100.0f;
        float envPdf = 0.0f;
        vec3 skysample =
            envSky(
            normalize(prd->direction),
            sunLightDir,
            make_float3(0., 0., 1.),
            40, // be careful
            .45,
            15.,
            1.030725f * 0.3f,
            params.elapsedTime,
            envPdf,
            upperBound,
            0.0

        );
        float misWeight = BRDFBasics::PowerHeuristic(prd->samplePdf,envPdf);

        misWeight = misWeight>0.0f?misWeight:0.0f;
        misWeight = envPdf>0.0f?misWeight:1.0f;
        misWeight = prd->depth>=1?misWeight:1.0f;
        misWeight = prd->samplePdf>0.0f?misWeight:1.0f;
        prd->radiance = misWeight * skysample ;
        if (params.show_background == false) {
            prd->radiance = prd->depth>=1?prd->radiance:make_float3(0,0,0);
        }

        prd->done      = true;
        prd->hitEnv    = true;
        return;
    }

    vec3 sigma_t, ss_alpha;
    prd->readMat(sigma_t, ss_alpha);

    vec3 transmittance;
    if (ss_alpha.x < 0.0f) { // is inside Glass
        transmittance = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
    } else {
        transmittance = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), false);
    }

    prd->attenuation *= transmittance;//DisneyBSDF::Transmission(prd->extinction,optixGetRayTmax());
    prd->attenuation2 *= transmittance;//DisneyBSDF::Transmission(prd->extinction,optixGetRayTmax());
    prd->origin += prd->direction * optixGetRayTmax();

    auto anisotropy = prd->anisotropy_queue[prd->curMatIdx]; 
    pbrt::HenyeyGreenstein hg{__half2float(anisotropy)}; float3 newdir;

    float2 uu = {rnd(prd->seed), rnd(prd->seed)}; 
    auto prob = hg.Sample_p(-prd->direction, newdir, uu);

    prd->direction = newdir; //DisneyBSDF::SampleScatterDirection(prd->seed);

    vec3 channelPDF = vec3(1.0f/3.0f);
    prd->channelPDF = channelPDF;
    if (ss_alpha.x < 0.0f) { // is inside Glass
        prd->maxDistance = DisneyBSDF::SampleDistance2(prd->seed, sigma_t, sigma_t, channelPDF);
        prd->isSS = false;
    } 
    else
    {
        prd->maxDistance =
            DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * ss_alpha, sigma_t, channelPDF);
        prd->channelPDF = channelPDF;
        prd->isSS = true;
    } 

    prd->depth++;

    if(length(prd->attenuation)<1e-7f){
        prd->done = true;
    }
}

extern "C" __global__ void __miss__occlusion()
{
    setPayloadOcclusion( false );
}

extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion( true );
}