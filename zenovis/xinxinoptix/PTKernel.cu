#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <sutil/vec_math.h>
#include <cuda/helpers.h>
#include "optixPathTracer.h"
#include "TraceStuff.h"

extern "C" {
__constant__ Params params;
}
//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
    const int    w   = params.width;
    const int    h   = params.height;
    //const float3 eye = params.eye;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframe_index;
    const CameraInfo cam = params.cam;

    unsigned int seed = tea<4>( idx.y*w + idx.x, subframe_index );

    float3 result = make_float3( 0.0f );
    int i = params.samples_per_launch;
    do
    {
        // The center of each pixel is at fraction (0.5,0.5)
        const float2 subpixel_jitter = make_float2( rnd( seed ), rnd( seed ) );

        const float2 d = 2.0f * make_float2(
                ( static_cast<float>( idx.x ) + subpixel_jitter.x ) / static_cast<float>( w ),
                ( static_cast<float>( idx.y ) + subpixel_jitter.y ) / static_cast<float>( h )
                ) - 1.0f;
        float3 ray_direction = normalize(cam.right * d.x + cam.up * d.y + cam.front);
        float3 ray_origin    = cam.eye;

        RadiancePRD prd;
        prd.emitted      = make_float3(0.f);
        prd.radiance     = make_float3(0.f);
        prd.attenuation  = make_float3(1.f);
        prd.attenuation2 = make_float3(1.f);
        prd.prob         = 1.0f;
        prd.prob2        = 1.0f;
        prd.countEmitted = true;
        prd.done         = false;
        prd.seed         = seed;
        prd.opacity      = 0;
        prd.flags        = 0;
        int depth = 0;
        for( ;; )
        {
            traceRadiance(
                    params.handle,
                    ray_origin,
                    ray_direction,
                    1e-5f,  // tmin       // TODO: smarter offset
                    1e16f,  // tmax
                    &prd );

            result += prd.emitted;
            result += prd.radiance * prd.attenuation2/prd.prob2;

            if( prd.done  || depth >= 5 ) // TODO RR, variable for depth
                break;

            ray_origin    = prd.origin;
            ray_direction = prd.direction;
            if(prd.opacity<0.99)
                ++depth;
        }
    }
    while( --i );

    const uint3    launch_index = optixGetLaunchIndex();
    const unsigned int image_index  = launch_index.y * params.width + launch_index.x;
    float3         accum_color  = result / static_cast<float>( params.samples_per_launch );

    if( subframe_index > 0 )
    {
        const float                 a = 1.0f / static_cast<float>( subframe_index+1 );
        const float3 accum_color_prev = make_float3( params.accum_buffer[ image_index ]);
        accum_color = lerp( accum_color_prev, accum_color, a );
    }
    /*if (launch_index.x == 0) {*/
        /*printf("%p\n", params.accum_buffer);*/
        /*printf("%p\n", params.frame_buffer);*/
    /*}*/
    params.accum_buffer[ image_index ] = make_float4( accum_color, 1.0f);
    params.frame_buffer[ image_index ] = make_color ( accum_color );
}


extern "C" __global__ void __miss__radiance()
{
    MissData* rt_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    RadiancePRD* prd = getPRD();

    prd->radiance = make_float3( rt_data->bg_color );
    prd->done      = true;
}
